/* Include files */

#include "dynamics_lib_sfun.h"
#include "c6_dynamics_lib.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c6_dynamics_lib(SFc6_dynamics_libInstanceStruct
  *chartInstance);
static void initialize_params_c6_dynamics_lib(SFc6_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_start_c6_dynamics_lib(SFc6_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_terminate_c6_dynamics_lib(SFc6_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_setup_runtime_resources_c6_dynamics_lib
  (SFc6_dynamics_libInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c6_dynamics_lib
  (SFc6_dynamics_libInstanceStruct *chartInstance);
static void enable_c6_dynamics_lib(SFc6_dynamics_libInstanceStruct
  *chartInstance);
static void disable_c6_dynamics_lib(SFc6_dynamics_libInstanceStruct
  *chartInstance);
static void sf_gateway_c6_dynamics_lib(SFc6_dynamics_libInstanceStruct
  *chartInstance);
static void ext_mode_exec_c6_dynamics_lib(SFc6_dynamics_libInstanceStruct
  *chartInstance);
static void c6_do_animation_call_c6_dynamics_lib(SFc6_dynamics_libInstanceStruct
  *chartInstance);
static const mxArray *get_sim_state_c6_dynamics_lib
  (SFc6_dynamics_libInstanceStruct *chartInstance);
static void set_sim_state_c6_dynamics_lib(SFc6_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c6_st);
static void initSimStructsc6_dynamics_lib(SFc6_dynamics_libInstanceStruct
  *chartInstance);
static void initSubchartIOPointersc6_dynamics_lib
  (SFc6_dynamics_libInstanceStruct *chartInstance);
static real32_T c6_eML_blk_kernel(SFc6_dynamics_libInstanceStruct *chartInstance,
  real_T c6_b_rearTrack, real32_T c6_b_rearTireStiffness);
static real32_T c6_emlrt_marshallIn(SFc6_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c6_nullptr, const char_T *c6_identifier);
static real32_T c6_b_emlrt_marshallIn(SFc6_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c6_u, const emlrtMsgIdentifier *c6_parentId);
static uint8_T c6_c_emlrt_marshallIn(SFc6_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c6_nullptr, const char_T *c6_identifier);
static uint8_T c6_d_emlrt_marshallIn(SFc6_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c6_u, const emlrtMsgIdentifier *c6_parentId);
static void init_dsm_address_info(SFc6_dynamics_libInstanceStruct *chartInstance);
static void init_simulink_io_address(SFc6_dynamics_libInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c6_dynamics_lib(SFc6_dynamics_libInstanceStruct
  *chartInstance)
{
  emlrtLicenseCheckR2022a(chartInstance->c6_fEmlrtCtx,
    "EMLRT:runTime:MexFunctionNeedsLicense", "distrib_computing_toolbox", 2);
  sf_is_first_init_cond(chartInstance->S);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  emlrtInitGPU(chartInstance->c6_fEmlrtCtx);
  hipGetLastError();
}

static void initialize_params_c6_dynamics_lib(SFc6_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void mdl_start_c6_dynamics_lib(SFc6_dynamics_libInstanceStruct
  *chartInstance)
{
  sim_mode_is_external(chartInstance->S);
}

static void mdl_terminate_c6_dynamics_lib(SFc6_dynamics_libInstanceStruct
  *chartInstance)
{
  hipError_t c6_errCode;
  c6_errCode = hipGetLastError();
  if (c6_errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(c6_errCode), (char_T *)
                       hipGetErrorName(c6_errCode), (char_T *)
                       hipGetErrorString(c6_errCode), (char_T *)
                       "SimGPUErrorChecks", chartInstance->c6_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c6_dynamics_lib
  (SFc6_dynamics_libInstanceStruct *chartInstance)
{
  sfSetAnimationVectors(chartInstance->S, chartInstance->c6_JITStateAnimation,
                        chartInstance->c6_JITTransitionAnimation);
}

static void mdl_cleanup_runtime_resources_c6_dynamics_lib
  (SFc6_dynamics_libInstanceStruct *chartInstance)
{
}

static void enable_c6_dynamics_lib(SFc6_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c6_dynamics_lib(SFc6_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void sf_gateway_c6_dynamics_lib(SFc6_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c6_JITTransitionAnimation[0] = 0U;
  *chartInstance->c6_antiRollRearTires = c6_eML_blk_kernel(chartInstance,
    *chartInstance->c6_rearTrack, *chartInstance->c6_rearTireStiffness);
  c6_do_animation_call_c6_dynamics_lib(chartInstance);
}

static void ext_mode_exec_c6_dynamics_lib(SFc6_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void c6_do_animation_call_c6_dynamics_lib(SFc6_dynamics_libInstanceStruct
  *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static const mxArray *get_sim_state_c6_dynamics_lib
  (SFc6_dynamics_libInstanceStruct *chartInstance)
{
  const mxArray *c6_b_y = NULL;
  const mxArray *c6_c_y = NULL;
  const mxArray *c6_st;
  const mxArray *c6_y = NULL;
  c6_st = NULL;
  c6_st = NULL;
  c6_y = NULL;
  sf_mex_assign(&c6_y, sf_mex_createcellmatrix(2, 1), false);
  c6_b_y = NULL;
  sf_mex_assign(&c6_b_y, sf_mex_create("y", chartInstance->c6_antiRollRearTires,
    1, 0U, 0U, 0U, 0), false);
  sf_mex_setcell(c6_y, 0, c6_b_y);
  c6_c_y = NULL;
  sf_mex_assign(&c6_c_y, sf_mex_create("y",
    &chartInstance->c6_is_active_c6_dynamics_lib, 3, 0U, 0U, 0U, 0), false);
  sf_mex_setcell(c6_y, 1, c6_c_y);
  sf_mex_assign(&c6_st, c6_y, false);
  return c6_st;
}

static void set_sim_state_c6_dynamics_lib(SFc6_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c6_st)
{
  const mxArray *c6_u;
  c6_u = sf_mex_dup(c6_st);
  *chartInstance->c6_antiRollRearTires = c6_emlrt_marshallIn(chartInstance,
    sf_mex_dup(sf_mex_getcell(c6_u, 0)), "antiRollRearTires");
  chartInstance->c6_is_active_c6_dynamics_lib = c6_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c6_u, 1)),
     "is_active_c6_dynamics_lib");
  sf_mex_destroy(&c6_u);
  sf_mex_destroy(&c6_st);
}

static void initSimStructsc6_dynamics_lib(SFc6_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void initSubchartIOPointersc6_dynamics_lib
  (SFc6_dynamics_libInstanceStruct *chartInstance)
{
}

const mxArray *sf_c6_dynamics_lib_get_eml_resolved_functions_info()
{
  const mxArray *c6_nameCaptureInfo = NULL;
  c6_nameCaptureInfo = NULL;
  sf_mex_assign(&c6_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c6_nameCaptureInfo;
}

static real32_T c6_eML_blk_kernel(SFc6_dynamics_libInstanceStruct *chartInstance,
  real_T c6_b_rearTrack, real32_T c6_b_rearTireStiffness)
{
  real_T c6_a;
  c6_a = c6_b_rearTrack / 1000.0;
  return static_cast<real32_T>(c6_a * c6_a * 1.5574077246549023) *
    c6_b_rearTireStiffness * 1000.0F / 2.0F;
}

static real32_T c6_emlrt_marshallIn(SFc6_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c6_nullptr, const char_T *c6_identifier)
{
  emlrtMsgIdentifier c6_thisId;
  real32_T c6_y;
  c6_thisId.fIdentifier = const_cast<const char_T *>(c6_identifier);
  c6_thisId.fParent = NULL;
  c6_thisId.bParentIsCell = false;
  c6_y = c6_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c6_nullptr), &c6_thisId);
  sf_mex_destroy(&c6_nullptr);
  return c6_y;
}

static real32_T c6_b_emlrt_marshallIn(SFc6_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c6_u, const emlrtMsgIdentifier *c6_parentId)
{
  real32_T c6_f;
  real32_T c6_y;
  sf_mex_import(c6_parentId, sf_mex_dup(c6_u), &c6_f, 0, 1, 0U, 0, 0U, 0);
  c6_y = c6_f;
  sf_mex_destroy(&c6_u);
  return c6_y;
}

static uint8_T c6_c_emlrt_marshallIn(SFc6_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c6_nullptr, const char_T *c6_identifier)
{
  emlrtMsgIdentifier c6_thisId;
  uint8_T c6_y;
  c6_thisId.fIdentifier = const_cast<const char_T *>(c6_identifier);
  c6_thisId.fParent = NULL;
  c6_thisId.bParentIsCell = false;
  c6_y = c6_d_emlrt_marshallIn(chartInstance, sf_mex_dup(c6_nullptr), &c6_thisId);
  sf_mex_destroy(&c6_nullptr);
  return c6_y;
}

static uint8_T c6_d_emlrt_marshallIn(SFc6_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c6_u, const emlrtMsgIdentifier *c6_parentId)
{
  uint8_T c6_b_u;
  uint8_T c6_y;
  sf_mex_import(c6_parentId, sf_mex_dup(c6_u), &c6_b_u, 1, 3, 0U, 0, 0U, 0);
  c6_y = c6_b_u;
  sf_mex_destroy(&c6_u);
  return c6_y;
}

static void init_dsm_address_info(SFc6_dynamics_libInstanceStruct *chartInstance)
{
}

static void init_simulink_io_address(SFc6_dynamics_libInstanceStruct
  *chartInstance)
{
  chartInstance->c6_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c6_rearTrack = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c6_antiRollRearTires = (real32_T *)
    ssGetOutputPortSignal_wrapper(chartInstance->S, 1);
  chartInstance->c6_rearTireStiffness = (real32_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c6_dynamics_lib_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(2669533888U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(3588063520U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(3314497866U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(1599914973U);
}

mxArray *sf_c6_dynamics_lib_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c6_dynamics_lib_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("chartInfo");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c6_dynamics_lib_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c6_dynamics_lib(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiAWYGJgYAPSHEDMxAABrFA+I1SMES7OAhdXAOKSyoJUkHhxUbJnCpDOS8wF8xN"
    "LKzzz0vLB5lswIMxnw2I+I5L5nFBxCPhgT5l+EQeQfg8k/SxY9Asi6ReA8hPzSjKD8nNyglITi0"
    "Iyi1KLofJ8SDT57lJwoEw/xP4AAv6SRPMXiJ9ZHJ+YXJJZlhqfbBafUgmMrszk4viczCQ0/wEAC"
    "N4fZA=="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c6_dynamics_lib_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "s4Y8RfVSyjyHTTXuB2WgM8G";
}

static void sf_opaque_initialize_c6_dynamics_lib(void *chartInstanceVar)
{
  initialize_params_c6_dynamics_lib((SFc6_dynamics_libInstanceStruct*)
    chartInstanceVar);
  initialize_c6_dynamics_lib((SFc6_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c6_dynamics_lib(void *chartInstanceVar)
{
  enable_c6_dynamics_lib((SFc6_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c6_dynamics_lib(void *chartInstanceVar)
{
  disable_c6_dynamics_lib((SFc6_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c6_dynamics_lib(void *chartInstanceVar)
{
  sf_gateway_c6_dynamics_lib((SFc6_dynamics_libInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c6_dynamics_lib(SimStruct* S)
{
  return get_sim_state_c6_dynamics_lib((SFc6_dynamics_libInstanceStruct *)
    sf_get_chart_instance_ptr(S));     /* raw sim ctx */
}

static void sf_opaque_set_sim_state_c6_dynamics_lib(SimStruct* S, const mxArray *
  st)
{
  set_sim_state_c6_dynamics_lib((SFc6_dynamics_libInstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c6_dynamics_lib(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc6_dynamics_libInstanceStruct*) chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_dynamics_lib_optimization_info();
    }

    mdl_cleanup_runtime_resources_c6_dynamics_lib
      ((SFc6_dynamics_libInstanceStruct*) chartInstanceVar);
    ((SFc6_dynamics_libInstanceStruct*) chartInstanceVar)->
      ~SFc6_dynamics_libInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c6_dynamics_lib(void *chartInstanceVar)
{
  mdl_start_c6_dynamics_lib((SFc6_dynamics_libInstanceStruct*) chartInstanceVar);
  if (chartInstanceVar) {
    sf_reset_warnings_ChartRunTimeInfo(((SFc6_dynamics_libInstanceStruct*)
      chartInstanceVar)->S);
  }
}

static void sf_opaque_mdl_terminate_c6_dynamics_lib(void *chartInstanceVar)
{
  mdl_terminate_c6_dynamics_lib((SFc6_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c6_dynamics_lib(SimStruct *S)
{
  mdlProcessParamsCommon(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c6_dynamics_lib((SFc6_dynamics_libInstanceStruct*)
      sf_get_chart_instance_ptr(S));
    initSubchartIOPointersc6_dynamics_lib((SFc6_dynamics_libInstanceStruct*)
      sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c6_dynamics_lib_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [21] = {
    "eNrtWM1v40QUn0SlYtHuUiSkZQUSe+QA0lI+1AOCtvnYRmpoVKdd4FJN7Od46HjsnY+k4Y/hwo0",
    "DEv8BR7hw5s/ghODGG8fJBickHkeCXYQrxx3bv/m99+Z9jUmt0yV43MXz8DVCdvH6Ip51Mj1eyM",
    "e1hXN6f4d8lI9/fIkQPwlgCMIzYchuiNshTNyjksaKuB+CxnAOKuFGs0R0RJiUxzIRggTh4wRpI",
    "rUTr2Kx4Uxct43wLbN6HDE/8qLE8OAYJ6TBmeCTv+NNje4hY5NJ8HUbINCRTMwwanM6XG8FqceN",
    "CPxrZWJnWynQnkmtqqpruGYph9YN+B2hNEUrqA36eppqaOgbNyNbfZU3QydxyhkV5W0dUeVBit6",
    "h4SIN8PfMaLReSV7kGzBBdSIZ5a2YNyIqy2J7HOXsoltzZzvrWDdhYIZDJobWutLEIFB/9JMStg",
    "obyQgkHcKZcOP1rXatm2yB535ZEqtZDJdUHvnoBwoCtxhEB1YexaWFPk7jhAVBBxw6qi/ZCNfIN",
    "W90bChVyhsmnnqSqoTNeFsjXFVVkbftiwblXLlh+0l6CiPgGX+TaloBO+V3ACvFgn6C3mEj1zH6",
    "jWBPDOTYRiICVt4rRwVUViQ+xYRfAs5iGwYQoJnnos8n2hRHRukkbmDoN09PS/ItYztCgwypD6X",
    "ztaRMAQqc+ZUjb8CUDSREo5V0pmXpGaYxWAlKVGhEc5zIa7Sxa2F4aisbCW5oCIbQBA1Zkmuhd1",
    "9SbkrKHCvMytY9LhRmWTdexNr4qQT2qR9BYGsg49DFPIsTlF1iZcvnEWo7YnrSBOVLlpaNJIMJH",
    "YuftVJ/ksKFuBbJWLRlEnt5F7PGrwAwa1ApsJQdYymVkzYKX05qCU/6WXZ3bRisnanmdGB94xEI",
    "rIZWV1u9qY9R1RLYbqJA22A99hW2I0IxpbEFnLSyGAimfTB52gfvrOiDX17og/fysf/hVTDBZpT",
    "56oqzQTbPw4V5bpfop2d9+DocWcKROW52fWsBX1vBSxauRb5b9b++Xy/w1fGvViNLdrpT4Nkp4H",
    "ZzW30Tf/f96z/9do/9fPLtH3fe/noTf22Jv5b9b3E/1N32K3fz8RuzXmmemUdLycu+e7LBD+4V/",
    "MCO1fufH5yHl97ky8lJv/+ZOd5/POwePMrm29sgb70g7+z+A9ufYeBmeUD6nSDf99gxNdN+3M5/",
    "sCDv7gZ73FrwJ0J+/WQ7/KuHZez1SsFedoxlmZ0nnOOOSfZxO6RW+HN1uR4cboef8vc26HW/oNf",
    "9rI+/ojZbw1UxMyzrV9X/XXHPC9//uPLrUKau1CviqvKRfxi3rX6u9fJ5f//hmnxGCu/vPcN6rM",
    "ujLn3Ts6bXL8Str3kzH38836s3IsaDFbuF/DE29OGqp/8R//7d0X6zPq5l7Zd/IP3ivSNB+QS3B",
    "9PtV367J+23ufkjCVSt3oP9G/Vk/h1/Q79yuxDfdjxmIkjG6p139z/Y36Y+/QkJyAy2",
    ""
  };

  static char newstr [1493] = "";
  newstr[0] = '\0';
  for (i = 0; i < 21; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c6_dynamics_lib(SimStruct *S)
{
  const char* newstr = sf_c6_dynamics_lib_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(2980998563U));
  ssSetChecksum1(S,(402049307U));
  ssSetChecksum2(S,(2856898153U));
  ssSetChecksum3(S,(2653687289U));
}

static void mdlRTW_c6_dynamics_lib(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c6_dynamics_lib(SimStruct *S)
{
  SFc6_dynamics_libInstanceStruct *chartInstance;
  chartInstance = (SFc6_dynamics_libInstanceStruct *)utMalloc(sizeof
    (SFc6_dynamics_libInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc6_dynamics_libInstanceStruct));
  chartInstance = new (chartInstance) SFc6_dynamics_libInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway = sf_opaque_gateway_c6_dynamics_lib;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c6_dynamics_lib;
  chartInstance->chartInfo.mdlStart = sf_opaque_mdl_start_c6_dynamics_lib;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c6_dynamics_lib;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c6_dynamics_lib;
  chartInstance->chartInfo.enableChart = sf_opaque_enable_c6_dynamics_lib;
  chartInstance->chartInfo.disableChart = sf_opaque_disable_c6_dynamics_lib;
  chartInstance->chartInfo.getSimState = sf_opaque_get_sim_state_c6_dynamics_lib;
  chartInstance->chartInfo.setSimState = sf_opaque_set_sim_state_c6_dynamics_lib;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c6_dynamics_lib;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c6_dynamics_lib;
  chartInstance->chartInfo.mdlSetWorkWidths = mdlSetWorkWidths_c6_dynamics_lib;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c6_dynamics_lib(chartInstance);
}

void c6_dynamics_lib_method_dispatcher(SimStruct *S, int_T method, void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c6_dynamics_lib(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c6_dynamics_lib(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c6_dynamics_lib(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c6_dynamics_lib_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
