/* Include files */

#include "dynamics_lib_sfun.h"
#include "c4_dynamics_lib.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c4_dynamics_lib(SFc4_dynamics_libInstanceStruct
  *chartInstance);
static void initialize_params_c4_dynamics_lib(SFc4_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_start_c4_dynamics_lib(SFc4_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_terminate_c4_dynamics_lib(SFc4_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_setup_runtime_resources_c4_dynamics_lib
  (SFc4_dynamics_libInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c4_dynamics_lib
  (SFc4_dynamics_libInstanceStruct *chartInstance);
static void enable_c4_dynamics_lib(SFc4_dynamics_libInstanceStruct
  *chartInstance);
static void disable_c4_dynamics_lib(SFc4_dynamics_libInstanceStruct
  *chartInstance);
static void sf_gateway_c4_dynamics_lib(SFc4_dynamics_libInstanceStruct
  *chartInstance);
static void ext_mode_exec_c4_dynamics_lib(SFc4_dynamics_libInstanceStruct
  *chartInstance);
static void c4_do_animation_call_c4_dynamics_lib(SFc4_dynamics_libInstanceStruct
  *chartInstance);
static const mxArray *get_sim_state_c4_dynamics_lib
  (SFc4_dynamics_libInstanceStruct *chartInstance);
static void set_sim_state_c4_dynamics_lib(SFc4_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c4_st);
static void initSimStructsc4_dynamics_lib(SFc4_dynamics_libInstanceStruct
  *chartInstance);
static void initSubchartIOPointersc4_dynamics_lib
  (SFc4_dynamics_libInstanceStruct *chartInstance);
static real_T c4_eML_blk_kernel(SFc4_dynamics_libInstanceStruct *chartInstance,
  real_T c4_b_rearARBStiffness, real_T c4_b_rearTrack, real_T
  c4_b_rearARBMotionRatio);
static real_T c4_emlrt_marshallIn(SFc4_dynamics_libInstanceStruct *chartInstance,
  const mxArray *c4_nullptr, const char_T *c4_identifier);
static real_T c4_b_emlrt_marshallIn(SFc4_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c4_u, const emlrtMsgIdentifier *c4_parentId);
static uint8_T c4_c_emlrt_marshallIn(SFc4_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c4_nullptr, const char_T *c4_identifier);
static uint8_T c4_d_emlrt_marshallIn(SFc4_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c4_u, const emlrtMsgIdentifier *c4_parentId);
static void init_dsm_address_info(SFc4_dynamics_libInstanceStruct *chartInstance);
static void init_simulink_io_address(SFc4_dynamics_libInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c4_dynamics_lib(SFc4_dynamics_libInstanceStruct
  *chartInstance)
{
  emlrtLicenseCheckR2022a(chartInstance->c4_fEmlrtCtx,
    "EMLRT:runTime:MexFunctionNeedsLicense", "distrib_computing_toolbox", 2);
  sf_is_first_init_cond(chartInstance->S);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  emlrtInitGPU(chartInstance->c4_fEmlrtCtx);
  hipGetLastError();
}

static void initialize_params_c4_dynamics_lib(SFc4_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void mdl_start_c4_dynamics_lib(SFc4_dynamics_libInstanceStruct
  *chartInstance)
{
  sim_mode_is_external(chartInstance->S);
}

static void mdl_terminate_c4_dynamics_lib(SFc4_dynamics_libInstanceStruct
  *chartInstance)
{
  hipError_t c4_errCode;
  c4_errCode = hipGetLastError();
  if (c4_errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(c4_errCode), (char_T *)
                       hipGetErrorName(c4_errCode), (char_T *)
                       hipGetErrorString(c4_errCode), (char_T *)
                       "SimGPUErrorChecks", chartInstance->c4_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c4_dynamics_lib
  (SFc4_dynamics_libInstanceStruct *chartInstance)
{
  sfSetAnimationVectors(chartInstance->S, chartInstance->c4_JITStateAnimation,
                        chartInstance->c4_JITTransitionAnimation);
}

static void mdl_cleanup_runtime_resources_c4_dynamics_lib
  (SFc4_dynamics_libInstanceStruct *chartInstance)
{
}

static void enable_c4_dynamics_lib(SFc4_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c4_dynamics_lib(SFc4_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void sf_gateway_c4_dynamics_lib(SFc4_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c4_JITTransitionAnimation[0] = 0U;
  *chartInstance->c4_antiRollRearARB = c4_eML_blk_kernel(chartInstance,
    *chartInstance->c4_rearARBStiffness, *chartInstance->c4_rearTrack,
    *chartInstance->c4_rearARBMotionRatio);
  c4_do_animation_call_c4_dynamics_lib(chartInstance);
}

static void ext_mode_exec_c4_dynamics_lib(SFc4_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void c4_do_animation_call_c4_dynamics_lib(SFc4_dynamics_libInstanceStruct
  *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static const mxArray *get_sim_state_c4_dynamics_lib
  (SFc4_dynamics_libInstanceStruct *chartInstance)
{
  const mxArray *c4_b_y = NULL;
  const mxArray *c4_c_y = NULL;
  const mxArray *c4_st;
  const mxArray *c4_y = NULL;
  c4_st = NULL;
  c4_st = NULL;
  c4_y = NULL;
  sf_mex_assign(&c4_y, sf_mex_createcellmatrix(2, 1), false);
  c4_b_y = NULL;
  sf_mex_assign(&c4_b_y, sf_mex_create("y", chartInstance->c4_antiRollRearARB, 0,
    0U, 0U, 0U, 0), false);
  sf_mex_setcell(c4_y, 0, c4_b_y);
  c4_c_y = NULL;
  sf_mex_assign(&c4_c_y, sf_mex_create("y",
    &chartInstance->c4_is_active_c4_dynamics_lib, 3, 0U, 0U, 0U, 0), false);
  sf_mex_setcell(c4_y, 1, c4_c_y);
  sf_mex_assign(&c4_st, c4_y, false);
  return c4_st;
}

static void set_sim_state_c4_dynamics_lib(SFc4_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c4_st)
{
  const mxArray *c4_u;
  c4_u = sf_mex_dup(c4_st);
  *chartInstance->c4_antiRollRearARB = c4_emlrt_marshallIn(chartInstance,
    sf_mex_dup(sf_mex_getcell(c4_u, 0)), "antiRollRearARB");
  chartInstance->c4_is_active_c4_dynamics_lib = c4_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c4_u, 1)),
     "is_active_c4_dynamics_lib");
  sf_mex_destroy(&c4_u);
  sf_mex_destroy(&c4_st);
}

static void initSimStructsc4_dynamics_lib(SFc4_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void initSubchartIOPointersc4_dynamics_lib
  (SFc4_dynamics_libInstanceStruct *chartInstance)
{
}

const mxArray *sf_c4_dynamics_lib_get_eml_resolved_functions_info()
{
  const mxArray *c4_nameCaptureInfo = NULL;
  c4_nameCaptureInfo = NULL;
  sf_mex_assign(&c4_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c4_nameCaptureInfo;
}

static real_T c4_eML_blk_kernel(SFc4_dynamics_libInstanceStruct *chartInstance,
  real_T c4_b_rearARBStiffness, real_T c4_b_rearTrack, real_T
  c4_b_rearARBMotionRatio)
{
  real_T c4_a;
  c4_a = c4_b_rearTrack / 1000.0;
  return c4_b_rearARBStiffness * 1000.0 * (c4_a * c4_a) * 1.5574077246549023 /
    (c4_b_rearARBMotionRatio * c4_b_rearARBMotionRatio);
}

static real_T c4_emlrt_marshallIn(SFc4_dynamics_libInstanceStruct *chartInstance,
  const mxArray *c4_nullptr, const char_T *c4_identifier)
{
  emlrtMsgIdentifier c4_thisId;
  real_T c4_y;
  c4_thisId.fIdentifier = const_cast<const char_T *>(c4_identifier);
  c4_thisId.fParent = NULL;
  c4_thisId.bParentIsCell = false;
  c4_y = c4_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c4_nullptr), &c4_thisId);
  sf_mex_destroy(&c4_nullptr);
  return c4_y;
}

static real_T c4_b_emlrt_marshallIn(SFc4_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c4_u, const emlrtMsgIdentifier *c4_parentId)
{
  real_T c4_d;
  real_T c4_y;
  sf_mex_import(c4_parentId, sf_mex_dup(c4_u), &c4_d, 1, 0, 0U, 0, 0U, 0);
  c4_y = c4_d;
  sf_mex_destroy(&c4_u);
  return c4_y;
}

static uint8_T c4_c_emlrt_marshallIn(SFc4_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c4_nullptr, const char_T *c4_identifier)
{
  emlrtMsgIdentifier c4_thisId;
  uint8_T c4_y;
  c4_thisId.fIdentifier = const_cast<const char_T *>(c4_identifier);
  c4_thisId.fParent = NULL;
  c4_thisId.bParentIsCell = false;
  c4_y = c4_d_emlrt_marshallIn(chartInstance, sf_mex_dup(c4_nullptr), &c4_thisId);
  sf_mex_destroy(&c4_nullptr);
  return c4_y;
}

static uint8_T c4_d_emlrt_marshallIn(SFc4_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c4_u, const emlrtMsgIdentifier *c4_parentId)
{
  uint8_T c4_b_u;
  uint8_T c4_y;
  sf_mex_import(c4_parentId, sf_mex_dup(c4_u), &c4_b_u, 1, 3, 0U, 0, 0U, 0);
  c4_y = c4_b_u;
  sf_mex_destroy(&c4_u);
  return c4_y;
}

static void init_dsm_address_info(SFc4_dynamics_libInstanceStruct *chartInstance)
{
}

static void init_simulink_io_address(SFc4_dynamics_libInstanceStruct
  *chartInstance)
{
  chartInstance->c4_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c4_rearARBStiffness = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c4_antiRollRearARB = (real_T *)ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c4_rearTrack = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c4_rearARBMotionRatio = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 2);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c4_dynamics_lib_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(3516175568U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(2841939300U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(3590592761U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(2804909940U);
}

mxArray *sf_c4_dynamics_lib_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c4_dynamics_lib_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("chartInfo");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c4_dynamics_lib_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c4_dynamics_lib(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiDmYGJgYAPRQMzEAAGsUD4jVIwRLs4CF1cA4pLKglSQeHFRsmcKkM5LzAXzE0s"
    "rPPPS8sHmWzAgzGfDYj4jkvmcUHEI+GBPmX4RB5B+ByT9LFj08yPpF4DyE/NKMoPyc3KCUhOLHI"
    "OcwP6AhRf57lFwoEw/xP4AAv6RRPMPiJ9ZHJ+YXJJZlhqfbBKfUgmMpszk4viczCQkc0EAADpoH"
    "hw="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c4_dynamics_lib_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sJciLkJhy6etiry8Gxy23lD";
}

static void sf_opaque_initialize_c4_dynamics_lib(void *chartInstanceVar)
{
  initialize_params_c4_dynamics_lib((SFc4_dynamics_libInstanceStruct*)
    chartInstanceVar);
  initialize_c4_dynamics_lib((SFc4_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c4_dynamics_lib(void *chartInstanceVar)
{
  enable_c4_dynamics_lib((SFc4_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c4_dynamics_lib(void *chartInstanceVar)
{
  disable_c4_dynamics_lib((SFc4_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c4_dynamics_lib(void *chartInstanceVar)
{
  sf_gateway_c4_dynamics_lib((SFc4_dynamics_libInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c4_dynamics_lib(SimStruct* S)
{
  return get_sim_state_c4_dynamics_lib((SFc4_dynamics_libInstanceStruct *)
    sf_get_chart_instance_ptr(S));     /* raw sim ctx */
}

static void sf_opaque_set_sim_state_c4_dynamics_lib(SimStruct* S, const mxArray *
  st)
{
  set_sim_state_c4_dynamics_lib((SFc4_dynamics_libInstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c4_dynamics_lib(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc4_dynamics_libInstanceStruct*) chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_dynamics_lib_optimization_info();
    }

    mdl_cleanup_runtime_resources_c4_dynamics_lib
      ((SFc4_dynamics_libInstanceStruct*) chartInstanceVar);
    ((SFc4_dynamics_libInstanceStruct*) chartInstanceVar)->
      ~SFc4_dynamics_libInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c4_dynamics_lib(void *chartInstanceVar)
{
  mdl_start_c4_dynamics_lib((SFc4_dynamics_libInstanceStruct*) chartInstanceVar);
  if (chartInstanceVar) {
    sf_reset_warnings_ChartRunTimeInfo(((SFc4_dynamics_libInstanceStruct*)
      chartInstanceVar)->S);
  }
}

static void sf_opaque_mdl_terminate_c4_dynamics_lib(void *chartInstanceVar)
{
  mdl_terminate_c4_dynamics_lib((SFc4_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c4_dynamics_lib(SimStruct *S)
{
  mdlProcessParamsCommon(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c4_dynamics_lib((SFc4_dynamics_libInstanceStruct*)
      sf_get_chart_instance_ptr(S));
    initSubchartIOPointersc4_dynamics_lib((SFc4_dynamics_libInstanceStruct*)
      sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c4_dynamics_lib_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [21] = {
    "eNrtWM2P20QUn6TLqkVl2wMSVCBRceKCVLaAioTo7uZjSbXLRutskbisJvZLPMp47M5HEvfP4c6",
    "dvwDxJ3Dkwp0jBw68cZw0dULicSRoEZYcZ2z/5vfem/c1JrXOOcHjAM9H7xKyj9ebeNbJ7HgjH9",
    "eWztn9PfJlPv75TUL8OIAhCM8MBmxK3A5hoi6VNFLE/RA0gktQMTeaxaIjBnF5LBMDkCB8nCCJp",
    "XbiVSwynIlR2wjfMqtvQ+aHXhgbHpzghDS4EDz9O97E6C4yNpkEX7cBAh3K2AzDNqfDzVaQetII",
    "wR8pEznbSoH2TGJVVeeGa5ZwaE3B7wilKVpBbdHX01RDQ0/djGz1Vd4cHUcJZ1SUt3VIlQcJeoe",
    "GqyTA3wuj0XoleZGvzwTVsWSUtyLeCKksi+1ylPMc3Zo721lHugl9MxwyMbTWlSYCgfqjn5Sw1a",
    "ARj0HSIVwIN17fateaZgu88MuSWM0ieErlsY9+oCBwi0F0YOVRXFro4TROWBC0z6GjepKNcY1c8",
    "0bHhlKlvGGimSepStiMtzXGVVUVedu+aFDOlRu2FydnMAae8TepphWwM34HsFIs6MXoHTZyHaPf",
    "CPbMQI5txCJg5b1yXEBlReIbTPgl4CyyYQABmnkh+mKibXFklI6jBoZ+8+ysJN8qtiM0yAH1oXS",
    "+lpQpQIEzv3LkDZiygYRotJLOtCw9wywGK0GJGhjRnMRyhDZ2LQwvbGUjwQ0NwRCaoCFLci307q",
    "eUm5IyRwqzsnWPK4VZ1o0XsTZ+KoF96ocQ2BrIOJxjnsUJyi6xsuXzGLUdM502QfmSJWUjyWBCx",
    "+JnrdRLE7gSIxFPRFvGkZd3MRv8CgCzBpUCS9kJllKZtlH4clJLeNbLsrtrw2DtTDWnfesbpyCw",
    "GlpdbfWmPkZVS2C7iQLtgvXYc2xHhGJKYwuYtrIYCLI++Ii86IP31vTBd5b64Lv52P/0OkixGWW",
    "+uuasn83zYGme2yX66XkfvglHVnBkgZtfP1rC19bwkqVrke9W/eX3bxT46ninVqut2OmtAs9eAb",
    "ef2+qLP3/47b2ffn3+vXh8+uGdg942/toKfy37b3E/1t32Kwf5+P15r7TIzOOV5GXf/XqLH7xT8",
    "AM7Vk98djZ6Eqafg2YyfXQ6TQ8f8mY2380t8tYL8s7v37f9GQZulgek3wnyfY8dUzPrx7P925K8",
    "+1vscWvJnwj5/fFu+LePqsYNlmV2GXOOOyZ5fHnykh9Xl+f+0W74GX93iz73Cvrcy/r3a2qzNFw",
    "XM8JqnFb1e1fc68L3P678OpSpJzcq4qrykX8Yt6t+rnXydX//wYZ8Rgrv332F9diUR136pVdNr1",
    "+IWz/zQT7+arFHb4SMB2t2CfljbOQH657+R/z7D0f7zfu3lrVf/mH0u4fHgvIUtwWzbVd+uyvtN",
    "7nFIwlUrd97/Rv1ZH7d1n/dLsS3HU+YCOKJ+viTw88Od6lPfwGWLgrW",
    ""
  };

  static char newstr [1481] = "";
  newstr[0] = '\0';
  for (i = 0; i < 21; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c4_dynamics_lib(SimStruct *S)
{
  const char* newstr = sf_c4_dynamics_lib_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(3853384761U));
  ssSetChecksum1(S,(2061551131U));
  ssSetChecksum2(S,(1195339426U));
  ssSetChecksum3(S,(1410207522U));
}

static void mdlRTW_c4_dynamics_lib(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c4_dynamics_lib(SimStruct *S)
{
  SFc4_dynamics_libInstanceStruct *chartInstance;
  chartInstance = (SFc4_dynamics_libInstanceStruct *)utMalloc(sizeof
    (SFc4_dynamics_libInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc4_dynamics_libInstanceStruct));
  chartInstance = new (chartInstance) SFc4_dynamics_libInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway = sf_opaque_gateway_c4_dynamics_lib;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c4_dynamics_lib;
  chartInstance->chartInfo.mdlStart = sf_opaque_mdl_start_c4_dynamics_lib;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c4_dynamics_lib;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c4_dynamics_lib;
  chartInstance->chartInfo.enableChart = sf_opaque_enable_c4_dynamics_lib;
  chartInstance->chartInfo.disableChart = sf_opaque_disable_c4_dynamics_lib;
  chartInstance->chartInfo.getSimState = sf_opaque_get_sim_state_c4_dynamics_lib;
  chartInstance->chartInfo.setSimState = sf_opaque_set_sim_state_c4_dynamics_lib;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c4_dynamics_lib;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c4_dynamics_lib;
  chartInstance->chartInfo.mdlSetWorkWidths = mdlSetWorkWidths_c4_dynamics_lib;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c4_dynamics_lib(chartInstance);
}

void c4_dynamics_lib_method_dispatcher(SimStruct *S, int_T method, void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c4_dynamics_lib(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c4_dynamics_lib(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c4_dynamics_lib(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c4_dynamics_lib_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
