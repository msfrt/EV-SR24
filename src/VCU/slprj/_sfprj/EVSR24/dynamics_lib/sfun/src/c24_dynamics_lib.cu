/* Include files */

#include "dynamics_lib_sfun.h"
#include "c24_dynamics_lib.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c24_dynamics_lib(SFc24_dynamics_libInstanceStruct
  *chartInstance);
static void initialize_params_c24_dynamics_lib(SFc24_dynamics_libInstanceStruct *
  chartInstance);
static void mdl_start_c24_dynamics_lib(SFc24_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_terminate_c24_dynamics_lib(SFc24_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_setup_runtime_resources_c24_dynamics_lib
  (SFc24_dynamics_libInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c24_dynamics_lib
  (SFc24_dynamics_libInstanceStruct *chartInstance);
static void enable_c24_dynamics_lib(SFc24_dynamics_libInstanceStruct
  *chartInstance);
static void disable_c24_dynamics_lib(SFc24_dynamics_libInstanceStruct
  *chartInstance);
static void sf_gateway_c24_dynamics_lib(SFc24_dynamics_libInstanceStruct
  *chartInstance);
static void ext_mode_exec_c24_dynamics_lib(SFc24_dynamics_libInstanceStruct
  *chartInstance);
static void c24_do_animation_call_c24_dynamics_lib
  (SFc24_dynamics_libInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c24_dynamics_lib
  (SFc24_dynamics_libInstanceStruct *chartInstance);
static void set_sim_state_c24_dynamics_lib(SFc24_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c24_st);
static void initSimStructsc24_dynamics_lib(SFc24_dynamics_libInstanceStruct
  *chartInstance);
static void initSubchartIOPointersc24_dynamics_lib
  (SFc24_dynamics_libInstanceStruct *chartInstance);
static real32_T c24_eML_blk_kernel(SFc24_dynamics_libInstanceStruct
  *chartInstance, real_T c24_b_totalMass, real32_T c24_b_totalMassDistribution,
  real_T c24_b_totalDownForce, real32_T c24_b_downForceDistribution, real32_T
  c24_b_totalRearWT, real_T c24_b_gravity);
static real32_T c24_emlrt_marshallIn(SFc24_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c24_nullptr, const char_T *c24_identifier);
static real32_T c24_b_emlrt_marshallIn(SFc24_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c24_u, const emlrtMsgIdentifier *c24_parentId);
static uint8_T c24_c_emlrt_marshallIn(SFc24_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c24_nullptr, const char_T *c24_identifier);
static uint8_T c24_d_emlrt_marshallIn(SFc24_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c24_u, const emlrtMsgIdentifier *c24_parentId);
static void init_dsm_address_info(SFc24_dynamics_libInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc24_dynamics_libInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c24_dynamics_lib(SFc24_dynamics_libInstanceStruct
  *chartInstance)
{
  emlrtLicenseCheckR2022a(chartInstance->c24_fEmlrtCtx,
    "EMLRT:runTime:MexFunctionNeedsLicense", "distrib_computing_toolbox", 2);
  sf_is_first_init_cond(chartInstance->S);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  emlrtInitGPU(chartInstance->c24_fEmlrtCtx);
  hipGetLastError();
}

static void initialize_params_c24_dynamics_lib(SFc24_dynamics_libInstanceStruct *
  chartInstance)
{
}

static void mdl_start_c24_dynamics_lib(SFc24_dynamics_libInstanceStruct
  *chartInstance)
{
  sim_mode_is_external(chartInstance->S);
}

static void mdl_terminate_c24_dynamics_lib(SFc24_dynamics_libInstanceStruct
  *chartInstance)
{
  hipError_t c24_errCode;
  c24_errCode = hipGetLastError();
  if (c24_errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(c24_errCode), (char_T *)
                       hipGetErrorName(c24_errCode), (char_T *)
                       hipGetErrorString(c24_errCode), (char_T *)
                       "SimGPUErrorChecks", chartInstance->c24_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c24_dynamics_lib
  (SFc24_dynamics_libInstanceStruct *chartInstance)
{
  sfSetAnimationVectors(chartInstance->S, chartInstance->c24_JITStateAnimation,
                        chartInstance->c24_JITTransitionAnimation);
}

static void mdl_cleanup_runtime_resources_c24_dynamics_lib
  (SFc24_dynamics_libInstanceStruct *chartInstance)
{
}

static void enable_c24_dynamics_lib(SFc24_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c24_dynamics_lib(SFc24_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void sf_gateway_c24_dynamics_lib(SFc24_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c24_JITTransitionAnimation[0] = 0U;
  *chartInstance->c24_dynamicLoadRR = c24_eML_blk_kernel(chartInstance,
    *chartInstance->c24_totalMass, *chartInstance->c24_totalMassDistribution,
    *chartInstance->c24_totalDownForce,
    *chartInstance->c24_downForceDistribution, *chartInstance->c24_totalRearWT, *
    chartInstance->c24_gravity);
  c24_do_animation_call_c24_dynamics_lib(chartInstance);
}

static void ext_mode_exec_c24_dynamics_lib(SFc24_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void c24_do_animation_call_c24_dynamics_lib
  (SFc24_dynamics_libInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static const mxArray *get_sim_state_c24_dynamics_lib
  (SFc24_dynamics_libInstanceStruct *chartInstance)
{
  const mxArray *c24_b_y = NULL;
  const mxArray *c24_c_y = NULL;
  const mxArray *c24_st;
  const mxArray *c24_y = NULL;
  c24_st = NULL;
  c24_st = NULL;
  c24_y = NULL;
  sf_mex_assign(&c24_y, sf_mex_createcellmatrix(2, 1), false);
  c24_b_y = NULL;
  sf_mex_assign(&c24_b_y, sf_mex_create("y", chartInstance->c24_dynamicLoadRR, 1,
    0U, 0U, 0U, 0), false);
  sf_mex_setcell(c24_y, 0, c24_b_y);
  c24_c_y = NULL;
  sf_mex_assign(&c24_c_y, sf_mex_create("y",
    &chartInstance->c24_is_active_c24_dynamics_lib, 3, 0U, 0U, 0U, 0), false);
  sf_mex_setcell(c24_y, 1, c24_c_y);
  sf_mex_assign(&c24_st, c24_y, false);
  return c24_st;
}

static void set_sim_state_c24_dynamics_lib(SFc24_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c24_st)
{
  const mxArray *c24_u;
  c24_u = sf_mex_dup(c24_st);
  *chartInstance->c24_dynamicLoadRR = c24_emlrt_marshallIn(chartInstance,
    sf_mex_dup(sf_mex_getcell(c24_u, 0)), "dynamicLoadRR");
  chartInstance->c24_is_active_c24_dynamics_lib = c24_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c24_u, 1)),
     "is_active_c24_dynamics_lib");
  sf_mex_destroy(&c24_u);
  sf_mex_destroy(&c24_st);
}

static void initSimStructsc24_dynamics_lib(SFc24_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void initSubchartIOPointersc24_dynamics_lib
  (SFc24_dynamics_libInstanceStruct *chartInstance)
{
}

const mxArray *sf_c24_dynamics_lib_get_eml_resolved_functions_info()
{
  const mxArray *c24_nameCaptureInfo = NULL;
  c24_nameCaptureInfo = NULL;
  sf_mex_assign(&c24_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c24_nameCaptureInfo;
}

static real32_T c24_eML_blk_kernel(SFc24_dynamics_libInstanceStruct
  *chartInstance, real_T c24_b_totalMass, real32_T c24_b_totalMassDistribution,
  real_T c24_b_totalDownForce, real32_T c24_b_downForceDistribution, real32_T
  c24_b_totalRearWT, real_T c24_b_gravity)
{
  return ((static_cast<real32_T>(c24_b_totalMass) * (100.0F -
            c24_b_totalMassDistribution) / 100.0F * 0.5F + static_cast<real32_T>
           (c24_b_totalDownForce) * (100.0F - c24_b_downForceDistribution) /
           100.0F / 2.0F) + c24_b_totalRearWT) * static_cast<real32_T>
    (c24_b_gravity);
}

static real32_T c24_emlrt_marshallIn(SFc24_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c24_nullptr, const char_T *c24_identifier)
{
  emlrtMsgIdentifier c24_thisId;
  real32_T c24_y;
  c24_thisId.fIdentifier = const_cast<const char_T *>(c24_identifier);
  c24_thisId.fParent = NULL;
  c24_thisId.bParentIsCell = false;
  c24_y = c24_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c24_nullptr),
    &c24_thisId);
  sf_mex_destroy(&c24_nullptr);
  return c24_y;
}

static real32_T c24_b_emlrt_marshallIn(SFc24_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c24_u, const emlrtMsgIdentifier *c24_parentId)
{
  real32_T c24_f;
  real32_T c24_y;
  sf_mex_import(c24_parentId, sf_mex_dup(c24_u), &c24_f, 0, 1, 0U, 0, 0U, 0);
  c24_y = c24_f;
  sf_mex_destroy(&c24_u);
  return c24_y;
}

static uint8_T c24_c_emlrt_marshallIn(SFc24_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c24_nullptr, const char_T *c24_identifier)
{
  emlrtMsgIdentifier c24_thisId;
  uint8_T c24_y;
  c24_thisId.fIdentifier = const_cast<const char_T *>(c24_identifier);
  c24_thisId.fParent = NULL;
  c24_thisId.bParentIsCell = false;
  c24_y = c24_d_emlrt_marshallIn(chartInstance, sf_mex_dup(c24_nullptr),
    &c24_thisId);
  sf_mex_destroy(&c24_nullptr);
  return c24_y;
}

static uint8_T c24_d_emlrt_marshallIn(SFc24_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c24_u, const emlrtMsgIdentifier *c24_parentId)
{
  uint8_T c24_b_u;
  uint8_T c24_y;
  sf_mex_import(c24_parentId, sf_mex_dup(c24_u), &c24_b_u, 1, 3, 0U, 0, 0U, 0);
  c24_y = c24_b_u;
  sf_mex_destroy(&c24_u);
  return c24_y;
}

static void init_dsm_address_info(SFc24_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc24_dynamics_libInstanceStruct
  *chartInstance)
{
  chartInstance->c24_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c24_dynamicLoadRR = (real32_T *)ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c24_totalMass = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c24_totalMassDistribution = (real32_T *)
    ssGetInputPortSignal_wrapper(chartInstance->S, 1);
  chartInstance->c24_totalDownForce = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 2);
  chartInstance->c24_downForceDistribution = (real32_T *)
    ssGetInputPortSignal_wrapper(chartInstance->S, 3);
  chartInstance->c24_totalRearWT = (real32_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 4);
  chartInstance->c24_gravity = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 5);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c24_dynamics_lib_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(4279468341U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(197420353U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(2965155608U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(3645350442U);
}

mxArray *sf_c24_dynamics_lib_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c24_dynamics_lib_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("chartInfo");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c24_dynamics_lib_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c24_dynamics_lib(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiDmYGJgYAPRQMzEAAGsUD4jVIwRLs4CF1cA4pLKglSQeHFRsmcKkM5LzAXzE0s"
    "rPPPS8sHmWzAgzGfDYj4jkvmcUHEI+GBPmX4RB5B+ByT9LFj08yLpF4DyUyqBPslM9slPTAkKgo"
    "YTLLzId4+CA2X6IfYHEPCPFJp/QPzM4vjE5JLMstT4ZCOTeKjviuNzMpMQ5oIAAKkLHbE="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c24_dynamics_lib_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sj7vP3DMWbyUIuJfHF7tAf";
}

static void sf_opaque_initialize_c24_dynamics_lib(void *chartInstanceVar)
{
  initialize_params_c24_dynamics_lib((SFc24_dynamics_libInstanceStruct*)
    chartInstanceVar);
  initialize_c24_dynamics_lib((SFc24_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c24_dynamics_lib(void *chartInstanceVar)
{
  enable_c24_dynamics_lib((SFc24_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c24_dynamics_lib(void *chartInstanceVar)
{
  disable_c24_dynamics_lib((SFc24_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c24_dynamics_lib(void *chartInstanceVar)
{
  sf_gateway_c24_dynamics_lib((SFc24_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c24_dynamics_lib(SimStruct* S)
{
  return get_sim_state_c24_dynamics_lib((SFc24_dynamics_libInstanceStruct *)
    sf_get_chart_instance_ptr(S));     /* raw sim ctx */
}

static void sf_opaque_set_sim_state_c24_dynamics_lib(SimStruct* S, const mxArray
  *st)
{
  set_sim_state_c24_dynamics_lib((SFc24_dynamics_libInstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c24_dynamics_lib(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc24_dynamics_libInstanceStruct*) chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_dynamics_lib_optimization_info();
    }

    mdl_cleanup_runtime_resources_c24_dynamics_lib
      ((SFc24_dynamics_libInstanceStruct*) chartInstanceVar);
    ((SFc24_dynamics_libInstanceStruct*) chartInstanceVar)->
      ~SFc24_dynamics_libInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c24_dynamics_lib(void *chartInstanceVar)
{
  mdl_start_c24_dynamics_lib((SFc24_dynamics_libInstanceStruct*)
    chartInstanceVar);
  if (chartInstanceVar) {
    sf_reset_warnings_ChartRunTimeInfo(((SFc24_dynamics_libInstanceStruct*)
      chartInstanceVar)->S);
  }
}

static void sf_opaque_mdl_terminate_c24_dynamics_lib(void *chartInstanceVar)
{
  mdl_terminate_c24_dynamics_lib((SFc24_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c24_dynamics_lib(SimStruct *S)
{
  mdlProcessParamsCommon(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c24_dynamics_lib((SFc24_dynamics_libInstanceStruct*)
      sf_get_chart_instance_ptr(S));
    initSubchartIOPointersc24_dynamics_lib((SFc24_dynamics_libInstanceStruct*)
      sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c24_dynamics_lib_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [21] = {
    "eNrtWM2P20QUn6yWqEVttRKooAqJHrkglS1QhBDsNh80aMNGm2wrcVlN7Od42PGMOx/Jhn8BiX+",
    "GS/8ADly5ceHcK0eOvHGcNDgh8TgStAhLXu/Y85v33m/e14TUOl2C1y28j94mpI7Pa3jvkdn1Wj",
    "6uLd2z9/vks3z88+uEBDKEEYi+jSJ2RfwuYZMeVTTRxP8SNIEz0JJbw6ToiEiWxzIRgQIR4AKpV",
    "MZLrmaJ5Uxctq0InGT9JGZB3I+l5eFDXJCGp4JP/05uak0PJTaZgsC0AUITK2lHcZvT0WYWlJk0",
    "YggutU28udJg+jZ1puqu5YalHFpXEHSENhRZ0Fvs7RtqoGGu/Eh29ur+HC2TlDMqynMdU92HFL3",
    "DwHka4t9Ta5C9knJR3pAJaqRilLcS3oipKovtcdSzi27NvXk2iWnC0I5GTIwcu8omINB+9JMSXE",
    "UNOQZFR3Aq/OQGzrrWVbbBC78siTUsgcdUHQfoBxpCvxhEB9Z9ilsLA1zGCwuCDjl09ECxMe6Rb",
    "97ouFCqlDdsMvMkXQmbyW2NcVd1RbntQDQo59oPO5DpCYyBZ/Kb1NAK2Jl8D7DWLBxI9A4XuZ7R",
    "bwV7aiHHNqQIWXmvHBdQWZH4GhN+CThLXBhAiDQvVF8stC2OrDYyaWDoN09OSspbxXaEARXRAEr",
    "na0WZBlQ48ytPuSHTLpAQjSyZzMrSK8xisBKU6MiK5kSqS+TYtzC84MpFgh8awhE0wUCW5Fro3Y",
    "8ptyV1TjRmZece5xqzrJ9cxLr4qQQOaBBD6Gog49DFPIsLlN1i7crnMVo7ZmbaBB0olpaNJIsJH",
    "YufY2kwTeFcXAo5EW0lk37exWzwKwDMGlQJLGUPsZSqaRuVL6e1gqeDLLv7NgyOZ2o4HTrf+BIE",
    "VkNnq6veNMCoaglsN1GhXbB99h22I0IzbbAFnLayGAhnfTB50Qfvr+mDD5b64IP8Dg4/vAin2I2",
    "yQF9wNnTr3Fta50aJfnreh2/CkRUcWeDmz/eW8LU1csnS083/ZGn+9b2/zq8X5O3N32UXWeHrZk",
    "HefgFfz/n69OOf7I+3n6U//PaG/OX758+Ldhf1qK3oUcv+d7hne37nllv5+J15z7TI0OOVJObmP",
    "triD7cL/uDG+tsH4979ZvfJcHresV9Fj9oPzHE0W+/aFn33CvrO3991fRoGcJYPVNAJ8/OPG1M7",
    "68uL+1nfwsf1Jb8i5PcvdsO/eVQmfm4W+HLjPHZOJA3Pzgr+XF2fu0e74Wfye1vsuVOw507Wx19",
    "Ql63hopgZVuO1qt/74l4Vef/jyu9DmbpSr4irKo/8w7hd7fOtl6/6/Hsb8hkpzD94ie3YlEd9+q",
    "aXza5fiV8/824+/nxxVm/EjIdrTgv5Z2zoo3Vf/yP+/Ycnf2/l45bjL/+B9Jv7x4LyKR4PZsev/",
    "HVPud/mFp8UUL3+DPZv1JPF7/hb+pUbhfh24wkToZzo9z84/Ohwl/r0J38cC2I=",
    ""
  };

  static char newstr [1489] = "";
  newstr[0] = '\0';
  for (i = 0; i < 21; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c24_dynamics_lib(SimStruct *S)
{
  const char* newstr = sf_c24_dynamics_lib_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(1975793210U));
  ssSetChecksum1(S,(1891112627U));
  ssSetChecksum2(S,(1863572878U));
  ssSetChecksum3(S,(3890711505U));
}

static void mdlRTW_c24_dynamics_lib(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c24_dynamics_lib(SimStruct *S)
{
  SFc24_dynamics_libInstanceStruct *chartInstance;
  chartInstance = (SFc24_dynamics_libInstanceStruct *)utMalloc(sizeof
    (SFc24_dynamics_libInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc24_dynamics_libInstanceStruct));
  chartInstance = new (chartInstance) SFc24_dynamics_libInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway = sf_opaque_gateway_c24_dynamics_lib;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c24_dynamics_lib;
  chartInstance->chartInfo.mdlStart = sf_opaque_mdl_start_c24_dynamics_lib;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c24_dynamics_lib;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c24_dynamics_lib;
  chartInstance->chartInfo.enableChart = sf_opaque_enable_c24_dynamics_lib;
  chartInstance->chartInfo.disableChart = sf_opaque_disable_c24_dynamics_lib;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c24_dynamics_lib;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c24_dynamics_lib;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c24_dynamics_lib;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c24_dynamics_lib;
  chartInstance->chartInfo.mdlSetWorkWidths = mdlSetWorkWidths_c24_dynamics_lib;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c24_dynamics_lib(chartInstance);
}

void c24_dynamics_lib_method_dispatcher(SimStruct *S, int_T method, void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c24_dynamics_lib(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c24_dynamics_lib(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c24_dynamics_lib(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c24_dynamics_lib_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
