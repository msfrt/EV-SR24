/* Include files */

#include "dynamics_lib_sfun.h"
#include "c16_dynamics_lib.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c16_dynamics_lib(SFc16_dynamics_libInstanceStruct
  *chartInstance);
static void initialize_params_c16_dynamics_lib(SFc16_dynamics_libInstanceStruct *
  chartInstance);
static void mdl_start_c16_dynamics_lib(SFc16_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_terminate_c16_dynamics_lib(SFc16_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_setup_runtime_resources_c16_dynamics_lib
  (SFc16_dynamics_libInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c16_dynamics_lib
  (SFc16_dynamics_libInstanceStruct *chartInstance);
static void enable_c16_dynamics_lib(SFc16_dynamics_libInstanceStruct
  *chartInstance);
static void disable_c16_dynamics_lib(SFc16_dynamics_libInstanceStruct
  *chartInstance);
static void sf_gateway_c16_dynamics_lib(SFc16_dynamics_libInstanceStruct
  *chartInstance);
static void ext_mode_exec_c16_dynamics_lib(SFc16_dynamics_libInstanceStruct
  *chartInstance);
static void c16_do_animation_call_c16_dynamics_lib
  (SFc16_dynamics_libInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c16_dynamics_lib
  (SFc16_dynamics_libInstanceStruct *chartInstance);
static void set_sim_state_c16_dynamics_lib(SFc16_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c16_st);
static void initSimStructsc16_dynamics_lib(SFc16_dynamics_libInstanceStruct
  *chartInstance);
static void initSubchartIOPointersc16_dynamics_lib
  (SFc16_dynamics_libInstanceStruct *chartInstance);
static real32_T c16_eML_blk_kernel(SFc16_dynamics_libInstanceStruct
  *chartInstance, real32_T c16_b_antiRollFrontAxleAndTire, real32_T
  c16_b_antiRollRearAxleAndTire);
static real32_T c16_emlrt_marshallIn(SFc16_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c16_nullptr, const char_T *c16_identifier);
static real32_T c16_b_emlrt_marshallIn(SFc16_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c16_u, const emlrtMsgIdentifier *c16_parentId);
static uint8_T c16_c_emlrt_marshallIn(SFc16_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c16_nullptr, const char_T *c16_identifier);
static uint8_T c16_d_emlrt_marshallIn(SFc16_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c16_u, const emlrtMsgIdentifier *c16_parentId);
static void init_dsm_address_info(SFc16_dynamics_libInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc16_dynamics_libInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c16_dynamics_lib(SFc16_dynamics_libInstanceStruct
  *chartInstance)
{
  emlrtLicenseCheckR2022a(chartInstance->c16_fEmlrtCtx,
    "EMLRT:runTime:MexFunctionNeedsLicense", "distrib_computing_toolbox", 2);
  sf_is_first_init_cond(chartInstance->S);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  emlrtInitGPU(chartInstance->c16_fEmlrtCtx);
  hipGetLastError();
}

static void initialize_params_c16_dynamics_lib(SFc16_dynamics_libInstanceStruct *
  chartInstance)
{
}

static void mdl_start_c16_dynamics_lib(SFc16_dynamics_libInstanceStruct
  *chartInstance)
{
  sim_mode_is_external(chartInstance->S);
}

static void mdl_terminate_c16_dynamics_lib(SFc16_dynamics_libInstanceStruct
  *chartInstance)
{
  hipError_t c16_errCode;
  c16_errCode = hipGetLastError();
  if (c16_errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(c16_errCode), (char_T *)
                       hipGetErrorName(c16_errCode), (char_T *)
                       hipGetErrorString(c16_errCode), (char_T *)
                       "SimGPUErrorChecks", chartInstance->c16_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c16_dynamics_lib
  (SFc16_dynamics_libInstanceStruct *chartInstance)
{
  sfSetAnimationVectors(chartInstance->S, chartInstance->c16_JITStateAnimation,
                        chartInstance->c16_JITTransitionAnimation);
}

static void mdl_cleanup_runtime_resources_c16_dynamics_lib
  (SFc16_dynamics_libInstanceStruct *chartInstance)
{
}

static void enable_c16_dynamics_lib(SFc16_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c16_dynamics_lib(SFc16_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void sf_gateway_c16_dynamics_lib(SFc16_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c16_JITTransitionAnimation[0] = 0U;
  *chartInstance->c16_antiRollTotal = c16_eML_blk_kernel(chartInstance,
    *chartInstance->c16_antiRollFrontAxleAndTire,
    *chartInstance->c16_antiRollRearAxleAndTire);
  c16_do_animation_call_c16_dynamics_lib(chartInstance);
}

static void ext_mode_exec_c16_dynamics_lib(SFc16_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void c16_do_animation_call_c16_dynamics_lib
  (SFc16_dynamics_libInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static const mxArray *get_sim_state_c16_dynamics_lib
  (SFc16_dynamics_libInstanceStruct *chartInstance)
{
  const mxArray *c16_b_y = NULL;
  const mxArray *c16_c_y = NULL;
  const mxArray *c16_st;
  const mxArray *c16_y = NULL;
  c16_st = NULL;
  c16_st = NULL;
  c16_y = NULL;
  sf_mex_assign(&c16_y, sf_mex_createcellmatrix(2, 1), false);
  c16_b_y = NULL;
  sf_mex_assign(&c16_b_y, sf_mex_create("y", chartInstance->c16_antiRollTotal, 1,
    0U, 0U, 0U, 0), false);
  sf_mex_setcell(c16_y, 0, c16_b_y);
  c16_c_y = NULL;
  sf_mex_assign(&c16_c_y, sf_mex_create("y",
    &chartInstance->c16_is_active_c16_dynamics_lib, 3, 0U, 0U, 0U, 0), false);
  sf_mex_setcell(c16_y, 1, c16_c_y);
  sf_mex_assign(&c16_st, c16_y, false);
  return c16_st;
}

static void set_sim_state_c16_dynamics_lib(SFc16_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c16_st)
{
  const mxArray *c16_u;
  c16_u = sf_mex_dup(c16_st);
  *chartInstance->c16_antiRollTotal = c16_emlrt_marshallIn(chartInstance,
    sf_mex_dup(sf_mex_getcell(c16_u, 0)), "antiRollTotal");
  chartInstance->c16_is_active_c16_dynamics_lib = c16_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c16_u, 1)),
     "is_active_c16_dynamics_lib");
  sf_mex_destroy(&c16_u);
  sf_mex_destroy(&c16_st);
}

static void initSimStructsc16_dynamics_lib(SFc16_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void initSubchartIOPointersc16_dynamics_lib
  (SFc16_dynamics_libInstanceStruct *chartInstance)
{
}

const mxArray *sf_c16_dynamics_lib_get_eml_resolved_functions_info()
{
  const mxArray *c16_nameCaptureInfo = NULL;
  c16_nameCaptureInfo = NULL;
  sf_mex_assign(&c16_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c16_nameCaptureInfo;
}

static real32_T c16_eML_blk_kernel(SFc16_dynamics_libInstanceStruct
  *chartInstance, real32_T c16_b_antiRollFrontAxleAndTire, real32_T
  c16_b_antiRollRearAxleAndTire)
{
  return c16_b_antiRollFrontAxleAndTire + c16_b_antiRollRearAxleAndTire;
}

static real32_T c16_emlrt_marshallIn(SFc16_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c16_nullptr, const char_T *c16_identifier)
{
  emlrtMsgIdentifier c16_thisId;
  real32_T c16_y;
  c16_thisId.fIdentifier = const_cast<const char_T *>(c16_identifier);
  c16_thisId.fParent = NULL;
  c16_thisId.bParentIsCell = false;
  c16_y = c16_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c16_nullptr),
    &c16_thisId);
  sf_mex_destroy(&c16_nullptr);
  return c16_y;
}

static real32_T c16_b_emlrt_marshallIn(SFc16_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c16_u, const emlrtMsgIdentifier *c16_parentId)
{
  real32_T c16_f;
  real32_T c16_y;
  sf_mex_import(c16_parentId, sf_mex_dup(c16_u), &c16_f, 0, 1, 0U, 0, 0U, 0);
  c16_y = c16_f;
  sf_mex_destroy(&c16_u);
  return c16_y;
}

static uint8_T c16_c_emlrt_marshallIn(SFc16_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c16_nullptr, const char_T *c16_identifier)
{
  emlrtMsgIdentifier c16_thisId;
  uint8_T c16_y;
  c16_thisId.fIdentifier = const_cast<const char_T *>(c16_identifier);
  c16_thisId.fParent = NULL;
  c16_thisId.bParentIsCell = false;
  c16_y = c16_d_emlrt_marshallIn(chartInstance, sf_mex_dup(c16_nullptr),
    &c16_thisId);
  sf_mex_destroy(&c16_nullptr);
  return c16_y;
}

static uint8_T c16_d_emlrt_marshallIn(SFc16_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c16_u, const emlrtMsgIdentifier *c16_parentId)
{
  uint8_T c16_b_u;
  uint8_T c16_y;
  sf_mex_import(c16_parentId, sf_mex_dup(c16_u), &c16_b_u, 1, 3, 0U, 0, 0U, 0);
  c16_y = c16_b_u;
  sf_mex_destroy(&c16_u);
  return c16_y;
}

static void init_dsm_address_info(SFc16_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc16_dynamics_libInstanceStruct
  *chartInstance)
{
  chartInstance->c16_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c16_antiRollFrontAxleAndTire = (real32_T *)
    ssGetInputPortSignal_wrapper(chartInstance->S, 0);
  chartInstance->c16_antiRollTotal = (real32_T *)ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c16_antiRollRearAxleAndTire = (real32_T *)
    ssGetInputPortSignal_wrapper(chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c16_dynamics_lib_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(174450961U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(3519221386U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(1871786123U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(1137890561U);
}

mxArray *sf_c16_dynamics_lib_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c16_dynamics_lib_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("chartInfo");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c16_dynamics_lib_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c16_dynamics_lib(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiDmYGJgYAPRQMzEAAGsUD4jVIwRLs4CF1cA4pLKglSQeHFRsmcKkM5LzAXzE0s"
    "rPPPS8sHmWzAgzGfDYj4jkvmcUHEI+GBPmX4RB5B+ByT9LFj08yLpF4DyE/NKMoPyc3JC8ksSc6"
    "DhBAsv8t2j4ECZfoj9AQT8I4XmHxA/szg+Mbkksyw1PtnQLD6lEhhPmcnF8TmZSQhzQQAA5CId8"
    "g=="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c16_dynamics_lib_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sDJSVGoPns7QjX5GxCg1f5C";
}

static void sf_opaque_initialize_c16_dynamics_lib(void *chartInstanceVar)
{
  initialize_params_c16_dynamics_lib((SFc16_dynamics_libInstanceStruct*)
    chartInstanceVar);
  initialize_c16_dynamics_lib((SFc16_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c16_dynamics_lib(void *chartInstanceVar)
{
  enable_c16_dynamics_lib((SFc16_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c16_dynamics_lib(void *chartInstanceVar)
{
  disable_c16_dynamics_lib((SFc16_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c16_dynamics_lib(void *chartInstanceVar)
{
  sf_gateway_c16_dynamics_lib((SFc16_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c16_dynamics_lib(SimStruct* S)
{
  return get_sim_state_c16_dynamics_lib((SFc16_dynamics_libInstanceStruct *)
    sf_get_chart_instance_ptr(S));     /* raw sim ctx */
}

static void sf_opaque_set_sim_state_c16_dynamics_lib(SimStruct* S, const mxArray
  *st)
{
  set_sim_state_c16_dynamics_lib((SFc16_dynamics_libInstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c16_dynamics_lib(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc16_dynamics_libInstanceStruct*) chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_dynamics_lib_optimization_info();
    }

    mdl_cleanup_runtime_resources_c16_dynamics_lib
      ((SFc16_dynamics_libInstanceStruct*) chartInstanceVar);
    ((SFc16_dynamics_libInstanceStruct*) chartInstanceVar)->
      ~SFc16_dynamics_libInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c16_dynamics_lib(void *chartInstanceVar)
{
  mdl_start_c16_dynamics_lib((SFc16_dynamics_libInstanceStruct*)
    chartInstanceVar);
  if (chartInstanceVar) {
    sf_reset_warnings_ChartRunTimeInfo(((SFc16_dynamics_libInstanceStruct*)
      chartInstanceVar)->S);
  }
}

static void sf_opaque_mdl_terminate_c16_dynamics_lib(void *chartInstanceVar)
{
  mdl_terminate_c16_dynamics_lib((SFc16_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c16_dynamics_lib(SimStruct *S)
{
  mdlProcessParamsCommon(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c16_dynamics_lib((SFc16_dynamics_libInstanceStruct*)
      sf_get_chart_instance_ptr(S));
    initSubchartIOPointersc16_dynamics_lib((SFc16_dynamics_libInstanceStruct*)
      sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c16_dynamics_lib_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [21] = {
    "eNrtWMuOG0UULVvDKImSaBZIoChSsmSDlAmaABKCmfgxGM0wJu0ZEEIalbuv7WKqqzr18IMvCAt",
    "+gJ9gi1ix5BPgG9iwhB232m3HtI3d1ZYgQbTVbld3nzr33rqvMqm0Tgket/F853VCdvF6Dc8qmR",
    "6vZOPKwjm9v0Pey8Y/3SAklBH0QQS212Nj4ncIG7eporEm/oegMTwBLbk1TIqW6MniWCZ6oECEO",
    "EEilfHi1Sy2nImrphWhY9afDlg4CAbS8ugxTkijM8Enf8ebWNNGxjpTEJomQGQGStr+oMlpf70V",
    "lBnVBhBeaRt720qDCWziVNWnlhuWcGiMIWwJbShaQW/QNzDUQM2M/Yzs9NXBDC3jhDMqitt6QHU",
    "ACXqHgfMkwu8za9B6BXmRr8sENVIxyhsxrw2oKoptc5TzFN2ae9vZxKYOXdvvM9F31lU2BoH6o5",
    "8UsFWvJoegaB/OhB9v6LRrjNMFnvtlQaxhMVxQdRSiH2iI/GIQHVgHFJcWOjiNFxYE7XJo6Y5iQ",
    "1wj37zRcqFUKm/YeOpJuhQ25W0McVV1Sd5mKGqUc+2H7cjkBIbAU/46NbQEdsrvAdaaRR2J3uEi",
    "1zP6rWBPLWTYmhQRK+6VwxwqLRIfY8IvAGexCwOI0Mxz0ecTbYojq42Maxj69ZOTgnzL2JYwoHo",
    "0hML5WlGmAQVO/cqTN2LaBRKi0Uom1bLwDNMYLAUlumdFfSTVFdrYtzA8t5WLBD80RH2og4E0yT",
    "XQuy8otwVljjVmZece5xqzrB8vYl38lAKHNBxA5Gog43CKeRYnKLrE2pXPI9R2yMykDjpULCkaS",
    "RYTOhY/Z6XOJIFzcSXkSDSVjIOsi1njVwCYNagSWMoeYylVkyYKX0xqBU87aXb3bRicnanhtOt8",
    "4xgEVkOnq6veNMSoaghsN1GgbbAB+wrbEaGZNtgCThppDERpH3xInvfBOyv64L2FPngvO8P9R5f",
    "RBLtRFupLzrpungcL89ws0E/P+vB1OLKEI3Pc7PrGAr6ygpcsXPN816t/fb+a46vip1IhS3a6le",
    "PZyeF2Mzs9++OLH76++8t37/746Jtnv4bfbuKvLPFX0t8O933Vb79yOxvfnfVK88w8XEpe7t0PN",
    "/jBazk/cGNd/yi4OJZtod/+5MvPDo7Htf5+76CWzndtg7zVnLyz+/ddf4aBm+YBFbaibN/jxtRO",
    "+/F0/7Yg7+4Ge1xf8CdCfvtgO/yrh0Xi5lbOXm6MZZk9kZx3pEm7v0U/Li/P/cPt8FP+9gZ97uT",
    "0uZP275fUZWm4zGeE5Tgt6/e+uJeF739c8XUoUk+qJXFl+cg/jNtWP986+bK//2BNPiO59/deYD",
    "3W5VGffulF0+tn4tfP3MvG78/36LUB49GKXUL2GBv53qqn/xH//t3TfrP+reHsl/0x+vlbR4LyC",
    "W4Lptuu7HZbuf/k5o8UUL167/Vv1JPZdVP/dTMX3248YiKSI/3m/sODh9vUpz8Be5sL1g==",
    ""
  };

  static char newstr [1497] = "";
  newstr[0] = '\0';
  for (i = 0; i < 21; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c16_dynamics_lib(SimStruct *S)
{
  const char* newstr = sf_c16_dynamics_lib_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(3126655368U));
  ssSetChecksum1(S,(2950241417U));
  ssSetChecksum2(S,(2369175609U));
  ssSetChecksum3(S,(2590239624U));
}

static void mdlRTW_c16_dynamics_lib(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c16_dynamics_lib(SimStruct *S)
{
  SFc16_dynamics_libInstanceStruct *chartInstance;
  chartInstance = (SFc16_dynamics_libInstanceStruct *)utMalloc(sizeof
    (SFc16_dynamics_libInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc16_dynamics_libInstanceStruct));
  chartInstance = new (chartInstance) SFc16_dynamics_libInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway = sf_opaque_gateway_c16_dynamics_lib;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c16_dynamics_lib;
  chartInstance->chartInfo.mdlStart = sf_opaque_mdl_start_c16_dynamics_lib;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c16_dynamics_lib;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c16_dynamics_lib;
  chartInstance->chartInfo.enableChart = sf_opaque_enable_c16_dynamics_lib;
  chartInstance->chartInfo.disableChart = sf_opaque_disable_c16_dynamics_lib;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c16_dynamics_lib;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c16_dynamics_lib;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c16_dynamics_lib;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c16_dynamics_lib;
  chartInstance->chartInfo.mdlSetWorkWidths = mdlSetWorkWidths_c16_dynamics_lib;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c16_dynamics_lib(chartInstance);
}

void c16_dynamics_lib_method_dispatcher(SimStruct *S, int_T method, void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c16_dynamics_lib(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c16_dynamics_lib(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c16_dynamics_lib(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c16_dynamics_lib_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
