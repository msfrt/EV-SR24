/* Include files */

#include "dynamics_lib_sfun.h"
#include "c7_dynamics_lib.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c7_dynamics_lib(SFc7_dynamics_libInstanceStruct
  *chartInstance);
static void initialize_params_c7_dynamics_lib(SFc7_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_start_c7_dynamics_lib(SFc7_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_terminate_c7_dynamics_lib(SFc7_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_setup_runtime_resources_c7_dynamics_lib
  (SFc7_dynamics_libInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c7_dynamics_lib
  (SFc7_dynamics_libInstanceStruct *chartInstance);
static void enable_c7_dynamics_lib(SFc7_dynamics_libInstanceStruct
  *chartInstance);
static void disable_c7_dynamics_lib(SFc7_dynamics_libInstanceStruct
  *chartInstance);
static void sf_gateway_c7_dynamics_lib(SFc7_dynamics_libInstanceStruct
  *chartInstance);
static void ext_mode_exec_c7_dynamics_lib(SFc7_dynamics_libInstanceStruct
  *chartInstance);
static void c7_do_animation_call_c7_dynamics_lib(SFc7_dynamics_libInstanceStruct
  *chartInstance);
static const mxArray *get_sim_state_c7_dynamics_lib
  (SFc7_dynamics_libInstanceStruct *chartInstance);
static void set_sim_state_c7_dynamics_lib(SFc7_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c7_st);
static void initSimStructsc7_dynamics_lib(SFc7_dynamics_libInstanceStruct
  *chartInstance);
static void initSubchartIOPointersc7_dynamics_lib
  (SFc7_dynamics_libInstanceStruct *chartInstance);
static real32_T c7_eML_blk_kernel(SFc7_dynamics_libInstanceStruct *chartInstance,
  real_T c7_b_frontNonSuspendedMass, real_T c7_b_MM5_ay, real32_T
  c7_b_frontNonSuspendedMassCGHeight, real_T c7_b_frontTrack);
static real32_T c7_emlrt_marshallIn(SFc7_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c7_nullptr, const char_T *c7_identifier);
static real32_T c7_b_emlrt_marshallIn(SFc7_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c7_u, const emlrtMsgIdentifier *c7_parentId);
static uint8_T c7_c_emlrt_marshallIn(SFc7_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c7_nullptr, const char_T *c7_identifier);
static uint8_T c7_d_emlrt_marshallIn(SFc7_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c7_u, const emlrtMsgIdentifier *c7_parentId);
static void init_dsm_address_info(SFc7_dynamics_libInstanceStruct *chartInstance);
static void init_simulink_io_address(SFc7_dynamics_libInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c7_dynamics_lib(SFc7_dynamics_libInstanceStruct
  *chartInstance)
{
  emlrtLicenseCheckR2022a(chartInstance->c7_fEmlrtCtx,
    "EMLRT:runTime:MexFunctionNeedsLicense", "distrib_computing_toolbox", 2);
  sf_is_first_init_cond(chartInstance->S);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  emlrtInitGPU(chartInstance->c7_fEmlrtCtx);
  hipGetLastError();
}

static void initialize_params_c7_dynamics_lib(SFc7_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void mdl_start_c7_dynamics_lib(SFc7_dynamics_libInstanceStruct
  *chartInstance)
{
  sim_mode_is_external(chartInstance->S);
}

static void mdl_terminate_c7_dynamics_lib(SFc7_dynamics_libInstanceStruct
  *chartInstance)
{
  hipError_t c7_errCode;
  c7_errCode = hipGetLastError();
  if (c7_errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(c7_errCode), (char_T *)
                       hipGetErrorName(c7_errCode), (char_T *)
                       hipGetErrorString(c7_errCode), (char_T *)
                       "SimGPUErrorChecks", chartInstance->c7_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c7_dynamics_lib
  (SFc7_dynamics_libInstanceStruct *chartInstance)
{
  sfSetAnimationVectors(chartInstance->S, chartInstance->c7_JITStateAnimation,
                        chartInstance->c7_JITTransitionAnimation);
}

static void mdl_cleanup_runtime_resources_c7_dynamics_lib
  (SFc7_dynamics_libInstanceStruct *chartInstance)
{
}

static void enable_c7_dynamics_lib(SFc7_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c7_dynamics_lib(SFc7_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void sf_gateway_c7_dynamics_lib(SFc7_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c7_JITTransitionAnimation[0] = 0U;
  *chartInstance->c7_frontNSMWT = c7_eML_blk_kernel(chartInstance,
    *chartInstance->c7_frontNonSuspendedMass, *chartInstance->c7_MM5_ay,
    *chartInstance->c7_frontNonSuspendedMassCGHeight,
    *chartInstance->c7_frontTrack);
  c7_do_animation_call_c7_dynamics_lib(chartInstance);
}

static void ext_mode_exec_c7_dynamics_lib(SFc7_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void c7_do_animation_call_c7_dynamics_lib(SFc7_dynamics_libInstanceStruct
  *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static const mxArray *get_sim_state_c7_dynamics_lib
  (SFc7_dynamics_libInstanceStruct *chartInstance)
{
  const mxArray *c7_b_y = NULL;
  const mxArray *c7_c_y = NULL;
  const mxArray *c7_st;
  const mxArray *c7_y = NULL;
  c7_st = NULL;
  c7_st = NULL;
  c7_y = NULL;
  sf_mex_assign(&c7_y, sf_mex_createcellmatrix(2, 1), false);
  c7_b_y = NULL;
  sf_mex_assign(&c7_b_y, sf_mex_create("y", chartInstance->c7_frontNSMWT, 1, 0U,
    0U, 0U, 0), false);
  sf_mex_setcell(c7_y, 0, c7_b_y);
  c7_c_y = NULL;
  sf_mex_assign(&c7_c_y, sf_mex_create("y",
    &chartInstance->c7_is_active_c7_dynamics_lib, 3, 0U, 0U, 0U, 0), false);
  sf_mex_setcell(c7_y, 1, c7_c_y);
  sf_mex_assign(&c7_st, c7_y, false);
  return c7_st;
}

static void set_sim_state_c7_dynamics_lib(SFc7_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c7_st)
{
  const mxArray *c7_u;
  c7_u = sf_mex_dup(c7_st);
  *chartInstance->c7_frontNSMWT = c7_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getcell(c7_u, 0)), "frontNSMWT");
  chartInstance->c7_is_active_c7_dynamics_lib = c7_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c7_u, 1)),
     "is_active_c7_dynamics_lib");
  sf_mex_destroy(&c7_u);
  sf_mex_destroy(&c7_st);
}

static void initSimStructsc7_dynamics_lib(SFc7_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void initSubchartIOPointersc7_dynamics_lib
  (SFc7_dynamics_libInstanceStruct *chartInstance)
{
}

const mxArray *sf_c7_dynamics_lib_get_eml_resolved_functions_info()
{
  const mxArray *c7_nameCaptureInfo = NULL;
  c7_nameCaptureInfo = NULL;
  sf_mex_assign(&c7_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c7_nameCaptureInfo;
}

static real32_T c7_eML_blk_kernel(SFc7_dynamics_libInstanceStruct *chartInstance,
  real_T c7_b_frontNonSuspendedMass, real_T c7_b_MM5_ay, real32_T
  c7_b_frontNonSuspendedMassCGHeight, real_T c7_b_frontTrack)
{
  return static_cast<real32_T>(c7_b_frontNonSuspendedMass * 2.0 * c7_b_MM5_ay) *
    c7_b_frontNonSuspendedMassCGHeight / static_cast<real32_T>(c7_b_frontTrack);
}

static real32_T c7_emlrt_marshallIn(SFc7_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c7_nullptr, const char_T *c7_identifier)
{
  emlrtMsgIdentifier c7_thisId;
  real32_T c7_y;
  c7_thisId.fIdentifier = const_cast<const char_T *>(c7_identifier);
  c7_thisId.fParent = NULL;
  c7_thisId.bParentIsCell = false;
  c7_y = c7_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c7_nullptr), &c7_thisId);
  sf_mex_destroy(&c7_nullptr);
  return c7_y;
}

static real32_T c7_b_emlrt_marshallIn(SFc7_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c7_u, const emlrtMsgIdentifier *c7_parentId)
{
  real32_T c7_f;
  real32_T c7_y;
  sf_mex_import(c7_parentId, sf_mex_dup(c7_u), &c7_f, 0, 1, 0U, 0, 0U, 0);
  c7_y = c7_f;
  sf_mex_destroy(&c7_u);
  return c7_y;
}

static uint8_T c7_c_emlrt_marshallIn(SFc7_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c7_nullptr, const char_T *c7_identifier)
{
  emlrtMsgIdentifier c7_thisId;
  uint8_T c7_y;
  c7_thisId.fIdentifier = const_cast<const char_T *>(c7_identifier);
  c7_thisId.fParent = NULL;
  c7_thisId.bParentIsCell = false;
  c7_y = c7_d_emlrt_marshallIn(chartInstance, sf_mex_dup(c7_nullptr), &c7_thisId);
  sf_mex_destroy(&c7_nullptr);
  return c7_y;
}

static uint8_T c7_d_emlrt_marshallIn(SFc7_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c7_u, const emlrtMsgIdentifier *c7_parentId)
{
  uint8_T c7_b_u;
  uint8_T c7_y;
  sf_mex_import(c7_parentId, sf_mex_dup(c7_u), &c7_b_u, 1, 3, 0U, 0, 0U, 0);
  c7_y = c7_b_u;
  sf_mex_destroy(&c7_u);
  return c7_y;
}

static void init_dsm_address_info(SFc7_dynamics_libInstanceStruct *chartInstance)
{
}

static void init_simulink_io_address(SFc7_dynamics_libInstanceStruct
  *chartInstance)
{
  chartInstance->c7_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c7_frontNonSuspendedMass = (real_T *)
    ssGetInputPortSignal_wrapper(chartInstance->S, 0);
  chartInstance->c7_frontNSMWT = (real32_T *)ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c7_MM5_ay = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c7_frontNonSuspendedMassCGHeight = (real32_T *)
    ssGetInputPortSignal_wrapper(chartInstance->S, 2);
  chartInstance->c7_frontTrack = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 3);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c7_dynamics_lib_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(34771236U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(3503627148U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(700666491U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(1655269721U);
}

mxArray *sf_c7_dynamics_lib_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c7_dynamics_lib_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("chartInfo");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c7_dynamics_lib_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c7_dynamics_lib(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiDmYGJgYAPRQMzEAAGsUD4jVIwRLs4CF1cA4pLKglSQeHFRsmcKkM5LzAXzE0s"
    "rPPPS8sHmWzAgzGfDYj4jkvmcUHEI+GBPmX4RB5B+ByT9LFj0cyHpF4Dy04ry80r8gn3DQyDifA"
    "wImnz3KDhQph9ifwAB/0ii+QfEzyyOT0wuySxLjU82j0+pBEZTZnJxfE5mEgOq/wBy/xwz"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c7_dynamics_lib_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "s9nDEkOrUZWchYTJDrDyuND";
}

static void sf_opaque_initialize_c7_dynamics_lib(void *chartInstanceVar)
{
  initialize_params_c7_dynamics_lib((SFc7_dynamics_libInstanceStruct*)
    chartInstanceVar);
  initialize_c7_dynamics_lib((SFc7_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c7_dynamics_lib(void *chartInstanceVar)
{
  enable_c7_dynamics_lib((SFc7_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c7_dynamics_lib(void *chartInstanceVar)
{
  disable_c7_dynamics_lib((SFc7_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c7_dynamics_lib(void *chartInstanceVar)
{
  sf_gateway_c7_dynamics_lib((SFc7_dynamics_libInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c7_dynamics_lib(SimStruct* S)
{
  return get_sim_state_c7_dynamics_lib((SFc7_dynamics_libInstanceStruct *)
    sf_get_chart_instance_ptr(S));     /* raw sim ctx */
}

static void sf_opaque_set_sim_state_c7_dynamics_lib(SimStruct* S, const mxArray *
  st)
{
  set_sim_state_c7_dynamics_lib((SFc7_dynamics_libInstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c7_dynamics_lib(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc7_dynamics_libInstanceStruct*) chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_dynamics_lib_optimization_info();
    }

    mdl_cleanup_runtime_resources_c7_dynamics_lib
      ((SFc7_dynamics_libInstanceStruct*) chartInstanceVar);
    ((SFc7_dynamics_libInstanceStruct*) chartInstanceVar)->
      ~SFc7_dynamics_libInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c7_dynamics_lib(void *chartInstanceVar)
{
  mdl_start_c7_dynamics_lib((SFc7_dynamics_libInstanceStruct*) chartInstanceVar);
  if (chartInstanceVar) {
    sf_reset_warnings_ChartRunTimeInfo(((SFc7_dynamics_libInstanceStruct*)
      chartInstanceVar)->S);
  }
}

static void sf_opaque_mdl_terminate_c7_dynamics_lib(void *chartInstanceVar)
{
  mdl_terminate_c7_dynamics_lib((SFc7_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c7_dynamics_lib(SimStruct *S)
{
  mdlProcessParamsCommon(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c7_dynamics_lib((SFc7_dynamics_libInstanceStruct*)
      sf_get_chart_instance_ptr(S));
    initSubchartIOPointersc7_dynamics_lib((SFc7_dynamics_libInstanceStruct*)
      sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c7_dynamics_lib_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [21] = {
    "eNrtWE+P20QUn0TLqq2WUqRKtBISe0RISKUItZUquts4KUGbboSzXdHLMrFf4lHGY3f+JBu+FNd",
    "+ASQOiDNfgBMXTogjbxwnDU5IPI4ELWIkr3ds/+b33pv3b0Jq7Q7BcR2v+7cI2cf7FbzqZDbeyu",
    "e1pWv2fI88zOc/XCMkSEIYgvDNYMAuidsQJu5SSWNF3IegMXwFKuFGs0S0xSApj2ViABJEgAuki",
    "dROvIrFhjMxahkRWGZ1HrEg8qPE8PAxLkjDU8Gnf8ebGt1FRo9JCHQLINSRTMwwanE63GwFqSeN",
    "CIKRMrGzrRRo36RWVdUxXLOUQ/MSgrZQmqIV1BZ9fU01NPSlm5Gtvsqfo5M45YyK8raOqPIhRe/",
    "QcJaG+PfUaLReSV7k6zNBdSIZ5c2YNyIqy2K7HOXsoFtzZzvrWHvQN8MhE0NrXWliEKg/+kkJWw",
    "0ayRgkHcKpcOMNrHbNy2yDF35ZEqtZDM+oPA7QDxSEbjGIDqx8ilsLPVzGCQuC9jm0VU+yMe6Ra",
    "95o21CqlDdMPPMkVQmb8TbHuKuqIm8rEA3KuXLD9pL0BMbAM36PaloBO+N3ACvFwl6C3mEj1zH6",
    "jWAvDOTYRiJCVt4rxwVUViSeYsIvAWexDQMI0cwL0RcLbYsjo3QSNzD0vZOTknyr2LbQIAc0gNL",
    "5WlKmAAXO/MqRN2TKBhKi0Uo607L0CrMYrAQlamCEN0nkCG3sWhhe2cpGghsawiF4oCFLck307m",
    "eUm5IyxwqzsnWPM4VZ1o0XsTZ+KoEDGkQQ2hrIOHQwz+ICZbdY2fJ5jNqOmZ56oALJ0rKRZDChY",
    "/GzVupNUzgTI5FMREsmsZ93MRv8CgCzBpUCS9ljLKVy2kLhy0kt4UUvy+6uDYO1M9Wc9q1vPAGB",
    "1dDqaqs3DTCqmgLbTRRoF6zPvsV2RCimNLaA02YWA2HWBx+RV33w3po++J2lPvhGPg/uXYRTbEZ",
    "ZoC4462fr3Fla56BEPz3vwzfhyAqOLHDz+4dL+NoaXrJ0L/Jdrf/1+70CXx2f1HAU7fQ22Yzbz2",
    "31468/0Y9+/+beg+8O3o0Pv/9lG39thb+W/W9xL+tu55Xr+fz9ea+0yMzjleRlv/1iix+8V/ADO",
    "1cPhNccncqz5+dB9HXvS096U/PUy9a7skXeekHe+fND259h4GZ5QAbtMD/32Dk1s348O78tybu/",
    "xR5Xl/yJkN8e7Ya/eVQmbq4V7GXnA5lgrfM7571VP64uz+HRbvgZf3eLPrcL+tzO+vcLarM0XBQ",
    "zwqp+Vf3eFfem8P2PK78PZerJXkVcVT7yD+N21c+1Tr7p39/ZkM9I4fsbr7Eem/KoS7/0uun1M3",
    "HrZz7I558vzuiNiPFwzSkhf42N/GDd2/+If//haL95/9a09st/GH3+6bGgfIrHgtmxK3/clfY3u",
    "cUrCVStP3v9G/Vkft/Wfx0U4tvOJ0yEyUR9/Mndz+7uUp/+BHxMCWE=",
    ""
  };

  static char newstr [1481] = "";
  newstr[0] = '\0';
  for (i = 0; i < 21; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c7_dynamics_lib(SimStruct *S)
{
  const char* newstr = sf_c7_dynamics_lib_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(1640950732U));
  ssSetChecksum1(S,(929101354U));
  ssSetChecksum2(S,(286043961U));
  ssSetChecksum3(S,(3854901357U));
}

static void mdlRTW_c7_dynamics_lib(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c7_dynamics_lib(SimStruct *S)
{
  SFc7_dynamics_libInstanceStruct *chartInstance;
  chartInstance = (SFc7_dynamics_libInstanceStruct *)utMalloc(sizeof
    (SFc7_dynamics_libInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc7_dynamics_libInstanceStruct));
  chartInstance = new (chartInstance) SFc7_dynamics_libInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway = sf_opaque_gateway_c7_dynamics_lib;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c7_dynamics_lib;
  chartInstance->chartInfo.mdlStart = sf_opaque_mdl_start_c7_dynamics_lib;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c7_dynamics_lib;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c7_dynamics_lib;
  chartInstance->chartInfo.enableChart = sf_opaque_enable_c7_dynamics_lib;
  chartInstance->chartInfo.disableChart = sf_opaque_disable_c7_dynamics_lib;
  chartInstance->chartInfo.getSimState = sf_opaque_get_sim_state_c7_dynamics_lib;
  chartInstance->chartInfo.setSimState = sf_opaque_set_sim_state_c7_dynamics_lib;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c7_dynamics_lib;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c7_dynamics_lib;
  chartInstance->chartInfo.mdlSetWorkWidths = mdlSetWorkWidths_c7_dynamics_lib;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c7_dynamics_lib(chartInstance);
}

void c7_dynamics_lib_method_dispatcher(SimStruct *S, int_T method, void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c7_dynamics_lib(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c7_dynamics_lib(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c7_dynamics_lib(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c7_dynamics_lib_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
