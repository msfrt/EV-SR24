/* Include files */

#include "dynamics_lib_sfun.h"
#include "c12_dynamics_lib.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c12_dynamics_lib(SFc12_dynamics_libInstanceStruct
  *chartInstance);
static void initialize_params_c12_dynamics_lib(SFc12_dynamics_libInstanceStruct *
  chartInstance);
static void mdl_start_c12_dynamics_lib(SFc12_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_terminate_c12_dynamics_lib(SFc12_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_setup_runtime_resources_c12_dynamics_lib
  (SFc12_dynamics_libInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c12_dynamics_lib
  (SFc12_dynamics_libInstanceStruct *chartInstance);
static void enable_c12_dynamics_lib(SFc12_dynamics_libInstanceStruct
  *chartInstance);
static void disable_c12_dynamics_lib(SFc12_dynamics_libInstanceStruct
  *chartInstance);
static void sf_gateway_c12_dynamics_lib(SFc12_dynamics_libInstanceStruct
  *chartInstance);
static void ext_mode_exec_c12_dynamics_lib(SFc12_dynamics_libInstanceStruct
  *chartInstance);
static void c12_do_animation_call_c12_dynamics_lib
  (SFc12_dynamics_libInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c12_dynamics_lib
  (SFc12_dynamics_libInstanceStruct *chartInstance);
static void set_sim_state_c12_dynamics_lib(SFc12_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c12_st);
static void initSimStructsc12_dynamics_lib(SFc12_dynamics_libInstanceStruct
  *chartInstance);
static void initSubchartIOPointersc12_dynamics_lib
  (SFc12_dynamics_libInstanceStruct *chartInstance);
static real_T c12_eML_blk_kernel(SFc12_dynamics_libInstanceStruct *chartInstance,
  real_T c12_b_rearTrack, real_T c12_b_rearWheelRate);
static real_T c12_emlrt_marshallIn(SFc12_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c12_nullptr, const char_T *c12_identifier);
static real_T c12_b_emlrt_marshallIn(SFc12_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c12_u, const emlrtMsgIdentifier *c12_parentId);
static uint8_T c12_c_emlrt_marshallIn(SFc12_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c12_nullptr, const char_T *c12_identifier);
static uint8_T c12_d_emlrt_marshallIn(SFc12_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c12_u, const emlrtMsgIdentifier *c12_parentId);
static void init_dsm_address_info(SFc12_dynamics_libInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc12_dynamics_libInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c12_dynamics_lib(SFc12_dynamics_libInstanceStruct
  *chartInstance)
{
  emlrtLicenseCheckR2022a(chartInstance->c12_fEmlrtCtx,
    "EMLRT:runTime:MexFunctionNeedsLicense", "distrib_computing_toolbox", 2);
  sf_is_first_init_cond(chartInstance->S);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  emlrtInitGPU(chartInstance->c12_fEmlrtCtx);
  hipGetLastError();
}

static void initialize_params_c12_dynamics_lib(SFc12_dynamics_libInstanceStruct *
  chartInstance)
{
}

static void mdl_start_c12_dynamics_lib(SFc12_dynamics_libInstanceStruct
  *chartInstance)
{
  sim_mode_is_external(chartInstance->S);
}

static void mdl_terminate_c12_dynamics_lib(SFc12_dynamics_libInstanceStruct
  *chartInstance)
{
  hipError_t c12_errCode;
  c12_errCode = hipGetLastError();
  if (c12_errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(c12_errCode), (char_T *)
                       hipGetErrorName(c12_errCode), (char_T *)
                       hipGetErrorString(c12_errCode), (char_T *)
                       "SimGPUErrorChecks", chartInstance->c12_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c12_dynamics_lib
  (SFc12_dynamics_libInstanceStruct *chartInstance)
{
  sfSetAnimationVectors(chartInstance->S, chartInstance->c12_JITStateAnimation,
                        chartInstance->c12_JITTransitionAnimation);
}

static void mdl_cleanup_runtime_resources_c12_dynamics_lib
  (SFc12_dynamics_libInstanceStruct *chartInstance)
{
}

static void enable_c12_dynamics_lib(SFc12_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c12_dynamics_lib(SFc12_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void sf_gateway_c12_dynamics_lib(SFc12_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c12_JITTransitionAnimation[0] = 0U;
  *chartInstance->c12_antiRollRearSprings = c12_eML_blk_kernel(chartInstance,
    *chartInstance->c12_rearTrack, *chartInstance->c12_rearWheelRate);
  c12_do_animation_call_c12_dynamics_lib(chartInstance);
}

static void ext_mode_exec_c12_dynamics_lib(SFc12_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void c12_do_animation_call_c12_dynamics_lib
  (SFc12_dynamics_libInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static const mxArray *get_sim_state_c12_dynamics_lib
  (SFc12_dynamics_libInstanceStruct *chartInstance)
{
  const mxArray *c12_b_y = NULL;
  const mxArray *c12_c_y = NULL;
  const mxArray *c12_st;
  const mxArray *c12_y = NULL;
  c12_st = NULL;
  c12_st = NULL;
  c12_y = NULL;
  sf_mex_assign(&c12_y, sf_mex_createcellmatrix(2, 1), false);
  c12_b_y = NULL;
  sf_mex_assign(&c12_b_y, sf_mex_create("y",
    chartInstance->c12_antiRollRearSprings, 0, 0U, 0U, 0U, 0), false);
  sf_mex_setcell(c12_y, 0, c12_b_y);
  c12_c_y = NULL;
  sf_mex_assign(&c12_c_y, sf_mex_create("y",
    &chartInstance->c12_is_active_c12_dynamics_lib, 3, 0U, 0U, 0U, 0), false);
  sf_mex_setcell(c12_y, 1, c12_c_y);
  sf_mex_assign(&c12_st, c12_y, false);
  return c12_st;
}

static void set_sim_state_c12_dynamics_lib(SFc12_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c12_st)
{
  const mxArray *c12_u;
  c12_u = sf_mex_dup(c12_st);
  *chartInstance->c12_antiRollRearSprings = c12_emlrt_marshallIn(chartInstance,
    sf_mex_dup(sf_mex_getcell(c12_u, 0)), "antiRollRearSprings");
  chartInstance->c12_is_active_c12_dynamics_lib = c12_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c12_u, 1)),
     "is_active_c12_dynamics_lib");
  sf_mex_destroy(&c12_u);
  sf_mex_destroy(&c12_st);
}

static void initSimStructsc12_dynamics_lib(SFc12_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void initSubchartIOPointersc12_dynamics_lib
  (SFc12_dynamics_libInstanceStruct *chartInstance)
{
}

const mxArray *sf_c12_dynamics_lib_get_eml_resolved_functions_info()
{
  const mxArray *c12_nameCaptureInfo = NULL;
  c12_nameCaptureInfo = NULL;
  sf_mex_assign(&c12_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c12_nameCaptureInfo;
}

static real_T c12_eML_blk_kernel(SFc12_dynamics_libInstanceStruct *chartInstance,
  real_T c12_b_rearTrack, real_T c12_b_rearWheelRate)
{
  real_T c12_a;
  c12_a = c12_b_rearTrack / 1000.0;
  return c12_a * c12_a * 1.5574077246549023 * (c12_b_rearWheelRate * 1000.0) /
    2.0;
}

static real_T c12_emlrt_marshallIn(SFc12_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c12_nullptr, const char_T *c12_identifier)
{
  emlrtMsgIdentifier c12_thisId;
  real_T c12_y;
  c12_thisId.fIdentifier = const_cast<const char_T *>(c12_identifier);
  c12_thisId.fParent = NULL;
  c12_thisId.bParentIsCell = false;
  c12_y = c12_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c12_nullptr),
    &c12_thisId);
  sf_mex_destroy(&c12_nullptr);
  return c12_y;
}

static real_T c12_b_emlrt_marshallIn(SFc12_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c12_u, const emlrtMsgIdentifier *c12_parentId)
{
  real_T c12_d;
  real_T c12_y;
  sf_mex_import(c12_parentId, sf_mex_dup(c12_u), &c12_d, 1, 0, 0U, 0, 0U, 0);
  c12_y = c12_d;
  sf_mex_destroy(&c12_u);
  return c12_y;
}

static uint8_T c12_c_emlrt_marshallIn(SFc12_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c12_nullptr, const char_T *c12_identifier)
{
  emlrtMsgIdentifier c12_thisId;
  uint8_T c12_y;
  c12_thisId.fIdentifier = const_cast<const char_T *>(c12_identifier);
  c12_thisId.fParent = NULL;
  c12_thisId.bParentIsCell = false;
  c12_y = c12_d_emlrt_marshallIn(chartInstance, sf_mex_dup(c12_nullptr),
    &c12_thisId);
  sf_mex_destroy(&c12_nullptr);
  return c12_y;
}

static uint8_T c12_d_emlrt_marshallIn(SFc12_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c12_u, const emlrtMsgIdentifier *c12_parentId)
{
  uint8_T c12_b_u;
  uint8_T c12_y;
  sf_mex_import(c12_parentId, sf_mex_dup(c12_u), &c12_b_u, 1, 3, 0U, 0, 0U, 0);
  c12_y = c12_b_u;
  sf_mex_destroy(&c12_u);
  return c12_y;
}

static void init_dsm_address_info(SFc12_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc12_dynamics_libInstanceStruct
  *chartInstance)
{
  chartInstance->c12_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c12_rearTrack = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c12_antiRollRearSprings = (real_T *)
    ssGetOutputPortSignal_wrapper(chartInstance->S, 1);
  chartInstance->c12_rearWheelRate = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c12_dynamics_lib_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(49291795U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(2150116786U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(2148149072U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(1297981808U);
}

mxArray *sf_c12_dynamics_lib_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c12_dynamics_lib_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("chartInfo");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c12_dynamics_lib_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c12_dynamics_lib(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiAWYGJgYAPSHEDMxAABrFA+I1SMES7OAhdXAOKSyoJUkHhxUbJnCpDOS8wF8xN"
    "LKzzz0vLB5lswIMxnw2I+I5L5nFBxCPhgT5l+EQeQfg8k/SxY9Asj6ReA8hPzSjKD8nNyglITi4"
    "ILijLz0osZYOEFo8l3l4IDZfoh9gcQ8JcUmr9A/Mzi+MTkksyy1PhkQ6P4lEpgfGUmF8fnZCYxo"
    "PgPANzoIHY="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c12_dynamics_lib_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "szELbKLmqydMgM0SPDKjlvF";
}

static void sf_opaque_initialize_c12_dynamics_lib(void *chartInstanceVar)
{
  initialize_params_c12_dynamics_lib((SFc12_dynamics_libInstanceStruct*)
    chartInstanceVar);
  initialize_c12_dynamics_lib((SFc12_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c12_dynamics_lib(void *chartInstanceVar)
{
  enable_c12_dynamics_lib((SFc12_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c12_dynamics_lib(void *chartInstanceVar)
{
  disable_c12_dynamics_lib((SFc12_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c12_dynamics_lib(void *chartInstanceVar)
{
  sf_gateway_c12_dynamics_lib((SFc12_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c12_dynamics_lib(SimStruct* S)
{
  return get_sim_state_c12_dynamics_lib((SFc12_dynamics_libInstanceStruct *)
    sf_get_chart_instance_ptr(S));     /* raw sim ctx */
}

static void sf_opaque_set_sim_state_c12_dynamics_lib(SimStruct* S, const mxArray
  *st)
{
  set_sim_state_c12_dynamics_lib((SFc12_dynamics_libInstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c12_dynamics_lib(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc12_dynamics_libInstanceStruct*) chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_dynamics_lib_optimization_info();
    }

    mdl_cleanup_runtime_resources_c12_dynamics_lib
      ((SFc12_dynamics_libInstanceStruct*) chartInstanceVar);
    ((SFc12_dynamics_libInstanceStruct*) chartInstanceVar)->
      ~SFc12_dynamics_libInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c12_dynamics_lib(void *chartInstanceVar)
{
  mdl_start_c12_dynamics_lib((SFc12_dynamics_libInstanceStruct*)
    chartInstanceVar);
  if (chartInstanceVar) {
    sf_reset_warnings_ChartRunTimeInfo(((SFc12_dynamics_libInstanceStruct*)
      chartInstanceVar)->S);
  }
}

static void sf_opaque_mdl_terminate_c12_dynamics_lib(void *chartInstanceVar)
{
  mdl_terminate_c12_dynamics_lib((SFc12_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c12_dynamics_lib(SimStruct *S)
{
  mdlProcessParamsCommon(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c12_dynamics_lib((SFc12_dynamics_libInstanceStruct*)
      sf_get_chart_instance_ptr(S));
    initSubchartIOPointersc12_dynamics_lib((SFc12_dynamics_libInstanceStruct*)
      sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c12_dynamics_lib_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [21] = {
    "eNrtWN2O20QUnkTLilalWgkkUFWJ3sEN0nYRggsEu80PRE3YqM4WiZvVxD6Jhx3PeOcn2fQdkJB",
    "4B16AByh3cMctD9ELJG645IzjpMEJiccrQYvwyvGO7W++c86cvzGpdXoEj9t4Hr9FyD5eX8WzTu",
    "bHK/m4tnLO7++Rj/PxzzcJCWUEYxCBHY3YFfE7hE36VNFEE/9D0AQegZbcGiZFR4xkeSwTI1AgQ",
    "pwglcp48WqWWM7ERduK0DHrL2MWxkEsLY8e4IQ0OhV89ne8qTV9ZGwyBaFpA0QmVtKO4zan4+1W",
    "UGbaiCG80DbxtpUGE9jUqap7lhuWcmhdQdgR2lC0gt6hb2CogYa58jOy01cHC7RMUs6oKG/rmOo",
    "AUvQOA2dphL+n1qD1SvIi35AJaqRilLcS3oipKovtc5Szh27Nve1sEtOEoR2PmRg76yqbgED90U",
    "9K2GrUkBNQdAynwo83dNq1rrIFXvplSaxhCTym6iREP9AQ+cUgOrAOKC4tDHAaLywIOuTQ0QPFJ",
    "rhGvnmj40KpUt6wydyTdCVsxtua4KrqirztUDQo59oPO5BpFybAM/4mNbQCds7vAdaaRQOJ3uEi",
    "1zP6rWCXFnJsQ4qIlffKSQGVFYkvMOGXgLPEhQFEaOal6MuJdsWR1UYmDQz9Zrdbkm8d2xEG1Ii",
    "GUDpfK8o0oMCZX3nyRky7QEI0WslkWpaeYR6DlaBEj6xoTqW6QBv7FobntnKR4IeGaAxNMJAluR",
    "Z692PKbUmZE41Z2bnHmcYs68eLWBc/lcAhDWOIXA1kHHqYZ3GCskusXfk8QW0nzMyaoEPF0rKRZ",
    "DGhY/FzVhrMUjgTF0JORVvJJMi7mC1+BYBZgyqBpewBllI1a6Pw5aRWcDnIsrtvw+DsTA2nQ+cb",
    "n4HAauh0ddWbhhhVLYHtJgp0HWzAnmA7IjTTBlvAWSuLgWjeB5PnffDehj74YKUPPsjP8P7ReTT",
    "DbpSF+pyzoZvncGWeWyX66UUfvg1H1nBkiVtc313B1zbwkpVrke9G/a/v1wt8dfyr1cianV4r8O",
    "wVcPu5nX7/4adnH3777PunT4Nfvht+884u/toafy373+F+rPvtV27n47uLXmmZmSdrycu9+/kOP",
    "3iz4AdurJ+0usOH3eRyFvXGvcOg33z4NZ+0s/kOdshbL8i7uH/P9WcYuFkeUGEnyvc9bkztvB93",
    "83+0Iu/+DnvcWPEnQn779Hr4N47L2Ov1gr3cGMsyeyQ5xx2TClKFganX/Lm6XPeOr4ef8/d36HW",
    "noNedrI8/py5bw3kxM6zHa1X/98W9LHz/48qvQ5m6Uq+Iq8pH/mHcdfXzrZcv+/uHW/IZKbx/8A",
    "LrsS2P+vRNL5pevxK/vubtfPzJcq/eiBmPNuwW8sfY0I82Pf2P+PcfnvZb9HEtZ7/8A+lX758Iy",
    "me4PZhvv/LbfeW+zS0fKaB68x7s36gny+/4O/qVW4X4duMpE5Gc6vfuH31wdJ369CcimRAQ",
    ""
  };

  static char newstr [1497] = "";
  newstr[0] = '\0';
  for (i = 0; i < 21; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c12_dynamics_lib(SimStruct *S)
{
  const char* newstr = sf_c12_dynamics_lib_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(3972510707U));
  ssSetChecksum1(S,(2800521271U));
  ssSetChecksum2(S,(3478372287U));
  ssSetChecksum3(S,(663577235U));
}

static void mdlRTW_c12_dynamics_lib(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c12_dynamics_lib(SimStruct *S)
{
  SFc12_dynamics_libInstanceStruct *chartInstance;
  chartInstance = (SFc12_dynamics_libInstanceStruct *)utMalloc(sizeof
    (SFc12_dynamics_libInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc12_dynamics_libInstanceStruct));
  chartInstance = new (chartInstance) SFc12_dynamics_libInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway = sf_opaque_gateway_c12_dynamics_lib;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c12_dynamics_lib;
  chartInstance->chartInfo.mdlStart = sf_opaque_mdl_start_c12_dynamics_lib;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c12_dynamics_lib;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c12_dynamics_lib;
  chartInstance->chartInfo.enableChart = sf_opaque_enable_c12_dynamics_lib;
  chartInstance->chartInfo.disableChart = sf_opaque_disable_c12_dynamics_lib;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c12_dynamics_lib;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c12_dynamics_lib;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c12_dynamics_lib;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c12_dynamics_lib;
  chartInstance->chartInfo.mdlSetWorkWidths = mdlSetWorkWidths_c12_dynamics_lib;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c12_dynamics_lib(chartInstance);
}

void c12_dynamics_lib_method_dispatcher(SimStruct *S, int_T method, void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c12_dynamics_lib(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c12_dynamics_lib(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c12_dynamics_lib(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c12_dynamics_lib_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
