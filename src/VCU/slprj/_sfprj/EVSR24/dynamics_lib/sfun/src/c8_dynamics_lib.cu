/* Include files */

#include "dynamics_lib_sfun.h"
#include "c8_dynamics_lib.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c8_dynamics_lib(SFc8_dynamics_libInstanceStruct
  *chartInstance);
static void initialize_params_c8_dynamics_lib(SFc8_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_start_c8_dynamics_lib(SFc8_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_terminate_c8_dynamics_lib(SFc8_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_setup_runtime_resources_c8_dynamics_lib
  (SFc8_dynamics_libInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c8_dynamics_lib
  (SFc8_dynamics_libInstanceStruct *chartInstance);
static void enable_c8_dynamics_lib(SFc8_dynamics_libInstanceStruct
  *chartInstance);
static void disable_c8_dynamics_lib(SFc8_dynamics_libInstanceStruct
  *chartInstance);
static void sf_gateway_c8_dynamics_lib(SFc8_dynamics_libInstanceStruct
  *chartInstance);
static void ext_mode_exec_c8_dynamics_lib(SFc8_dynamics_libInstanceStruct
  *chartInstance);
static void c8_do_animation_call_c8_dynamics_lib(SFc8_dynamics_libInstanceStruct
  *chartInstance);
static const mxArray *get_sim_state_c8_dynamics_lib
  (SFc8_dynamics_libInstanceStruct *chartInstance);
static void set_sim_state_c8_dynamics_lib(SFc8_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c8_st);
static void initSimStructsc8_dynamics_lib(SFc8_dynamics_libInstanceStruct
  *chartInstance);
static void initSubchartIOPointersc8_dynamics_lib
  (SFc8_dynamics_libInstanceStruct *chartInstance);
static real32_T c8_eML_blk_kernel(SFc8_dynamics_libInstanceStruct *chartInstance,
  real_T c8_b_rearNonSuspendedMass, real_T c8_b_MM5_ay, real32_T
  c8_b_rearNonSuspendedMassCGHeight, real_T c8_b_rearTrack);
static real32_T c8_emlrt_marshallIn(SFc8_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c8_nullptr, const char_T *c8_identifier);
static real32_T c8_b_emlrt_marshallIn(SFc8_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c8_u, const emlrtMsgIdentifier *c8_parentId);
static uint8_T c8_c_emlrt_marshallIn(SFc8_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c8_nullptr, const char_T *c8_identifier);
static uint8_T c8_d_emlrt_marshallIn(SFc8_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c8_u, const emlrtMsgIdentifier *c8_parentId);
static void init_dsm_address_info(SFc8_dynamics_libInstanceStruct *chartInstance);
static void init_simulink_io_address(SFc8_dynamics_libInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c8_dynamics_lib(SFc8_dynamics_libInstanceStruct
  *chartInstance)
{
  emlrtLicenseCheckR2022a(chartInstance->c8_fEmlrtCtx,
    "EMLRT:runTime:MexFunctionNeedsLicense", "distrib_computing_toolbox", 2);
  sf_is_first_init_cond(chartInstance->S);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  emlrtInitGPU(chartInstance->c8_fEmlrtCtx);
  hipGetLastError();
}

static void initialize_params_c8_dynamics_lib(SFc8_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void mdl_start_c8_dynamics_lib(SFc8_dynamics_libInstanceStruct
  *chartInstance)
{
  sim_mode_is_external(chartInstance->S);
}

static void mdl_terminate_c8_dynamics_lib(SFc8_dynamics_libInstanceStruct
  *chartInstance)
{
  hipError_t c8_errCode;
  c8_errCode = hipGetLastError();
  if (c8_errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(c8_errCode), (char_T *)
                       hipGetErrorName(c8_errCode), (char_T *)
                       hipGetErrorString(c8_errCode), (char_T *)
                       "SimGPUErrorChecks", chartInstance->c8_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c8_dynamics_lib
  (SFc8_dynamics_libInstanceStruct *chartInstance)
{
  sfSetAnimationVectors(chartInstance->S, chartInstance->c8_JITStateAnimation,
                        chartInstance->c8_JITTransitionAnimation);
}

static void mdl_cleanup_runtime_resources_c8_dynamics_lib
  (SFc8_dynamics_libInstanceStruct *chartInstance)
{
}

static void enable_c8_dynamics_lib(SFc8_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c8_dynamics_lib(SFc8_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void sf_gateway_c8_dynamics_lib(SFc8_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c8_JITTransitionAnimation[0] = 0U;
  *chartInstance->c8_rearNSMWT = c8_eML_blk_kernel(chartInstance,
    *chartInstance->c8_rearNonSuspendedMass, *chartInstance->c8_MM5_ay,
    *chartInstance->c8_rearNonSuspendedMassCGHeight,
    *chartInstance->c8_rearTrack);
  c8_do_animation_call_c8_dynamics_lib(chartInstance);
}

static void ext_mode_exec_c8_dynamics_lib(SFc8_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void c8_do_animation_call_c8_dynamics_lib(SFc8_dynamics_libInstanceStruct
  *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static const mxArray *get_sim_state_c8_dynamics_lib
  (SFc8_dynamics_libInstanceStruct *chartInstance)
{
  const mxArray *c8_b_y = NULL;
  const mxArray *c8_c_y = NULL;
  const mxArray *c8_st;
  const mxArray *c8_y = NULL;
  c8_st = NULL;
  c8_st = NULL;
  c8_y = NULL;
  sf_mex_assign(&c8_y, sf_mex_createcellmatrix(2, 1), false);
  c8_b_y = NULL;
  sf_mex_assign(&c8_b_y, sf_mex_create("y", chartInstance->c8_rearNSMWT, 1, 0U,
    0U, 0U, 0), false);
  sf_mex_setcell(c8_y, 0, c8_b_y);
  c8_c_y = NULL;
  sf_mex_assign(&c8_c_y, sf_mex_create("y",
    &chartInstance->c8_is_active_c8_dynamics_lib, 3, 0U, 0U, 0U, 0), false);
  sf_mex_setcell(c8_y, 1, c8_c_y);
  sf_mex_assign(&c8_st, c8_y, false);
  return c8_st;
}

static void set_sim_state_c8_dynamics_lib(SFc8_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c8_st)
{
  const mxArray *c8_u;
  c8_u = sf_mex_dup(c8_st);
  *chartInstance->c8_rearNSMWT = c8_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getcell(c8_u, 0)), "rearNSMWT");
  chartInstance->c8_is_active_c8_dynamics_lib = c8_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c8_u, 1)),
     "is_active_c8_dynamics_lib");
  sf_mex_destroy(&c8_u);
  sf_mex_destroy(&c8_st);
}

static void initSimStructsc8_dynamics_lib(SFc8_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void initSubchartIOPointersc8_dynamics_lib
  (SFc8_dynamics_libInstanceStruct *chartInstance)
{
}

const mxArray *sf_c8_dynamics_lib_get_eml_resolved_functions_info()
{
  const mxArray *c8_nameCaptureInfo = NULL;
  c8_nameCaptureInfo = NULL;
  sf_mex_assign(&c8_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c8_nameCaptureInfo;
}

static real32_T c8_eML_blk_kernel(SFc8_dynamics_libInstanceStruct *chartInstance,
  real_T c8_b_rearNonSuspendedMass, real_T c8_b_MM5_ay, real32_T
  c8_b_rearNonSuspendedMassCGHeight, real_T c8_b_rearTrack)
{
  return static_cast<real32_T>(c8_b_rearNonSuspendedMass * 2.0 * c8_b_MM5_ay) *
    c8_b_rearNonSuspendedMassCGHeight / static_cast<real32_T>(c8_b_rearTrack);
}

static real32_T c8_emlrt_marshallIn(SFc8_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c8_nullptr, const char_T *c8_identifier)
{
  emlrtMsgIdentifier c8_thisId;
  real32_T c8_y;
  c8_thisId.fIdentifier = const_cast<const char_T *>(c8_identifier);
  c8_thisId.fParent = NULL;
  c8_thisId.bParentIsCell = false;
  c8_y = c8_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c8_nullptr), &c8_thisId);
  sf_mex_destroy(&c8_nullptr);
  return c8_y;
}

static real32_T c8_b_emlrt_marshallIn(SFc8_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c8_u, const emlrtMsgIdentifier *c8_parentId)
{
  real32_T c8_f;
  real32_T c8_y;
  sf_mex_import(c8_parentId, sf_mex_dup(c8_u), &c8_f, 0, 1, 0U, 0, 0U, 0);
  c8_y = c8_f;
  sf_mex_destroy(&c8_u);
  return c8_y;
}

static uint8_T c8_c_emlrt_marshallIn(SFc8_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c8_nullptr, const char_T *c8_identifier)
{
  emlrtMsgIdentifier c8_thisId;
  uint8_T c8_y;
  c8_thisId.fIdentifier = const_cast<const char_T *>(c8_identifier);
  c8_thisId.fParent = NULL;
  c8_thisId.bParentIsCell = false;
  c8_y = c8_d_emlrt_marshallIn(chartInstance, sf_mex_dup(c8_nullptr), &c8_thisId);
  sf_mex_destroy(&c8_nullptr);
  return c8_y;
}

static uint8_T c8_d_emlrt_marshallIn(SFc8_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c8_u, const emlrtMsgIdentifier *c8_parentId)
{
  uint8_T c8_b_u;
  uint8_T c8_y;
  sf_mex_import(c8_parentId, sf_mex_dup(c8_u), &c8_b_u, 1, 3, 0U, 0, 0U, 0);
  c8_y = c8_b_u;
  sf_mex_destroy(&c8_u);
  return c8_y;
}

static void init_dsm_address_info(SFc8_dynamics_libInstanceStruct *chartInstance)
{
}

static void init_simulink_io_address(SFc8_dynamics_libInstanceStruct
  *chartInstance)
{
  chartInstance->c8_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c8_rearNonSuspendedMass = (real_T *)
    ssGetInputPortSignal_wrapper(chartInstance->S, 0);
  chartInstance->c8_rearNSMWT = (real32_T *)ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c8_MM5_ay = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c8_rearNonSuspendedMassCGHeight = (real32_T *)
    ssGetInputPortSignal_wrapper(chartInstance->S, 2);
  chartInstance->c8_rearTrack = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 3);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c8_dynamics_lib_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(4255991394U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(2081089437U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(1405941125U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(469218498U);
}

mxArray *sf_c8_dynamics_lib_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c8_dynamics_lib_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("chartInfo");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c8_dynamics_lib_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c8_dynamics_lib(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiDmYGJgYAPRQMzEAAGsUD4jVIwRLs4CF1cA4pLKglSQeHFRsmcKkM5LzAXzE0s"
    "rPPPS8sHmWzAgzGfDYj4jkvmcUHEI+GBPmX4RB5B+ByT9LFj0cyLpF4Dyi1ITi/yCfcNDoOJ8SD"
    "T57lFwoEw/xP4AAv6RRPMPiJ9ZHJ+YXJJZlhqfbBGfUgmMpszk4viczCQ0/wEA+YMbsw=="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c8_dynamics_lib_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sBgkeG5mq506CydKSQt6XiD";
}

static void sf_opaque_initialize_c8_dynamics_lib(void *chartInstanceVar)
{
  initialize_params_c8_dynamics_lib((SFc8_dynamics_libInstanceStruct*)
    chartInstanceVar);
  initialize_c8_dynamics_lib((SFc8_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c8_dynamics_lib(void *chartInstanceVar)
{
  enable_c8_dynamics_lib((SFc8_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c8_dynamics_lib(void *chartInstanceVar)
{
  disable_c8_dynamics_lib((SFc8_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c8_dynamics_lib(void *chartInstanceVar)
{
  sf_gateway_c8_dynamics_lib((SFc8_dynamics_libInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c8_dynamics_lib(SimStruct* S)
{
  return get_sim_state_c8_dynamics_lib((SFc8_dynamics_libInstanceStruct *)
    sf_get_chart_instance_ptr(S));     /* raw sim ctx */
}

static void sf_opaque_set_sim_state_c8_dynamics_lib(SimStruct* S, const mxArray *
  st)
{
  set_sim_state_c8_dynamics_lib((SFc8_dynamics_libInstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c8_dynamics_lib(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc8_dynamics_libInstanceStruct*) chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_dynamics_lib_optimization_info();
    }

    mdl_cleanup_runtime_resources_c8_dynamics_lib
      ((SFc8_dynamics_libInstanceStruct*) chartInstanceVar);
    ((SFc8_dynamics_libInstanceStruct*) chartInstanceVar)->
      ~SFc8_dynamics_libInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c8_dynamics_lib(void *chartInstanceVar)
{
  mdl_start_c8_dynamics_lib((SFc8_dynamics_libInstanceStruct*) chartInstanceVar);
  if (chartInstanceVar) {
    sf_reset_warnings_ChartRunTimeInfo(((SFc8_dynamics_libInstanceStruct*)
      chartInstanceVar)->S);
  }
}

static void sf_opaque_mdl_terminate_c8_dynamics_lib(void *chartInstanceVar)
{
  mdl_terminate_c8_dynamics_lib((SFc8_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c8_dynamics_lib(SimStruct *S)
{
  mdlProcessParamsCommon(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c8_dynamics_lib((SFc8_dynamics_libInstanceStruct*)
      sf_get_chart_instance_ptr(S));
    initSubchartIOPointersc8_dynamics_lib((SFc8_dynamics_libInstanceStruct*)
      sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c8_dynamics_lib_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [21] = {
    "eNrtWE2P20QYnoRl1UVLqQRSqYTUCi5ckJairXpAdHfzUSI23YCzLUKI1cR+E48yHnvnI9nwO/g",
    "L/QuoPXPhzImfwQ2OvOM4aXBC4nEkaBEjeb1j+5n3Y96PZ0IqrTbBcR2v++8Ssov3a3hVyXS8ns",
    "0rC9f0+Q75NJv//AYhfhzAAIRn+n12RdyGMFGHShop4j4EjeArUDE3msWiJfpxcSwTfZAgfFwgi",
    "aV2kqtYZDgTw6YRvpWsnoTMD70wNjw4wQVpcCb45O/kJkZ3UGKdSfB1EyDQoYzNIGxyOljvBanH",
    "tRD8oTKRs68UaM8k1lTVNlyzhEPjCvyWUJqiF9QGez1NNdT0lZuTrb3Km6HjKOGMiuK+DqnyIMH",
    "o0HCeBPj3zGj0XkG5KK/HBNWxZJQ3Il4LqSyK7XDUs41hzZ39rCNdh54ZDJgYWO9KE4FA+zFOCv",
    "iqX4tHIOkAzoSbXN9a17hKN3gelwWxmkXwmMpjH+NAQeCWgxjAyqO4tdDFZZywIGiPQ0t1JRvhH",
    "rnWjZZNpVJ1w0TTSFKlsKncxgh3VZWU2/RFjXKu3LDdODmFEfBUfp1qWgI7le8AVooF3Rijw2au",
    "Y/YbwS4NZNhaLAJWPCpHOVTaJB5hwS8AZ5FNAwjQzXPV5wttyiOjdBzVMPXrp6cF5S1jW0KD7FM",
    "fCtdrSZkCVDiNK0e5AVM2kRCNXtKplYVXmOZgKShRfSPq41gO0ceujeGFr2wmuKEhGEAdNKRFro",
    "HR/ZhyU1DnSGFVtuFxrrDKuslFrM2fUmCf+iEEtgcyDm2ss7hA0S1Wtn0eo7Ujpid1UL5kSdFMM",
    "ljQsflZL3UnCZyLoYjHoinjyMtYzJq4AsCqQaXAVnaCrVROmqh8Ma0lXHbT6u5KGKyfqea0Z2Pj",
    "IQjshtZW272pj1nVEEg3UaFtsB77HumIUExppICTRpoDQcqDj8gLHryzgge/tcCDb2Rz//5FMEE",
    "yynx1wVkvXedgYZ39Anx6xsPX4cgSjsxxs/uHC/jKCrlk4Z6Xt1f96/c7OXlVfFLBkffTm2Q9bj",
    "fz1QfPv+0+ffu1/nc//PjTL+//vr9JfmVJfiX93+KeVd3OK9ez+XszrjSvzKOl4mW//XxDHNzMx",
    "YGdq5PBEB4eRpeHB/dqk+AL70t972tWT9e7tkHfak7f2fM7lp9h4qZ1QPqtIDv32Dk1Uz6ent8W",
    "9N3d4I+9hXgi5LcH2+HfOSqSN3s5f+2lNYLKR177SXdFHJfX587Rdvip/M4Ge27l7LmV8vcLaqs",
    "0XOQrwrJ9ZePeFfeqyPsfV3wfivSTnZK4svLIP4zb1j7XPvmqf3+wpp6R3Pc3XmI71tVRF770st",
    "n1K3HjM7ez+WfzM3otZDxYcUrIXiOR7696+x+J7z8c/Tfjbw3rv+yH0W8+ORaUT/BYMD12ZY870",
    "v4mN3+FrEWtPnv9G/1kdt/Ev/Zz+W3nYyaCeKw++vju4d1t+tOfbaUHpg==",
    ""
  };

  static char newstr [1485] = "";
  newstr[0] = '\0';
  for (i = 0; i < 21; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c8_dynamics_lib(SimStruct *S)
{
  const char* newstr = sf_c8_dynamics_lib_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(1415362851U));
  ssSetChecksum1(S,(1711477669U));
  ssSetChecksum2(S,(3283522654U));
  ssSetChecksum3(S,(217522898U));
}

static void mdlRTW_c8_dynamics_lib(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c8_dynamics_lib(SimStruct *S)
{
  SFc8_dynamics_libInstanceStruct *chartInstance;
  chartInstance = (SFc8_dynamics_libInstanceStruct *)utMalloc(sizeof
    (SFc8_dynamics_libInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc8_dynamics_libInstanceStruct));
  chartInstance = new (chartInstance) SFc8_dynamics_libInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway = sf_opaque_gateway_c8_dynamics_lib;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c8_dynamics_lib;
  chartInstance->chartInfo.mdlStart = sf_opaque_mdl_start_c8_dynamics_lib;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c8_dynamics_lib;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c8_dynamics_lib;
  chartInstance->chartInfo.enableChart = sf_opaque_enable_c8_dynamics_lib;
  chartInstance->chartInfo.disableChart = sf_opaque_disable_c8_dynamics_lib;
  chartInstance->chartInfo.getSimState = sf_opaque_get_sim_state_c8_dynamics_lib;
  chartInstance->chartInfo.setSimState = sf_opaque_set_sim_state_c8_dynamics_lib;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c8_dynamics_lib;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c8_dynamics_lib;
  chartInstance->chartInfo.mdlSetWorkWidths = mdlSetWorkWidths_c8_dynamics_lib;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c8_dynamics_lib(chartInstance);
}

void c8_dynamics_lib_method_dispatcher(SimStruct *S, int_T method, void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c8_dynamics_lib(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c8_dynamics_lib(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c8_dynamics_lib(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c8_dynamics_lib_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
