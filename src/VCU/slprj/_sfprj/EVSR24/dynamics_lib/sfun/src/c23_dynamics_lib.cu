/* Include files */

#include "dynamics_lib_sfun.h"
#include "c23_dynamics_lib.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c23_dynamics_lib(SFc23_dynamics_libInstanceStruct
  *chartInstance);
static void initialize_params_c23_dynamics_lib(SFc23_dynamics_libInstanceStruct *
  chartInstance);
static void mdl_start_c23_dynamics_lib(SFc23_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_terminate_c23_dynamics_lib(SFc23_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_setup_runtime_resources_c23_dynamics_lib
  (SFc23_dynamics_libInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c23_dynamics_lib
  (SFc23_dynamics_libInstanceStruct *chartInstance);
static void enable_c23_dynamics_lib(SFc23_dynamics_libInstanceStruct
  *chartInstance);
static void disable_c23_dynamics_lib(SFc23_dynamics_libInstanceStruct
  *chartInstance);
static void sf_gateway_c23_dynamics_lib(SFc23_dynamics_libInstanceStruct
  *chartInstance);
static void ext_mode_exec_c23_dynamics_lib(SFc23_dynamics_libInstanceStruct
  *chartInstance);
static void c23_do_animation_call_c23_dynamics_lib
  (SFc23_dynamics_libInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c23_dynamics_lib
  (SFc23_dynamics_libInstanceStruct *chartInstance);
static void set_sim_state_c23_dynamics_lib(SFc23_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c23_st);
static void initSimStructsc23_dynamics_lib(SFc23_dynamics_libInstanceStruct
  *chartInstance);
static void initSubchartIOPointersc23_dynamics_lib
  (SFc23_dynamics_libInstanceStruct *chartInstance);
static real32_T c23_eML_blk_kernel(SFc23_dynamics_libInstanceStruct
  *chartInstance, real_T c23_b_totalMass, real32_T c23_b_totalMassDistribution,
  real_T c23_b_totalDownForce, real32_T c23_b_downForceDistribution, real32_T
  c23_b_totalRearWT, real_T c23_b_gravity);
static real32_T c23_emlrt_marshallIn(SFc23_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c23_nullptr, const char_T *c23_identifier);
static real32_T c23_b_emlrt_marshallIn(SFc23_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c23_u, const emlrtMsgIdentifier *c23_parentId);
static uint8_T c23_c_emlrt_marshallIn(SFc23_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c23_nullptr, const char_T *c23_identifier);
static uint8_T c23_d_emlrt_marshallIn(SFc23_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c23_u, const emlrtMsgIdentifier *c23_parentId);
static void init_dsm_address_info(SFc23_dynamics_libInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc23_dynamics_libInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c23_dynamics_lib(SFc23_dynamics_libInstanceStruct
  *chartInstance)
{
  emlrtLicenseCheckR2022a(chartInstance->c23_fEmlrtCtx,
    "EMLRT:runTime:MexFunctionNeedsLicense", "distrib_computing_toolbox", 2);
  sf_is_first_init_cond(chartInstance->S);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  emlrtInitGPU(chartInstance->c23_fEmlrtCtx);
  hipGetLastError();
}

static void initialize_params_c23_dynamics_lib(SFc23_dynamics_libInstanceStruct *
  chartInstance)
{
}

static void mdl_start_c23_dynamics_lib(SFc23_dynamics_libInstanceStruct
  *chartInstance)
{
  sim_mode_is_external(chartInstance->S);
}

static void mdl_terminate_c23_dynamics_lib(SFc23_dynamics_libInstanceStruct
  *chartInstance)
{
  hipError_t c23_errCode;
  c23_errCode = hipGetLastError();
  if (c23_errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(c23_errCode), (char_T *)
                       hipGetErrorName(c23_errCode), (char_T *)
                       hipGetErrorString(c23_errCode), (char_T *)
                       "SimGPUErrorChecks", chartInstance->c23_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c23_dynamics_lib
  (SFc23_dynamics_libInstanceStruct *chartInstance)
{
  sfSetAnimationVectors(chartInstance->S, chartInstance->c23_JITStateAnimation,
                        chartInstance->c23_JITTransitionAnimation);
}

static void mdl_cleanup_runtime_resources_c23_dynamics_lib
  (SFc23_dynamics_libInstanceStruct *chartInstance)
{
}

static void enable_c23_dynamics_lib(SFc23_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c23_dynamics_lib(SFc23_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void sf_gateway_c23_dynamics_lib(SFc23_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c23_JITTransitionAnimation[0] = 0U;
  *chartInstance->c23_dynamicLoadRL = c23_eML_blk_kernel(chartInstance,
    *chartInstance->c23_totalMass, *chartInstance->c23_totalMassDistribution,
    *chartInstance->c23_totalDownForce,
    *chartInstance->c23_downForceDistribution, *chartInstance->c23_totalRearWT, *
    chartInstance->c23_gravity);
  c23_do_animation_call_c23_dynamics_lib(chartInstance);
}

static void ext_mode_exec_c23_dynamics_lib(SFc23_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void c23_do_animation_call_c23_dynamics_lib
  (SFc23_dynamics_libInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static const mxArray *get_sim_state_c23_dynamics_lib
  (SFc23_dynamics_libInstanceStruct *chartInstance)
{
  const mxArray *c23_b_y = NULL;
  const mxArray *c23_c_y = NULL;
  const mxArray *c23_st;
  const mxArray *c23_y = NULL;
  c23_st = NULL;
  c23_st = NULL;
  c23_y = NULL;
  sf_mex_assign(&c23_y, sf_mex_createcellmatrix(2, 1), false);
  c23_b_y = NULL;
  sf_mex_assign(&c23_b_y, sf_mex_create("y", chartInstance->c23_dynamicLoadRL, 1,
    0U, 0U, 0U, 0), false);
  sf_mex_setcell(c23_y, 0, c23_b_y);
  c23_c_y = NULL;
  sf_mex_assign(&c23_c_y, sf_mex_create("y",
    &chartInstance->c23_is_active_c23_dynamics_lib, 3, 0U, 0U, 0U, 0), false);
  sf_mex_setcell(c23_y, 1, c23_c_y);
  sf_mex_assign(&c23_st, c23_y, false);
  return c23_st;
}

static void set_sim_state_c23_dynamics_lib(SFc23_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c23_st)
{
  const mxArray *c23_u;
  c23_u = sf_mex_dup(c23_st);
  *chartInstance->c23_dynamicLoadRL = c23_emlrt_marshallIn(chartInstance,
    sf_mex_dup(sf_mex_getcell(c23_u, 0)), "dynamicLoadRL");
  chartInstance->c23_is_active_c23_dynamics_lib = c23_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c23_u, 1)),
     "is_active_c23_dynamics_lib");
  sf_mex_destroy(&c23_u);
  sf_mex_destroy(&c23_st);
}

static void initSimStructsc23_dynamics_lib(SFc23_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void initSubchartIOPointersc23_dynamics_lib
  (SFc23_dynamics_libInstanceStruct *chartInstance)
{
}

const mxArray *sf_c23_dynamics_lib_get_eml_resolved_functions_info()
{
  const mxArray *c23_nameCaptureInfo = NULL;
  c23_nameCaptureInfo = NULL;
  sf_mex_assign(&c23_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c23_nameCaptureInfo;
}

static real32_T c23_eML_blk_kernel(SFc23_dynamics_libInstanceStruct
  *chartInstance, real_T c23_b_totalMass, real32_T c23_b_totalMassDistribution,
  real_T c23_b_totalDownForce, real32_T c23_b_downForceDistribution, real32_T
  c23_b_totalRearWT, real_T c23_b_gravity)
{
  return ((static_cast<real32_T>(c23_b_totalMass) * (100.0F -
            c23_b_totalMassDistribution) / 100.0F * 0.5F + static_cast<real32_T>
           (c23_b_totalDownForce) * (100.0F - c23_b_downForceDistribution) /
           100.0F / 2.0F) - c23_b_totalRearWT) * static_cast<real32_T>
    (c23_b_gravity);
}

static real32_T c23_emlrt_marshallIn(SFc23_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c23_nullptr, const char_T *c23_identifier)
{
  emlrtMsgIdentifier c23_thisId;
  real32_T c23_y;
  c23_thisId.fIdentifier = const_cast<const char_T *>(c23_identifier);
  c23_thisId.fParent = NULL;
  c23_thisId.bParentIsCell = false;
  c23_y = c23_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c23_nullptr),
    &c23_thisId);
  sf_mex_destroy(&c23_nullptr);
  return c23_y;
}

static real32_T c23_b_emlrt_marshallIn(SFc23_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c23_u, const emlrtMsgIdentifier *c23_parentId)
{
  real32_T c23_f;
  real32_T c23_y;
  sf_mex_import(c23_parentId, sf_mex_dup(c23_u), &c23_f, 0, 1, 0U, 0, 0U, 0);
  c23_y = c23_f;
  sf_mex_destroy(&c23_u);
  return c23_y;
}

static uint8_T c23_c_emlrt_marshallIn(SFc23_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c23_nullptr, const char_T *c23_identifier)
{
  emlrtMsgIdentifier c23_thisId;
  uint8_T c23_y;
  c23_thisId.fIdentifier = const_cast<const char_T *>(c23_identifier);
  c23_thisId.fParent = NULL;
  c23_thisId.bParentIsCell = false;
  c23_y = c23_d_emlrt_marshallIn(chartInstance, sf_mex_dup(c23_nullptr),
    &c23_thisId);
  sf_mex_destroy(&c23_nullptr);
  return c23_y;
}

static uint8_T c23_d_emlrt_marshallIn(SFc23_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c23_u, const emlrtMsgIdentifier *c23_parentId)
{
  uint8_T c23_b_u;
  uint8_T c23_y;
  sf_mex_import(c23_parentId, sf_mex_dup(c23_u), &c23_b_u, 1, 3, 0U, 0, 0U, 0);
  c23_y = c23_b_u;
  sf_mex_destroy(&c23_u);
  return c23_y;
}

static void init_dsm_address_info(SFc23_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc23_dynamics_libInstanceStruct
  *chartInstance)
{
  chartInstance->c23_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c23_dynamicLoadRL = (real32_T *)ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c23_totalMass = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c23_totalMassDistribution = (real32_T *)
    ssGetInputPortSignal_wrapper(chartInstance->S, 1);
  chartInstance->c23_totalDownForce = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 2);
  chartInstance->c23_downForceDistribution = (real32_T *)
    ssGetInputPortSignal_wrapper(chartInstance->S, 3);
  chartInstance->c23_totalRearWT = (real32_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 4);
  chartInstance->c23_gravity = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 5);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c23_dynamics_lib_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(103796622U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(832532404U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(3650621661U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(2062090635U);
}

mxArray *sf_c23_dynamics_lib_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c23_dynamics_lib_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("chartInfo");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c23_dynamics_lib_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c23_dynamics_lib(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiDmYGJgYAPRQMzEAAGsUD4jVIwRLs4CF1cA4pLKglSQeHFRsmcKkM5LzAXzE0s"
    "rPPPS8sHmWzAgzGfDYj4jkvmcUHEI+GBPmX4RB5B+ByT9LFj08yLpF4DyUyqBPslM9slPTAnygY"
    "YTLLzId4+CA2X6IfYHEPCPFJp/QPzM4vjE5JLMstT4ZCPjeKjviuNzMpMQ5oIAAKNnHao="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c23_dynamics_lib_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sN9KeYWMcIt6DcjNgjMRWlF";
}

static void sf_opaque_initialize_c23_dynamics_lib(void *chartInstanceVar)
{
  initialize_params_c23_dynamics_lib((SFc23_dynamics_libInstanceStruct*)
    chartInstanceVar);
  initialize_c23_dynamics_lib((SFc23_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c23_dynamics_lib(void *chartInstanceVar)
{
  enable_c23_dynamics_lib((SFc23_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c23_dynamics_lib(void *chartInstanceVar)
{
  disable_c23_dynamics_lib((SFc23_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c23_dynamics_lib(void *chartInstanceVar)
{
  sf_gateway_c23_dynamics_lib((SFc23_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c23_dynamics_lib(SimStruct* S)
{
  return get_sim_state_c23_dynamics_lib((SFc23_dynamics_libInstanceStruct *)
    sf_get_chart_instance_ptr(S));     /* raw sim ctx */
}

static void sf_opaque_set_sim_state_c23_dynamics_lib(SimStruct* S, const mxArray
  *st)
{
  set_sim_state_c23_dynamics_lib((SFc23_dynamics_libInstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c23_dynamics_lib(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc23_dynamics_libInstanceStruct*) chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_dynamics_lib_optimization_info();
    }

    mdl_cleanup_runtime_resources_c23_dynamics_lib
      ((SFc23_dynamics_libInstanceStruct*) chartInstanceVar);
    ((SFc23_dynamics_libInstanceStruct*) chartInstanceVar)->
      ~SFc23_dynamics_libInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c23_dynamics_lib(void *chartInstanceVar)
{
  mdl_start_c23_dynamics_lib((SFc23_dynamics_libInstanceStruct*)
    chartInstanceVar);
  if (chartInstanceVar) {
    sf_reset_warnings_ChartRunTimeInfo(((SFc23_dynamics_libInstanceStruct*)
      chartInstanceVar)->S);
  }
}

static void sf_opaque_mdl_terminate_c23_dynamics_lib(void *chartInstanceVar)
{
  mdl_terminate_c23_dynamics_lib((SFc23_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c23_dynamics_lib(SimStruct *S)
{
  mdlProcessParamsCommon(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c23_dynamics_lib((SFc23_dynamics_libInstanceStruct*)
      sf_get_chart_instance_ptr(S));
    initSubchartIOPointersc23_dynamics_lib((SFc23_dynamics_libInstanceStruct*)
      sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c23_dynamics_lib_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [21] = {
    "eNrtWE9vG0UUH1vBaqu2ygGJqkKiRy5IbSIQSKhN6j/Fqp1YWScRcIgmu8/eaWZnN/PHjjly4si",
    "RD8EH4CPwEfgQHLggceTNeu2atbF31hK0iJU2m9md37z3fvP+jUml3SV43cf74AEhNXzewrtKpt",
    "c72biycE/f75DPs/HPdwjx4wCGIDwzGLAb4nYJE/WopJEi7pegEZyAirnRLBZtMYiLY5kYgATh4",
    "wJJLLWTXMUiw5m4ahnhW8nqPGR+6IWx4cFzXJAGx4JP/k5uYnQPJTaYBF+3AAIdytgMwxanw/Us",
    "SD2uh+BfKRM5c6VAeyaxpqqu4ZolHJo34LeF0hRZUBvs9TTVUNc3biRbe5U3Q8dRwhkVxbkOqfI",
    "gQe/QcJoE+PfYaGSvoFyUd8kE1bFklDcjXg+pLIrtcdSzi27NnXnWkW7ApRkOmRhadqWJQKD96C",
    "cFuBrU4xFIOoRj4SbXt9Y1b9INnvtlQaxmEZxReeijHygI3GIQHVh5FLcW+riMExYEveTQVn3JR",
    "rhHrnmjbUOpVN4w0dSTVClsKrc5wl1VJeW2fFGnnCs3bD9OOjACnspvUE1LYKfyHcBKsaAfo3fY",
    "yHWMfiPYtYEMW49FwIp75SiHSovEESb8AnAW2TCAAGmeqz5faFMcGaXjqI6h3+h0CspbxraFBjm",
    "gPhTO15IyBahw6leOcgOmbCAhGlnSqZWFV5jGYCkoUQMjGuNYXiHHroXhNVc2EtzQEAyhARrSJN",
    "dE7z6j3BTUOVKYla17nCrMsm5yEWvjpxTYp34Iga2BjEMX8ywuUHSLlS2fh2jtiOlJA5QvWVI0k",
    "gwmdCx+lqX+JIFTcSXisWjJOPKyLmaNXwFg1qBSYCl7jqVUTlqofDGtJVz30+zu2jBYnqnm9NL6",
    "xgsQWA2trbZ6Ux+jqimw3USFtsF67BtsR4RiSmMLOGmmMRBM+2Dyug/eWdEH7y70wbvZ7e/tXwQ",
    "T7EaZry44u7TrPF5Y526BfnrWh6/DkSUcmeNmzw8X8JUVcsnC087/dGH+7epf59dy8qqzd+lFlv",
    "i6l5O3k8PXMr6uf/zh6YPvf/3u92+9/Scvvz7L253Xo7KkRyX93+J+qrqdW+5n4/dnPdM8Q4+Wk",
    "pid+8UGf3gv5w92rI4+ewlfnnf9tv6k4b86Gr7qnpzzVrrerQ36VnP6zt4/sn0aBnCaD6TfDrLz",
    "jx1TM+3L8/tZ28DH7QW/IuS3Z9vh3z0oEj/3cnzZcRY7nZgGJ52cP5fX59HBdvip/N4Gex7m7Hm",
    "Y9vEX1GZruMhnhuV4Lev3rri3Rd7/uOL7UKSu1Eriysoj/zBuW/tc6+XbPv/xmnxGcvN332A71u",
    "VRl77pTbPrF+LWz3yQjZ/Oz+r1kPFgxWkh+4wN/WDV1/+If//hyN+sf2ta/rIfSL/aPxSUT/B4M",
    "D1+Za970v42N/8kgarVZ7B/o57Mf8ff0K/czcW3HY+ZCOKx+ujJ3sd729SnPwEhtQpG",
    ""
  };

  static char newstr [1493] = "";
  newstr[0] = '\0';
  for (i = 0; i < 21; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c23_dynamics_lib(SimStruct *S)
{
  const char* newstr = sf_c23_dynamics_lib_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(1050193009U));
  ssSetChecksum1(S,(2297204760U));
  ssSetChecksum2(S,(861111029U));
  ssSetChecksum3(S,(1448823601U));
}

static void mdlRTW_c23_dynamics_lib(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c23_dynamics_lib(SimStruct *S)
{
  SFc23_dynamics_libInstanceStruct *chartInstance;
  chartInstance = (SFc23_dynamics_libInstanceStruct *)utMalloc(sizeof
    (SFc23_dynamics_libInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc23_dynamics_libInstanceStruct));
  chartInstance = new (chartInstance) SFc23_dynamics_libInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway = sf_opaque_gateway_c23_dynamics_lib;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c23_dynamics_lib;
  chartInstance->chartInfo.mdlStart = sf_opaque_mdl_start_c23_dynamics_lib;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c23_dynamics_lib;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c23_dynamics_lib;
  chartInstance->chartInfo.enableChart = sf_opaque_enable_c23_dynamics_lib;
  chartInstance->chartInfo.disableChart = sf_opaque_disable_c23_dynamics_lib;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c23_dynamics_lib;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c23_dynamics_lib;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c23_dynamics_lib;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c23_dynamics_lib;
  chartInstance->chartInfo.mdlSetWorkWidths = mdlSetWorkWidths_c23_dynamics_lib;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c23_dynamics_lib(chartInstance);
}

void c23_dynamics_lib_method_dispatcher(SimStruct *S, int_T method, void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c23_dynamics_lib(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c23_dynamics_lib(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c23_dynamics_lib(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c23_dynamics_lib_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
