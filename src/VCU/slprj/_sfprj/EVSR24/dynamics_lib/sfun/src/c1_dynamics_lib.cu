/* Include files */

#include "dynamics_lib_sfun.h"
#include "c1_dynamics_lib.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c1_dynamics_lib(SFc1_dynamics_libInstanceStruct
  *chartInstance);
static void initialize_params_c1_dynamics_lib(SFc1_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_start_c1_dynamics_lib(SFc1_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_terminate_c1_dynamics_lib(SFc1_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_setup_runtime_resources_c1_dynamics_lib
  (SFc1_dynamics_libInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c1_dynamics_lib
  (SFc1_dynamics_libInstanceStruct *chartInstance);
static void enable_c1_dynamics_lib(SFc1_dynamics_libInstanceStruct
  *chartInstance);
static void disable_c1_dynamics_lib(SFc1_dynamics_libInstanceStruct
  *chartInstance);
static void sf_gateway_c1_dynamics_lib(SFc1_dynamics_libInstanceStruct
  *chartInstance);
static void ext_mode_exec_c1_dynamics_lib(SFc1_dynamics_libInstanceStruct
  *chartInstance);
static void c1_do_animation_call_c1_dynamics_lib(SFc1_dynamics_libInstanceStruct
  *chartInstance);
static const mxArray *get_sim_state_c1_dynamics_lib
  (SFc1_dynamics_libInstanceStruct *chartInstance);
static void set_sim_state_c1_dynamics_lib(SFc1_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c1_st);
static void initSimStructsc1_dynamics_lib(SFc1_dynamics_libInstanceStruct
  *chartInstance);
static void initSubchartIOPointersc1_dynamics_lib
  (SFc1_dynamics_libInstanceStruct *chartInstance);
static real_T c1_eML_blk_kernel(SFc1_dynamics_libInstanceStruct *chartInstance,
  real_T c1_b_wheelBase, real_T c1_b_suspendedMassWeightDistribution, real_T
  c1_b_suspendedMassCGCoordinatesZ, real_T c1_b_frontRollCenterHeight, real_T
  c1_b_rearRollCenterHeight);
static real_T c1_emlrt_marshallIn(SFc1_dynamics_libInstanceStruct *chartInstance,
  const mxArray *c1_nullptr, const char_T *c1_identifier);
static real_T c1_b_emlrt_marshallIn(SFc1_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c1_u, const emlrtMsgIdentifier *c1_parentId);
static uint8_T c1_c_emlrt_marshallIn(SFc1_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c1_nullptr, const char_T *c1_identifier);
static uint8_T c1_d_emlrt_marshallIn(SFc1_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c1_u, const emlrtMsgIdentifier *c1_parentId);
static void init_dsm_address_info(SFc1_dynamics_libInstanceStruct *chartInstance);
static void init_simulink_io_address(SFc1_dynamics_libInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c1_dynamics_lib(SFc1_dynamics_libInstanceStruct
  *chartInstance)
{
  emlrtLicenseCheckR2022a(chartInstance->c1_fEmlrtCtx,
    "EMLRT:runTime:MexFunctionNeedsLicense", "distrib_computing_toolbox", 2);
  sf_is_first_init_cond(chartInstance->S);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  emlrtInitGPU(chartInstance->c1_fEmlrtCtx);
  hipGetLastError();
}

static void initialize_params_c1_dynamics_lib(SFc1_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void mdl_start_c1_dynamics_lib(SFc1_dynamics_libInstanceStruct
  *chartInstance)
{
  sim_mode_is_external(chartInstance->S);
}

static void mdl_terminate_c1_dynamics_lib(SFc1_dynamics_libInstanceStruct
  *chartInstance)
{
  hipError_t c1_errCode;
  c1_errCode = hipGetLastError();
  if (c1_errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(c1_errCode), (char_T *)
                       hipGetErrorName(c1_errCode), (char_T *)
                       hipGetErrorString(c1_errCode), (char_T *)
                       "SimGPUErrorChecks", chartInstance->c1_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c1_dynamics_lib
  (SFc1_dynamics_libInstanceStruct *chartInstance)
{
  sfSetAnimationVectors(chartInstance->S, chartInstance->c1_JITStateAnimation,
                        chartInstance->c1_JITTransitionAnimation);
}

static void mdl_cleanup_runtime_resources_c1_dynamics_lib
  (SFc1_dynamics_libInstanceStruct *chartInstance)
{
}

static void enable_c1_dynamics_lib(SFc1_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c1_dynamics_lib(SFc1_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void sf_gateway_c1_dynamics_lib(SFc1_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c1_JITTransitionAnimation[0] = 0U;
  *chartInstance->c1_deltaZ = c1_eML_blk_kernel(chartInstance,
    *chartInstance->c1_wheelBase,
    *chartInstance->c1_suspendedMassWeightDistribution,
    *chartInstance->c1_suspendedMassCGCoordinatesZ,
    *chartInstance->c1_frontRollCenterHeight,
    *chartInstance->c1_rearRollCenterHeight);
  c1_do_animation_call_c1_dynamics_lib(chartInstance);
}

static void ext_mode_exec_c1_dynamics_lib(SFc1_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void c1_do_animation_call_c1_dynamics_lib(SFc1_dynamics_libInstanceStruct
  *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static const mxArray *get_sim_state_c1_dynamics_lib
  (SFc1_dynamics_libInstanceStruct *chartInstance)
{
  const mxArray *c1_b_y = NULL;
  const mxArray *c1_c_y = NULL;
  const mxArray *c1_st;
  const mxArray *c1_y = NULL;
  c1_st = NULL;
  c1_st = NULL;
  c1_y = NULL;
  sf_mex_assign(&c1_y, sf_mex_createcellmatrix(2, 1), false);
  c1_b_y = NULL;
  sf_mex_assign(&c1_b_y, sf_mex_create("y", chartInstance->c1_deltaZ, 0, 0U, 0U,
    0U, 0), false);
  sf_mex_setcell(c1_y, 0, c1_b_y);
  c1_c_y = NULL;
  sf_mex_assign(&c1_c_y, sf_mex_create("y",
    &chartInstance->c1_is_active_c1_dynamics_lib, 3, 0U, 0U, 0U, 0), false);
  sf_mex_setcell(c1_y, 1, c1_c_y);
  sf_mex_assign(&c1_st, c1_y, false);
  return c1_st;
}

static void set_sim_state_c1_dynamics_lib(SFc1_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c1_st)
{
  const mxArray *c1_u;
  c1_u = sf_mex_dup(c1_st);
  *chartInstance->c1_deltaZ = c1_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getcell(c1_u, 0)), "deltaZ");
  chartInstance->c1_is_active_c1_dynamics_lib = c1_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c1_u, 1)),
     "is_active_c1_dynamics_lib");
  sf_mex_destroy(&c1_u);
  sf_mex_destroy(&c1_st);
}

static void initSimStructsc1_dynamics_lib(SFc1_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void initSubchartIOPointersc1_dynamics_lib
  (SFc1_dynamics_libInstanceStruct *chartInstance)
{
}

const mxArray *sf_c1_dynamics_lib_get_eml_resolved_functions_info()
{
  const mxArray *c1_nameCaptureInfo = NULL;
  c1_nameCaptureInfo = NULL;
  sf_mex_assign(&c1_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c1_nameCaptureInfo;
}

static real_T c1_eML_blk_kernel(SFc1_dynamics_libInstanceStruct *chartInstance,
  real_T c1_b_wheelBase, real_T c1_b_suspendedMassWeightDistribution, real_T
  c1_b_suspendedMassCGCoordinatesZ, real_T c1_b_frontRollCenterHeight, real_T
  c1_b_rearRollCenterHeight)
{
  return c1_b_suspendedMassCGCoordinatesZ - ((c1_b_rearRollCenterHeight -
    c1_b_frontRollCenterHeight) / c1_b_wheelBase * ((100.0 -
    c1_b_suspendedMassWeightDistribution) / 100.0 * c1_b_wheelBase) +
    c1_b_frontRollCenterHeight);
}

static real_T c1_emlrt_marshallIn(SFc1_dynamics_libInstanceStruct *chartInstance,
  const mxArray *c1_nullptr, const char_T *c1_identifier)
{
  emlrtMsgIdentifier c1_thisId;
  real_T c1_y;
  c1_thisId.fIdentifier = const_cast<const char_T *>(c1_identifier);
  c1_thisId.fParent = NULL;
  c1_thisId.bParentIsCell = false;
  c1_y = c1_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c1_nullptr), &c1_thisId);
  sf_mex_destroy(&c1_nullptr);
  return c1_y;
}

static real_T c1_b_emlrt_marshallIn(SFc1_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c1_u, const emlrtMsgIdentifier *c1_parentId)
{
  real_T c1_d;
  real_T c1_y;
  sf_mex_import(c1_parentId, sf_mex_dup(c1_u), &c1_d, 1, 0, 0U, 0, 0U, 0);
  c1_y = c1_d;
  sf_mex_destroy(&c1_u);
  return c1_y;
}

static uint8_T c1_c_emlrt_marshallIn(SFc1_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c1_nullptr, const char_T *c1_identifier)
{
  emlrtMsgIdentifier c1_thisId;
  uint8_T c1_y;
  c1_thisId.fIdentifier = const_cast<const char_T *>(c1_identifier);
  c1_thisId.fParent = NULL;
  c1_thisId.bParentIsCell = false;
  c1_y = c1_d_emlrt_marshallIn(chartInstance, sf_mex_dup(c1_nullptr), &c1_thisId);
  sf_mex_destroy(&c1_nullptr);
  return c1_y;
}

static uint8_T c1_d_emlrt_marshallIn(SFc1_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c1_u, const emlrtMsgIdentifier *c1_parentId)
{
  uint8_T c1_b_u;
  uint8_T c1_y;
  sf_mex_import(c1_parentId, sf_mex_dup(c1_u), &c1_b_u, 1, 3, 0U, 0, 0U, 0);
  c1_y = c1_b_u;
  sf_mex_destroy(&c1_u);
  return c1_y;
}

static void init_dsm_address_info(SFc1_dynamics_libInstanceStruct *chartInstance)
{
}

static void init_simulink_io_address(SFc1_dynamics_libInstanceStruct
  *chartInstance)
{
  chartInstance->c1_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c1_wheelBase = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c1_deltaZ = (real_T *)ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c1_suspendedMassWeightDistribution = (real_T *)
    ssGetInputPortSignal_wrapper(chartInstance->S, 1);
  chartInstance->c1_suspendedMassCGCoordinatesZ = (real_T *)
    ssGetInputPortSignal_wrapper(chartInstance->S, 2);
  chartInstance->c1_frontRollCenterHeight = (real_T *)
    ssGetInputPortSignal_wrapper(chartInstance->S, 3);
  chartInstance->c1_rearRollCenterHeight = (real_T *)
    ssGetInputPortSignal_wrapper(chartInstance->S, 4);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c1_dynamics_lib_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(114917123U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(904154887U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(1546643183U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(4111119175U);
}

mxArray *sf_c1_dynamics_lib_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c1_dynamics_lib_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("chartInfo");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c1_dynamics_lib_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c1_dynamics_lib(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0izig62fBop8NSb8AlJ+SmlOSGAUNH2g4ke8OBQfK9E"
    "PsDyDgD0k0f4D4mcXxicklmWWp8cmG8SmVwOjJTC6Oz8lMQjIXBACSpRq3"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c1_dynamics_lib_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sfSTahroaTAfMgdzbanaRZD";
}

static void sf_opaque_initialize_c1_dynamics_lib(void *chartInstanceVar)
{
  initialize_params_c1_dynamics_lib((SFc1_dynamics_libInstanceStruct*)
    chartInstanceVar);
  initialize_c1_dynamics_lib((SFc1_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c1_dynamics_lib(void *chartInstanceVar)
{
  enable_c1_dynamics_lib((SFc1_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c1_dynamics_lib(void *chartInstanceVar)
{
  disable_c1_dynamics_lib((SFc1_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c1_dynamics_lib(void *chartInstanceVar)
{
  sf_gateway_c1_dynamics_lib((SFc1_dynamics_libInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c1_dynamics_lib(SimStruct* S)
{
  return get_sim_state_c1_dynamics_lib((SFc1_dynamics_libInstanceStruct *)
    sf_get_chart_instance_ptr(S));     /* raw sim ctx */
}

static void sf_opaque_set_sim_state_c1_dynamics_lib(SimStruct* S, const mxArray *
  st)
{
  set_sim_state_c1_dynamics_lib((SFc1_dynamics_libInstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c1_dynamics_lib(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc1_dynamics_libInstanceStruct*) chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_dynamics_lib_optimization_info();
    }

    mdl_cleanup_runtime_resources_c1_dynamics_lib
      ((SFc1_dynamics_libInstanceStruct*) chartInstanceVar);
    ((SFc1_dynamics_libInstanceStruct*) chartInstanceVar)->
      ~SFc1_dynamics_libInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c1_dynamics_lib(void *chartInstanceVar)
{
  mdl_start_c1_dynamics_lib((SFc1_dynamics_libInstanceStruct*) chartInstanceVar);
  if (chartInstanceVar) {
    sf_reset_warnings_ChartRunTimeInfo(((SFc1_dynamics_libInstanceStruct*)
      chartInstanceVar)->S);
  }
}

static void sf_opaque_mdl_terminate_c1_dynamics_lib(void *chartInstanceVar)
{
  mdl_terminate_c1_dynamics_lib((SFc1_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c1_dynamics_lib(SimStruct *S)
{
  mdlProcessParamsCommon(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c1_dynamics_lib((SFc1_dynamics_libInstanceStruct*)
      sf_get_chart_instance_ptr(S));
    initSubchartIOPointersc1_dynamics_lib((SFc1_dynamics_libInstanceStruct*)
      sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c1_dynamics_lib_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [21] = {
    "eNrtWE2P20QYnkS7EUWl9IBUKiHRIxektoDEAcFu8wGRNt2wzhZphbSa2K/t0Y7H7nwkm/4Dzvw",
    "GTlw48wu4cOdnwA0JDrzjOGnqhMTjSNAiLDnW2PPM+zHvxzMhjf6A4HUL74/fJqSFz9fwbpL5dV",
    "iMGyv3/P0B+aQY//Q6IX4aQATCM2HIronbJUwypJImirhfgiZwBirlRrNU9EWYVscyEYIE4eMCW",
    "Sq1k1zFEsOZuOoZ4VvJ6quY+bEXp4YHj3BBGpwKPvs7uZnRQ5TYYRJ83QMIdCxTE8U9TqPtXpB6",
    "2o7Bv1ImcfaVAu2ZzJqqBoZrlnHoXoPfF0pT9ILaYa+nqYa2vnZzsrVXeQt0mmScUVHd1zFVHmQ",
    "YHRrOswB/T41G71WUi/LGTFCdSkZ5N+HtmMqq2CFHPQcY1tzZzzrRHRibKGIist6VJgGB9mOcVP",
    "BV2E4nIGkEp8JNrm+t617nG7yMy4pYzRJ4QuWxj3GgIHDLQQxg5VHcWhjhMk5YEHTMoa9Gkk1wj",
    "1zrRt+mUq26YZJ5JKla2Fxud4K7qmrK7fmiTTlXbthRmp3ABHguv0M1rYGdy3cAK8WCUYrRYTPX",
    "MfuNYE8NFNh2KgJWPSonJVTeJB5jwa8AZ4lNAwjQzUvVlwvtyiOjdJq0MfU7JycV5a1j+0KDDKk",
    "Pleu1pEwBKpzHlaPcgCmbSIhGL+ncysorzHOwFpSo0IjONJVX6GPXxvDcVzYT3NAQRNABDXmR62",
    "J0P6HcVNQ5UViVbXicK6yybnIRa/OnFtinfgyB7YGMwwDrLC5QdYuVbZ/HaO2E6VkHlC9ZVjWTD",
    "BZ0bH7WS6NZBufiSqRT0ZNp4hUsZktcAWDVoFJgK3uErVTOeqh8Na0lPB3l1d2VMFg/U83p2MbG",
    "5yCwG1pbbfemPmZVVyDdRIX2wXrsGdIRoZjSSAFn3TwHgpwHH5HnPPhgAw9+c4UH3y7G/oPLYIZ",
    "klPnqkrNxvs79lXVuVuDTCx6+DUfWcGSJWzzfW8E3NsglK8+c96/Mv9F8cf5hSV5z8c5eBX7VX2",
    "+U5B2U8K3CZz9/8/WfZ9+rD3/87jfz+I9vvyzbXdajsaZHgyzk/9B0O7fcKsbvLDjTskJP1oqYn",
    "fvFjni4U4qHO3ld9EYUST0dHYeDKHg2poKeXXTm+7RD32ZJ38X7e5anYQLn9UD6/aA4/9gxNXNe",
    "Xt7P1g5/3FiJK0J+/Ww//FtHZfwmf7VK/mrldYZrevFiHNfX497Rfvi5/OEOO+6W7Lib8/dLaqs",
    "0XJYrwnqe1o13V9yrIu9/XPV9qNJPDmvi6soj/zBuX/tc++SrPv/+lnpGSvNvv8R2bKujLnzpZb",
    "PrF+LGY94txp8uz+jtmPFgwymh+IxEPtz09T8S3787+m/B27rWf8UfoxcfHAvKZ3gsmB+7itdDa",
    "f+TW36SQNXms9e/0U/IBh6+ia/cLOW3HU+ZCNKpev/Bw48e7tOf/gIlNgnY",
    ""
  };

  static char newstr [1485] = "";
  newstr[0] = '\0';
  for (i = 0; i < 21; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c1_dynamics_lib(SimStruct *S)
{
  const char* newstr = sf_c1_dynamics_lib_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(4233923021U));
  ssSetChecksum1(S,(879995730U));
  ssSetChecksum2(S,(1978770874U));
  ssSetChecksum3(S,(1368652622U));
}

static void mdlRTW_c1_dynamics_lib(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c1_dynamics_lib(SimStruct *S)
{
  SFc1_dynamics_libInstanceStruct *chartInstance;
  chartInstance = (SFc1_dynamics_libInstanceStruct *)utMalloc(sizeof
    (SFc1_dynamics_libInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc1_dynamics_libInstanceStruct));
  chartInstance = new (chartInstance) SFc1_dynamics_libInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway = sf_opaque_gateway_c1_dynamics_lib;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c1_dynamics_lib;
  chartInstance->chartInfo.mdlStart = sf_opaque_mdl_start_c1_dynamics_lib;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c1_dynamics_lib;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c1_dynamics_lib;
  chartInstance->chartInfo.enableChart = sf_opaque_enable_c1_dynamics_lib;
  chartInstance->chartInfo.disableChart = sf_opaque_disable_c1_dynamics_lib;
  chartInstance->chartInfo.getSimState = sf_opaque_get_sim_state_c1_dynamics_lib;
  chartInstance->chartInfo.setSimState = sf_opaque_set_sim_state_c1_dynamics_lib;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c1_dynamics_lib;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c1_dynamics_lib;
  chartInstance->chartInfo.mdlSetWorkWidths = mdlSetWorkWidths_c1_dynamics_lib;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c1_dynamics_lib(chartInstance);
}

void c1_dynamics_lib_method_dispatcher(SimStruct *S, int_T method, void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c1_dynamics_lib(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c1_dynamics_lib(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c1_dynamics_lib(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c1_dynamics_lib_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
