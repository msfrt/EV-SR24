/* Include files */

#include "dynamics_lib_sfun.h"
#include "c21_dynamics_lib.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c21_dynamics_lib(SFc21_dynamics_libInstanceStruct
  *chartInstance);
static void initialize_params_c21_dynamics_lib(SFc21_dynamics_libInstanceStruct *
  chartInstance);
static void mdl_start_c21_dynamics_lib(SFc21_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_terminate_c21_dynamics_lib(SFc21_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_setup_runtime_resources_c21_dynamics_lib
  (SFc21_dynamics_libInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c21_dynamics_lib
  (SFc21_dynamics_libInstanceStruct *chartInstance);
static void enable_c21_dynamics_lib(SFc21_dynamics_libInstanceStruct
  *chartInstance);
static void disable_c21_dynamics_lib(SFc21_dynamics_libInstanceStruct
  *chartInstance);
static void sf_gateway_c21_dynamics_lib(SFc21_dynamics_libInstanceStruct
  *chartInstance);
static void ext_mode_exec_c21_dynamics_lib(SFc21_dynamics_libInstanceStruct
  *chartInstance);
static void c21_do_animation_call_c21_dynamics_lib
  (SFc21_dynamics_libInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c21_dynamics_lib
  (SFc21_dynamics_libInstanceStruct *chartInstance);
static void set_sim_state_c21_dynamics_lib(SFc21_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c21_st);
static void initSimStructsc21_dynamics_lib(SFc21_dynamics_libInstanceStruct
  *chartInstance);
static void initSubchartIOPointersc21_dynamics_lib
  (SFc21_dynamics_libInstanceStruct *chartInstance);
static real32_T c21_eML_blk_kernel(SFc21_dynamics_libInstanceStruct
  *chartInstance, real_T c21_b_totalMass, real32_T c21_b_totalMassDistribution,
  real_T c21_b_totalDownForce, real32_T c21_b_downForceDistribution, real32_T
  c21_b_totalFrontWT, real_T c21_b_gravity);
static real32_T c21_emlrt_marshallIn(SFc21_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c21_nullptr, const char_T *c21_identifier);
static real32_T c21_b_emlrt_marshallIn(SFc21_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c21_u, const emlrtMsgIdentifier *c21_parentId);
static uint8_T c21_c_emlrt_marshallIn(SFc21_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c21_nullptr, const char_T *c21_identifier);
static uint8_T c21_d_emlrt_marshallIn(SFc21_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c21_u, const emlrtMsgIdentifier *c21_parentId);
static void init_dsm_address_info(SFc21_dynamics_libInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc21_dynamics_libInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c21_dynamics_lib(SFc21_dynamics_libInstanceStruct
  *chartInstance)
{
  emlrtLicenseCheckR2022a(chartInstance->c21_fEmlrtCtx,
    "EMLRT:runTime:MexFunctionNeedsLicense", "distrib_computing_toolbox", 2);
  sf_is_first_init_cond(chartInstance->S);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  emlrtInitGPU(chartInstance->c21_fEmlrtCtx);
  hipGetLastError();
}

static void initialize_params_c21_dynamics_lib(SFc21_dynamics_libInstanceStruct *
  chartInstance)
{
}

static void mdl_start_c21_dynamics_lib(SFc21_dynamics_libInstanceStruct
  *chartInstance)
{
  sim_mode_is_external(chartInstance->S);
}

static void mdl_terminate_c21_dynamics_lib(SFc21_dynamics_libInstanceStruct
  *chartInstance)
{
  hipError_t c21_errCode;
  c21_errCode = hipGetLastError();
  if (c21_errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(c21_errCode), (char_T *)
                       hipGetErrorName(c21_errCode), (char_T *)
                       hipGetErrorString(c21_errCode), (char_T *)
                       "SimGPUErrorChecks", chartInstance->c21_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c21_dynamics_lib
  (SFc21_dynamics_libInstanceStruct *chartInstance)
{
  sfSetAnimationVectors(chartInstance->S, chartInstance->c21_JITStateAnimation,
                        chartInstance->c21_JITTransitionAnimation);
}

static void mdl_cleanup_runtime_resources_c21_dynamics_lib
  (SFc21_dynamics_libInstanceStruct *chartInstance)
{
}

static void enable_c21_dynamics_lib(SFc21_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c21_dynamics_lib(SFc21_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void sf_gateway_c21_dynamics_lib(SFc21_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c21_JITTransitionAnimation[0] = 0U;
  *chartInstance->c21_dynamicLoadFL = c21_eML_blk_kernel(chartInstance,
    *chartInstance->c21_totalMass, *chartInstance->c21_totalMassDistribution,
    *chartInstance->c21_totalDownForce,
    *chartInstance->c21_downForceDistribution, *chartInstance->c21_totalFrontWT,
    *chartInstance->c21_gravity);
  c21_do_animation_call_c21_dynamics_lib(chartInstance);
}

static void ext_mode_exec_c21_dynamics_lib(SFc21_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void c21_do_animation_call_c21_dynamics_lib
  (SFc21_dynamics_libInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static const mxArray *get_sim_state_c21_dynamics_lib
  (SFc21_dynamics_libInstanceStruct *chartInstance)
{
  const mxArray *c21_b_y = NULL;
  const mxArray *c21_c_y = NULL;
  const mxArray *c21_st;
  const mxArray *c21_y = NULL;
  c21_st = NULL;
  c21_st = NULL;
  c21_y = NULL;
  sf_mex_assign(&c21_y, sf_mex_createcellmatrix(2, 1), false);
  c21_b_y = NULL;
  sf_mex_assign(&c21_b_y, sf_mex_create("y", chartInstance->c21_dynamicLoadFL, 1,
    0U, 0U, 0U, 0), false);
  sf_mex_setcell(c21_y, 0, c21_b_y);
  c21_c_y = NULL;
  sf_mex_assign(&c21_c_y, sf_mex_create("y",
    &chartInstance->c21_is_active_c21_dynamics_lib, 3, 0U, 0U, 0U, 0), false);
  sf_mex_setcell(c21_y, 1, c21_c_y);
  sf_mex_assign(&c21_st, c21_y, false);
  return c21_st;
}

static void set_sim_state_c21_dynamics_lib(SFc21_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c21_st)
{
  const mxArray *c21_u;
  c21_u = sf_mex_dup(c21_st);
  *chartInstance->c21_dynamicLoadFL = c21_emlrt_marshallIn(chartInstance,
    sf_mex_dup(sf_mex_getcell(c21_u, 0)), "dynamicLoadFL");
  chartInstance->c21_is_active_c21_dynamics_lib = c21_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c21_u, 1)),
     "is_active_c21_dynamics_lib");
  sf_mex_destroy(&c21_u);
  sf_mex_destroy(&c21_st);
}

static void initSimStructsc21_dynamics_lib(SFc21_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void initSubchartIOPointersc21_dynamics_lib
  (SFc21_dynamics_libInstanceStruct *chartInstance)
{
}

const mxArray *sf_c21_dynamics_lib_get_eml_resolved_functions_info()
{
  const mxArray *c21_nameCaptureInfo = NULL;
  c21_nameCaptureInfo = NULL;
  sf_mex_assign(&c21_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c21_nameCaptureInfo;
}

static real32_T c21_eML_blk_kernel(SFc21_dynamics_libInstanceStruct
  *chartInstance, real_T c21_b_totalMass, real32_T c21_b_totalMassDistribution,
  real_T c21_b_totalDownForce, real32_T c21_b_downForceDistribution, real32_T
  c21_b_totalFrontWT, real_T c21_b_gravity)
{
  return ((static_cast<real32_T>(c21_b_totalMass) * c21_b_totalMassDistribution /
           100.0F * 0.5F + static_cast<real32_T>(c21_b_totalDownForce) *
           c21_b_downForceDistribution / 100.0F / 2.0F) - c21_b_totalFrontWT) *
    static_cast<real32_T>(c21_b_gravity);
}

static real32_T c21_emlrt_marshallIn(SFc21_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c21_nullptr, const char_T *c21_identifier)
{
  emlrtMsgIdentifier c21_thisId;
  real32_T c21_y;
  c21_thisId.fIdentifier = const_cast<const char_T *>(c21_identifier);
  c21_thisId.fParent = NULL;
  c21_thisId.bParentIsCell = false;
  c21_y = c21_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c21_nullptr),
    &c21_thisId);
  sf_mex_destroy(&c21_nullptr);
  return c21_y;
}

static real32_T c21_b_emlrt_marshallIn(SFc21_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c21_u, const emlrtMsgIdentifier *c21_parentId)
{
  real32_T c21_f;
  real32_T c21_y;
  sf_mex_import(c21_parentId, sf_mex_dup(c21_u), &c21_f, 0, 1, 0U, 0, 0U, 0);
  c21_y = c21_f;
  sf_mex_destroy(&c21_u);
  return c21_y;
}

static uint8_T c21_c_emlrt_marshallIn(SFc21_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c21_nullptr, const char_T *c21_identifier)
{
  emlrtMsgIdentifier c21_thisId;
  uint8_T c21_y;
  c21_thisId.fIdentifier = const_cast<const char_T *>(c21_identifier);
  c21_thisId.fParent = NULL;
  c21_thisId.bParentIsCell = false;
  c21_y = c21_d_emlrt_marshallIn(chartInstance, sf_mex_dup(c21_nullptr),
    &c21_thisId);
  sf_mex_destroy(&c21_nullptr);
  return c21_y;
}

static uint8_T c21_d_emlrt_marshallIn(SFc21_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c21_u, const emlrtMsgIdentifier *c21_parentId)
{
  uint8_T c21_b_u;
  uint8_T c21_y;
  sf_mex_import(c21_parentId, sf_mex_dup(c21_u), &c21_b_u, 1, 3, 0U, 0, 0U, 0);
  c21_y = c21_b_u;
  sf_mex_destroy(&c21_u);
  return c21_y;
}

static void init_dsm_address_info(SFc21_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc21_dynamics_libInstanceStruct
  *chartInstance)
{
  chartInstance->c21_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c21_dynamicLoadFL = (real32_T *)ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c21_totalMass = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c21_totalMassDistribution = (real32_T *)
    ssGetInputPortSignal_wrapper(chartInstance->S, 1);
  chartInstance->c21_totalDownForce = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 2);
  chartInstance->c21_downForceDistribution = (real32_T *)
    ssGetInputPortSignal_wrapper(chartInstance->S, 3);
  chartInstance->c21_totalFrontWT = (real32_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 4);
  chartInstance->c21_gravity = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 5);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c21_dynamics_lib_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(2673803551U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(3444221098U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(1266948562U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(2428351321U);
}

mxArray *sf_c21_dynamics_lib_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c21_dynamics_lib_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("chartInfo");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c21_dynamics_lib_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c21_dynamics_lib(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiDmYGJgYAPRQMzEAAGsUD4jVIwRLs4CF1cA4pLKglSQeHFRsmcKkM5LzAXzE0s"
    "rPPPS8sHmWzAgzGfDYj4jkvmcUHEI+GBPmX4RB5B+ByT9LFj08yLpF4DyUyqBPslM9slPTHHzgY"
    "YTLLzId4+CA2X6IfYHEPCPFJp/QPzM4vjE5JLMstT4ZCPDeKjviuNzMpMQ5oIAAJgTHZw="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c21_dynamics_lib_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sY4GhcpqgfuFhGlNJtBVvd";
}

static void sf_opaque_initialize_c21_dynamics_lib(void *chartInstanceVar)
{
  initialize_params_c21_dynamics_lib((SFc21_dynamics_libInstanceStruct*)
    chartInstanceVar);
  initialize_c21_dynamics_lib((SFc21_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c21_dynamics_lib(void *chartInstanceVar)
{
  enable_c21_dynamics_lib((SFc21_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c21_dynamics_lib(void *chartInstanceVar)
{
  disable_c21_dynamics_lib((SFc21_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c21_dynamics_lib(void *chartInstanceVar)
{
  sf_gateway_c21_dynamics_lib((SFc21_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c21_dynamics_lib(SimStruct* S)
{
  return get_sim_state_c21_dynamics_lib((SFc21_dynamics_libInstanceStruct *)
    sf_get_chart_instance_ptr(S));     /* raw sim ctx */
}

static void sf_opaque_set_sim_state_c21_dynamics_lib(SimStruct* S, const mxArray
  *st)
{
  set_sim_state_c21_dynamics_lib((SFc21_dynamics_libInstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c21_dynamics_lib(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc21_dynamics_libInstanceStruct*) chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_dynamics_lib_optimization_info();
    }

    mdl_cleanup_runtime_resources_c21_dynamics_lib
      ((SFc21_dynamics_libInstanceStruct*) chartInstanceVar);
    ((SFc21_dynamics_libInstanceStruct*) chartInstanceVar)->
      ~SFc21_dynamics_libInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c21_dynamics_lib(void *chartInstanceVar)
{
  mdl_start_c21_dynamics_lib((SFc21_dynamics_libInstanceStruct*)
    chartInstanceVar);
  if (chartInstanceVar) {
    sf_reset_warnings_ChartRunTimeInfo(((SFc21_dynamics_libInstanceStruct*)
      chartInstanceVar)->S);
  }
}

static void sf_opaque_mdl_terminate_c21_dynamics_lib(void *chartInstanceVar)
{
  mdl_terminate_c21_dynamics_lib((SFc21_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c21_dynamics_lib(SimStruct *S)
{
  mdlProcessParamsCommon(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c21_dynamics_lib((SFc21_dynamics_libInstanceStruct*)
      sf_get_chart_instance_ptr(S));
    initSubchartIOPointersc21_dynamics_lib((SFc21_dynamics_libInstanceStruct*)
      sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c21_dynamics_lib_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [21] = {
    "eNrtWM1v40QUn4QS7cLuqgcEaIXEcuOCtFtAAoGgbT5KULqNSFoEl2piP8ejjsfufCQNF/4EpD1",
    "x5MqNI38CF+78GRw58sZ2ssEJiceRYBdhyXXHnt+8937zviak1j0leN3D+/B1Qhr4vIV3nWTXi/",
    "m4tnRn7/fIx/n4l5cI8WIfxiAGJgjYDXG7hIn6VNJIEfdL0Ai+ABVzo1ksuiKIy2OZCECC8HCBJ",
    "JbaSa5ikeFMXHWM8Kxk9WXIvHAQxob7x7gg9c8En/2d3MToPkpsMQme7gD4OpSxGYcdTsebWZB6",
    "2gzBu1ImcuZKgR6YxJqqTg3XLOHQvgGvK5SmyILaYu9AUw1NfeNGsrVXDeboOEo4o6I81yFVA0j",
    "QOzScJz7+PTMa2SspF+WNmKA6lozydsSbIZVlsX2Oep6iW3NnnnWkWzAy4zETY8uuNBEItB/9pA",
    "RXQTOegKRjOBNucj1rXfsm3eCFX5bEahbBBZVHHvqBAt8tBtGB1YDi1sIQl3HCgqAjDl01lGyCe",
    "+SaN7o2lCrlDRNlnqQqYVO57Qnuqqoot+OJJuVcuWGHcdKDCfBUfotqWgGbyXcAK8X8YYzeYSPX",
    "MfqNYNcGcmwzFj4r75WTAiotEo8x4ZeAs8iGAfhI80L1xULb4sgoHUdNDP1Wr1dS3iq2KzTIgHp",
    "QOl9LyhSgwqlfOcr1mbKBhGhkSadWll4hi8FKUKICI1rTWF4hx66F4SlXNhLc0OCPoQUa0iTXRu",
    "++oNyU1DlSmJWte5wrzLJuchFr46cS2KNeCL6tgYzDKeZZXKDsFitbPo/Q2gnTsxYoT7KkbCQZT",
    "OhY/CxLw1kC5+JKxFPRkXE0yLuYDX4FgFmDSoGl7BhLqZx1UPlyWku4HqbZ3bVhsDxTzenI+sYJ",
    "CKyG1lZbvamHUdUW2G6iQrtgB+wbbEeEYkpjCzhrpzHgZ30wedoH763pg/eX+uD9/PYOHl36M+x",
    "GmacuORvZdR4urXOnRD8978M34cgKjixw8+fbS/jaGrlk6Wnnf7A0/3b9r/MbBXn1+bv0Iit83S",
    "3I2yvgGzlfP377/Vs/vPDdRz/9+qF8+cmTZtHuoh61FT1q6f8W93Pd7dxyLx+/Me+ZFhl6spLE7",
    "NzPtvjDqwV/sGP11XsnoZdcjwPTCU/448/18cXEz9a7tUXfekHf+fsHtk/DAE7zgfS6fn7+sWNq",
    "sr68uJ+NLXzcXvIrQn7/dDf8K4dl4udugS87zmOnF1O/0yv4c3V9Hhzuhs/k97fYc79gz/20j7+",
    "kNlvDZTEzrMZrVb93xT0v8v7Hld+HMnWlURFXVR75h3G72udaL5/3+Q835DNSmL//DNuxKY+69E",
    "3Pml2/Ebd+5s18/MnirN4MGffXnBbyz9jQB+u+/kf8+w9H/l7Lx23LX/4D6dfvHgnKZ3g8yI5f+",
    "eu+tL/NLT5JoGr9GezfqCeL3/G39Ct3CvFtx1Mm/Hiq3nl08P7BLvXpT1/lCiI=",
    ""
  };

  static char newstr [1489] = "";
  newstr[0] = '\0';
  for (i = 0; i < 21; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c21_dynamics_lib(SimStruct *S)
{
  const char* newstr = sf_c21_dynamics_lib_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(563642281U));
  ssSetChecksum1(S,(999228320U));
  ssSetChecksum2(S,(1916390832U));
  ssSetChecksum3(S,(1133679115U));
}

static void mdlRTW_c21_dynamics_lib(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c21_dynamics_lib(SimStruct *S)
{
  SFc21_dynamics_libInstanceStruct *chartInstance;
  chartInstance = (SFc21_dynamics_libInstanceStruct *)utMalloc(sizeof
    (SFc21_dynamics_libInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc21_dynamics_libInstanceStruct));
  chartInstance = new (chartInstance) SFc21_dynamics_libInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway = sf_opaque_gateway_c21_dynamics_lib;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c21_dynamics_lib;
  chartInstance->chartInfo.mdlStart = sf_opaque_mdl_start_c21_dynamics_lib;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c21_dynamics_lib;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c21_dynamics_lib;
  chartInstance->chartInfo.enableChart = sf_opaque_enable_c21_dynamics_lib;
  chartInstance->chartInfo.disableChart = sf_opaque_disable_c21_dynamics_lib;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c21_dynamics_lib;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c21_dynamics_lib;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c21_dynamics_lib;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c21_dynamics_lib;
  chartInstance->chartInfo.mdlSetWorkWidths = mdlSetWorkWidths_c21_dynamics_lib;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c21_dynamics_lib(chartInstance);
}

void c21_dynamics_lib_method_dispatcher(SimStruct *S, int_T method, void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c21_dynamics_lib(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c21_dynamics_lib(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c21_dynamics_lib(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c21_dynamics_lib_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
