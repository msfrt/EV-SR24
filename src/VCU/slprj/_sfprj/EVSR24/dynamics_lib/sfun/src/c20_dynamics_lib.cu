/* Include files */

#include "dynamics_lib_sfun.h"
#include "c20_dynamics_lib.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c20_dynamics_lib(SFc20_dynamics_libInstanceStruct
  *chartInstance);
static void initialize_params_c20_dynamics_lib(SFc20_dynamics_libInstanceStruct *
  chartInstance);
static void mdl_start_c20_dynamics_lib(SFc20_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_terminate_c20_dynamics_lib(SFc20_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_setup_runtime_resources_c20_dynamics_lib
  (SFc20_dynamics_libInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c20_dynamics_lib
  (SFc20_dynamics_libInstanceStruct *chartInstance);
static void enable_c20_dynamics_lib(SFc20_dynamics_libInstanceStruct
  *chartInstance);
static void disable_c20_dynamics_lib(SFc20_dynamics_libInstanceStruct
  *chartInstance);
static void sf_gateway_c20_dynamics_lib(SFc20_dynamics_libInstanceStruct
  *chartInstance);
static void ext_mode_exec_c20_dynamics_lib(SFc20_dynamics_libInstanceStruct
  *chartInstance);
static void c20_do_animation_call_c20_dynamics_lib
  (SFc20_dynamics_libInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c20_dynamics_lib
  (SFc20_dynamics_libInstanceStruct *chartInstance);
static void set_sim_state_c20_dynamics_lib(SFc20_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c20_st);
static void initSimStructsc20_dynamics_lib(SFc20_dynamics_libInstanceStruct
  *chartInstance);
static void initSubchartIOPointersc20_dynamics_lib
  (SFc20_dynamics_libInstanceStruct *chartInstance);
static real32_T c20_eML_blk_kernel(SFc20_dynamics_libInstanceStruct
  *chartInstance, real32_T c20_b_rearNSMWT, real32_T c20_b_rearElWT, real_T
  c20_b_rearGeoWT);
static real32_T c20_emlrt_marshallIn(SFc20_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c20_nullptr, const char_T *c20_identifier);
static real32_T c20_b_emlrt_marshallIn(SFc20_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c20_u, const emlrtMsgIdentifier *c20_parentId);
static uint8_T c20_c_emlrt_marshallIn(SFc20_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c20_nullptr, const char_T *c20_identifier);
static uint8_T c20_d_emlrt_marshallIn(SFc20_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c20_u, const emlrtMsgIdentifier *c20_parentId);
static void init_dsm_address_info(SFc20_dynamics_libInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc20_dynamics_libInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c20_dynamics_lib(SFc20_dynamics_libInstanceStruct
  *chartInstance)
{
  emlrtLicenseCheckR2022a(chartInstance->c20_fEmlrtCtx,
    "EMLRT:runTime:MexFunctionNeedsLicense", "distrib_computing_toolbox", 2);
  sf_is_first_init_cond(chartInstance->S);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  emlrtInitGPU(chartInstance->c20_fEmlrtCtx);
  hipGetLastError();
}

static void initialize_params_c20_dynamics_lib(SFc20_dynamics_libInstanceStruct *
  chartInstance)
{
}

static void mdl_start_c20_dynamics_lib(SFc20_dynamics_libInstanceStruct
  *chartInstance)
{
  sim_mode_is_external(chartInstance->S);
}

static void mdl_terminate_c20_dynamics_lib(SFc20_dynamics_libInstanceStruct
  *chartInstance)
{
  hipError_t c20_errCode;
  c20_errCode = hipGetLastError();
  if (c20_errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(c20_errCode), (char_T *)
                       hipGetErrorName(c20_errCode), (char_T *)
                       hipGetErrorString(c20_errCode), (char_T *)
                       "SimGPUErrorChecks", chartInstance->c20_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c20_dynamics_lib
  (SFc20_dynamics_libInstanceStruct *chartInstance)
{
  sfSetAnimationVectors(chartInstance->S, chartInstance->c20_JITStateAnimation,
                        chartInstance->c20_JITTransitionAnimation);
}

static void mdl_cleanup_runtime_resources_c20_dynamics_lib
  (SFc20_dynamics_libInstanceStruct *chartInstance)
{
}

static void enable_c20_dynamics_lib(SFc20_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c20_dynamics_lib(SFc20_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void sf_gateway_c20_dynamics_lib(SFc20_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c20_JITTransitionAnimation[0] = 0U;
  *chartInstance->c20_rearTotalWT = c20_eML_blk_kernel(chartInstance,
    *chartInstance->c20_rearNSMWT, *chartInstance->c20_rearElWT,
    *chartInstance->c20_rearGeoWT);
  c20_do_animation_call_c20_dynamics_lib(chartInstance);
}

static void ext_mode_exec_c20_dynamics_lib(SFc20_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void c20_do_animation_call_c20_dynamics_lib
  (SFc20_dynamics_libInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static const mxArray *get_sim_state_c20_dynamics_lib
  (SFc20_dynamics_libInstanceStruct *chartInstance)
{
  const mxArray *c20_b_y = NULL;
  const mxArray *c20_c_y = NULL;
  const mxArray *c20_st;
  const mxArray *c20_y = NULL;
  c20_st = NULL;
  c20_st = NULL;
  c20_y = NULL;
  sf_mex_assign(&c20_y, sf_mex_createcellmatrix(2, 1), false);
  c20_b_y = NULL;
  sf_mex_assign(&c20_b_y, sf_mex_create("y", chartInstance->c20_rearTotalWT, 1,
    0U, 0U, 0U, 0), false);
  sf_mex_setcell(c20_y, 0, c20_b_y);
  c20_c_y = NULL;
  sf_mex_assign(&c20_c_y, sf_mex_create("y",
    &chartInstance->c20_is_active_c20_dynamics_lib, 3, 0U, 0U, 0U, 0), false);
  sf_mex_setcell(c20_y, 1, c20_c_y);
  sf_mex_assign(&c20_st, c20_y, false);
  return c20_st;
}

static void set_sim_state_c20_dynamics_lib(SFc20_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c20_st)
{
  const mxArray *c20_u;
  c20_u = sf_mex_dup(c20_st);
  *chartInstance->c20_rearTotalWT = c20_emlrt_marshallIn(chartInstance,
    sf_mex_dup(sf_mex_getcell(c20_u, 0)), "rearTotalWT");
  chartInstance->c20_is_active_c20_dynamics_lib = c20_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c20_u, 1)),
     "is_active_c20_dynamics_lib");
  sf_mex_destroy(&c20_u);
  sf_mex_destroy(&c20_st);
}

static void initSimStructsc20_dynamics_lib(SFc20_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void initSubchartIOPointersc20_dynamics_lib
  (SFc20_dynamics_libInstanceStruct *chartInstance)
{
}

const mxArray *sf_c20_dynamics_lib_get_eml_resolved_functions_info()
{
  const mxArray *c20_nameCaptureInfo = NULL;
  c20_nameCaptureInfo = NULL;
  sf_mex_assign(&c20_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c20_nameCaptureInfo;
}

static real32_T c20_eML_blk_kernel(SFc20_dynamics_libInstanceStruct
  *chartInstance, real32_T c20_b_rearNSMWT, real32_T c20_b_rearElWT, real_T
  c20_b_rearGeoWT)
{
  return (c20_b_rearNSMWT + c20_b_rearElWT) + static_cast<real32_T>
    (c20_b_rearGeoWT);
}

static real32_T c20_emlrt_marshallIn(SFc20_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c20_nullptr, const char_T *c20_identifier)
{
  emlrtMsgIdentifier c20_thisId;
  real32_T c20_y;
  c20_thisId.fIdentifier = const_cast<const char_T *>(c20_identifier);
  c20_thisId.fParent = NULL;
  c20_thisId.bParentIsCell = false;
  c20_y = c20_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c20_nullptr),
    &c20_thisId);
  sf_mex_destroy(&c20_nullptr);
  return c20_y;
}

static real32_T c20_b_emlrt_marshallIn(SFc20_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c20_u, const emlrtMsgIdentifier *c20_parentId)
{
  real32_T c20_f;
  real32_T c20_y;
  sf_mex_import(c20_parentId, sf_mex_dup(c20_u), &c20_f, 0, 1, 0U, 0, 0U, 0);
  c20_y = c20_f;
  sf_mex_destroy(&c20_u);
  return c20_y;
}

static uint8_T c20_c_emlrt_marshallIn(SFc20_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c20_nullptr, const char_T *c20_identifier)
{
  emlrtMsgIdentifier c20_thisId;
  uint8_T c20_y;
  c20_thisId.fIdentifier = const_cast<const char_T *>(c20_identifier);
  c20_thisId.fParent = NULL;
  c20_thisId.bParentIsCell = false;
  c20_y = c20_d_emlrt_marshallIn(chartInstance, sf_mex_dup(c20_nullptr),
    &c20_thisId);
  sf_mex_destroy(&c20_nullptr);
  return c20_y;
}

static uint8_T c20_d_emlrt_marshallIn(SFc20_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c20_u, const emlrtMsgIdentifier *c20_parentId)
{
  uint8_T c20_b_u;
  uint8_T c20_y;
  sf_mex_import(c20_parentId, sf_mex_dup(c20_u), &c20_b_u, 1, 3, 0U, 0, 0U, 0);
  c20_y = c20_b_u;
  sf_mex_destroy(&c20_u);
  return c20_y;
}

static void init_dsm_address_info(SFc20_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc20_dynamics_libInstanceStruct
  *chartInstance)
{
  chartInstance->c20_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c20_rearNSMWT = (real32_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c20_rearTotalWT = (real32_T *)ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c20_rearElWT = (real32_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c20_rearGeoWT = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 2);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c20_dynamics_lib_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(3249489784U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(3351134557U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(413153145U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(1910199332U);
}

mxArray *sf_c20_dynamics_lib_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c20_dynamics_lib_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("chartInfo");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c20_dynamics_lib_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c20_dynamics_lib(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiDmYGJgYAPRQMzEAAGsUD4jVIwRLs4CF1cA4pLKglSQeHFRsmcKkM5LzAXzE0s"
    "rPPPS8sHmWzAgzGfDYj4jkvmcUHEI+GBPmX4RB5B+ByT9LFj0cyPpF4Dyi1ITi0LySxJzwkMYYO"
    "EEo8l3j4IDZfoh9gcQ8I8Umn9A/Mzi+MTkksyy1PhkI4P4lEpgPGUmF8fnZCYxoPgPAAHDHPk="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c20_dynamics_lib_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sklQdNloCdlTDXMMCBdhIRH";
}

static void sf_opaque_initialize_c20_dynamics_lib(void *chartInstanceVar)
{
  initialize_params_c20_dynamics_lib((SFc20_dynamics_libInstanceStruct*)
    chartInstanceVar);
  initialize_c20_dynamics_lib((SFc20_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c20_dynamics_lib(void *chartInstanceVar)
{
  enable_c20_dynamics_lib((SFc20_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c20_dynamics_lib(void *chartInstanceVar)
{
  disable_c20_dynamics_lib((SFc20_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c20_dynamics_lib(void *chartInstanceVar)
{
  sf_gateway_c20_dynamics_lib((SFc20_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c20_dynamics_lib(SimStruct* S)
{
  return get_sim_state_c20_dynamics_lib((SFc20_dynamics_libInstanceStruct *)
    sf_get_chart_instance_ptr(S));     /* raw sim ctx */
}

static void sf_opaque_set_sim_state_c20_dynamics_lib(SimStruct* S, const mxArray
  *st)
{
  set_sim_state_c20_dynamics_lib((SFc20_dynamics_libInstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c20_dynamics_lib(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc20_dynamics_libInstanceStruct*) chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_dynamics_lib_optimization_info();
    }

    mdl_cleanup_runtime_resources_c20_dynamics_lib
      ((SFc20_dynamics_libInstanceStruct*) chartInstanceVar);
    ((SFc20_dynamics_libInstanceStruct*) chartInstanceVar)->
      ~SFc20_dynamics_libInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c20_dynamics_lib(void *chartInstanceVar)
{
  mdl_start_c20_dynamics_lib((SFc20_dynamics_libInstanceStruct*)
    chartInstanceVar);
  if (chartInstanceVar) {
    sf_reset_warnings_ChartRunTimeInfo(((SFc20_dynamics_libInstanceStruct*)
      chartInstanceVar)->S);
  }
}

static void sf_opaque_mdl_terminate_c20_dynamics_lib(void *chartInstanceVar)
{
  mdl_terminate_c20_dynamics_lib((SFc20_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c20_dynamics_lib(SimStruct *S)
{
  mdlProcessParamsCommon(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c20_dynamics_lib((SFc20_dynamics_libInstanceStruct*)
      sf_get_chart_instance_ptr(S));
    initSubchartIOPointersc20_dynamics_lib((SFc20_dynamics_libInstanceStruct*)
      sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c20_dynamics_lib_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [21] = {
    "eNrtWN2O20QUnmSXVUtLtRdIRRUSveQGaVmExAUqu+skNNKmG5psi7hZzdon8SjjGXd+kg0PwiM",
    "gHqFvAI/AG3CHuINLzjhOGpyQeBwJWoQlr3dsf/PNOT4/34TU2h2Cxz08P3uPkAO83sKzTmbHW/",
    "m4tnTO7u+Tz/PxT28TEsoIhiB6djBgN8TvEDbpUkUTTfwPQRN4Clpya5gUbTGQ5bFMDECBCHGCV",
    "CrjxatZYjkTo5YVoWPWz2MWxr1YWh6d4YQ0uhB8+ne8qTVdZGwwBaFpAUQmVtIO4xanw81eUGYS",
    "xBCOtE28faXB9GzqTNUdyw1LOTRvIGwLbSh6QW+xt2eogcDc+DnZ2at7c7RMUs6oKO/rmOoepBg",
    "dBi7TCP9eWIPeK8mLfNdMUCMVo7yZ8CCmqiy2y3GdHQxr7u1nk5gGXNvhkImh866yCQi0H+OkhK",
    "8GgRyDokO4EH68obOueZN94EVclsQalsAzqk5DjAMNkV8OYgDrHsVPC32cxgsLgl5zaOu+YmP8R",
    "r51o+1SqVLdsMksknQlbMbbHONX1RV5W6EIKOfaD9uX6TmMgWf8DWpoBeyM3wOsNYv6EqPDZa5n",
    "9lvBXljIsYEUESsfleMCKmsST7Dgl4CzxKUBROjmxdIXE23LI6uNTAJM/cb5eUm+VWxbGFADGkL",
    "peq0o04ALzuLKkzdi2iUSotFLJrOy9AyzHKwEJXpgRWMi1Qh97NsYXvnKZYIfGqIhNMBAVuSaGN",
    "3PKLcl15xorMouPC41Vlk/XsS6/KkEDmkYQ+R6IOPQwTqLE5T9xNq1z1O0dszMtAE6VCwtm0kWC",
    "zo2P+el/jSFSzESciJaSia9XMVsiCsArBpUCWxlZ9hK1bSFiy+3agUv+ll19xUMzs/UcHrtYuNL",
    "ENgNna2ue9MQs6opUG7ignbB9ti3KEeEZtqgBJw2sxyIMh18Ql7p4P01OvhwSQcf5md4fHQVTVG",
    "NslBfcXbt5jlamuduCT091+GbcGQFRxa4+fXDJXxtDS9Zuhb5btf/+v5ega+Od2q12oqf3inw7B",
    "dwB7mffm2ff6+/U5e//B7s/XD/x5fb+Gsr/LXsf4d7Wffbr9zLx+/PtdKiMo9Xipd79/GWOLhfi",
    "AM31iP+VfSEyyDi/cbXnU5wFsXtp4+z+W5tWW+9sN75/YdOn2HiZnVAhe0o3/e4MbUzPZ7t35bW",
    "e7DFH7eX4omQ377YDf/uSZm8uVPw152sRlDVl4by5/2VOK6+nocnu+Fn/N0t9jwo2PMg0+9X1FV",
    "puCpWhNU8rRr3vrg3he9/XPnvUKaf7FXEVeUj/zBuV/t8++Sb/v7RhnpGCu8fvsZ2bKqjPnrpdb",
    "PrZ+KnZz7Ix48We/QgZjxas0vIH6OQH6x7+h+J7z88/TfXb03nv/yH0W8+ORWUT3FbMNt25be7y",
    "v0mt3iEqkWv33v9G/1kft2mv+4W8tuNJ0xEcqI/+vj40+Nd+tOf6X4KuQ==",
    ""
  };

  static char newstr [1485] = "";
  newstr[0] = '\0';
  for (i = 0; i < 21; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c20_dynamics_lib(SimStruct *S)
{
  const char* newstr = sf_c20_dynamics_lib_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(2790017518U));
  ssSetChecksum1(S,(1433572979U));
  ssSetChecksum2(S,(54786023U));
  ssSetChecksum3(S,(3100055464U));
}

static void mdlRTW_c20_dynamics_lib(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c20_dynamics_lib(SimStruct *S)
{
  SFc20_dynamics_libInstanceStruct *chartInstance;
  chartInstance = (SFc20_dynamics_libInstanceStruct *)utMalloc(sizeof
    (SFc20_dynamics_libInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc20_dynamics_libInstanceStruct));
  chartInstance = new (chartInstance) SFc20_dynamics_libInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway = sf_opaque_gateway_c20_dynamics_lib;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c20_dynamics_lib;
  chartInstance->chartInfo.mdlStart = sf_opaque_mdl_start_c20_dynamics_lib;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c20_dynamics_lib;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c20_dynamics_lib;
  chartInstance->chartInfo.enableChart = sf_opaque_enable_c20_dynamics_lib;
  chartInstance->chartInfo.disableChart = sf_opaque_disable_c20_dynamics_lib;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c20_dynamics_lib;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c20_dynamics_lib;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c20_dynamics_lib;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c20_dynamics_lib;
  chartInstance->chartInfo.mdlSetWorkWidths = mdlSetWorkWidths_c20_dynamics_lib;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c20_dynamics_lib(chartInstance);
}

void c20_dynamics_lib_method_dispatcher(SimStruct *S, int_T method, void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c20_dynamics_lib(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c20_dynamics_lib(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c20_dynamics_lib(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c20_dynamics_lib_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
