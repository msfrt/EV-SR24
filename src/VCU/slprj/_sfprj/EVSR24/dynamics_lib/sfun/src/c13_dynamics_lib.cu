/* Include files */

#include "dynamics_lib_sfun.h"
#include "c13_dynamics_lib.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c13_dynamics_lib(SFc13_dynamics_libInstanceStruct
  *chartInstance);
static void initialize_params_c13_dynamics_lib(SFc13_dynamics_libInstanceStruct *
  chartInstance);
static void mdl_start_c13_dynamics_lib(SFc13_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_terminate_c13_dynamics_lib(SFc13_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_setup_runtime_resources_c13_dynamics_lib
  (SFc13_dynamics_libInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c13_dynamics_lib
  (SFc13_dynamics_libInstanceStruct *chartInstance);
static void enable_c13_dynamics_lib(SFc13_dynamics_libInstanceStruct
  *chartInstance);
static void disable_c13_dynamics_lib(SFc13_dynamics_libInstanceStruct
  *chartInstance);
static void sf_gateway_c13_dynamics_lib(SFc13_dynamics_libInstanceStruct
  *chartInstance);
static void ext_mode_exec_c13_dynamics_lib(SFc13_dynamics_libInstanceStruct
  *chartInstance);
static void c13_do_animation_call_c13_dynamics_lib
  (SFc13_dynamics_libInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c13_dynamics_lib
  (SFc13_dynamics_libInstanceStruct *chartInstance);
static void set_sim_state_c13_dynamics_lib(SFc13_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c13_st);
static void initSimStructsc13_dynamics_lib(SFc13_dynamics_libInstanceStruct
  *chartInstance);
static void initSubchartIOPointersc13_dynamics_lib
  (SFc13_dynamics_libInstanceStruct *chartInstance);
static real_T c13_eML_blk_kernel(SFc13_dynamics_libInstanceStruct *chartInstance,
  real_T c13_b_frontSpringStiffness, real_T c13_b_frontSpringMotionRatio);
static real_T c13_emlrt_marshallIn(SFc13_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c13_nullptr, const char_T *c13_identifier);
static real_T c13_b_emlrt_marshallIn(SFc13_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c13_u, const emlrtMsgIdentifier *c13_parentId);
static uint8_T c13_c_emlrt_marshallIn(SFc13_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c13_nullptr, const char_T *c13_identifier);
static uint8_T c13_d_emlrt_marshallIn(SFc13_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c13_u, const emlrtMsgIdentifier *c13_parentId);
static void init_dsm_address_info(SFc13_dynamics_libInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc13_dynamics_libInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c13_dynamics_lib(SFc13_dynamics_libInstanceStruct
  *chartInstance)
{
  emlrtLicenseCheckR2022a(chartInstance->c13_fEmlrtCtx,
    "EMLRT:runTime:MexFunctionNeedsLicense", "distrib_computing_toolbox", 2);
  sf_is_first_init_cond(chartInstance->S);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  emlrtInitGPU(chartInstance->c13_fEmlrtCtx);
  hipGetLastError();
}

static void initialize_params_c13_dynamics_lib(SFc13_dynamics_libInstanceStruct *
  chartInstance)
{
}

static void mdl_start_c13_dynamics_lib(SFc13_dynamics_libInstanceStruct
  *chartInstance)
{
  sim_mode_is_external(chartInstance->S);
}

static void mdl_terminate_c13_dynamics_lib(SFc13_dynamics_libInstanceStruct
  *chartInstance)
{
  hipError_t c13_errCode;
  c13_errCode = hipGetLastError();
  if (c13_errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(c13_errCode), (char_T *)
                       hipGetErrorName(c13_errCode), (char_T *)
                       hipGetErrorString(c13_errCode), (char_T *)
                       "SimGPUErrorChecks", chartInstance->c13_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c13_dynamics_lib
  (SFc13_dynamics_libInstanceStruct *chartInstance)
{
  sfSetAnimationVectors(chartInstance->S, chartInstance->c13_JITStateAnimation,
                        chartInstance->c13_JITTransitionAnimation);
}

static void mdl_cleanup_runtime_resources_c13_dynamics_lib
  (SFc13_dynamics_libInstanceStruct *chartInstance)
{
}

static void enable_c13_dynamics_lib(SFc13_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c13_dynamics_lib(SFc13_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void sf_gateway_c13_dynamics_lib(SFc13_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c13_JITTransitionAnimation[0] = 0U;
  *chartInstance->c13_frontWheelRate = c13_eML_blk_kernel(chartInstance,
    *chartInstance->c13_frontSpringStiffness,
    *chartInstance->c13_frontSpringMotionRatio);
  c13_do_animation_call_c13_dynamics_lib(chartInstance);
}

static void ext_mode_exec_c13_dynamics_lib(SFc13_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void c13_do_animation_call_c13_dynamics_lib
  (SFc13_dynamics_libInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static const mxArray *get_sim_state_c13_dynamics_lib
  (SFc13_dynamics_libInstanceStruct *chartInstance)
{
  const mxArray *c13_b_y = NULL;
  const mxArray *c13_c_y = NULL;
  const mxArray *c13_st;
  const mxArray *c13_y = NULL;
  c13_st = NULL;
  c13_st = NULL;
  c13_y = NULL;
  sf_mex_assign(&c13_y, sf_mex_createcellmatrix(2, 1), false);
  c13_b_y = NULL;
  sf_mex_assign(&c13_b_y, sf_mex_create("y", chartInstance->c13_frontWheelRate,
    0, 0U, 0U, 0U, 0), false);
  sf_mex_setcell(c13_y, 0, c13_b_y);
  c13_c_y = NULL;
  sf_mex_assign(&c13_c_y, sf_mex_create("y",
    &chartInstance->c13_is_active_c13_dynamics_lib, 3, 0U, 0U, 0U, 0), false);
  sf_mex_setcell(c13_y, 1, c13_c_y);
  sf_mex_assign(&c13_st, c13_y, false);
  return c13_st;
}

static void set_sim_state_c13_dynamics_lib(SFc13_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c13_st)
{
  const mxArray *c13_u;
  c13_u = sf_mex_dup(c13_st);
  *chartInstance->c13_frontWheelRate = c13_emlrt_marshallIn(chartInstance,
    sf_mex_dup(sf_mex_getcell(c13_u, 0)), "frontWheelRate");
  chartInstance->c13_is_active_c13_dynamics_lib = c13_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c13_u, 1)),
     "is_active_c13_dynamics_lib");
  sf_mex_destroy(&c13_u);
  sf_mex_destroy(&c13_st);
}

static void initSimStructsc13_dynamics_lib(SFc13_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void initSubchartIOPointersc13_dynamics_lib
  (SFc13_dynamics_libInstanceStruct *chartInstance)
{
}

const mxArray *sf_c13_dynamics_lib_get_eml_resolved_functions_info()
{
  const mxArray *c13_nameCaptureInfo = NULL;
  c13_nameCaptureInfo = NULL;
  sf_mex_assign(&c13_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c13_nameCaptureInfo;
}

static real_T c13_eML_blk_kernel(SFc13_dynamics_libInstanceStruct *chartInstance,
  real_T c13_b_frontSpringStiffness, real_T c13_b_frontSpringMotionRatio)
{
  return c13_b_frontSpringStiffness / (c13_b_frontSpringMotionRatio *
    c13_b_frontSpringMotionRatio);
}

static real_T c13_emlrt_marshallIn(SFc13_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c13_nullptr, const char_T *c13_identifier)
{
  emlrtMsgIdentifier c13_thisId;
  real_T c13_y;
  c13_thisId.fIdentifier = const_cast<const char_T *>(c13_identifier);
  c13_thisId.fParent = NULL;
  c13_thisId.bParentIsCell = false;
  c13_y = c13_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c13_nullptr),
    &c13_thisId);
  sf_mex_destroy(&c13_nullptr);
  return c13_y;
}

static real_T c13_b_emlrt_marshallIn(SFc13_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c13_u, const emlrtMsgIdentifier *c13_parentId)
{
  real_T c13_d;
  real_T c13_y;
  sf_mex_import(c13_parentId, sf_mex_dup(c13_u), &c13_d, 1, 0, 0U, 0, 0U, 0);
  c13_y = c13_d;
  sf_mex_destroy(&c13_u);
  return c13_y;
}

static uint8_T c13_c_emlrt_marshallIn(SFc13_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c13_nullptr, const char_T *c13_identifier)
{
  emlrtMsgIdentifier c13_thisId;
  uint8_T c13_y;
  c13_thisId.fIdentifier = const_cast<const char_T *>(c13_identifier);
  c13_thisId.fParent = NULL;
  c13_thisId.bParentIsCell = false;
  c13_y = c13_d_emlrt_marshallIn(chartInstance, sf_mex_dup(c13_nullptr),
    &c13_thisId);
  sf_mex_destroy(&c13_nullptr);
  return c13_y;
}

static uint8_T c13_d_emlrt_marshallIn(SFc13_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c13_u, const emlrtMsgIdentifier *c13_parentId)
{
  uint8_T c13_b_u;
  uint8_T c13_y;
  sf_mex_import(c13_parentId, sf_mex_dup(c13_u), &c13_b_u, 1, 3, 0U, 0, 0U, 0);
  c13_y = c13_b_u;
  sf_mex_destroy(&c13_u);
  return c13_y;
}

static void init_dsm_address_info(SFc13_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc13_dynamics_libInstanceStruct
  *chartInstance)
{
  chartInstance->c13_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c13_frontSpringStiffness = (real_T *)
    ssGetInputPortSignal_wrapper(chartInstance->S, 0);
  chartInstance->c13_frontWheelRate = (real_T *)ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c13_frontSpringMotionRatio = (real_T *)
    ssGetInputPortSignal_wrapper(chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c13_dynamics_lib_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(3010871865U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(3322241292U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(3655968327U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(1438991017U);
}

mxArray *sf_c13_dynamics_lib_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c13_dynamics_lib_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("chartInfo");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c13_dynamics_lib_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c13_dynamics_lib(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiDmYGJgYAPRQMzEAAGsUD4jVIwRLs4CF1cA4pLKglSQeHFRsmcKkM5LzAXzE0s"
    "rPPPS8sHmWzAgzGfDYj4jkvmcUHEI+GBPmX4RB5B+ByT9LFj08yHpF4Dy04ry80rCM1JTc4ISS1"
    "IhYrDwIt89Cg6U6YfYH0DAP1Jo/gHxM4vjE5NLMstS45MNjeNTKoHxlJlcHJ+TmYQwFwQAP3MeU"
    "g=="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c13_dynamics_lib_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "swZnH3WG6KgyPoyYFsXa5UE";
}

static void sf_opaque_initialize_c13_dynamics_lib(void *chartInstanceVar)
{
  initialize_params_c13_dynamics_lib((SFc13_dynamics_libInstanceStruct*)
    chartInstanceVar);
  initialize_c13_dynamics_lib((SFc13_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c13_dynamics_lib(void *chartInstanceVar)
{
  enable_c13_dynamics_lib((SFc13_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c13_dynamics_lib(void *chartInstanceVar)
{
  disable_c13_dynamics_lib((SFc13_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c13_dynamics_lib(void *chartInstanceVar)
{
  sf_gateway_c13_dynamics_lib((SFc13_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c13_dynamics_lib(SimStruct* S)
{
  return get_sim_state_c13_dynamics_lib((SFc13_dynamics_libInstanceStruct *)
    sf_get_chart_instance_ptr(S));     /* raw sim ctx */
}

static void sf_opaque_set_sim_state_c13_dynamics_lib(SimStruct* S, const mxArray
  *st)
{
  set_sim_state_c13_dynamics_lib((SFc13_dynamics_libInstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c13_dynamics_lib(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc13_dynamics_libInstanceStruct*) chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_dynamics_lib_optimization_info();
    }

    mdl_cleanup_runtime_resources_c13_dynamics_lib
      ((SFc13_dynamics_libInstanceStruct*) chartInstanceVar);
    ((SFc13_dynamics_libInstanceStruct*) chartInstanceVar)->
      ~SFc13_dynamics_libInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c13_dynamics_lib(void *chartInstanceVar)
{
  mdl_start_c13_dynamics_lib((SFc13_dynamics_libInstanceStruct*)
    chartInstanceVar);
  if (chartInstanceVar) {
    sf_reset_warnings_ChartRunTimeInfo(((SFc13_dynamics_libInstanceStruct*)
      chartInstanceVar)->S);
  }
}

static void sf_opaque_mdl_terminate_c13_dynamics_lib(void *chartInstanceVar)
{
  mdl_terminate_c13_dynamics_lib((SFc13_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c13_dynamics_lib(SimStruct *S)
{
  mdlProcessParamsCommon(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c13_dynamics_lib((SFc13_dynamics_libInstanceStruct*)
      sf_get_chart_instance_ptr(S));
    initSubchartIOPointersc13_dynamics_lib((SFc13_dynamics_libInstanceStruct*)
      sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c13_dynamics_lib_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [21] = {
    "eNrtWN2O20QUnkTLqq3Kai+QQBVVK64QElLbVREXCHabnzZilw2bbBd6s5rYJ/ZoxzPu/CQb7pF",
    "4Bp6AF0DiEXgELvoQXCKuOOM4aeqExOOVoEU4cpyx/c13zpnzNyG1zhHBYwfPT98jZBuv1/Csk+",
    "nxVj6uLZzT+1vks3z86w1CAhlCBKJnh0N2SfwOYZMuVTTRxP8QNIET0JJbw6ToiKEsj2ViCApEg",
    "BOkUhkvXs0Sy5m4aFsROGZ9FrMg7sXS8vARTkjDY8Enf8ebWtNFxiZTEJg2QGhiJW0UtzmN1ltB",
    "mXEjhuBC28TbVhpMz6ZOVX1kuWEph9YlBB2hDUUr6A369gw10DCXfkZ2+ureDC2TlDMqyts6pro",
    "HKXqHgdM0xO9ja9B6JXmRb8AENVIxylsJb8RUlcV2Ocp5hG7Nve1sEtOEgY0iJiJnXWUTEKg/+k",
    "kJWw0bcgSKRnAs/HgDp13rMlvguV+WxBqWwFOqDgL0Aw2hXwyiA+sexaWFPk7jhQVBBxw6uq/YC",
    "NfIN290XChVyhs2mXqSroTNeFsjXFVdkbcdiAblXPth+zI9hBHwjL9JDa2AnfJ7gLVmYV+id7jI",
    "9Yx+K9hzCzm2IUXIynvlqIDKisRXmPBLwFniwgBCNPNc9PlEm+LIaiOTBoZ+8/CwJN8ytiMMqCE",
    "NoHS+VpRpQIEzv/LkDZl2gYRotJLJtCw9wzQGK0GJHlrRHEt1gTb2LQwvbeUiwQ8NYQRNMJAluR",
    "Z691PKbUmZE41Z2bnHqcYs68eLWBc/lcABDWIIXQ1kHI4wz+IEZZdYu/J5gNqOmJk0QQeKpWUjy",
    "WJCx+LnrNSfpHAqLoQci7aSSS/vYtb4FQBmDaoElrJHWErVpI3Cl5NawfN+lt19GwZnZ2o4HTjf",
    "eAwCq6HT1VVvGmBUtQS2myjQVbA99h22I0IzbbAFnLSyGAizPnifvOyDt1b0wbsLffBufgb3987",
    "DCXajLNDnnA3cPPcW5rlZop+e9eHrcGQJR+a42fXDBXxtBS9ZuBb5rtdffb9e4Kvjp1YjS3Z6u8",
    "CzVcBt53aKRj/dePHDB99/9OPtkz9f/Pz1Jv7aEn8t++1wv9T99is7+fj9Wa80z8yjpeTl3n2yw",
    "Q/eLfiBG+vxM/Fk7+zxJ19Gk66cfNvW39CHp61svmsb5K0X5J3dv+v6MwzcLA+ooBPm+x43pnba",
    "j2f7twV5tzfY4/qCPxHy+xdXw7+zXyZudgr2cuOhksKcxQD8BBPcq35cXZ67+1fDT/m7G/S5VdD",
    "nVta/n1OXpeG8mBGW47Sq3/vi3hS+/3Hl16FMPalXxFXlI/8w7qr6+dbJN/39e2vyGSm8v/sa67",
    "Euj/r0S6+bXr8Rv37mTj7+fL5Hb8SMhyt2CfljbOSHq57+R/z7D0/7zfq3lrNf/sfos70DQfkEt",
    "wXTbVd+u6vcf3LzRwqoXr33+jfqyey6qf+6WYhvNx4zEcqx/vj+g4cPrlKf/gLpgAti",
    ""
  };

  static char newstr [1493] = "";
  newstr[0] = '\0';
  for (i = 0; i < 21; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c13_dynamics_lib(SimStruct *S)
{
  const char* newstr = sf_c13_dynamics_lib_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(178812519U));
  ssSetChecksum1(S,(2233632988U));
  ssSetChecksum2(S,(1377671722U));
  ssSetChecksum3(S,(1370873082U));
}

static void mdlRTW_c13_dynamics_lib(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c13_dynamics_lib(SimStruct *S)
{
  SFc13_dynamics_libInstanceStruct *chartInstance;
  chartInstance = (SFc13_dynamics_libInstanceStruct *)utMalloc(sizeof
    (SFc13_dynamics_libInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc13_dynamics_libInstanceStruct));
  chartInstance = new (chartInstance) SFc13_dynamics_libInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway = sf_opaque_gateway_c13_dynamics_lib;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c13_dynamics_lib;
  chartInstance->chartInfo.mdlStart = sf_opaque_mdl_start_c13_dynamics_lib;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c13_dynamics_lib;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c13_dynamics_lib;
  chartInstance->chartInfo.enableChart = sf_opaque_enable_c13_dynamics_lib;
  chartInstance->chartInfo.disableChart = sf_opaque_disable_c13_dynamics_lib;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c13_dynamics_lib;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c13_dynamics_lib;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c13_dynamics_lib;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c13_dynamics_lib;
  chartInstance->chartInfo.mdlSetWorkWidths = mdlSetWorkWidths_c13_dynamics_lib;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c13_dynamics_lib(chartInstance);
}

void c13_dynamics_lib_method_dispatcher(SimStruct *S, int_T method, void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c13_dynamics_lib(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c13_dynamics_lib(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c13_dynamics_lib(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c13_dynamics_lib_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
