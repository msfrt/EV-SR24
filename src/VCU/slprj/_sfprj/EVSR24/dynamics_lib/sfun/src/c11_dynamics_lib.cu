/* Include files */

#include "dynamics_lib_sfun.h"
#include "c11_dynamics_lib.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c11_dynamics_lib(SFc11_dynamics_libInstanceStruct
  *chartInstance);
static void initialize_params_c11_dynamics_lib(SFc11_dynamics_libInstanceStruct *
  chartInstance);
static void mdl_start_c11_dynamics_lib(SFc11_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_terminate_c11_dynamics_lib(SFc11_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_setup_runtime_resources_c11_dynamics_lib
  (SFc11_dynamics_libInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c11_dynamics_lib
  (SFc11_dynamics_libInstanceStruct *chartInstance);
static void enable_c11_dynamics_lib(SFc11_dynamics_libInstanceStruct
  *chartInstance);
static void disable_c11_dynamics_lib(SFc11_dynamics_libInstanceStruct
  *chartInstance);
static void sf_gateway_c11_dynamics_lib(SFc11_dynamics_libInstanceStruct
  *chartInstance);
static void ext_mode_exec_c11_dynamics_lib(SFc11_dynamics_libInstanceStruct
  *chartInstance);
static void c11_do_animation_call_c11_dynamics_lib
  (SFc11_dynamics_libInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c11_dynamics_lib
  (SFc11_dynamics_libInstanceStruct *chartInstance);
static void set_sim_state_c11_dynamics_lib(SFc11_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c11_st);
static void initSimStructsc11_dynamics_lib(SFc11_dynamics_libInstanceStruct
  *chartInstance);
static void initSubchartIOPointersc11_dynamics_lib
  (SFc11_dynamics_libInstanceStruct *chartInstance);
static real_T c11_eML_blk_kernel(SFc11_dynamics_libInstanceStruct *chartInstance,
  real_T c11_b_frontTrack, real_T c11_b_frontWheelRate);
static real_T c11_emlrt_marshallIn(SFc11_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c11_nullptr, const char_T *c11_identifier);
static real_T c11_b_emlrt_marshallIn(SFc11_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c11_u, const emlrtMsgIdentifier *c11_parentId);
static uint8_T c11_c_emlrt_marshallIn(SFc11_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c11_nullptr, const char_T *c11_identifier);
static uint8_T c11_d_emlrt_marshallIn(SFc11_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c11_u, const emlrtMsgIdentifier *c11_parentId);
static void init_dsm_address_info(SFc11_dynamics_libInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc11_dynamics_libInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c11_dynamics_lib(SFc11_dynamics_libInstanceStruct
  *chartInstance)
{
  emlrtLicenseCheckR2022a(chartInstance->c11_fEmlrtCtx,
    "EMLRT:runTime:MexFunctionNeedsLicense", "distrib_computing_toolbox", 2);
  sf_is_first_init_cond(chartInstance->S);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  emlrtInitGPU(chartInstance->c11_fEmlrtCtx);
  hipGetLastError();
}

static void initialize_params_c11_dynamics_lib(SFc11_dynamics_libInstanceStruct *
  chartInstance)
{
}

static void mdl_start_c11_dynamics_lib(SFc11_dynamics_libInstanceStruct
  *chartInstance)
{
  sim_mode_is_external(chartInstance->S);
}

static void mdl_terminate_c11_dynamics_lib(SFc11_dynamics_libInstanceStruct
  *chartInstance)
{
  hipError_t c11_errCode;
  c11_errCode = hipGetLastError();
  if (c11_errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(c11_errCode), (char_T *)
                       hipGetErrorName(c11_errCode), (char_T *)
                       hipGetErrorString(c11_errCode), (char_T *)
                       "SimGPUErrorChecks", chartInstance->c11_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c11_dynamics_lib
  (SFc11_dynamics_libInstanceStruct *chartInstance)
{
  sfSetAnimationVectors(chartInstance->S, chartInstance->c11_JITStateAnimation,
                        chartInstance->c11_JITTransitionAnimation);
}

static void mdl_cleanup_runtime_resources_c11_dynamics_lib
  (SFc11_dynamics_libInstanceStruct *chartInstance)
{
}

static void enable_c11_dynamics_lib(SFc11_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c11_dynamics_lib(SFc11_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void sf_gateway_c11_dynamics_lib(SFc11_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c11_JITTransitionAnimation[0] = 0U;
  *chartInstance->c11_antiRollFrontSprings = c11_eML_blk_kernel(chartInstance,
    *chartInstance->c11_frontTrack, *chartInstance->c11_frontWheelRate);
  c11_do_animation_call_c11_dynamics_lib(chartInstance);
}

static void ext_mode_exec_c11_dynamics_lib(SFc11_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void c11_do_animation_call_c11_dynamics_lib
  (SFc11_dynamics_libInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static const mxArray *get_sim_state_c11_dynamics_lib
  (SFc11_dynamics_libInstanceStruct *chartInstance)
{
  const mxArray *c11_b_y = NULL;
  const mxArray *c11_c_y = NULL;
  const mxArray *c11_st;
  const mxArray *c11_y = NULL;
  c11_st = NULL;
  c11_st = NULL;
  c11_y = NULL;
  sf_mex_assign(&c11_y, sf_mex_createcellmatrix(2, 1), false);
  c11_b_y = NULL;
  sf_mex_assign(&c11_b_y, sf_mex_create("y",
    chartInstance->c11_antiRollFrontSprings, 0, 0U, 0U, 0U, 0), false);
  sf_mex_setcell(c11_y, 0, c11_b_y);
  c11_c_y = NULL;
  sf_mex_assign(&c11_c_y, sf_mex_create("y",
    &chartInstance->c11_is_active_c11_dynamics_lib, 3, 0U, 0U, 0U, 0), false);
  sf_mex_setcell(c11_y, 1, c11_c_y);
  sf_mex_assign(&c11_st, c11_y, false);
  return c11_st;
}

static void set_sim_state_c11_dynamics_lib(SFc11_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c11_st)
{
  const mxArray *c11_u;
  c11_u = sf_mex_dup(c11_st);
  *chartInstance->c11_antiRollFrontSprings = c11_emlrt_marshallIn(chartInstance,
    sf_mex_dup(sf_mex_getcell(c11_u, 0)), "antiRollFrontSprings");
  chartInstance->c11_is_active_c11_dynamics_lib = c11_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c11_u, 1)),
     "is_active_c11_dynamics_lib");
  sf_mex_destroy(&c11_u);
  sf_mex_destroy(&c11_st);
}

static void initSimStructsc11_dynamics_lib(SFc11_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void initSubchartIOPointersc11_dynamics_lib
  (SFc11_dynamics_libInstanceStruct *chartInstance)
{
}

const mxArray *sf_c11_dynamics_lib_get_eml_resolved_functions_info()
{
  const mxArray *c11_nameCaptureInfo = NULL;
  c11_nameCaptureInfo = NULL;
  sf_mex_assign(&c11_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c11_nameCaptureInfo;
}

static real_T c11_eML_blk_kernel(SFc11_dynamics_libInstanceStruct *chartInstance,
  real_T c11_b_frontTrack, real_T c11_b_frontWheelRate)
{
  real_T c11_a;
  c11_a = c11_b_frontTrack / 1000.0;
  return c11_a * c11_a * 1.5574077246549023 * (c11_b_frontWheelRate * 1000.0) /
    2.0;
}

static real_T c11_emlrt_marshallIn(SFc11_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c11_nullptr, const char_T *c11_identifier)
{
  emlrtMsgIdentifier c11_thisId;
  real_T c11_y;
  c11_thisId.fIdentifier = const_cast<const char_T *>(c11_identifier);
  c11_thisId.fParent = NULL;
  c11_thisId.bParentIsCell = false;
  c11_y = c11_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c11_nullptr),
    &c11_thisId);
  sf_mex_destroy(&c11_nullptr);
  return c11_y;
}

static real_T c11_b_emlrt_marshallIn(SFc11_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c11_u, const emlrtMsgIdentifier *c11_parentId)
{
  real_T c11_d;
  real_T c11_y;
  sf_mex_import(c11_parentId, sf_mex_dup(c11_u), &c11_d, 1, 0, 0U, 0, 0U, 0);
  c11_y = c11_d;
  sf_mex_destroy(&c11_u);
  return c11_y;
}

static uint8_T c11_c_emlrt_marshallIn(SFc11_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c11_nullptr, const char_T *c11_identifier)
{
  emlrtMsgIdentifier c11_thisId;
  uint8_T c11_y;
  c11_thisId.fIdentifier = const_cast<const char_T *>(c11_identifier);
  c11_thisId.fParent = NULL;
  c11_thisId.bParentIsCell = false;
  c11_y = c11_d_emlrt_marshallIn(chartInstance, sf_mex_dup(c11_nullptr),
    &c11_thisId);
  sf_mex_destroy(&c11_nullptr);
  return c11_y;
}

static uint8_T c11_d_emlrt_marshallIn(SFc11_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c11_u, const emlrtMsgIdentifier *c11_parentId)
{
  uint8_T c11_b_u;
  uint8_T c11_y;
  sf_mex_import(c11_parentId, sf_mex_dup(c11_u), &c11_b_u, 1, 3, 0U, 0, 0U, 0);
  c11_y = c11_b_u;
  sf_mex_destroy(&c11_u);
  return c11_y;
}

static void init_dsm_address_info(SFc11_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc11_dynamics_libInstanceStruct
  *chartInstance)
{
  chartInstance->c11_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c11_frontTrack = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c11_antiRollFrontSprings = (real_T *)
    ssGetOutputPortSignal_wrapper(chartInstance->S, 1);
  chartInstance->c11_frontWheelRate = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c11_dynamics_lib_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(1117483395U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(3344270233U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(2357246692U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(479213669U);
}

mxArray *sf_c11_dynamics_lib_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c11_dynamics_lib_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("chartInfo");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c11_dynamics_lib_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c11_dynamics_lib(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiAWYGJgYAPSHEDMxAABrFA+I1SMES7OAhdXAOKSyoJUkHhxUbJnCpDOS8wF8xN"
    "LKzzz0vLB5lswIMxnw2I+I5L5nFBxCPhgT5l+EQeQfg8k/SxY9Isg6ReA8hPzSjKD8nNy3Iry80"
    "qCC4oy89KLYeEFo8l3l4IDZfoh9gcQ8JcUmr9A/Mzi+MTkksyy1PhkQ8P4lEpgfGUmF8fnZCYhz"
    "AUBAFUoIPY="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c11_dynamics_lib_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sguz5aP2MdkthwhiEaEoDKC";
}

static void sf_opaque_initialize_c11_dynamics_lib(void *chartInstanceVar)
{
  initialize_params_c11_dynamics_lib((SFc11_dynamics_libInstanceStruct*)
    chartInstanceVar);
  initialize_c11_dynamics_lib((SFc11_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c11_dynamics_lib(void *chartInstanceVar)
{
  enable_c11_dynamics_lib((SFc11_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c11_dynamics_lib(void *chartInstanceVar)
{
  disable_c11_dynamics_lib((SFc11_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c11_dynamics_lib(void *chartInstanceVar)
{
  sf_gateway_c11_dynamics_lib((SFc11_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c11_dynamics_lib(SimStruct* S)
{
  return get_sim_state_c11_dynamics_lib((SFc11_dynamics_libInstanceStruct *)
    sf_get_chart_instance_ptr(S));     /* raw sim ctx */
}

static void sf_opaque_set_sim_state_c11_dynamics_lib(SimStruct* S, const mxArray
  *st)
{
  set_sim_state_c11_dynamics_lib((SFc11_dynamics_libInstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c11_dynamics_lib(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc11_dynamics_libInstanceStruct*) chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_dynamics_lib_optimization_info();
    }

    mdl_cleanup_runtime_resources_c11_dynamics_lib
      ((SFc11_dynamics_libInstanceStruct*) chartInstanceVar);
    ((SFc11_dynamics_libInstanceStruct*) chartInstanceVar)->
      ~SFc11_dynamics_libInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c11_dynamics_lib(void *chartInstanceVar)
{
  mdl_start_c11_dynamics_lib((SFc11_dynamics_libInstanceStruct*)
    chartInstanceVar);
  if (chartInstanceVar) {
    sf_reset_warnings_ChartRunTimeInfo(((SFc11_dynamics_libInstanceStruct*)
      chartInstanceVar)->S);
  }
}

static void sf_opaque_mdl_terminate_c11_dynamics_lib(void *chartInstanceVar)
{
  mdl_terminate_c11_dynamics_lib((SFc11_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c11_dynamics_lib(SimStruct *S)
{
  mdlProcessParamsCommon(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c11_dynamics_lib((SFc11_dynamics_libInstanceStruct*)
      sf_get_chart_instance_ptr(S));
    initSubchartIOPointersc11_dynamics_lib((SFc11_dynamics_libInstanceStruct*)
      sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c11_dynamics_lib_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [21] = {
    "eNrtWE+P20QUn0TLiqJS7aECVCFRCQnBAam7qBIHBLtNHIjYsFGdLRKX1cR+iUcZz7jzJ9n0wld",
    "AfAz4Chw4coIb4lvAkRPijeOkwQmJJ5GgRXjleMf2b37vvXn/xqTW7hA8buF5+hohh3h9Ec86mR",
    "0vFOPa0jm7f0A+KMY/vERIJGMYggjtYMCuid8hbNqliqaa+B+CpvAQtOTWMCnaYiCrY5kYgAIR4",
    "QSZVMaLV7PUciZGLSsix6w/T1iUhIm0PH6AE9L4QvDp3/Fm1nSRsckURKYFEJtESTtMWpwON1tB",
    "mUkjgWikbeptKw0mtJlTVXcsNyzjEFxD1BbaULSC3qJvaKiBhrn2M7LTV4dztEwzzqiobuuE6hA",
    "y9A4Dl1mMvxfWoPUq8iJfnwlqpGKUBylvJFRVxXY5ytlBt+bedjapaULfDodMDJ11lU1BoP7oJx",
    "VsNWjIMSg6hAvhxxs57YLrfIEXflkRa1gKj6g6i9APNMR+MYgOrEOKSws9nMYLC4L2ObR1T7Exr",
    "pFv3mi7UNopb9h05kl6J2zOG4xxVfWOvK1INCjn2g/bk9k5jIHn/E1q6A7YGb8HWGsW9yR6h4tc",
    "z+i3gj22UGAbUsSsuleOS6i8SHyGCb8CnKUuDCBGMy9EX0y0LY6sNjJtYOg3z88r8q1i28KAGtA",
    "IKudrRZkGFDj3K0/emGkXSIhGK5lcy8ozzGJwJyjRAyuaE6lGaGPfwvDUVi4S/NAQD6EJBvIkF6",
    "B3P6LcVpQ51ZiVnXtcasyyfryIdfGzEziiUQKxq4GMQwfzLE5QdYm1K59nqO2YmWkTdKRYVjWSL",
    "CZ0LH7OSr1pBpdiJOREtJRMw6KL2eBXAJg1qBJYyh5gKVXTFgpfTWoFj3t5dvdtGJydqeG073zj",
    "YxBYDZ2urnrTCKMqENhuokD7YEP2BNsRoZk22AJOgzwG4lkfTJ72wQdr+uCjpT74qDij4+OreIr",
    "dKIv0FWd9N8+9pXluVuin5334JhxZwZEFbn59ewlfW8NLlq5lvhv1v75fL/HV8a9WIyt2ernEc1",
    "DCHRZ2Gn/561ff/PTOHz9//eNb33375ivb+Gsr/LX8f4f7vu63X7lVjF+f90qLzDxeSV7u3U+2+",
    "MGrJT9wYz20T+7T7kknHplkkrCABrL5aSOf72iLvPWSvPP7d11/hoGb5wEVteNi3+PG1M76cTf/",
    "+0vyHm6xx40lfyLkt4/2w98+rWKv2yV7uTGWZfZQco7ZSJgwUxiZuuzPu8t193Q//Iy/u0WvOyW",
    "97uR9/BV12RquyplhNV539X9f3PPC9z+u+jpUqSv1HXG78pF/GLevfr718nl//96GfEZK7x89w3",
    "psyqM+fdOzptcvxK+veaMYf7jYqzcSxuM1u4XiMTb0g3VP/yP+/bun/eZ9XODsV3wg/eK9M0H5F",
    "LcHs+1Xcbur3Le5xSMFVK/fg/0b9WTxHX9Lv3KzFN9uPGEilhP97vHJ/ZN96tOftX8Pyg==",
    ""
  };

  static char newstr [1497] = "";
  newstr[0] = '\0';
  for (i = 0; i < 21; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c11_dynamics_lib(SimStruct *S)
{
  const char* newstr = sf_c11_dynamics_lib_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(2431614838U));
  ssSetChecksum1(S,(4264153002U));
  ssSetChecksum2(S,(651072214U));
  ssSetChecksum3(S,(371436476U));
}

static void mdlRTW_c11_dynamics_lib(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c11_dynamics_lib(SimStruct *S)
{
  SFc11_dynamics_libInstanceStruct *chartInstance;
  chartInstance = (SFc11_dynamics_libInstanceStruct *)utMalloc(sizeof
    (SFc11_dynamics_libInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc11_dynamics_libInstanceStruct));
  chartInstance = new (chartInstance) SFc11_dynamics_libInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway = sf_opaque_gateway_c11_dynamics_lib;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c11_dynamics_lib;
  chartInstance->chartInfo.mdlStart = sf_opaque_mdl_start_c11_dynamics_lib;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c11_dynamics_lib;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c11_dynamics_lib;
  chartInstance->chartInfo.enableChart = sf_opaque_enable_c11_dynamics_lib;
  chartInstance->chartInfo.disableChart = sf_opaque_disable_c11_dynamics_lib;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c11_dynamics_lib;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c11_dynamics_lib;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c11_dynamics_lib;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c11_dynamics_lib;
  chartInstance->chartInfo.mdlSetWorkWidths = mdlSetWorkWidths_c11_dynamics_lib;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c11_dynamics_lib(chartInstance);
}

void c11_dynamics_lib_method_dispatcher(SimStruct *S, int_T method, void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c11_dynamics_lib(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c11_dynamics_lib(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c11_dynamics_lib(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c11_dynamics_lib_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
