/* Include files */

#include "dynamics_lib_sfun.h"
#include "c22_dynamics_lib.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c22_dynamics_lib(SFc22_dynamics_libInstanceStruct
  *chartInstance);
static void initialize_params_c22_dynamics_lib(SFc22_dynamics_libInstanceStruct *
  chartInstance);
static void mdl_start_c22_dynamics_lib(SFc22_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_terminate_c22_dynamics_lib(SFc22_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_setup_runtime_resources_c22_dynamics_lib
  (SFc22_dynamics_libInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c22_dynamics_lib
  (SFc22_dynamics_libInstanceStruct *chartInstance);
static void enable_c22_dynamics_lib(SFc22_dynamics_libInstanceStruct
  *chartInstance);
static void disable_c22_dynamics_lib(SFc22_dynamics_libInstanceStruct
  *chartInstance);
static void sf_gateway_c22_dynamics_lib(SFc22_dynamics_libInstanceStruct
  *chartInstance);
static void ext_mode_exec_c22_dynamics_lib(SFc22_dynamics_libInstanceStruct
  *chartInstance);
static void c22_do_animation_call_c22_dynamics_lib
  (SFc22_dynamics_libInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c22_dynamics_lib
  (SFc22_dynamics_libInstanceStruct *chartInstance);
static void set_sim_state_c22_dynamics_lib(SFc22_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c22_st);
static void initSimStructsc22_dynamics_lib(SFc22_dynamics_libInstanceStruct
  *chartInstance);
static void initSubchartIOPointersc22_dynamics_lib
  (SFc22_dynamics_libInstanceStruct *chartInstance);
static real32_T c22_eML_blk_kernel(SFc22_dynamics_libInstanceStruct
  *chartInstance, real_T c22_b_totalMass, real32_T c22_b_totalMassDistribution,
  real_T c22_b_totalDownForce, real32_T c22_b_downForceDistribution, real32_T
  c22_b_totalFrontWT, real_T c22_b_gravity);
static real32_T c22_emlrt_marshallIn(SFc22_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c22_nullptr, const char_T *c22_identifier);
static real32_T c22_b_emlrt_marshallIn(SFc22_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c22_u, const emlrtMsgIdentifier *c22_parentId);
static uint8_T c22_c_emlrt_marshallIn(SFc22_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c22_nullptr, const char_T *c22_identifier);
static uint8_T c22_d_emlrt_marshallIn(SFc22_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c22_u, const emlrtMsgIdentifier *c22_parentId);
static void init_dsm_address_info(SFc22_dynamics_libInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc22_dynamics_libInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c22_dynamics_lib(SFc22_dynamics_libInstanceStruct
  *chartInstance)
{
  emlrtLicenseCheckR2022a(chartInstance->c22_fEmlrtCtx,
    "EMLRT:runTime:MexFunctionNeedsLicense", "distrib_computing_toolbox", 2);
  sf_is_first_init_cond(chartInstance->S);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  emlrtInitGPU(chartInstance->c22_fEmlrtCtx);
  hipGetLastError();
}

static void initialize_params_c22_dynamics_lib(SFc22_dynamics_libInstanceStruct *
  chartInstance)
{
}

static void mdl_start_c22_dynamics_lib(SFc22_dynamics_libInstanceStruct
  *chartInstance)
{
  sim_mode_is_external(chartInstance->S);
}

static void mdl_terminate_c22_dynamics_lib(SFc22_dynamics_libInstanceStruct
  *chartInstance)
{
  hipError_t c22_errCode;
  c22_errCode = hipGetLastError();
  if (c22_errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(c22_errCode), (char_T *)
                       hipGetErrorName(c22_errCode), (char_T *)
                       hipGetErrorString(c22_errCode), (char_T *)
                       "SimGPUErrorChecks", chartInstance->c22_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c22_dynamics_lib
  (SFc22_dynamics_libInstanceStruct *chartInstance)
{
  sfSetAnimationVectors(chartInstance->S, chartInstance->c22_JITStateAnimation,
                        chartInstance->c22_JITTransitionAnimation);
}

static void mdl_cleanup_runtime_resources_c22_dynamics_lib
  (SFc22_dynamics_libInstanceStruct *chartInstance)
{
}

static void enable_c22_dynamics_lib(SFc22_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c22_dynamics_lib(SFc22_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void sf_gateway_c22_dynamics_lib(SFc22_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c22_JITTransitionAnimation[0] = 0U;
  *chartInstance->c22_dynamicLoadFR = c22_eML_blk_kernel(chartInstance,
    *chartInstance->c22_totalMass, *chartInstance->c22_totalMassDistribution,
    *chartInstance->c22_totalDownForce,
    *chartInstance->c22_downForceDistribution, *chartInstance->c22_totalFrontWT,
    *chartInstance->c22_gravity);
  c22_do_animation_call_c22_dynamics_lib(chartInstance);
}

static void ext_mode_exec_c22_dynamics_lib(SFc22_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void c22_do_animation_call_c22_dynamics_lib
  (SFc22_dynamics_libInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static const mxArray *get_sim_state_c22_dynamics_lib
  (SFc22_dynamics_libInstanceStruct *chartInstance)
{
  const mxArray *c22_b_y = NULL;
  const mxArray *c22_c_y = NULL;
  const mxArray *c22_st;
  const mxArray *c22_y = NULL;
  c22_st = NULL;
  c22_st = NULL;
  c22_y = NULL;
  sf_mex_assign(&c22_y, sf_mex_createcellmatrix(2, 1), false);
  c22_b_y = NULL;
  sf_mex_assign(&c22_b_y, sf_mex_create("y", chartInstance->c22_dynamicLoadFR, 1,
    0U, 0U, 0U, 0), false);
  sf_mex_setcell(c22_y, 0, c22_b_y);
  c22_c_y = NULL;
  sf_mex_assign(&c22_c_y, sf_mex_create("y",
    &chartInstance->c22_is_active_c22_dynamics_lib, 3, 0U, 0U, 0U, 0), false);
  sf_mex_setcell(c22_y, 1, c22_c_y);
  sf_mex_assign(&c22_st, c22_y, false);
  return c22_st;
}

static void set_sim_state_c22_dynamics_lib(SFc22_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c22_st)
{
  const mxArray *c22_u;
  c22_u = sf_mex_dup(c22_st);
  *chartInstance->c22_dynamicLoadFR = c22_emlrt_marshallIn(chartInstance,
    sf_mex_dup(sf_mex_getcell(c22_u, 0)), "dynamicLoadFR");
  chartInstance->c22_is_active_c22_dynamics_lib = c22_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c22_u, 1)),
     "is_active_c22_dynamics_lib");
  sf_mex_destroy(&c22_u);
  sf_mex_destroy(&c22_st);
}

static void initSimStructsc22_dynamics_lib(SFc22_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void initSubchartIOPointersc22_dynamics_lib
  (SFc22_dynamics_libInstanceStruct *chartInstance)
{
}

const mxArray *sf_c22_dynamics_lib_get_eml_resolved_functions_info()
{
  const mxArray *c22_nameCaptureInfo = NULL;
  c22_nameCaptureInfo = NULL;
  sf_mex_assign(&c22_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c22_nameCaptureInfo;
}

static real32_T c22_eML_blk_kernel(SFc22_dynamics_libInstanceStruct
  *chartInstance, real_T c22_b_totalMass, real32_T c22_b_totalMassDistribution,
  real_T c22_b_totalDownForce, real32_T c22_b_downForceDistribution, real32_T
  c22_b_totalFrontWT, real_T c22_b_gravity)
{
  return ((static_cast<real32_T>(c22_b_totalMass) * c22_b_totalMassDistribution /
           100.0F * 0.5F + static_cast<real32_T>(c22_b_totalDownForce) *
           c22_b_downForceDistribution / 100.0F / 2.0F) + c22_b_totalFrontWT) *
    static_cast<real32_T>(c22_b_gravity);
}

static real32_T c22_emlrt_marshallIn(SFc22_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c22_nullptr, const char_T *c22_identifier)
{
  emlrtMsgIdentifier c22_thisId;
  real32_T c22_y;
  c22_thisId.fIdentifier = const_cast<const char_T *>(c22_identifier);
  c22_thisId.fParent = NULL;
  c22_thisId.bParentIsCell = false;
  c22_y = c22_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c22_nullptr),
    &c22_thisId);
  sf_mex_destroy(&c22_nullptr);
  return c22_y;
}

static real32_T c22_b_emlrt_marshallIn(SFc22_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c22_u, const emlrtMsgIdentifier *c22_parentId)
{
  real32_T c22_f;
  real32_T c22_y;
  sf_mex_import(c22_parentId, sf_mex_dup(c22_u), &c22_f, 0, 1, 0U, 0, 0U, 0);
  c22_y = c22_f;
  sf_mex_destroy(&c22_u);
  return c22_y;
}

static uint8_T c22_c_emlrt_marshallIn(SFc22_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c22_nullptr, const char_T *c22_identifier)
{
  emlrtMsgIdentifier c22_thisId;
  uint8_T c22_y;
  c22_thisId.fIdentifier = const_cast<const char_T *>(c22_identifier);
  c22_thisId.fParent = NULL;
  c22_thisId.bParentIsCell = false;
  c22_y = c22_d_emlrt_marshallIn(chartInstance, sf_mex_dup(c22_nullptr),
    &c22_thisId);
  sf_mex_destroy(&c22_nullptr);
  return c22_y;
}

static uint8_T c22_d_emlrt_marshallIn(SFc22_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c22_u, const emlrtMsgIdentifier *c22_parentId)
{
  uint8_T c22_b_u;
  uint8_T c22_y;
  sf_mex_import(c22_parentId, sf_mex_dup(c22_u), &c22_b_u, 1, 3, 0U, 0, 0U, 0);
  c22_y = c22_b_u;
  sf_mex_destroy(&c22_u);
  return c22_y;
}

static void init_dsm_address_info(SFc22_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc22_dynamics_libInstanceStruct
  *chartInstance)
{
  chartInstance->c22_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c22_dynamicLoadFR = (real32_T *)ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c22_totalMass = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c22_totalMassDistribution = (real32_T *)
    ssGetInputPortSignal_wrapper(chartInstance->S, 1);
  chartInstance->c22_totalDownForce = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 2);
  chartInstance->c22_downForceDistribution = (real32_T *)
    ssGetInputPortSignal_wrapper(chartInstance->S, 3);
  chartInstance->c22_totalFrontWT = (real32_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 4);
  chartInstance->c22_gravity = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 5);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c22_dynamics_lib_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(2691231917U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(3488839858U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(2052365322U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(932011506U);
}

mxArray *sf_c22_dynamics_lib_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c22_dynamics_lib_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("chartInfo");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c22_dynamics_lib_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c22_dynamics_lib(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiDmYGJgYAPRQMzEAAGsUD4jVIwRLs4CF1cA4pLKglSQeHFRsmcKkM5LzAXzE0s"
    "rPPPS8sHmWzAgzGfDYj4jkvmcUHEI+GBPmX4RB5B+ByT9LFj08yLpF4DyUyqBPslM9slPTHELgo"
    "YTLLzId4+CA2X6IfYHEPCPFJp/QPzM4vjE5JLMstT4ZCOjeKjviuNzMpMQ5oIAAJ23HaM="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c22_dynamics_lib_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sxNq7wXLKWBjWUIBjayGiPB";
}

static void sf_opaque_initialize_c22_dynamics_lib(void *chartInstanceVar)
{
  initialize_params_c22_dynamics_lib((SFc22_dynamics_libInstanceStruct*)
    chartInstanceVar);
  initialize_c22_dynamics_lib((SFc22_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c22_dynamics_lib(void *chartInstanceVar)
{
  enable_c22_dynamics_lib((SFc22_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c22_dynamics_lib(void *chartInstanceVar)
{
  disable_c22_dynamics_lib((SFc22_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c22_dynamics_lib(void *chartInstanceVar)
{
  sf_gateway_c22_dynamics_lib((SFc22_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c22_dynamics_lib(SimStruct* S)
{
  return get_sim_state_c22_dynamics_lib((SFc22_dynamics_libInstanceStruct *)
    sf_get_chart_instance_ptr(S));     /* raw sim ctx */
}

static void sf_opaque_set_sim_state_c22_dynamics_lib(SimStruct* S, const mxArray
  *st)
{
  set_sim_state_c22_dynamics_lib((SFc22_dynamics_libInstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c22_dynamics_lib(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc22_dynamics_libInstanceStruct*) chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_dynamics_lib_optimization_info();
    }

    mdl_cleanup_runtime_resources_c22_dynamics_lib
      ((SFc22_dynamics_libInstanceStruct*) chartInstanceVar);
    ((SFc22_dynamics_libInstanceStruct*) chartInstanceVar)->
      ~SFc22_dynamics_libInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c22_dynamics_lib(void *chartInstanceVar)
{
  mdl_start_c22_dynamics_lib((SFc22_dynamics_libInstanceStruct*)
    chartInstanceVar);
  if (chartInstanceVar) {
    sf_reset_warnings_ChartRunTimeInfo(((SFc22_dynamics_libInstanceStruct*)
      chartInstanceVar)->S);
  }
}

static void sf_opaque_mdl_terminate_c22_dynamics_lib(void *chartInstanceVar)
{
  mdl_terminate_c22_dynamics_lib((SFc22_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c22_dynamics_lib(SimStruct *S)
{
  mdlProcessParamsCommon(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c22_dynamics_lib((SFc22_dynamics_libInstanceStruct*)
      sf_get_chart_instance_ptr(S));
    initSubchartIOPointersc22_dynamics_lib((SFc22_dynamics_libInstanceStruct*)
      sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c22_dynamics_lib_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [21] = {
    "eNrtWE2P20QYnkRL1FZttQckUIVEj1yQyiIEB0R381Uist2oyXYRElpN7Df2dMdj73zkA4k/gbh",
    "x41dw5sRP4MABiT/AEfXEO46TBickHkeCFmHJ6x17nnnf95n3a0IqnVOC1128j98kpIbPG3hXyf",
    "x6LRtXVu75+wPycTb+6RYhXuxDAKJvRiM2JW6XMFGPShop4n4JGsETUDE3msWiI0ZxcSwTI5AgP",
    "FwgiaV2kqtYZDgTV20jPCtZXYTMC/thbLhfxwWpfyb47O/kJkb3UGKTSfB0G8DXoYxNELY5Dbaz",
    "IPWkEYJ3pUzkzJUC3TeJNVWdGq5ZwqE1Ba8jlKbIgtphb19TDQ09dSPZ2qv6C3QcJZxRUZzrkKo",
    "+JOgdGs4TH/+eGY3sFZSL8oZMUB1LRnkr4o2QyqLYHkc9T9GtuTPPOtJNGJogYCKw7EoTgUD70U",
    "8KcDVqxGOQNIAz4SbXs9a1pukGL/2yIFazCJ5SeeKhHyjw3WIQHVj1KW4tDHAZJywIOuTQUQPJx",
    "rhHrnmjY0OpVN4w0dyTVClsKrc1xl1VJeW2PdGgnCs37CBOujAGnspvUk1LYOfyHcBKMX8Qo3fY",
    "yHWMfiPYtYEM24iFz4p75TiHSovEY0z4BeAssmEAPtK8VH250K44MkrHUQNDv9ntFpS3ju0IDXJ",
    "EPSicryVlClDh1K8c5fpM2UBCNLKkUysLrzCPwVJQokZGNCexvEKOXQvDC65sJLihwQ+gCRrSJN",
    "dC735KuSmoc6QwK1v3OFeYZd3kItbGTymwR70QfFsDGYdTzLO4QNEtVrZ8nqC1Y6ZnTVCeZEnRS",
    "DKY0LH4WZYGswTOxZWIJ6It46ifdTFb/AoAswaVAktZHUupnLVR+WJaS7gepNndtWGwPFPN6dD6",
    "xiMQWA2trbZ6Uw+jqiWw3USF9sH22VfYjgjFlMYWcNZKY8Cf98HkRR98sKEPPlzpgw+z2zs6uvR",
    "n2I0yT11yNrTrPFhZ53aBfnrRh2/DkTUcWeIWz3dW8JUNcsnK087/aGX+zepf59dy8qqLd+lF1v",
    "i6k5N3kMPXMr6+/SX69dZ33wfffPn86+c//qbyduf1qKzpUUn/t7gfqm7nlrvZ+K1Fz7TM0OO1J",
    "GbnfrrDH97I+YMdq+nj6w8nn3c/u6g/uzjv1J/R2SPWq6fr3dihbzWn7+L9fdunYQCn+UB6HT87",
    "/9gxNfO+PL+ftR183FzxK0J+f7gf/vXjIvFzJ8eXHWex042p336S8+fy+tw/3g8/l9/bYc+9nD3",
    "30j7+ktpsDZf5zLAer2X93hX3qsj7H1d8H4rUlVpJXFl55B/G7Wufa7181ec/2JLPSG7+4Utsx7",
    "Y86tI3vWx2/Uzc+pm3s/Eny7N6I2Tc33BayD5jQz/a9PU/4t9/OPK36N9alr/sB9Iv3j8RlM/we",
    "DA/fmWve9L+Nrf8JIGqzWewf6OeLH/H39Gv3M7Ftx1PmPDjiXr3vaMPjvapT38CaSINGg==",
    ""
  };

  static char newstr [1497] = "";
  newstr[0] = '\0';
  for (i = 0; i < 21; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c22_dynamics_lib(SimStruct *S)
{
  const char* newstr = sf_c22_dynamics_lib_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(3765296276U));
  ssSetChecksum1(S,(1738709770U));
  ssSetChecksum2(S,(2113559954U));
  ssSetChecksum3(S,(1944306426U));
}

static void mdlRTW_c22_dynamics_lib(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c22_dynamics_lib(SimStruct *S)
{
  SFc22_dynamics_libInstanceStruct *chartInstance;
  chartInstance = (SFc22_dynamics_libInstanceStruct *)utMalloc(sizeof
    (SFc22_dynamics_libInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc22_dynamics_libInstanceStruct));
  chartInstance = new (chartInstance) SFc22_dynamics_libInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway = sf_opaque_gateway_c22_dynamics_lib;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c22_dynamics_lib;
  chartInstance->chartInfo.mdlStart = sf_opaque_mdl_start_c22_dynamics_lib;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c22_dynamics_lib;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c22_dynamics_lib;
  chartInstance->chartInfo.enableChart = sf_opaque_enable_c22_dynamics_lib;
  chartInstance->chartInfo.disableChart = sf_opaque_disable_c22_dynamics_lib;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c22_dynamics_lib;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c22_dynamics_lib;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c22_dynamics_lib;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c22_dynamics_lib;
  chartInstance->chartInfo.mdlSetWorkWidths = mdlSetWorkWidths_c22_dynamics_lib;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c22_dynamics_lib(chartInstance);
}

void c22_dynamics_lib_method_dispatcher(SimStruct *S, int_T method, void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c22_dynamics_lib(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c22_dynamics_lib(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c22_dynamics_lib(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c22_dynamics_lib_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
