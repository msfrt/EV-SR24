/* Include files */

#include "dynamics_lib_sfun.h"
#include "c2_dynamics_lib.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c2_dynamics_lib(SFc2_dynamics_libInstanceStruct
  *chartInstance);
static void initialize_params_c2_dynamics_lib(SFc2_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_start_c2_dynamics_lib(SFc2_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_terminate_c2_dynamics_lib(SFc2_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_setup_runtime_resources_c2_dynamics_lib
  (SFc2_dynamics_libInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c2_dynamics_lib
  (SFc2_dynamics_libInstanceStruct *chartInstance);
static void enable_c2_dynamics_lib(SFc2_dynamics_libInstanceStruct
  *chartInstance);
static void disable_c2_dynamics_lib(SFc2_dynamics_libInstanceStruct
  *chartInstance);
static void sf_gateway_c2_dynamics_lib(SFc2_dynamics_libInstanceStruct
  *chartInstance);
static void ext_mode_exec_c2_dynamics_lib(SFc2_dynamics_libInstanceStruct
  *chartInstance);
static void c2_do_animation_call_c2_dynamics_lib(SFc2_dynamics_libInstanceStruct
  *chartInstance);
static const mxArray *get_sim_state_c2_dynamics_lib
  (SFc2_dynamics_libInstanceStruct *chartInstance);
static void set_sim_state_c2_dynamics_lib(SFc2_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c2_st);
static void initSimStructsc2_dynamics_lib(SFc2_dynamics_libInstanceStruct
  *chartInstance);
static void initSubchartIOPointersc2_dynamics_lib
  (SFc2_dynamics_libInstanceStruct *chartInstance);
static real_T c2_eML_blk_kernel(SFc2_dynamics_libInstanceStruct *chartInstance,
  real_T c2_b_rearSpringStiffness, real_T c2_b_rearSpringMotionRatio);
static real_T c2_emlrt_marshallIn(SFc2_dynamics_libInstanceStruct *chartInstance,
  const mxArray *c2_nullptr, const char_T *c2_identifier);
static real_T c2_b_emlrt_marshallIn(SFc2_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c2_u, const emlrtMsgIdentifier *c2_parentId);
static uint8_T c2_c_emlrt_marshallIn(SFc2_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c2_nullptr, const char_T *c2_identifier);
static uint8_T c2_d_emlrt_marshallIn(SFc2_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c2_u, const emlrtMsgIdentifier *c2_parentId);
static void init_dsm_address_info(SFc2_dynamics_libInstanceStruct *chartInstance);
static void init_simulink_io_address(SFc2_dynamics_libInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c2_dynamics_lib(SFc2_dynamics_libInstanceStruct
  *chartInstance)
{
  emlrtLicenseCheckR2022a(chartInstance->c2_fEmlrtCtx,
    "EMLRT:runTime:MexFunctionNeedsLicense", "distrib_computing_toolbox", 2);
  sf_is_first_init_cond(chartInstance->S);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  emlrtInitGPU(chartInstance->c2_fEmlrtCtx);
  hipGetLastError();
}

static void initialize_params_c2_dynamics_lib(SFc2_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void mdl_start_c2_dynamics_lib(SFc2_dynamics_libInstanceStruct
  *chartInstance)
{
  sim_mode_is_external(chartInstance->S);
}

static void mdl_terminate_c2_dynamics_lib(SFc2_dynamics_libInstanceStruct
  *chartInstance)
{
  hipError_t c2_errCode;
  c2_errCode = hipGetLastError();
  if (c2_errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(c2_errCode), (char_T *)
                       hipGetErrorName(c2_errCode), (char_T *)
                       hipGetErrorString(c2_errCode), (char_T *)
                       "SimGPUErrorChecks", chartInstance->c2_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c2_dynamics_lib
  (SFc2_dynamics_libInstanceStruct *chartInstance)
{
  sfSetAnimationVectors(chartInstance->S, chartInstance->c2_JITStateAnimation,
                        chartInstance->c2_JITTransitionAnimation);
}

static void mdl_cleanup_runtime_resources_c2_dynamics_lib
  (SFc2_dynamics_libInstanceStruct *chartInstance)
{
}

static void enable_c2_dynamics_lib(SFc2_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c2_dynamics_lib(SFc2_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void sf_gateway_c2_dynamics_lib(SFc2_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c2_JITTransitionAnimation[0] = 0U;
  *chartInstance->c2_rearWheelRate = c2_eML_blk_kernel(chartInstance,
    *chartInstance->c2_rearSpringStiffness,
    *chartInstance->c2_rearSpringMotionRatio);
  c2_do_animation_call_c2_dynamics_lib(chartInstance);
}

static void ext_mode_exec_c2_dynamics_lib(SFc2_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void c2_do_animation_call_c2_dynamics_lib(SFc2_dynamics_libInstanceStruct
  *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static const mxArray *get_sim_state_c2_dynamics_lib
  (SFc2_dynamics_libInstanceStruct *chartInstance)
{
  const mxArray *c2_b_y = NULL;
  const mxArray *c2_c_y = NULL;
  const mxArray *c2_st;
  const mxArray *c2_y = NULL;
  c2_st = NULL;
  c2_st = NULL;
  c2_y = NULL;
  sf_mex_assign(&c2_y, sf_mex_createcellmatrix(2, 1), false);
  c2_b_y = NULL;
  sf_mex_assign(&c2_b_y, sf_mex_create("y", chartInstance->c2_rearWheelRate, 0,
    0U, 0U, 0U, 0), false);
  sf_mex_setcell(c2_y, 0, c2_b_y);
  c2_c_y = NULL;
  sf_mex_assign(&c2_c_y, sf_mex_create("y",
    &chartInstance->c2_is_active_c2_dynamics_lib, 3, 0U, 0U, 0U, 0), false);
  sf_mex_setcell(c2_y, 1, c2_c_y);
  sf_mex_assign(&c2_st, c2_y, false);
  return c2_st;
}

static void set_sim_state_c2_dynamics_lib(SFc2_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c2_st)
{
  const mxArray *c2_u;
  c2_u = sf_mex_dup(c2_st);
  *chartInstance->c2_rearWheelRate = c2_emlrt_marshallIn(chartInstance,
    sf_mex_dup(sf_mex_getcell(c2_u, 0)), "rearWheelRate");
  chartInstance->c2_is_active_c2_dynamics_lib = c2_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c2_u, 1)),
     "is_active_c2_dynamics_lib");
  sf_mex_destroy(&c2_u);
  sf_mex_destroy(&c2_st);
}

static void initSimStructsc2_dynamics_lib(SFc2_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void initSubchartIOPointersc2_dynamics_lib
  (SFc2_dynamics_libInstanceStruct *chartInstance)
{
}

const mxArray *sf_c2_dynamics_lib_get_eml_resolved_functions_info()
{
  const mxArray *c2_nameCaptureInfo = NULL;
  c2_nameCaptureInfo = NULL;
  sf_mex_assign(&c2_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c2_nameCaptureInfo;
}

static real_T c2_eML_blk_kernel(SFc2_dynamics_libInstanceStruct *chartInstance,
  real_T c2_b_rearSpringStiffness, real_T c2_b_rearSpringMotionRatio)
{
  return c2_b_rearSpringStiffness / (c2_b_rearSpringMotionRatio *
    c2_b_rearSpringMotionRatio);
}

static real_T c2_emlrt_marshallIn(SFc2_dynamics_libInstanceStruct *chartInstance,
  const mxArray *c2_nullptr, const char_T *c2_identifier)
{
  emlrtMsgIdentifier c2_thisId;
  real_T c2_y;
  c2_thisId.fIdentifier = const_cast<const char_T *>(c2_identifier);
  c2_thisId.fParent = NULL;
  c2_thisId.bParentIsCell = false;
  c2_y = c2_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c2_nullptr), &c2_thisId);
  sf_mex_destroy(&c2_nullptr);
  return c2_y;
}

static real_T c2_b_emlrt_marshallIn(SFc2_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c2_u, const emlrtMsgIdentifier *c2_parentId)
{
  real_T c2_d;
  real_T c2_y;
  sf_mex_import(c2_parentId, sf_mex_dup(c2_u), &c2_d, 1, 0, 0U, 0, 0U, 0);
  c2_y = c2_d;
  sf_mex_destroy(&c2_u);
  return c2_y;
}

static uint8_T c2_c_emlrt_marshallIn(SFc2_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c2_nullptr, const char_T *c2_identifier)
{
  emlrtMsgIdentifier c2_thisId;
  uint8_T c2_y;
  c2_thisId.fIdentifier = const_cast<const char_T *>(c2_identifier);
  c2_thisId.fParent = NULL;
  c2_thisId.bParentIsCell = false;
  c2_y = c2_d_emlrt_marshallIn(chartInstance, sf_mex_dup(c2_nullptr), &c2_thisId);
  sf_mex_destroy(&c2_nullptr);
  return c2_y;
}

static uint8_T c2_d_emlrt_marshallIn(SFc2_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c2_u, const emlrtMsgIdentifier *c2_parentId)
{
  uint8_T c2_b_u;
  uint8_T c2_y;
  sf_mex_import(c2_parentId, sf_mex_dup(c2_u), &c2_b_u, 1, 3, 0U, 0, 0U, 0);
  c2_y = c2_b_u;
  sf_mex_destroy(&c2_u);
  return c2_y;
}

static void init_dsm_address_info(SFc2_dynamics_libInstanceStruct *chartInstance)
{
}

static void init_simulink_io_address(SFc2_dynamics_libInstanceStruct
  *chartInstance)
{
  chartInstance->c2_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c2_rearSpringStiffness = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c2_rearWheelRate = (real_T *)ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c2_rearSpringMotionRatio = (real_T *)
    ssGetInputPortSignal_wrapper(chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c2_dynamics_lib_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(144108107U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(2729250480U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(3658348481U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(1914598297U);
}

mxArray *sf_c2_dynamics_lib_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c2_dynamics_lib_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("chartInfo");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c2_dynamics_lib_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c2_dynamics_lib(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiDmYGJgYAPRQMzEAAGsUD4jVIwRLs4CF1cA4pLKglSQeHFRsmcKkM5LzAXzE0s"
    "rPPPS8sHmWzAgzGfDYj4jkvmcUHEI+GBPmX4RB5B+ByT9LFj08yLpF4Dyi1ITi8IzUlNzghJLUq"
    "HhBAsv8t2j4ECZfoj9AQT8I4nmHxA/szg+Mbkksyw1PtkoPqUSGE2ZycXxOZlJSOaCAADHMB2d"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c2_dynamics_lib_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "ss7rWVwETCFpW2ESquArRkC";
}

static void sf_opaque_initialize_c2_dynamics_lib(void *chartInstanceVar)
{
  initialize_params_c2_dynamics_lib((SFc2_dynamics_libInstanceStruct*)
    chartInstanceVar);
  initialize_c2_dynamics_lib((SFc2_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c2_dynamics_lib(void *chartInstanceVar)
{
  enable_c2_dynamics_lib((SFc2_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c2_dynamics_lib(void *chartInstanceVar)
{
  disable_c2_dynamics_lib((SFc2_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c2_dynamics_lib(void *chartInstanceVar)
{
  sf_gateway_c2_dynamics_lib((SFc2_dynamics_libInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c2_dynamics_lib(SimStruct* S)
{
  return get_sim_state_c2_dynamics_lib((SFc2_dynamics_libInstanceStruct *)
    sf_get_chart_instance_ptr(S));     /* raw sim ctx */
}

static void sf_opaque_set_sim_state_c2_dynamics_lib(SimStruct* S, const mxArray *
  st)
{
  set_sim_state_c2_dynamics_lib((SFc2_dynamics_libInstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c2_dynamics_lib(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc2_dynamics_libInstanceStruct*) chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_dynamics_lib_optimization_info();
    }

    mdl_cleanup_runtime_resources_c2_dynamics_lib
      ((SFc2_dynamics_libInstanceStruct*) chartInstanceVar);
    ((SFc2_dynamics_libInstanceStruct*) chartInstanceVar)->
      ~SFc2_dynamics_libInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c2_dynamics_lib(void *chartInstanceVar)
{
  mdl_start_c2_dynamics_lib((SFc2_dynamics_libInstanceStruct*) chartInstanceVar);
  if (chartInstanceVar) {
    sf_reset_warnings_ChartRunTimeInfo(((SFc2_dynamics_libInstanceStruct*)
      chartInstanceVar)->S);
  }
}

static void sf_opaque_mdl_terminate_c2_dynamics_lib(void *chartInstanceVar)
{
  mdl_terminate_c2_dynamics_lib((SFc2_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c2_dynamics_lib(SimStruct *S)
{
  mdlProcessParamsCommon(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c2_dynamics_lib((SFc2_dynamics_libInstanceStruct*)
      sf_get_chart_instance_ptr(S));
    initSubchartIOPointersc2_dynamics_lib((SFc2_dynamics_libInstanceStruct*)
      sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c2_dynamics_lib_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [21] = {
    "eNrtWE9v40QUn0Sl2l1tlz0gLSutRI9ckJYiBEgI2nUcCGppVKetxIFqYr/Eo4zH7vzJHy58BCS",
    "+A1+AE0eOiBs3PgZHbvDGcbLBCYknkWAX4chxxvZvfu+9ef8mpNY6I3g8wPP91wnZx+sdPOtker",
    "xSjGsL5/T+HvmwGP90j5AwjaAPIjC9HhsTt0OYpE0lTRRxPwRN4AJUyo1mqWiJXlody0QPJIgQJ",
    "8hSqZ14FUsMZ2LQNCK0zOo6ZmEcxKnh0TOckEbngk/+jjczuo2MDSYh1E2ASMcyNf24yWl/vRWk",
    "HnkxhANlEmdbKdCByayq6sxwzTIO/hjCllCaohXUBn0DTTV4euxmZKuvCmboNMk4o6K6rWOqAsj",
    "QOzRcZhF+nxuN1qvIi3xdJqhOJaPcT7gXU1kV2+Yo5xm6NXe2s050A7qm32eib60rTQIC9Uc/qW",
    "CrnpcOQdI+nAs33tBq54/zBZ77ZUWsZglcUXkSoh8oiNxiEB1YBRSXFjo4jRMWBO1yaKmOZENcI",
    "9e80bKhtFXeMMnUk9RW2JzXH+Kqqi15m6HwKOfKDdtJs1MYAs/5G1TTLbBTfgewUizqpOgdNnId",
    "o98IdmugwHqpiFh1rxyWUHmR+BwTfgU4S2wYQIRmnos+n2hTHBml08TD0G+cnlbkW8a2hAbZoyF",
    "UzteSMgUocO5XjrwRUzaQEI1W0rmWlWeYxuBWUKJ6RjRGqRygjV0Lw3Nb2UhwQ0PUhwZoyJOcj9",
    "59RbmpKHOiMCtb97hUmGXdeBFr42crcEjDGCJbAxmHM8yzOEHVJVa2fJ6gtkOmJw1QoWRZ1Ugym",
    "NCx+FkrdSYZXIqBSEeiKdMkKLqYNX4FgFmDSoGl7BmWUjlpovDVpJZw28mzu2vDYO1MNadd6xuf",
    "gMBqaHW11ZuGGFW+wHYTBdoFG7CvsB0RiimNLeDEz2MgyvvgY/K8D95b0Qe/utAHPyzG4dFNNMF",
    "mlIXqhrNuPs/ThXnuV+inZ334OhxZwpE5bnZ9cwFfW8FLFq5lvrv1v75fL/HV8VOrkSU7HZR49k",
    "q4/cJWf3zwzdeHtS9//OX7b8fdnz/7bhN/bYm/lv+2uB/qbvuVB8X4yaxXmmfm4VLysu9+usEPH",
    "pX84FFeK9+T11cjv+M1s+sjP7g1J/Ji4OXz3dkgb70k7+z+oe3PMHDzPCDDVlTse+yYmmk/nu/f",
    "FuTd32CPuwv+RMhvH++Gf+24StwclOx1kOcIKq9jAH6B+a3kx9vLc3i8G37K396gz+OSPo/z/v2",
    "G2iwNN+WMsByn2/q9K+5l4fsfV30dqtST+pa4bfnIP4zbVT/XOvmyv/90TT4jpfcfvsB6rMujLv",
    "3Si6bXr8Stn3mjGH8036N7MePRil1C8Rgb+d6qp/8R//7d0X6z/s239iv+GP3inRNB+QS3BdNtV",
    "3G7Le1/cvNH2LWo1Xuvf6OezK6b+q/7pfi24xETUTpSb7199O7RLvXpT7uTCv4=",
    ""
  };

  static char newstr [1489] = "";
  newstr[0] = '\0';
  for (i = 0; i < 21; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c2_dynamics_lib(SimStruct *S)
{
  const char* newstr = sf_c2_dynamics_lib_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(2140092927U));
  ssSetChecksum1(S,(3227386144U));
  ssSetChecksum2(S,(2022879953U));
  ssSetChecksum3(S,(2773142114U));
}

static void mdlRTW_c2_dynamics_lib(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c2_dynamics_lib(SimStruct *S)
{
  SFc2_dynamics_libInstanceStruct *chartInstance;
  chartInstance = (SFc2_dynamics_libInstanceStruct *)utMalloc(sizeof
    (SFc2_dynamics_libInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc2_dynamics_libInstanceStruct));
  chartInstance = new (chartInstance) SFc2_dynamics_libInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway = sf_opaque_gateway_c2_dynamics_lib;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c2_dynamics_lib;
  chartInstance->chartInfo.mdlStart = sf_opaque_mdl_start_c2_dynamics_lib;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c2_dynamics_lib;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c2_dynamics_lib;
  chartInstance->chartInfo.enableChart = sf_opaque_enable_c2_dynamics_lib;
  chartInstance->chartInfo.disableChart = sf_opaque_disable_c2_dynamics_lib;
  chartInstance->chartInfo.getSimState = sf_opaque_get_sim_state_c2_dynamics_lib;
  chartInstance->chartInfo.setSimState = sf_opaque_set_sim_state_c2_dynamics_lib;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c2_dynamics_lib;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c2_dynamics_lib;
  chartInstance->chartInfo.mdlSetWorkWidths = mdlSetWorkWidths_c2_dynamics_lib;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c2_dynamics_lib(chartInstance);
}

void c2_dynamics_lib_method_dispatcher(SimStruct *S, int_T method, void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c2_dynamics_lib(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c2_dynamics_lib(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c2_dynamics_lib(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c2_dynamics_lib_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
