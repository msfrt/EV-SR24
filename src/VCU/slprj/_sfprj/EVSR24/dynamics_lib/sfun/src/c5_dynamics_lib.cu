/* Include files */

#include "dynamics_lib_sfun.h"
#include "c5_dynamics_lib.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c5_dynamics_lib(SFc5_dynamics_libInstanceStruct
  *chartInstance);
static void initialize_params_c5_dynamics_lib(SFc5_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_start_c5_dynamics_lib(SFc5_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_terminate_c5_dynamics_lib(SFc5_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_setup_runtime_resources_c5_dynamics_lib
  (SFc5_dynamics_libInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c5_dynamics_lib
  (SFc5_dynamics_libInstanceStruct *chartInstance);
static void enable_c5_dynamics_lib(SFc5_dynamics_libInstanceStruct
  *chartInstance);
static void disable_c5_dynamics_lib(SFc5_dynamics_libInstanceStruct
  *chartInstance);
static void sf_gateway_c5_dynamics_lib(SFc5_dynamics_libInstanceStruct
  *chartInstance);
static void ext_mode_exec_c5_dynamics_lib(SFc5_dynamics_libInstanceStruct
  *chartInstance);
static void c5_do_animation_call_c5_dynamics_lib(SFc5_dynamics_libInstanceStruct
  *chartInstance);
static const mxArray *get_sim_state_c5_dynamics_lib
  (SFc5_dynamics_libInstanceStruct *chartInstance);
static void set_sim_state_c5_dynamics_lib(SFc5_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c5_st);
static void initSimStructsc5_dynamics_lib(SFc5_dynamics_libInstanceStruct
  *chartInstance);
static void initSubchartIOPointersc5_dynamics_lib
  (SFc5_dynamics_libInstanceStruct *chartInstance);
static real32_T c5_eML_blk_kernel(SFc5_dynamics_libInstanceStruct *chartInstance,
  real_T c5_b_frontTrack, real32_T c5_b_frontTireStiffness);
static real32_T c5_emlrt_marshallIn(SFc5_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c5_nullptr, const char_T *c5_identifier);
static real32_T c5_b_emlrt_marshallIn(SFc5_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c5_u, const emlrtMsgIdentifier *c5_parentId);
static uint8_T c5_c_emlrt_marshallIn(SFc5_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c5_nullptr, const char_T *c5_identifier);
static uint8_T c5_d_emlrt_marshallIn(SFc5_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c5_u, const emlrtMsgIdentifier *c5_parentId);
static void init_dsm_address_info(SFc5_dynamics_libInstanceStruct *chartInstance);
static void init_simulink_io_address(SFc5_dynamics_libInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c5_dynamics_lib(SFc5_dynamics_libInstanceStruct
  *chartInstance)
{
  emlrtLicenseCheckR2022a(chartInstance->c5_fEmlrtCtx,
    "EMLRT:runTime:MexFunctionNeedsLicense", "distrib_computing_toolbox", 2);
  sf_is_first_init_cond(chartInstance->S);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  emlrtInitGPU(chartInstance->c5_fEmlrtCtx);
  hipGetLastError();
}

static void initialize_params_c5_dynamics_lib(SFc5_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void mdl_start_c5_dynamics_lib(SFc5_dynamics_libInstanceStruct
  *chartInstance)
{
  sim_mode_is_external(chartInstance->S);
}

static void mdl_terminate_c5_dynamics_lib(SFc5_dynamics_libInstanceStruct
  *chartInstance)
{
  hipError_t c5_errCode;
  c5_errCode = hipGetLastError();
  if (c5_errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(c5_errCode), (char_T *)
                       hipGetErrorName(c5_errCode), (char_T *)
                       hipGetErrorString(c5_errCode), (char_T *)
                       "SimGPUErrorChecks", chartInstance->c5_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c5_dynamics_lib
  (SFc5_dynamics_libInstanceStruct *chartInstance)
{
  sfSetAnimationVectors(chartInstance->S, chartInstance->c5_JITStateAnimation,
                        chartInstance->c5_JITTransitionAnimation);
}

static void mdl_cleanup_runtime_resources_c5_dynamics_lib
  (SFc5_dynamics_libInstanceStruct *chartInstance)
{
}

static void enable_c5_dynamics_lib(SFc5_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c5_dynamics_lib(SFc5_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void sf_gateway_c5_dynamics_lib(SFc5_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c5_JITTransitionAnimation[0] = 0U;
  *chartInstance->c5_antiRollFrontTires = c5_eML_blk_kernel(chartInstance,
    *chartInstance->c5_frontTrack, *chartInstance->c5_frontTireStiffness);
  c5_do_animation_call_c5_dynamics_lib(chartInstance);
}

static void ext_mode_exec_c5_dynamics_lib(SFc5_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void c5_do_animation_call_c5_dynamics_lib(SFc5_dynamics_libInstanceStruct
  *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static const mxArray *get_sim_state_c5_dynamics_lib
  (SFc5_dynamics_libInstanceStruct *chartInstance)
{
  const mxArray *c5_b_y = NULL;
  const mxArray *c5_c_y = NULL;
  const mxArray *c5_st;
  const mxArray *c5_y = NULL;
  c5_st = NULL;
  c5_st = NULL;
  c5_y = NULL;
  sf_mex_assign(&c5_y, sf_mex_createcellmatrix(2, 1), false);
  c5_b_y = NULL;
  sf_mex_assign(&c5_b_y, sf_mex_create("y", chartInstance->c5_antiRollFrontTires,
    1, 0U, 0U, 0U, 0), false);
  sf_mex_setcell(c5_y, 0, c5_b_y);
  c5_c_y = NULL;
  sf_mex_assign(&c5_c_y, sf_mex_create("y",
    &chartInstance->c5_is_active_c5_dynamics_lib, 3, 0U, 0U, 0U, 0), false);
  sf_mex_setcell(c5_y, 1, c5_c_y);
  sf_mex_assign(&c5_st, c5_y, false);
  return c5_st;
}

static void set_sim_state_c5_dynamics_lib(SFc5_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c5_st)
{
  const mxArray *c5_u;
  c5_u = sf_mex_dup(c5_st);
  *chartInstance->c5_antiRollFrontTires = c5_emlrt_marshallIn(chartInstance,
    sf_mex_dup(sf_mex_getcell(c5_u, 0)), "antiRollFrontTires");
  chartInstance->c5_is_active_c5_dynamics_lib = c5_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c5_u, 1)),
     "is_active_c5_dynamics_lib");
  sf_mex_destroy(&c5_u);
  sf_mex_destroy(&c5_st);
}

static void initSimStructsc5_dynamics_lib(SFc5_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void initSubchartIOPointersc5_dynamics_lib
  (SFc5_dynamics_libInstanceStruct *chartInstance)
{
}

const mxArray *sf_c5_dynamics_lib_get_eml_resolved_functions_info()
{
  const mxArray *c5_nameCaptureInfo = NULL;
  c5_nameCaptureInfo = NULL;
  sf_mex_assign(&c5_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c5_nameCaptureInfo;
}

static real32_T c5_eML_blk_kernel(SFc5_dynamics_libInstanceStruct *chartInstance,
  real_T c5_b_frontTrack, real32_T c5_b_frontTireStiffness)
{
  real_T c5_a;
  c5_a = c5_b_frontTrack / 1000.0;
  return static_cast<real32_T>(c5_a * c5_a * 1.5574077246549023) *
    c5_b_frontTireStiffness * 1000.0F / 2.0F;
}

static real32_T c5_emlrt_marshallIn(SFc5_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c5_nullptr, const char_T *c5_identifier)
{
  emlrtMsgIdentifier c5_thisId;
  real32_T c5_y;
  c5_thisId.fIdentifier = const_cast<const char_T *>(c5_identifier);
  c5_thisId.fParent = NULL;
  c5_thisId.bParentIsCell = false;
  c5_y = c5_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c5_nullptr), &c5_thisId);
  sf_mex_destroy(&c5_nullptr);
  return c5_y;
}

static real32_T c5_b_emlrt_marshallIn(SFc5_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c5_u, const emlrtMsgIdentifier *c5_parentId)
{
  real32_T c5_f;
  real32_T c5_y;
  sf_mex_import(c5_parentId, sf_mex_dup(c5_u), &c5_f, 0, 1, 0U, 0, 0U, 0);
  c5_y = c5_f;
  sf_mex_destroy(&c5_u);
  return c5_y;
}

static uint8_T c5_c_emlrt_marshallIn(SFc5_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c5_nullptr, const char_T *c5_identifier)
{
  emlrtMsgIdentifier c5_thisId;
  uint8_T c5_y;
  c5_thisId.fIdentifier = const_cast<const char_T *>(c5_identifier);
  c5_thisId.fParent = NULL;
  c5_thisId.bParentIsCell = false;
  c5_y = c5_d_emlrt_marshallIn(chartInstance, sf_mex_dup(c5_nullptr), &c5_thisId);
  sf_mex_destroy(&c5_nullptr);
  return c5_y;
}

static uint8_T c5_d_emlrt_marshallIn(SFc5_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c5_u, const emlrtMsgIdentifier *c5_parentId)
{
  uint8_T c5_b_u;
  uint8_T c5_y;
  sf_mex_import(c5_parentId, sf_mex_dup(c5_u), &c5_b_u, 1, 3, 0U, 0, 0U, 0);
  c5_y = c5_b_u;
  sf_mex_destroy(&c5_u);
  return c5_y;
}

static void init_dsm_address_info(SFc5_dynamics_libInstanceStruct *chartInstance)
{
}

static void init_simulink_io_address(SFc5_dynamics_libInstanceStruct
  *chartInstance)
{
  chartInstance->c5_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c5_frontTrack = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c5_antiRollFrontTires = (real32_T *)
    ssGetOutputPortSignal_wrapper(chartInstance->S, 1);
  chartInstance->c5_frontTireStiffness = (real32_T *)
    ssGetInputPortSignal_wrapper(chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c5_dynamics_lib_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(3107291137U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(4055247592U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(3894949891U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(3894335377U);
}

mxArray *sf_c5_dynamics_lib_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c5_dynamics_lib_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("chartInfo");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c5_dynamics_lib_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c5_dynamics_lib(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiAWYGJgYAPSHEDMxAABrFA+I1SMES7OAhdXAOKSyoJUkHhxUbJnCpDOS8wF8xN"
    "LKzzz0vLB5lswIMxnw2I+I5L5nFBxCPhgT5l+EQeQfg8k/SxY9Ash6ReA8hPzSjKD8nNy3Iry80"
    "pCMotSiyHyfAwImnx3KThQph9ifwABf0mi+QvEzyyOT0wuySxLjU82jU+pBEZXZnJxfE5mEgOq/"
    "wCB7R/k"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c5_dynamics_lib_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sKWXgFHw0v60WmuNXaku1L";
}

static void sf_opaque_initialize_c5_dynamics_lib(void *chartInstanceVar)
{
  initialize_params_c5_dynamics_lib((SFc5_dynamics_libInstanceStruct*)
    chartInstanceVar);
  initialize_c5_dynamics_lib((SFc5_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c5_dynamics_lib(void *chartInstanceVar)
{
  enable_c5_dynamics_lib((SFc5_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c5_dynamics_lib(void *chartInstanceVar)
{
  disable_c5_dynamics_lib((SFc5_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c5_dynamics_lib(void *chartInstanceVar)
{
  sf_gateway_c5_dynamics_lib((SFc5_dynamics_libInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c5_dynamics_lib(SimStruct* S)
{
  return get_sim_state_c5_dynamics_lib((SFc5_dynamics_libInstanceStruct *)
    sf_get_chart_instance_ptr(S));     /* raw sim ctx */
}

static void sf_opaque_set_sim_state_c5_dynamics_lib(SimStruct* S, const mxArray *
  st)
{
  set_sim_state_c5_dynamics_lib((SFc5_dynamics_libInstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c5_dynamics_lib(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc5_dynamics_libInstanceStruct*) chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_dynamics_lib_optimization_info();
    }

    mdl_cleanup_runtime_resources_c5_dynamics_lib
      ((SFc5_dynamics_libInstanceStruct*) chartInstanceVar);
    ((SFc5_dynamics_libInstanceStruct*) chartInstanceVar)->
      ~SFc5_dynamics_libInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c5_dynamics_lib(void *chartInstanceVar)
{
  mdl_start_c5_dynamics_lib((SFc5_dynamics_libInstanceStruct*) chartInstanceVar);
  if (chartInstanceVar) {
    sf_reset_warnings_ChartRunTimeInfo(((SFc5_dynamics_libInstanceStruct*)
      chartInstanceVar)->S);
  }
}

static void sf_opaque_mdl_terminate_c5_dynamics_lib(void *chartInstanceVar)
{
  mdl_terminate_c5_dynamics_lib((SFc5_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c5_dynamics_lib(SimStruct *S)
{
  mdlProcessParamsCommon(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c5_dynamics_lib((SFc5_dynamics_libInstanceStruct*)
      sf_get_chart_instance_ptr(S));
    initSubchartIOPointersc5_dynamics_lib((SFc5_dynamics_libInstanceStruct*)
      sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c5_dynamics_lib_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [21] = {
    "eNrtWM2P20QUn0TLiqJSVggBlZDoEQ5I26IiDgh2mw8asR9Rk92tuKwm9os9ynjszkc++A/4O7h",
    "x5w4XxJ/AjQs3LpwQR97YThqckHgcCVpUrxzv2P7N770372tMap1TgsctPI/eJmQfry/jWSfZ8V",
    "I+ri2d2f098kk+/ukVQrzYhwBEzwyHbErcDmGiLpU0UsT9EDSCR6BibjSLRUcM4/JYJoYgQXg4Q",
    "RJL7cSrWGQ4E6O2EZ5lVlch88JeGBvuP8AJqX8u+OyfeBOju8jYZBI83QbwdShjE4RtToPNVpB6",
    "0gjBGykTOdtKge6ZxKqqTg3XLOHQmoLXEUpTtILaom9PUw0NPXUzstVX9eboOEo4o6K8rUOqepC",
    "gd2i4SHz8PTcarVeSF/kGTFAdS0Z5K+KNkMqy2C5HOU/RrbmznXWkmzAwQcBEYK0rTQQC9Uc/KW",
    "GrYSMeg6QBnAs3Xs9q15qmC7zwy5JYzSK4pPLYQz9Q4LvFIDqw6lFcWujjNE5YEHTAoaP6ko1xj",
    "VzzRseGUqW8YaLMk1QlbMrbGuOqqoq8bU80KOfKDduPkxMYA0/5m1TTCtiM3wGsFPP7MXqHjVzH",
    "6DeCPTGQYxux8Fl5rxwXUGmROMOEXwLOIhsG4KOZF6IvJtoWR0bpOGpg6DdPTkryrWI7QoMcUg9",
    "K52tJmQIUOPUrR16fKRtIiEYr6VTL0jNkMVgJStTQiOYkliO0sWtheGorGwluaPADaIKGNMm10L",
    "svKTclZY4UZmXrHhcKs6wbL2Jt/FQCe9QLwbc1kHE4xTyLE5RdYmXL5zFqO2Z61gTlSZaUjSSDC",
    "R2Ln7VSf5bAhRiJeCLaMo56eRezwa8AMGtQKbCUPcBSKmdtFL6c1BKe9NPs7towWDtTzenA+sbn",
    "ILAaWl1t9aYeRlVLYLuJAu2C7bGvsB0RiimNLeCslcaAn/XB5GkfvLemD35tqQ8+yMfe/Wt/hs0",
    "o89Q1Z4N0nsOleW6W6KfnffgmHFnBkQVufn1vCV9bw0uWrkW+G/W/v18v8NXxr1YjK3Z6tcCzV8",
    "Dt57b67fL9H85+mc6+Ib/++PUf3363jb+2wl9L/7e47+tu+5Vb+fidea+0yMzjleRl3324xQ/eL",
    "PiBHasvrh4H7YeTw/FHh1eROXtMR+buSTbfwRZ56wV55/fv2P4MAzfNA9Lr+Pm+x46pyfpxO//H",
    "S/Lub7HHjSV/IuT3z3bDv3FUxl6vF+xlx1iW2aOYc8xGQvdxP6RW/bm6XHeOdsNn/N0tet0u6HU",
    "77eOvqc3WcF3MDKv6VfV/V9zzwvcCV34dytSVekVcVT7yL+N21c+1Xj7v7x9uyGek8P7BM6zHpj",
    "zq0jc9a3r9TNz6mnfz8aeLvXojZNxfs1vIH2NDP1z39H/i33862u+tfNyy9ss/kH754bGgfIbbg",
    "2z7ld/uSvttbvFIAlXr92D/RT1ZfMff0q/cLMS3HU+Y8OOJ+uDuvfv3dqlPfwHiOA4g",
    ""
  };

  static char newstr [1493] = "";
  newstr[0] = '\0';
  for (i = 0; i < 21; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c5_dynamics_lib(SimStruct *S)
{
  const char* newstr = sf_c5_dynamics_lib_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(3240711915U));
  ssSetChecksum1(S,(2037964622U));
  ssSetChecksum2(S,(3336831139U));
  ssSetChecksum3(S,(3047880329U));
}

static void mdlRTW_c5_dynamics_lib(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c5_dynamics_lib(SimStruct *S)
{
  SFc5_dynamics_libInstanceStruct *chartInstance;
  chartInstance = (SFc5_dynamics_libInstanceStruct *)utMalloc(sizeof
    (SFc5_dynamics_libInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc5_dynamics_libInstanceStruct));
  chartInstance = new (chartInstance) SFc5_dynamics_libInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway = sf_opaque_gateway_c5_dynamics_lib;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c5_dynamics_lib;
  chartInstance->chartInfo.mdlStart = sf_opaque_mdl_start_c5_dynamics_lib;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c5_dynamics_lib;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c5_dynamics_lib;
  chartInstance->chartInfo.enableChart = sf_opaque_enable_c5_dynamics_lib;
  chartInstance->chartInfo.disableChart = sf_opaque_disable_c5_dynamics_lib;
  chartInstance->chartInfo.getSimState = sf_opaque_get_sim_state_c5_dynamics_lib;
  chartInstance->chartInfo.setSimState = sf_opaque_set_sim_state_c5_dynamics_lib;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c5_dynamics_lib;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c5_dynamics_lib;
  chartInstance->chartInfo.mdlSetWorkWidths = mdlSetWorkWidths_c5_dynamics_lib;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c5_dynamics_lib(chartInstance);
}

void c5_dynamics_lib_method_dispatcher(SimStruct *S, int_T method, void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c5_dynamics_lib(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c5_dynamics_lib(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c5_dynamics_lib(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c5_dynamics_lib_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
