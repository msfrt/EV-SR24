/* Include files */

#include "dynamics_lib_sfun.h"
#include "c14_dynamics_lib.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c14_dynamics_lib(SFc14_dynamics_libInstanceStruct
  *chartInstance);
static void initialize_params_c14_dynamics_lib(SFc14_dynamics_libInstanceStruct *
  chartInstance);
static void mdl_start_c14_dynamics_lib(SFc14_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_terminate_c14_dynamics_lib(SFc14_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_setup_runtime_resources_c14_dynamics_lib
  (SFc14_dynamics_libInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c14_dynamics_lib
  (SFc14_dynamics_libInstanceStruct *chartInstance);
static void enable_c14_dynamics_lib(SFc14_dynamics_libInstanceStruct
  *chartInstance);
static void disable_c14_dynamics_lib(SFc14_dynamics_libInstanceStruct
  *chartInstance);
static void sf_gateway_c14_dynamics_lib(SFc14_dynamics_libInstanceStruct
  *chartInstance);
static void ext_mode_exec_c14_dynamics_lib(SFc14_dynamics_libInstanceStruct
  *chartInstance);
static void c14_do_animation_call_c14_dynamics_lib
  (SFc14_dynamics_libInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c14_dynamics_lib
  (SFc14_dynamics_libInstanceStruct *chartInstance);
static void set_sim_state_c14_dynamics_lib(SFc14_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c14_st);
static void initSimStructsc14_dynamics_lib(SFc14_dynamics_libInstanceStruct
  *chartInstance);
static void initSubchartIOPointersc14_dynamics_lib
  (SFc14_dynamics_libInstanceStruct *chartInstance);
static real32_T c14_eML_blk_kernel(SFc14_dynamics_libInstanceStruct
  *chartInstance, real_T c14_b_antiRollFrontSprings, real32_T
  c14_b_antiRollFrontARB, real32_T c14_b_antiRollFrontTires);
static real32_T c14_emlrt_marshallIn(SFc14_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c14_nullptr, const char_T *c14_identifier);
static real32_T c14_b_emlrt_marshallIn(SFc14_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c14_u, const emlrtMsgIdentifier *c14_parentId);
static uint8_T c14_c_emlrt_marshallIn(SFc14_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c14_nullptr, const char_T *c14_identifier);
static uint8_T c14_d_emlrt_marshallIn(SFc14_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c14_u, const emlrtMsgIdentifier *c14_parentId);
static void init_dsm_address_info(SFc14_dynamics_libInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc14_dynamics_libInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c14_dynamics_lib(SFc14_dynamics_libInstanceStruct
  *chartInstance)
{
  emlrtLicenseCheckR2022a(chartInstance->c14_fEmlrtCtx,
    "EMLRT:runTime:MexFunctionNeedsLicense", "distrib_computing_toolbox", 2);
  sf_is_first_init_cond(chartInstance->S);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  emlrtInitGPU(chartInstance->c14_fEmlrtCtx);
  hipGetLastError();
}

static void initialize_params_c14_dynamics_lib(SFc14_dynamics_libInstanceStruct *
  chartInstance)
{
}

static void mdl_start_c14_dynamics_lib(SFc14_dynamics_libInstanceStruct
  *chartInstance)
{
  sim_mode_is_external(chartInstance->S);
}

static void mdl_terminate_c14_dynamics_lib(SFc14_dynamics_libInstanceStruct
  *chartInstance)
{
  hipError_t c14_errCode;
  c14_errCode = hipGetLastError();
  if (c14_errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(c14_errCode), (char_T *)
                       hipGetErrorName(c14_errCode), (char_T *)
                       hipGetErrorString(c14_errCode), (char_T *)
                       "SimGPUErrorChecks", chartInstance->c14_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c14_dynamics_lib
  (SFc14_dynamics_libInstanceStruct *chartInstance)
{
  sfSetAnimationVectors(chartInstance->S, chartInstance->c14_JITStateAnimation,
                        chartInstance->c14_JITTransitionAnimation);
}

static void mdl_cleanup_runtime_resources_c14_dynamics_lib
  (SFc14_dynamics_libInstanceStruct *chartInstance)
{
}

static void enable_c14_dynamics_lib(SFc14_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c14_dynamics_lib(SFc14_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void sf_gateway_c14_dynamics_lib(SFc14_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c14_JITTransitionAnimation[0] = 0U;
  *chartInstance->c14_antiRollFrontAxleAndTire = c14_eML_blk_kernel
    (chartInstance, *chartInstance->c14_antiRollFrontSprings,
     *chartInstance->c14_antiRollFrontARB,
     *chartInstance->c14_antiRollFrontTires);
  c14_do_animation_call_c14_dynamics_lib(chartInstance);
}

static void ext_mode_exec_c14_dynamics_lib(SFc14_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void c14_do_animation_call_c14_dynamics_lib
  (SFc14_dynamics_libInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static const mxArray *get_sim_state_c14_dynamics_lib
  (SFc14_dynamics_libInstanceStruct *chartInstance)
{
  const mxArray *c14_b_y = NULL;
  const mxArray *c14_c_y = NULL;
  const mxArray *c14_st;
  const mxArray *c14_y = NULL;
  c14_st = NULL;
  c14_st = NULL;
  c14_y = NULL;
  sf_mex_assign(&c14_y, sf_mex_createcellmatrix(2, 1), false);
  c14_b_y = NULL;
  sf_mex_assign(&c14_b_y, sf_mex_create("y",
    chartInstance->c14_antiRollFrontAxleAndTire, 1, 0U, 0U, 0U, 0), false);
  sf_mex_setcell(c14_y, 0, c14_b_y);
  c14_c_y = NULL;
  sf_mex_assign(&c14_c_y, sf_mex_create("y",
    &chartInstance->c14_is_active_c14_dynamics_lib, 3, 0U, 0U, 0U, 0), false);
  sf_mex_setcell(c14_y, 1, c14_c_y);
  sf_mex_assign(&c14_st, c14_y, false);
  return c14_st;
}

static void set_sim_state_c14_dynamics_lib(SFc14_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c14_st)
{
  const mxArray *c14_u;
  c14_u = sf_mex_dup(c14_st);
  *chartInstance->c14_antiRollFrontAxleAndTire = c14_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c14_u, 0)),
     "antiRollFrontAxleAndTire");
  chartInstance->c14_is_active_c14_dynamics_lib = c14_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c14_u, 1)),
     "is_active_c14_dynamics_lib");
  sf_mex_destroy(&c14_u);
  sf_mex_destroy(&c14_st);
}

static void initSimStructsc14_dynamics_lib(SFc14_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void initSubchartIOPointersc14_dynamics_lib
  (SFc14_dynamics_libInstanceStruct *chartInstance)
{
}

const mxArray *sf_c14_dynamics_lib_get_eml_resolved_functions_info()
{
  const mxArray *c14_nameCaptureInfo = NULL;
  c14_nameCaptureInfo = NULL;
  sf_mex_assign(&c14_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c14_nameCaptureInfo;
}

static real32_T c14_eML_blk_kernel(SFc14_dynamics_libInstanceStruct
  *chartInstance, real_T c14_b_antiRollFrontSprings, real32_T
  c14_b_antiRollFrontARB, real32_T c14_b_antiRollFrontTires)
{
  return (static_cast<real32_T>(c14_b_antiRollFrontSprings) +
          c14_b_antiRollFrontARB) * c14_b_antiRollFrontTires /
    ((static_cast<real32_T>(c14_b_antiRollFrontSprings) + c14_b_antiRollFrontARB)
     + c14_b_antiRollFrontTires);
}

static real32_T c14_emlrt_marshallIn(SFc14_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c14_nullptr, const char_T *c14_identifier)
{
  emlrtMsgIdentifier c14_thisId;
  real32_T c14_y;
  c14_thisId.fIdentifier = const_cast<const char_T *>(c14_identifier);
  c14_thisId.fParent = NULL;
  c14_thisId.bParentIsCell = false;
  c14_y = c14_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c14_nullptr),
    &c14_thisId);
  sf_mex_destroy(&c14_nullptr);
  return c14_y;
}

static real32_T c14_b_emlrt_marshallIn(SFc14_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c14_u, const emlrtMsgIdentifier *c14_parentId)
{
  real32_T c14_f;
  real32_T c14_y;
  sf_mex_import(c14_parentId, sf_mex_dup(c14_u), &c14_f, 0, 1, 0U, 0, 0U, 0);
  c14_y = c14_f;
  sf_mex_destroy(&c14_u);
  return c14_y;
}

static uint8_T c14_c_emlrt_marshallIn(SFc14_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c14_nullptr, const char_T *c14_identifier)
{
  emlrtMsgIdentifier c14_thisId;
  uint8_T c14_y;
  c14_thisId.fIdentifier = const_cast<const char_T *>(c14_identifier);
  c14_thisId.fParent = NULL;
  c14_thisId.bParentIsCell = false;
  c14_y = c14_d_emlrt_marshallIn(chartInstance, sf_mex_dup(c14_nullptr),
    &c14_thisId);
  sf_mex_destroy(&c14_nullptr);
  return c14_y;
}

static uint8_T c14_d_emlrt_marshallIn(SFc14_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c14_u, const emlrtMsgIdentifier *c14_parentId)
{
  uint8_T c14_b_u;
  uint8_T c14_y;
  sf_mex_import(c14_parentId, sf_mex_dup(c14_u), &c14_b_u, 1, 3, 0U, 0, 0U, 0);
  c14_y = c14_b_u;
  sf_mex_destroy(&c14_u);
  return c14_y;
}

static void init_dsm_address_info(SFc14_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc14_dynamics_libInstanceStruct
  *chartInstance)
{
  chartInstance->c14_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c14_antiRollFrontAxleAndTire = (real32_T *)
    ssGetOutputPortSignal_wrapper(chartInstance->S, 1);
  chartInstance->c14_antiRollFrontSprings = (real_T *)
    ssGetInputPortSignal_wrapper(chartInstance->S, 0);
  chartInstance->c14_antiRollFrontARB = (real32_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c14_antiRollFrontTires = (real32_T *)
    ssGetInputPortSignal_wrapper(chartInstance->S, 2);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c14_dynamics_lib_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(1060234072U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(980335451U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(3189463588U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(4073111086U);
}

mxArray *sf_c14_dynamics_lib_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c14_dynamics_lib_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("chartInfo");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c14_dynamics_lib_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c14_dynamics_lib(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiAWYGJgYAPSHEDMxAABrFA+I1SMES7OAhdXAOKSyoJUkHhxUbJnCpDOS8wF8xN"
    "LKzzz0vLB5lswIMxnw2I+I5L5nFBxCPhgT5l+EQeQfg8k/SxY9Esg6ReA8hPzSjKD8nNy3Iry80"
    "ocK3JSHfNSQjKLUvmg6ihzl4IDZfoh9gcQ8JcUmr9A/Mzi+MTkksyy1PhkQ5P4lEpgfGUmF8fnZ"
    "CYhzAUBAIuPIkw="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c14_dynamics_lib_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "szgPCLVfBNLzBqi6PctNnzB";
}

static void sf_opaque_initialize_c14_dynamics_lib(void *chartInstanceVar)
{
  initialize_params_c14_dynamics_lib((SFc14_dynamics_libInstanceStruct*)
    chartInstanceVar);
  initialize_c14_dynamics_lib((SFc14_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c14_dynamics_lib(void *chartInstanceVar)
{
  enable_c14_dynamics_lib((SFc14_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c14_dynamics_lib(void *chartInstanceVar)
{
  disable_c14_dynamics_lib((SFc14_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c14_dynamics_lib(void *chartInstanceVar)
{
  sf_gateway_c14_dynamics_lib((SFc14_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c14_dynamics_lib(SimStruct* S)
{
  return get_sim_state_c14_dynamics_lib((SFc14_dynamics_libInstanceStruct *)
    sf_get_chart_instance_ptr(S));     /* raw sim ctx */
}

static void sf_opaque_set_sim_state_c14_dynamics_lib(SimStruct* S, const mxArray
  *st)
{
  set_sim_state_c14_dynamics_lib((SFc14_dynamics_libInstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c14_dynamics_lib(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc14_dynamics_libInstanceStruct*) chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_dynamics_lib_optimization_info();
    }

    mdl_cleanup_runtime_resources_c14_dynamics_lib
      ((SFc14_dynamics_libInstanceStruct*) chartInstanceVar);
    ((SFc14_dynamics_libInstanceStruct*) chartInstanceVar)->
      ~SFc14_dynamics_libInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c14_dynamics_lib(void *chartInstanceVar)
{
  mdl_start_c14_dynamics_lib((SFc14_dynamics_libInstanceStruct*)
    chartInstanceVar);
  if (chartInstanceVar) {
    sf_reset_warnings_ChartRunTimeInfo(((SFc14_dynamics_libInstanceStruct*)
      chartInstanceVar)->S);
  }
}

static void sf_opaque_mdl_terminate_c14_dynamics_lib(void *chartInstanceVar)
{
  mdl_terminate_c14_dynamics_lib((SFc14_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c14_dynamics_lib(SimStruct *S)
{
  mdlProcessParamsCommon(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c14_dynamics_lib((SFc14_dynamics_libInstanceStruct*)
      sf_get_chart_instance_ptr(S));
    initSubchartIOPointersc14_dynamics_lib((SFc14_dynamics_libInstanceStruct*)
      sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c14_dynamics_lib_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [21] = {
    "eNrtWMtu20YUHSmu0RRp4EWBBEaAZNlNgCR9IIuitZ6NANkWItkFujFG5KU48HBIz0MPf0f2/ZU",
    "uuugn9CcKdFmgm96hKEWlVIlDAW1SlABFDckz59479zUklc4pweM+nicPCTnE64d4Vsn8+CAbV1",
    "bO+f0D8lU2/vkjQrzYhxGIvgkCNiVuhzBRj0oaKeJ+CBrBa1AxN5rFoiOCuDiWiQAkCA8nSGKpn",
    "XgViwxn4rpthGeZ1Xch88J+GBvu13FC6p8LPvs73sToHjI2mQRPtwF8HcrYjMI2p6PtVpB60gjB",
    "u1YmcraVAt03iVVVnRquWcKhNQWvI5SmaAW1Q9++phoaeupmZKuv6i/QcZRwRkVxW4dU9SFB79B",
    "wkfj4e240Wq8gL/INmaA6lozyVsQbIZVFsT2Ocp6iW3NnO+tIN2FoRiMmRta60kQgUH/0kwK2Ch",
    "rxGCQdwblw4/Wsdq1pusBLvyyI1SyCSyprHvqBAt8tBtGBVZ/i0sIAp3HCgqBDDh01kGyMa+SaN",
    "zo2lErlDRPNPUmVwqa8rTGuqirJ2/ZEg3Ku3LCDOOnCGHjK36SalsDO+R3ASjF/EKN32Mh1jH4j",
    "2I2BDNuIhc+Ke+U4h0qLxBkm/AJwFtkwAB/NvBR9OdGuODJKx1EDQ7/Z7RbkW8d2hAYZUA8K52t",
    "JmQIUOPUrR16fKRtIiEYr6VTLwjPMY7AUlKjAiOYkltdoY9fC8NZWNhLc0OCPoAka0iTXQu++pN",
    "wUlDlSmJWte1wozLJuvIi18VMK7FEvBN/WQMbhFPMsTlB0iZUtnzXUdsz0rAnKkywpGkkGEzoWP",
    "2ulwSyBC3Et4oloyzjqZ13MFr8CwKxBpcBSVsdSKmdtFL6Y1BJuBml2d20YrJ2p5nRofeNbEFgN",
    "ra62elMPo6olsN1EgfbB9tkttiNCMaWxBZy10hjw530wedsHH2zog49W+uCj7PSef37lz7AbZZ6",
    "64mxo53m2Ms+9Av30og/fhiNrOLLELa6fruArG3jJyjXPd7f61/fv5PiqeKdSqazZ6eMcz0EOd5",
    "jZ6dXLX5/+9Iax4x/+mAQPHt3s4q+s8VfS/xb3Y9Vtv3I/Gz9a9ErLzDxeS1723Vc7/OBBzg/sW",
    "N2Oeo3uZVA/697Wb9iXPU+fidt6Ot/RDnmrOXkX95/Y/gwDN80D0uv42b7HjqmZ9+N2/pcr8h7u",
    "sMfdFX8i5Ldv9sN/clLEXg9z9rJjLMvsdcw5ZiOha1MONeEPcFu06s/l5Xpysh9+zt/boddxTq/",
    "jtI+/ojZbw1U+M6zHa1n/d8W9L3z/44qvQ5G6cqckriwf+Ydx++rnWi/f9/efbclnJPf+0Tusx7",
    "Y86tI3vWt6/ULc+prH2fjr5V69ETLub9gtZI+xoQ82Pf2P+PfvjvZb9HEta7/sA+n3n9UE5TPcH",
    "sy3X9ntnrTf5paPJFC1eQ/2b9ST5Xf8Hf3KvVx82/GECT+eqKfPX3zxYp/69CdDZw76",
    ""
  };

  static char newstr [1493] = "";
  newstr[0] = '\0';
  for (i = 0; i < 21; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c14_dynamics_lib(SimStruct *S)
{
  const char* newstr = sf_c14_dynamics_lib_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(770521160U));
  ssSetChecksum1(S,(1768527044U));
  ssSetChecksum2(S,(2012978970U));
  ssSetChecksum3(S,(1897666406U));
}

static void mdlRTW_c14_dynamics_lib(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c14_dynamics_lib(SimStruct *S)
{
  SFc14_dynamics_libInstanceStruct *chartInstance;
  chartInstance = (SFc14_dynamics_libInstanceStruct *)utMalloc(sizeof
    (SFc14_dynamics_libInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc14_dynamics_libInstanceStruct));
  chartInstance = new (chartInstance) SFc14_dynamics_libInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway = sf_opaque_gateway_c14_dynamics_lib;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c14_dynamics_lib;
  chartInstance->chartInfo.mdlStart = sf_opaque_mdl_start_c14_dynamics_lib;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c14_dynamics_lib;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c14_dynamics_lib;
  chartInstance->chartInfo.enableChart = sf_opaque_enable_c14_dynamics_lib;
  chartInstance->chartInfo.disableChart = sf_opaque_disable_c14_dynamics_lib;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c14_dynamics_lib;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c14_dynamics_lib;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c14_dynamics_lib;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c14_dynamics_lib;
  chartInstance->chartInfo.mdlSetWorkWidths = mdlSetWorkWidths_c14_dynamics_lib;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c14_dynamics_lib(chartInstance);
}

void c14_dynamics_lib_method_dispatcher(SimStruct *S, int_T method, void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c14_dynamics_lib(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c14_dynamics_lib(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c14_dynamics_lib(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c14_dynamics_lib_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
