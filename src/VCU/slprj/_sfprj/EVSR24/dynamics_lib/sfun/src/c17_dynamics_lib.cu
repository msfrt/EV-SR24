/* Include files */

#include "dynamics_lib_sfun.h"
#include "c17_dynamics_lib.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c17_dynamics_lib(SFc17_dynamics_libInstanceStruct
  *chartInstance);
static void initialize_params_c17_dynamics_lib(SFc17_dynamics_libInstanceStruct *
  chartInstance);
static void mdl_start_c17_dynamics_lib(SFc17_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_terminate_c17_dynamics_lib(SFc17_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_setup_runtime_resources_c17_dynamics_lib
  (SFc17_dynamics_libInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c17_dynamics_lib
  (SFc17_dynamics_libInstanceStruct *chartInstance);
static void enable_c17_dynamics_lib(SFc17_dynamics_libInstanceStruct
  *chartInstance);
static void disable_c17_dynamics_lib(SFc17_dynamics_libInstanceStruct
  *chartInstance);
static void sf_gateway_c17_dynamics_lib(SFc17_dynamics_libInstanceStruct
  *chartInstance);
static void ext_mode_exec_c17_dynamics_lib(SFc17_dynamics_libInstanceStruct
  *chartInstance);
static void c17_do_animation_call_c17_dynamics_lib
  (SFc17_dynamics_libInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c17_dynamics_lib
  (SFc17_dynamics_libInstanceStruct *chartInstance);
static void set_sim_state_c17_dynamics_lib(SFc17_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c17_st);
static void initSimStructsc17_dynamics_lib(SFc17_dynamics_libInstanceStruct
  *chartInstance);
static void initSubchartIOPointersc17_dynamics_lib
  (SFc17_dynamics_libInstanceStruct *chartInstance);
static real32_T c17_eML_blk_kernel(SFc17_dynamics_libInstanceStruct
  *chartInstance, real_T c17_b_suspendedMass, real_T c17_b_MM5_ay, real_T
  c17_b_deltaZ, real32_T c17_b_antiRollFrontAxleAndTire, real32_T
  c17_b_antiRollTotal, real_T c17_b_frontTrack);
static real32_T c17_emlrt_marshallIn(SFc17_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c17_nullptr, const char_T *c17_identifier);
static real32_T c17_b_emlrt_marshallIn(SFc17_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c17_u, const emlrtMsgIdentifier *c17_parentId);
static uint8_T c17_c_emlrt_marshallIn(SFc17_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c17_nullptr, const char_T *c17_identifier);
static uint8_T c17_d_emlrt_marshallIn(SFc17_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c17_u, const emlrtMsgIdentifier *c17_parentId);
static void init_dsm_address_info(SFc17_dynamics_libInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc17_dynamics_libInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c17_dynamics_lib(SFc17_dynamics_libInstanceStruct
  *chartInstance)
{
  emlrtLicenseCheckR2022a(chartInstance->c17_fEmlrtCtx,
    "EMLRT:runTime:MexFunctionNeedsLicense", "distrib_computing_toolbox", 2);
  sf_is_first_init_cond(chartInstance->S);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  emlrtInitGPU(chartInstance->c17_fEmlrtCtx);
  hipGetLastError();
}

static void initialize_params_c17_dynamics_lib(SFc17_dynamics_libInstanceStruct *
  chartInstance)
{
}

static void mdl_start_c17_dynamics_lib(SFc17_dynamics_libInstanceStruct
  *chartInstance)
{
  sim_mode_is_external(chartInstance->S);
}

static void mdl_terminate_c17_dynamics_lib(SFc17_dynamics_libInstanceStruct
  *chartInstance)
{
  hipError_t c17_errCode;
  c17_errCode = hipGetLastError();
  if (c17_errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(c17_errCode), (char_T *)
                       hipGetErrorName(c17_errCode), (char_T *)
                       hipGetErrorString(c17_errCode), (char_T *)
                       "SimGPUErrorChecks", chartInstance->c17_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c17_dynamics_lib
  (SFc17_dynamics_libInstanceStruct *chartInstance)
{
  sfSetAnimationVectors(chartInstance->S, chartInstance->c17_JITStateAnimation,
                        chartInstance->c17_JITTransitionAnimation);
}

static void mdl_cleanup_runtime_resources_c17_dynamics_lib
  (SFc17_dynamics_libInstanceStruct *chartInstance)
{
}

static void enable_c17_dynamics_lib(SFc17_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c17_dynamics_lib(SFc17_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void sf_gateway_c17_dynamics_lib(SFc17_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c17_JITTransitionAnimation[0] = 0U;
  *chartInstance->c17_frontElWT = c17_eML_blk_kernel(chartInstance,
    *chartInstance->c17_suspendedMass, *chartInstance->c17_MM5_ay,
    *chartInstance->c17_deltaZ, *chartInstance->c17_antiRollFrontAxleAndTire,
    *chartInstance->c17_antiRollTotal, *chartInstance->c17_frontTrack);
  c17_do_animation_call_c17_dynamics_lib(chartInstance);
}

static void ext_mode_exec_c17_dynamics_lib(SFc17_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void c17_do_animation_call_c17_dynamics_lib
  (SFc17_dynamics_libInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static const mxArray *get_sim_state_c17_dynamics_lib
  (SFc17_dynamics_libInstanceStruct *chartInstance)
{
  const mxArray *c17_b_y = NULL;
  const mxArray *c17_c_y = NULL;
  const mxArray *c17_st;
  const mxArray *c17_y = NULL;
  c17_st = NULL;
  c17_st = NULL;
  c17_y = NULL;
  sf_mex_assign(&c17_y, sf_mex_createcellmatrix(2, 1), false);
  c17_b_y = NULL;
  sf_mex_assign(&c17_b_y, sf_mex_create("y", chartInstance->c17_frontElWT, 1, 0U,
    0U, 0U, 0), false);
  sf_mex_setcell(c17_y, 0, c17_b_y);
  c17_c_y = NULL;
  sf_mex_assign(&c17_c_y, sf_mex_create("y",
    &chartInstance->c17_is_active_c17_dynamics_lib, 3, 0U, 0U, 0U, 0), false);
  sf_mex_setcell(c17_y, 1, c17_c_y);
  sf_mex_assign(&c17_st, c17_y, false);
  return c17_st;
}

static void set_sim_state_c17_dynamics_lib(SFc17_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c17_st)
{
  const mxArray *c17_u;
  c17_u = sf_mex_dup(c17_st);
  *chartInstance->c17_frontElWT = c17_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getcell(c17_u, 0)), "frontElWT");
  chartInstance->c17_is_active_c17_dynamics_lib = c17_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c17_u, 1)),
     "is_active_c17_dynamics_lib");
  sf_mex_destroy(&c17_u);
  sf_mex_destroy(&c17_st);
}

static void initSimStructsc17_dynamics_lib(SFc17_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void initSubchartIOPointersc17_dynamics_lib
  (SFc17_dynamics_libInstanceStruct *chartInstance)
{
}

const mxArray *sf_c17_dynamics_lib_get_eml_resolved_functions_info()
{
  const mxArray *c17_nameCaptureInfo = NULL;
  c17_nameCaptureInfo = NULL;
  sf_mex_assign(&c17_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c17_nameCaptureInfo;
}

static real32_T c17_eML_blk_kernel(SFc17_dynamics_libInstanceStruct
  *chartInstance, real_T c17_b_suspendedMass, real_T c17_b_MM5_ay, real_T
  c17_b_deltaZ, real32_T c17_b_antiRollFrontAxleAndTire, real32_T
  c17_b_antiRollTotal, real_T c17_b_frontTrack)
{
  return static_cast<real32_T>(c17_b_suspendedMass * c17_b_MM5_ay * c17_b_deltaZ)
    * (c17_b_antiRollFrontAxleAndTire / c17_b_antiRollTotal) /
    static_cast<real32_T>(c17_b_frontTrack);
}

static real32_T c17_emlrt_marshallIn(SFc17_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c17_nullptr, const char_T *c17_identifier)
{
  emlrtMsgIdentifier c17_thisId;
  real32_T c17_y;
  c17_thisId.fIdentifier = const_cast<const char_T *>(c17_identifier);
  c17_thisId.fParent = NULL;
  c17_thisId.bParentIsCell = false;
  c17_y = c17_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c17_nullptr),
    &c17_thisId);
  sf_mex_destroy(&c17_nullptr);
  return c17_y;
}

static real32_T c17_b_emlrt_marshallIn(SFc17_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c17_u, const emlrtMsgIdentifier *c17_parentId)
{
  real32_T c17_f;
  real32_T c17_y;
  sf_mex_import(c17_parentId, sf_mex_dup(c17_u), &c17_f, 0, 1, 0U, 0, 0U, 0);
  c17_y = c17_f;
  sf_mex_destroy(&c17_u);
  return c17_y;
}

static uint8_T c17_c_emlrt_marshallIn(SFc17_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c17_nullptr, const char_T *c17_identifier)
{
  emlrtMsgIdentifier c17_thisId;
  uint8_T c17_y;
  c17_thisId.fIdentifier = const_cast<const char_T *>(c17_identifier);
  c17_thisId.fParent = NULL;
  c17_thisId.bParentIsCell = false;
  c17_y = c17_d_emlrt_marshallIn(chartInstance, sf_mex_dup(c17_nullptr),
    &c17_thisId);
  sf_mex_destroy(&c17_nullptr);
  return c17_y;
}

static uint8_T c17_d_emlrt_marshallIn(SFc17_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c17_u, const emlrtMsgIdentifier *c17_parentId)
{
  uint8_T c17_b_u;
  uint8_T c17_y;
  sf_mex_import(c17_parentId, sf_mex_dup(c17_u), &c17_b_u, 1, 3, 0U, 0, 0U, 0);
  c17_y = c17_b_u;
  sf_mex_destroy(&c17_u);
  return c17_y;
}

static void init_dsm_address_info(SFc17_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc17_dynamics_libInstanceStruct
  *chartInstance)
{
  chartInstance->c17_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c17_frontElWT = (real32_T *)ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c17_suspendedMass = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c17_MM5_ay = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c17_deltaZ = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 2);
  chartInstance->c17_antiRollFrontAxleAndTire = (real32_T *)
    ssGetInputPortSignal_wrapper(chartInstance->S, 3);
  chartInstance->c17_antiRollTotal = (real32_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 4);
  chartInstance->c17_frontTrack = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 5);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c17_dynamics_lib_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(2358284852U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(2715706970U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(3374312784U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(329161484U);
}

mxArray *sf_c17_dynamics_lib_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c17_dynamics_lib_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("chartInfo");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c17_dynamics_lib_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c17_dynamics_lib(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiDmYGJgYAPRQMzEAAGsUD4jVIwRLs4CF1cA4pLKglSQeHFRsmcKkM5LzAXzE0s"
    "rPPPS8sHmWzAgzGfDYj4jkvmcUHEI+GBPmX4RB5B+ByT9LFj0cyLpF4Dy04ry80pcc8JDoOJ8SD"
    "T57lFwoEw/xP4AAv6RQvMPiJ9ZHJ+YXJJZlhqfbGgen1IJjKfM5OL4nMwkVP8BADitHCc="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c17_dynamics_lib_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sKZiLMSeDjbdnuSb2DtURk";
}

static void sf_opaque_initialize_c17_dynamics_lib(void *chartInstanceVar)
{
  initialize_params_c17_dynamics_lib((SFc17_dynamics_libInstanceStruct*)
    chartInstanceVar);
  initialize_c17_dynamics_lib((SFc17_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c17_dynamics_lib(void *chartInstanceVar)
{
  enable_c17_dynamics_lib((SFc17_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c17_dynamics_lib(void *chartInstanceVar)
{
  disable_c17_dynamics_lib((SFc17_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c17_dynamics_lib(void *chartInstanceVar)
{
  sf_gateway_c17_dynamics_lib((SFc17_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c17_dynamics_lib(SimStruct* S)
{
  return get_sim_state_c17_dynamics_lib((SFc17_dynamics_libInstanceStruct *)
    sf_get_chart_instance_ptr(S));     /* raw sim ctx */
}

static void sf_opaque_set_sim_state_c17_dynamics_lib(SimStruct* S, const mxArray
  *st)
{
  set_sim_state_c17_dynamics_lib((SFc17_dynamics_libInstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c17_dynamics_lib(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc17_dynamics_libInstanceStruct*) chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_dynamics_lib_optimization_info();
    }

    mdl_cleanup_runtime_resources_c17_dynamics_lib
      ((SFc17_dynamics_libInstanceStruct*) chartInstanceVar);
    ((SFc17_dynamics_libInstanceStruct*) chartInstanceVar)->
      ~SFc17_dynamics_libInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c17_dynamics_lib(void *chartInstanceVar)
{
  mdl_start_c17_dynamics_lib((SFc17_dynamics_libInstanceStruct*)
    chartInstanceVar);
  if (chartInstanceVar) {
    sf_reset_warnings_ChartRunTimeInfo(((SFc17_dynamics_libInstanceStruct*)
      chartInstanceVar)->S);
  }
}

static void sf_opaque_mdl_terminate_c17_dynamics_lib(void *chartInstanceVar)
{
  mdl_terminate_c17_dynamics_lib((SFc17_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c17_dynamics_lib(SimStruct *S)
{
  mdlProcessParamsCommon(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c17_dynamics_lib((SFc17_dynamics_libInstanceStruct*)
      sf_get_chart_instance_ptr(S));
    initSubchartIOPointersc17_dynamics_lib((SFc17_dynamics_libInstanceStruct*)
      sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c17_dynamics_lib_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [21] = {
    "eNrtWN2O20QUnkRL1EWl7AWiqEJib5C4QWoXIRBCdLf5gYgNG9XZVqqQVhP7JB4yHrvzk2x4EF6",
    "BR4BHAPECfQzu4IILzthOGpyQeBwJWoQlr3fs+eac8835m5Bat0fwuoX36VuENPB5A+86ya5X8n",
    "Ft5c7eH5BP8/FPrxLixwGMQXhmNGLXxO0SJupTSSNF3C9BI3gIKuZGs1h0xSguj2ViBBKEjwsks",
    "dROchWLDGdi0jHCt5LV45D5oRfGhgcPcEEaXAg+/zu5idF9lNhiEnzdAQh0KGMzDjucjrezIPWs",
    "GYI/USZy5kqB9kxiTVU9wzVLOLSvwe8KpSmyoHbY62mqoamv3Ui29ipvgY6jhDMqynMdUuVBgt6",
    "h4TIJ8O+F0cheSbkob8gE1bFklLcj3gypLIvtc9Szh27NnXnWkW7B0IzHTIwtu9JEINB+9JMSXI",
    "2a8RQkHcOFcJPrW+va1+kGL/2yJFazCB5ReeajHygI3GIQHVh5FLcWBriMExYEHXLoqoFkU9wj1",
    "7zRtaFUKW+YKPMkVQmbym1PcVdVRbkdXzQp58oNO4iTc5gCT+W3qKYVsJl8B7BSLBjE6B02ch2j",
    "3wj21ECObcYiYOW9clpApUXiK0z4JeAssmEAAdK8VH250K44MkrHURNDv3V+XlLeOrYrNMgR9aF",
    "0vpaUKUCFU79ylBswZQMJ0ciSTq0svUIWg5WgRI2MaM1iOUGOXQvDc65sJLihIRhDCzSkSa6N3v",
    "2IclNS50hhVrbucakwy7rJRayNn0pgn/ohBLYGMg49zLO4QNktVrZ8nqG1U6bnLVC+ZEnZSDKY0",
    "LH4WZYG8wQuxUTEM9GRceTlXcwWvwLArEGlwFL2AEupnHdQ+XJaS3g6SLO7a8Ngeaaa06H1jc9B",
    "YDW0ttrqTX2MqrbAdhMV2gfrsW+xHRGKKY0t4LydxkCQ9cHkeR98sKEPPlrpg4/y27/30VUwx26",
    "U+eqKs6Fd5+7KOjdL9NOLPnwbjqzhyBK3eL63gq9tkEtWnnb+xyvzD+t/nd8oyKsv3qUXWePrtY",
    "K8gwK+kfPlHRz/8rr4/tlvn/z89Xc/vPtH0e6iHrU1PWrp/xb3Y93t3HIrH7+96JmWGXq6lsTs3",
    "C92+MObBX+wY/XlE3be86D1zTAQxhuetPTlw0m23o0d+tYL+i7eH9s+DQM4zQfS7wb5+ceOqcn6",
    "8uJ+NnbwcbjiV4T8en8//BunZeLnsMCXHY9kLHSbPx5s8Ofq+hyf7ofP5Pd32HOnYM+dtI+/ojZ",
    "bw1UxM6zbV9XvXXEvi7z/ceX3oUxdaVTEVZVH/mHcvva51suXff7dLfmMFOYfvcB2bMujLn3Ti2",
    "bXM+LWz7yTjz9bntWbIePBhtNC/hkb+tGmr/8R//7dkb/b+bht+ct/IH3ywZmgfI7Hg+z4lb/uS",
    "/vb3PKTBKo2n8H+jXqy/B1/R79ysxDfdjxjIohn6v17Jx+e7FOf/gTokQmS",
    ""
  };

  static char newstr [1485] = "";
  newstr[0] = '\0';
  for (i = 0; i < 21; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c17_dynamics_lib(SimStruct *S)
{
  const char* newstr = sf_c17_dynamics_lib_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(3407873107U));
  ssSetChecksum1(S,(3634851343U));
  ssSetChecksum2(S,(1556691703U));
  ssSetChecksum3(S,(4230330262U));
}

static void mdlRTW_c17_dynamics_lib(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c17_dynamics_lib(SimStruct *S)
{
  SFc17_dynamics_libInstanceStruct *chartInstance;
  chartInstance = (SFc17_dynamics_libInstanceStruct *)utMalloc(sizeof
    (SFc17_dynamics_libInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc17_dynamics_libInstanceStruct));
  chartInstance = new (chartInstance) SFc17_dynamics_libInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway = sf_opaque_gateway_c17_dynamics_lib;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c17_dynamics_lib;
  chartInstance->chartInfo.mdlStart = sf_opaque_mdl_start_c17_dynamics_lib;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c17_dynamics_lib;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c17_dynamics_lib;
  chartInstance->chartInfo.enableChart = sf_opaque_enable_c17_dynamics_lib;
  chartInstance->chartInfo.disableChart = sf_opaque_disable_c17_dynamics_lib;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c17_dynamics_lib;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c17_dynamics_lib;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c17_dynamics_lib;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c17_dynamics_lib;
  chartInstance->chartInfo.mdlSetWorkWidths = mdlSetWorkWidths_c17_dynamics_lib;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c17_dynamics_lib(chartInstance);
}

void c17_dynamics_lib_method_dispatcher(SimStruct *S, int_T method, void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c17_dynamics_lib(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c17_dynamics_lib(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c17_dynamics_lib(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c17_dynamics_lib_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
