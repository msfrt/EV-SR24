/* Include files */

#include "dynamics_lib_sfun.h"
#include "c19_dynamics_lib.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c19_dynamics_lib(SFc19_dynamics_libInstanceStruct
  *chartInstance);
static void initialize_params_c19_dynamics_lib(SFc19_dynamics_libInstanceStruct *
  chartInstance);
static void mdl_start_c19_dynamics_lib(SFc19_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_terminate_c19_dynamics_lib(SFc19_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_setup_runtime_resources_c19_dynamics_lib
  (SFc19_dynamics_libInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c19_dynamics_lib
  (SFc19_dynamics_libInstanceStruct *chartInstance);
static void enable_c19_dynamics_lib(SFc19_dynamics_libInstanceStruct
  *chartInstance);
static void disable_c19_dynamics_lib(SFc19_dynamics_libInstanceStruct
  *chartInstance);
static void sf_gateway_c19_dynamics_lib(SFc19_dynamics_libInstanceStruct
  *chartInstance);
static void ext_mode_exec_c19_dynamics_lib(SFc19_dynamics_libInstanceStruct
  *chartInstance);
static void c19_do_animation_call_c19_dynamics_lib
  (SFc19_dynamics_libInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c19_dynamics_lib
  (SFc19_dynamics_libInstanceStruct *chartInstance);
static void set_sim_state_c19_dynamics_lib(SFc19_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c19_st);
static void initSimStructsc19_dynamics_lib(SFc19_dynamics_libInstanceStruct
  *chartInstance);
static void initSubchartIOPointersc19_dynamics_lib
  (SFc19_dynamics_libInstanceStruct *chartInstance);
static real32_T c19_eML_blk_kernel(SFc19_dynamics_libInstanceStruct
  *chartInstance, real32_T c19_b_frontNSMWT, real32_T c19_b_frontElWT, real_T
  c19_b_frontGeoWT);
static real32_T c19_emlrt_marshallIn(SFc19_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c19_nullptr, const char_T *c19_identifier);
static real32_T c19_b_emlrt_marshallIn(SFc19_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c19_u, const emlrtMsgIdentifier *c19_parentId);
static uint8_T c19_c_emlrt_marshallIn(SFc19_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c19_nullptr, const char_T *c19_identifier);
static uint8_T c19_d_emlrt_marshallIn(SFc19_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c19_u, const emlrtMsgIdentifier *c19_parentId);
static void init_dsm_address_info(SFc19_dynamics_libInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc19_dynamics_libInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c19_dynamics_lib(SFc19_dynamics_libInstanceStruct
  *chartInstance)
{
  emlrtLicenseCheckR2022a(chartInstance->c19_fEmlrtCtx,
    "EMLRT:runTime:MexFunctionNeedsLicense", "distrib_computing_toolbox", 2);
  sf_is_first_init_cond(chartInstance->S);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  emlrtInitGPU(chartInstance->c19_fEmlrtCtx);
  hipGetLastError();
}

static void initialize_params_c19_dynamics_lib(SFc19_dynamics_libInstanceStruct *
  chartInstance)
{
}

static void mdl_start_c19_dynamics_lib(SFc19_dynamics_libInstanceStruct
  *chartInstance)
{
  sim_mode_is_external(chartInstance->S);
}

static void mdl_terminate_c19_dynamics_lib(SFc19_dynamics_libInstanceStruct
  *chartInstance)
{
  hipError_t c19_errCode;
  c19_errCode = hipGetLastError();
  if (c19_errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(c19_errCode), (char_T *)
                       hipGetErrorName(c19_errCode), (char_T *)
                       hipGetErrorString(c19_errCode), (char_T *)
                       "SimGPUErrorChecks", chartInstance->c19_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c19_dynamics_lib
  (SFc19_dynamics_libInstanceStruct *chartInstance)
{
  sfSetAnimationVectors(chartInstance->S, chartInstance->c19_JITStateAnimation,
                        chartInstance->c19_JITTransitionAnimation);
}

static void mdl_cleanup_runtime_resources_c19_dynamics_lib
  (SFc19_dynamics_libInstanceStruct *chartInstance)
{
}

static void enable_c19_dynamics_lib(SFc19_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c19_dynamics_lib(SFc19_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void sf_gateway_c19_dynamics_lib(SFc19_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c19_JITTransitionAnimation[0] = 0U;
  *chartInstance->c19_frontTotalWT = c19_eML_blk_kernel(chartInstance,
    *chartInstance->c19_frontNSMWT, *chartInstance->c19_frontElWT,
    *chartInstance->c19_frontGeoWT);
  c19_do_animation_call_c19_dynamics_lib(chartInstance);
}

static void ext_mode_exec_c19_dynamics_lib(SFc19_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void c19_do_animation_call_c19_dynamics_lib
  (SFc19_dynamics_libInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static const mxArray *get_sim_state_c19_dynamics_lib
  (SFc19_dynamics_libInstanceStruct *chartInstance)
{
  const mxArray *c19_b_y = NULL;
  const mxArray *c19_c_y = NULL;
  const mxArray *c19_st;
  const mxArray *c19_y = NULL;
  c19_st = NULL;
  c19_st = NULL;
  c19_y = NULL;
  sf_mex_assign(&c19_y, sf_mex_createcellmatrix(2, 1), false);
  c19_b_y = NULL;
  sf_mex_assign(&c19_b_y, sf_mex_create("y", chartInstance->c19_frontTotalWT, 1,
    0U, 0U, 0U, 0), false);
  sf_mex_setcell(c19_y, 0, c19_b_y);
  c19_c_y = NULL;
  sf_mex_assign(&c19_c_y, sf_mex_create("y",
    &chartInstance->c19_is_active_c19_dynamics_lib, 3, 0U, 0U, 0U, 0), false);
  sf_mex_setcell(c19_y, 1, c19_c_y);
  sf_mex_assign(&c19_st, c19_y, false);
  return c19_st;
}

static void set_sim_state_c19_dynamics_lib(SFc19_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c19_st)
{
  const mxArray *c19_u;
  c19_u = sf_mex_dup(c19_st);
  *chartInstance->c19_frontTotalWT = c19_emlrt_marshallIn(chartInstance,
    sf_mex_dup(sf_mex_getcell(c19_u, 0)), "frontTotalWT");
  chartInstance->c19_is_active_c19_dynamics_lib = c19_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c19_u, 1)),
     "is_active_c19_dynamics_lib");
  sf_mex_destroy(&c19_u);
  sf_mex_destroy(&c19_st);
}

static void initSimStructsc19_dynamics_lib(SFc19_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void initSubchartIOPointersc19_dynamics_lib
  (SFc19_dynamics_libInstanceStruct *chartInstance)
{
}

const mxArray *sf_c19_dynamics_lib_get_eml_resolved_functions_info()
{
  const mxArray *c19_nameCaptureInfo = NULL;
  c19_nameCaptureInfo = NULL;
  sf_mex_assign(&c19_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c19_nameCaptureInfo;
}

static real32_T c19_eML_blk_kernel(SFc19_dynamics_libInstanceStruct
  *chartInstance, real32_T c19_b_frontNSMWT, real32_T c19_b_frontElWT, real_T
  c19_b_frontGeoWT)
{
  return (c19_b_frontNSMWT + c19_b_frontElWT) + static_cast<real32_T>
    (c19_b_frontGeoWT);
}

static real32_T c19_emlrt_marshallIn(SFc19_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c19_nullptr, const char_T *c19_identifier)
{
  emlrtMsgIdentifier c19_thisId;
  real32_T c19_y;
  c19_thisId.fIdentifier = const_cast<const char_T *>(c19_identifier);
  c19_thisId.fParent = NULL;
  c19_thisId.bParentIsCell = false;
  c19_y = c19_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c19_nullptr),
    &c19_thisId);
  sf_mex_destroy(&c19_nullptr);
  return c19_y;
}

static real32_T c19_b_emlrt_marshallIn(SFc19_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c19_u, const emlrtMsgIdentifier *c19_parentId)
{
  real32_T c19_f;
  real32_T c19_y;
  sf_mex_import(c19_parentId, sf_mex_dup(c19_u), &c19_f, 0, 1, 0U, 0, 0U, 0);
  c19_y = c19_f;
  sf_mex_destroy(&c19_u);
  return c19_y;
}

static uint8_T c19_c_emlrt_marshallIn(SFc19_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c19_nullptr, const char_T *c19_identifier)
{
  emlrtMsgIdentifier c19_thisId;
  uint8_T c19_y;
  c19_thisId.fIdentifier = const_cast<const char_T *>(c19_identifier);
  c19_thisId.fParent = NULL;
  c19_thisId.bParentIsCell = false;
  c19_y = c19_d_emlrt_marshallIn(chartInstance, sf_mex_dup(c19_nullptr),
    &c19_thisId);
  sf_mex_destroy(&c19_nullptr);
  return c19_y;
}

static uint8_T c19_d_emlrt_marshallIn(SFc19_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c19_u, const emlrtMsgIdentifier *c19_parentId)
{
  uint8_T c19_b_u;
  uint8_T c19_y;
  sf_mex_import(c19_parentId, sf_mex_dup(c19_u), &c19_b_u, 1, 3, 0U, 0, 0U, 0);
  c19_y = c19_b_u;
  sf_mex_destroy(&c19_u);
  return c19_y;
}

static void init_dsm_address_info(SFc19_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc19_dynamics_libInstanceStruct
  *chartInstance)
{
  chartInstance->c19_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c19_frontNSMWT = (real32_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c19_frontTotalWT = (real32_T *)ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c19_frontElWT = (real32_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c19_frontGeoWT = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 2);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c19_dynamics_lib_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(2281896112U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(3041804723U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(2116558729U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(822124920U);
}

mxArray *sf_c19_dynamics_lib_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c19_dynamics_lib_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("chartInfo");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c19_dynamics_lib_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c19_dynamics_lib(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiDmYGJgYAPRQMzEAAGsUD4jVIwRLs4CF1cA4pLKglSQeHFRsmcKkM5LzAXzE0s"
    "rPPPS8sHmWzAgzGfDYj4jkvmcUHEI+GBPmX4RB5B+ByT9LFj08yDpF4Dy04ry80pC8ksSc8JDYO"
    "EEo8l3j4IDZfoh9gcQ8I8Umn9A/Mzi+MTkksyy1PhkQ8v4lEpgPGUmF8fnZCYhzAUBAHsWHYI="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c19_dynamics_lib_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "s3L65rUiZohatpj0LGvJmUG";
}

static void sf_opaque_initialize_c19_dynamics_lib(void *chartInstanceVar)
{
  initialize_params_c19_dynamics_lib((SFc19_dynamics_libInstanceStruct*)
    chartInstanceVar);
  initialize_c19_dynamics_lib((SFc19_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c19_dynamics_lib(void *chartInstanceVar)
{
  enable_c19_dynamics_lib((SFc19_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c19_dynamics_lib(void *chartInstanceVar)
{
  disable_c19_dynamics_lib((SFc19_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c19_dynamics_lib(void *chartInstanceVar)
{
  sf_gateway_c19_dynamics_lib((SFc19_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c19_dynamics_lib(SimStruct* S)
{
  return get_sim_state_c19_dynamics_lib((SFc19_dynamics_libInstanceStruct *)
    sf_get_chart_instance_ptr(S));     /* raw sim ctx */
}

static void sf_opaque_set_sim_state_c19_dynamics_lib(SimStruct* S, const mxArray
  *st)
{
  set_sim_state_c19_dynamics_lib((SFc19_dynamics_libInstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c19_dynamics_lib(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc19_dynamics_libInstanceStruct*) chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_dynamics_lib_optimization_info();
    }

    mdl_cleanup_runtime_resources_c19_dynamics_lib
      ((SFc19_dynamics_libInstanceStruct*) chartInstanceVar);
    ((SFc19_dynamics_libInstanceStruct*) chartInstanceVar)->
      ~SFc19_dynamics_libInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c19_dynamics_lib(void *chartInstanceVar)
{
  mdl_start_c19_dynamics_lib((SFc19_dynamics_libInstanceStruct*)
    chartInstanceVar);
  if (chartInstanceVar) {
    sf_reset_warnings_ChartRunTimeInfo(((SFc19_dynamics_libInstanceStruct*)
      chartInstanceVar)->S);
  }
}

static void sf_opaque_mdl_terminate_c19_dynamics_lib(void *chartInstanceVar)
{
  mdl_terminate_c19_dynamics_lib((SFc19_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c19_dynamics_lib(SimStruct *S)
{
  mdlProcessParamsCommon(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c19_dynamics_lib((SFc19_dynamics_libInstanceStruct*)
      sf_get_chart_instance_ptr(S));
    initSubchartIOPointersc19_dynamics_lib((SFc19_dynamics_libInstanceStruct*)
      sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c19_dynamics_lib_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [21] = {
    "eNrtWFuP20QUnqTLqkWl2gckUIVEH1ElpG1LUZEQ7DaXJSjLRiRppb6sJvZJPOx4xjuXXEBIvPP",
    "KG78DiQd+AD+BV/5BH3nkjOOkwQmJxytBi7DkOGP7m++cM+c2JpXWKcHjFp6P3iZkH6/X8ayS+f",
    "FaNq6snPP7e+TjbPzr64QEMoQRiK4dDtmU+B3Cxh2qaKyJ/yFoDF+CltwaJkVLDGVxLBNDUCACn",
    "CCRynjxahZbzsRF04rAMeunEQuibiQtDx/jhDQ8E3z2d7yJNR1krDMFgWkChCZS0o6iJqej7VZQ",
    "ZlKLILjQNva2lQbTtYlTVZ9abljCoTGFoCW0oWgFvUPfrqEGambqZ2Snr+4u0DJOOKOiuK0jqru",
    "QoHcY6Cch/p5Zg9YryIt8AyaokYpR3oh5LaKqKLbDUc5TdGvubWcTmzoM7GjExMhZV9kYBOqPfl",
    "LAVsOaHIOiIzgTfryB064xTRd46ZcFsYbF8ISq4wD9QEPoF4PowLpLcWmhh9N4YUHQAYeW7ik2x",
    "jXyzRstF0ql8oaN556kS2FT3sYYV1WX5G0GokY5137YnkzaMAae8tepoSWwc34PsNYs7En0Dhe5",
    "ntFvBbu0kGFrUoSsuFeOc6i0SHyBCb8AnMUuDCBEMy9FX060K46sNjKuYejX2+2CfOvYljCghjS",
    "AwvlaUaYBBU79ypM3ZNoFEqLRSibVsvAM8xgsBSV6aEV9ItUF2ti3MLywlYsEPzSEI6iDgTTJNd",
    "C7n1BuC8oca8zKzj36GrOsHy9iXfyUAgc0iCB0NZBxOMU8ixMUXWLtyucxajtmZlYHHSiWFI0ki",
    "wkdi5+zUm+WQF9cCDkRTSXjbtbFbPErAMwaVAksZY+xlKpZE4UvJrWCy16a3X0bBmdnajgdON84",
    "AYHV0OnqqjcNMKoaAttNFOgq2C77GtsRoZk22ALOGmkMhGkffERe9MF7G/rgg5U++CA7g3sfnYc",
    "z7EZZoM85G7h5DlfmuVmgn1704dtwZA1HlrjF9b0VfGUDL1m55vluVP/6/rUcXxXvVCqVNTu9ke",
    "PZy+H2MztdTr8VH3z3+/O7P/1w98dvvv9lF39ljb+S/ne4n6t++5Vb2fidRa+0zMzjteTl3v1sh",
    "x+8lfMDN9YP2h8+VH32TEbUJF8dtk/Gn8f9k3S+6zvkrebkXdy/4/ozDNw0D6igFWb7Hjemdt6P",
    "p/u3FXn3d9jjxoo/EfL806vh3zwqEjc3c/Zy46GSwvSkofxpL+/H5eW5c3Q1/Jy/s0Of2zl9bqf",
    "9+zl1WRrO8xlhPU7L+r0v7lXh+x9XfB2K1JNrJXFl+cg/jLuqfr518lV//3BLPiO59w9eYj225V",
    "Gffull0+s34tfPvJuNP1nu0WsR4+GGXUL2GBv54aan/xH//sPTfov+reHsl30YffbgWFA+w23Bf",
    "NuV3e4o901u+UgB1Zv3Xv9GPVlcy/RfEyZCOdHv37v/8P5V6tOfGgsLQg==",
    ""
  };

  static char newstr [1485] = "";
  newstr[0] = '\0';
  for (i = 0; i < 21; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c19_dynamics_lib(SimStruct *S)
{
  const char* newstr = sf_c19_dynamics_lib_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(1853716593U));
  ssSetChecksum1(S,(4041310260U));
  ssSetChecksum2(S,(714388778U));
  ssSetChecksum3(S,(3163257755U));
}

static void mdlRTW_c19_dynamics_lib(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c19_dynamics_lib(SimStruct *S)
{
  SFc19_dynamics_libInstanceStruct *chartInstance;
  chartInstance = (SFc19_dynamics_libInstanceStruct *)utMalloc(sizeof
    (SFc19_dynamics_libInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc19_dynamics_libInstanceStruct));
  chartInstance = new (chartInstance) SFc19_dynamics_libInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway = sf_opaque_gateway_c19_dynamics_lib;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c19_dynamics_lib;
  chartInstance->chartInfo.mdlStart = sf_opaque_mdl_start_c19_dynamics_lib;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c19_dynamics_lib;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c19_dynamics_lib;
  chartInstance->chartInfo.enableChart = sf_opaque_enable_c19_dynamics_lib;
  chartInstance->chartInfo.disableChart = sf_opaque_disable_c19_dynamics_lib;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c19_dynamics_lib;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c19_dynamics_lib;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c19_dynamics_lib;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c19_dynamics_lib;
  chartInstance->chartInfo.mdlSetWorkWidths = mdlSetWorkWidths_c19_dynamics_lib;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c19_dynamics_lib(chartInstance);
}

void c19_dynamics_lib_method_dispatcher(SimStruct *S, int_T method, void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c19_dynamics_lib(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c19_dynamics_lib(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c19_dynamics_lib(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c19_dynamics_lib_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
