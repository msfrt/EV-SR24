/* Include files */

#include "dynamics_lib_sfun.h"
#include "c10_dynamics_lib.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c10_dynamics_lib(SFc10_dynamics_libInstanceStruct
  *chartInstance);
static void initialize_params_c10_dynamics_lib(SFc10_dynamics_libInstanceStruct *
  chartInstance);
static void mdl_start_c10_dynamics_lib(SFc10_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_terminate_c10_dynamics_lib(SFc10_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_setup_runtime_resources_c10_dynamics_lib
  (SFc10_dynamics_libInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c10_dynamics_lib
  (SFc10_dynamics_libInstanceStruct *chartInstance);
static void enable_c10_dynamics_lib(SFc10_dynamics_libInstanceStruct
  *chartInstance);
static void disable_c10_dynamics_lib(SFc10_dynamics_libInstanceStruct
  *chartInstance);
static void sf_gateway_c10_dynamics_lib(SFc10_dynamics_libInstanceStruct
  *chartInstance);
static void ext_mode_exec_c10_dynamics_lib(SFc10_dynamics_libInstanceStruct
  *chartInstance);
static void c10_do_animation_call_c10_dynamics_lib
  (SFc10_dynamics_libInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c10_dynamics_lib
  (SFc10_dynamics_libInstanceStruct *chartInstance);
static void set_sim_state_c10_dynamics_lib(SFc10_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c10_st);
static void initSimStructsc10_dynamics_lib(SFc10_dynamics_libInstanceStruct
  *chartInstance);
static void initSubchartIOPointersc10_dynamics_lib
  (SFc10_dynamics_libInstanceStruct *chartInstance);
static real_T c10_eML_blk_kernel(SFc10_dynamics_libInstanceStruct *chartInstance,
  real_T c10_b_suspendedMass, real_T c10_b_suspendedMassWeightDistribution,
  real_T c10_b_MM5_ay, real_T c10_b_rearRollCenterHeight, real_T c10_b_rearTrack);
static real_T c10_emlrt_marshallIn(SFc10_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c10_nullptr, const char_T *c10_identifier);
static real_T c10_b_emlrt_marshallIn(SFc10_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c10_u, const emlrtMsgIdentifier *c10_parentId);
static uint8_T c10_c_emlrt_marshallIn(SFc10_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c10_nullptr, const char_T *c10_identifier);
static uint8_T c10_d_emlrt_marshallIn(SFc10_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c10_u, const emlrtMsgIdentifier *c10_parentId);
static void init_dsm_address_info(SFc10_dynamics_libInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc10_dynamics_libInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c10_dynamics_lib(SFc10_dynamics_libInstanceStruct
  *chartInstance)
{
  emlrtLicenseCheckR2022a(chartInstance->c10_fEmlrtCtx,
    "EMLRT:runTime:MexFunctionNeedsLicense", "distrib_computing_toolbox", 2);
  sf_is_first_init_cond(chartInstance->S);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  emlrtInitGPU(chartInstance->c10_fEmlrtCtx);
  hipGetLastError();
}

static void initialize_params_c10_dynamics_lib(SFc10_dynamics_libInstanceStruct *
  chartInstance)
{
}

static void mdl_start_c10_dynamics_lib(SFc10_dynamics_libInstanceStruct
  *chartInstance)
{
  sim_mode_is_external(chartInstance->S);
}

static void mdl_terminate_c10_dynamics_lib(SFc10_dynamics_libInstanceStruct
  *chartInstance)
{
  hipError_t c10_errCode;
  c10_errCode = hipGetLastError();
  if (c10_errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(c10_errCode), (char_T *)
                       hipGetErrorName(c10_errCode), (char_T *)
                       hipGetErrorString(c10_errCode), (char_T *)
                       "SimGPUErrorChecks", chartInstance->c10_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c10_dynamics_lib
  (SFc10_dynamics_libInstanceStruct *chartInstance)
{
  sfSetAnimationVectors(chartInstance->S, chartInstance->c10_JITStateAnimation,
                        chartInstance->c10_JITTransitionAnimation);
}

static void mdl_cleanup_runtime_resources_c10_dynamics_lib
  (SFc10_dynamics_libInstanceStruct *chartInstance)
{
}

static void enable_c10_dynamics_lib(SFc10_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c10_dynamics_lib(SFc10_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void sf_gateway_c10_dynamics_lib(SFc10_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c10_JITTransitionAnimation[0] = 0U;
  *chartInstance->c10_rearGeoWT = c10_eML_blk_kernel(chartInstance,
    *chartInstance->c10_suspendedMass,
    *chartInstance->c10_suspendedMassWeightDistribution,
    *chartInstance->c10_MM5_ay, *chartInstance->c10_rearRollCenterHeight,
    *chartInstance->c10_rearTrack);
  c10_do_animation_call_c10_dynamics_lib(chartInstance);
}

static void ext_mode_exec_c10_dynamics_lib(SFc10_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void c10_do_animation_call_c10_dynamics_lib
  (SFc10_dynamics_libInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static const mxArray *get_sim_state_c10_dynamics_lib
  (SFc10_dynamics_libInstanceStruct *chartInstance)
{
  const mxArray *c10_b_y = NULL;
  const mxArray *c10_c_y = NULL;
  const mxArray *c10_st;
  const mxArray *c10_y = NULL;
  c10_st = NULL;
  c10_st = NULL;
  c10_y = NULL;
  sf_mex_assign(&c10_y, sf_mex_createcellmatrix(2, 1), false);
  c10_b_y = NULL;
  sf_mex_assign(&c10_b_y, sf_mex_create("y", chartInstance->c10_rearGeoWT, 0, 0U,
    0U, 0U, 0), false);
  sf_mex_setcell(c10_y, 0, c10_b_y);
  c10_c_y = NULL;
  sf_mex_assign(&c10_c_y, sf_mex_create("y",
    &chartInstance->c10_is_active_c10_dynamics_lib, 3, 0U, 0U, 0U, 0), false);
  sf_mex_setcell(c10_y, 1, c10_c_y);
  sf_mex_assign(&c10_st, c10_y, false);
  return c10_st;
}

static void set_sim_state_c10_dynamics_lib(SFc10_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c10_st)
{
  const mxArray *c10_u;
  c10_u = sf_mex_dup(c10_st);
  *chartInstance->c10_rearGeoWT = c10_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getcell(c10_u, 0)), "rearGeoWT");
  chartInstance->c10_is_active_c10_dynamics_lib = c10_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c10_u, 1)),
     "is_active_c10_dynamics_lib");
  sf_mex_destroy(&c10_u);
  sf_mex_destroy(&c10_st);
}

static void initSimStructsc10_dynamics_lib(SFc10_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void initSubchartIOPointersc10_dynamics_lib
  (SFc10_dynamics_libInstanceStruct *chartInstance)
{
}

const mxArray *sf_c10_dynamics_lib_get_eml_resolved_functions_info()
{
  const mxArray *c10_nameCaptureInfo = NULL;
  c10_nameCaptureInfo = NULL;
  sf_mex_assign(&c10_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c10_nameCaptureInfo;
}

static real_T c10_eML_blk_kernel(SFc10_dynamics_libInstanceStruct *chartInstance,
  real_T c10_b_suspendedMass, real_T c10_b_suspendedMassWeightDistribution,
  real_T c10_b_MM5_ay, real_T c10_b_rearRollCenterHeight, real_T c10_b_rearTrack)
{
  return c10_b_suspendedMass * ((100.0 - c10_b_suspendedMassWeightDistribution) /
    100.0) * c10_b_MM5_ay * c10_b_rearRollCenterHeight / c10_b_rearTrack;
}

static real_T c10_emlrt_marshallIn(SFc10_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c10_nullptr, const char_T *c10_identifier)
{
  emlrtMsgIdentifier c10_thisId;
  real_T c10_y;
  c10_thisId.fIdentifier = const_cast<const char_T *>(c10_identifier);
  c10_thisId.fParent = NULL;
  c10_thisId.bParentIsCell = false;
  c10_y = c10_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c10_nullptr),
    &c10_thisId);
  sf_mex_destroy(&c10_nullptr);
  return c10_y;
}

static real_T c10_b_emlrt_marshallIn(SFc10_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c10_u, const emlrtMsgIdentifier *c10_parentId)
{
  real_T c10_d;
  real_T c10_y;
  sf_mex_import(c10_parentId, sf_mex_dup(c10_u), &c10_d, 1, 0, 0U, 0, 0U, 0);
  c10_y = c10_d;
  sf_mex_destroy(&c10_u);
  return c10_y;
}

static uint8_T c10_c_emlrt_marshallIn(SFc10_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c10_nullptr, const char_T *c10_identifier)
{
  emlrtMsgIdentifier c10_thisId;
  uint8_T c10_y;
  c10_thisId.fIdentifier = const_cast<const char_T *>(c10_identifier);
  c10_thisId.fParent = NULL;
  c10_thisId.bParentIsCell = false;
  c10_y = c10_d_emlrt_marshallIn(chartInstance, sf_mex_dup(c10_nullptr),
    &c10_thisId);
  sf_mex_destroy(&c10_nullptr);
  return c10_y;
}

static uint8_T c10_d_emlrt_marshallIn(SFc10_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c10_u, const emlrtMsgIdentifier *c10_parentId)
{
  uint8_T c10_b_u;
  uint8_T c10_y;
  sf_mex_import(c10_parentId, sf_mex_dup(c10_u), &c10_b_u, 1, 3, 0U, 0, 0U, 0);
  c10_y = c10_b_u;
  sf_mex_destroy(&c10_u);
  return c10_y;
}

static void init_dsm_address_info(SFc10_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc10_dynamics_libInstanceStruct
  *chartInstance)
{
  chartInstance->c10_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c10_suspendedMass = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c10_rearGeoWT = (real_T *)ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c10_suspendedMassWeightDistribution = (real_T *)
    ssGetInputPortSignal_wrapper(chartInstance->S, 1);
  chartInstance->c10_MM5_ay = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 2);
  chartInstance->c10_rearRollCenterHeight = (real_T *)
    ssGetInputPortSignal_wrapper(chartInstance->S, 3);
  chartInstance->c10_rearTrack = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 4);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c10_dynamics_lib_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(202376942U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(337017653U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(948754008U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(1146897419U);
}

mxArray *sf_c10_dynamics_lib_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c10_dynamics_lib_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("chartInfo");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c10_dynamics_lib_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c10_dynamics_lib(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiDmYGJgYAPRQMzEAAGsUD4jVIwRLs4CF1cA4pLKglSQeHFRsmcKkM5LzAXzE0s"
    "rPPPS8sHmWzAgzGfDYj4jkvmcUHEI+GBPmX4RB5B+ByT9LFj0cyLpF4Dyi1ITi9xT88NDoOJ8SD"
    "T57lFwoEw/xP4AAv6RQvMPiJ9ZHJ+YXJJZlhqfbGgQn1IJjKfM5OL4nMwkVP8BACOwHAs="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c10_dynamics_lib_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sWyfhtqFOhK8Ezx2ndVFl";
}

static void sf_opaque_initialize_c10_dynamics_lib(void *chartInstanceVar)
{
  initialize_params_c10_dynamics_lib((SFc10_dynamics_libInstanceStruct*)
    chartInstanceVar);
  initialize_c10_dynamics_lib((SFc10_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c10_dynamics_lib(void *chartInstanceVar)
{
  enable_c10_dynamics_lib((SFc10_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c10_dynamics_lib(void *chartInstanceVar)
{
  disable_c10_dynamics_lib((SFc10_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c10_dynamics_lib(void *chartInstanceVar)
{
  sf_gateway_c10_dynamics_lib((SFc10_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c10_dynamics_lib(SimStruct* S)
{
  return get_sim_state_c10_dynamics_lib((SFc10_dynamics_libInstanceStruct *)
    sf_get_chart_instance_ptr(S));     /* raw sim ctx */
}

static void sf_opaque_set_sim_state_c10_dynamics_lib(SimStruct* S, const mxArray
  *st)
{
  set_sim_state_c10_dynamics_lib((SFc10_dynamics_libInstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c10_dynamics_lib(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc10_dynamics_libInstanceStruct*) chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_dynamics_lib_optimization_info();
    }

    mdl_cleanup_runtime_resources_c10_dynamics_lib
      ((SFc10_dynamics_libInstanceStruct*) chartInstanceVar);
    ((SFc10_dynamics_libInstanceStruct*) chartInstanceVar)->
      ~SFc10_dynamics_libInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c10_dynamics_lib(void *chartInstanceVar)
{
  mdl_start_c10_dynamics_lib((SFc10_dynamics_libInstanceStruct*)
    chartInstanceVar);
  if (chartInstanceVar) {
    sf_reset_warnings_ChartRunTimeInfo(((SFc10_dynamics_libInstanceStruct*)
      chartInstanceVar)->S);
  }
}

static void sf_opaque_mdl_terminate_c10_dynamics_lib(void *chartInstanceVar)
{
  mdl_terminate_c10_dynamics_lib((SFc10_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c10_dynamics_lib(SimStruct *S)
{
  mdlProcessParamsCommon(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c10_dynamics_lib((SFc10_dynamics_libInstanceStruct*)
      sf_get_chart_instance_ptr(S));
    initSubchartIOPointersc10_dynamics_lib((SFc10_dynamics_libInstanceStruct*)
      sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c10_dynamics_lib_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [21] = {
    "eNrtWNuKI0UYroSZYUfWZQRFWQQXr7wRZkeEvVh0ZnNYgzNOMJkZ8MKh0v0nXUx1dU8dcti3UAR",
    "fxUfwBQS99Mo78XIv/avTycZOTKo6oLtiQ6ep7vrqP9R/+Cqk0jojeN3D+/gdQvbweQfvKpleu/",
    "m4snBP3++Qx/n4x9cICZIQBiA6pt9nY+J3CRO3qaSxIv6XoDF8CSrhRrNEtEQ/cccy0QcJIsAF0",
    "kRqL7mKxYYzcdM0IrCS1VXEgqgTJYaHT3BBGp4LPvk7uanRbZRYZxIC3QQIdSQTM4ianA7We0Hq",
    "US2C4EaZ2NtXCnTHpNZUdWa4ZimHxhiCllCaohfUBns7mmqo6bGfk629qjNDJ3HKGRXuvo6o6kC",
    "K0aHhIg3x99xo9J6jXJTXY4LqRDLKGzGvRVS6Ytsc9TzDsObeftaxrkPPDAZMDKx3pYlBoP0YJw",
    "6+6teSIUg6gHPhJzew1jXG2QbP49IRq1kMl1SeBBgHCkK/HMQAVh2KWwtdXMYLC4L2OLRUV7Ih7",
    "pFv3WjZVCpVN0w8jSRVCpvJbQxxV1VJuc1A1Cjnyg/bTdJTGALP5NeppiWwU/keYKVY2E0wOmzm",
    "ema/EezWQI6tJSJk7lE5LKCyJvEFFnwHOIttGkCIbp6rPl9oUx4ZpZO4hqlfPz11lLeMbQkNsk8",
    "DcK7XkjIFqHAWV55yQ6ZsIiEavaQzK51XmOZgKShRfSPqo0TeoI99G8MLX9lM8ENDOIA6aMiKXA",
    "Oj+5Jy46hzrLAq2/C4UFhl/eQi1uZPKXBAgwhC2wMZhzOss7iA6xYr2z5P0Noh05M6qECy1DWTD",
    "BZ0bH7WS91JChfiRiQj0ZRJ3MlZzJq4AsCqQaXAVvYEW6mcNFF5N60l3Haz6u5LGKyfqea0Z2Pj",
    "KQjshtZW271pgFnVEEg3UaFtsB32DOmIUExppICTRpYD4ZQHkxc8eGcFDz5Y4MEH+R08PLwOJ8h",
    "GWaCuOevZdQ4X1rnrwKdnPHwdjizhyBw3e36wgK+skEsWnnb+o4X5+9W/zt8tyKvO3tlrdm5YwL",
    "9ekLdTwO/l/vqNf//dT8/f//VbevnG7798/U3R7qIelSU9KmQm/4eq37nlXj5+d8aZ5hV6uFTE7",
    "NzPNsTDW4V4sGN1NelH+rZ5Hn3+qPFsfCTCyybP17uzQd9qQd/Z+weWp2ECZ/VABq0wP//YMTVT",
    "Xl7cz70N/thfiCtC/vh0O/ybxy75s1/w135WK6h8CslVd0U8l9fnwfF2+Kn89gZ77hfsuZ/x+Gt",
    "qqzVcFyvDsn1l494X96rI+x/nvg8ufWW3JK6sPPIP47a1z7dfvurzD9fUM1KYf/AS27Gujvrwpp",
    "fNrp+JH595Lx9/Mj+r1yLGwxWnhfwzEvr+qq//kfh+7um/t/Nxw/ov/4P0q49OBOUTPB5Mj1/56",
    "7a0/83NPyFrUavPYP9GPyEr+PgqvnK3kN92PGIiTEbqw4dHHx9t05/+BPapCqM=",
    ""
  };

  static char newstr [1489] = "";
  newstr[0] = '\0';
  for (i = 0; i < 21; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c10_dynamics_lib(SimStruct *S)
{
  const char* newstr = sf_c10_dynamics_lib_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(2492951783U));
  ssSetChecksum1(S,(3777165525U));
  ssSetChecksum2(S,(290873746U));
  ssSetChecksum3(S,(2438912494U));
}

static void mdlRTW_c10_dynamics_lib(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c10_dynamics_lib(SimStruct *S)
{
  SFc10_dynamics_libInstanceStruct *chartInstance;
  chartInstance = (SFc10_dynamics_libInstanceStruct *)utMalloc(sizeof
    (SFc10_dynamics_libInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc10_dynamics_libInstanceStruct));
  chartInstance = new (chartInstance) SFc10_dynamics_libInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway = sf_opaque_gateway_c10_dynamics_lib;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c10_dynamics_lib;
  chartInstance->chartInfo.mdlStart = sf_opaque_mdl_start_c10_dynamics_lib;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c10_dynamics_lib;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c10_dynamics_lib;
  chartInstance->chartInfo.enableChart = sf_opaque_enable_c10_dynamics_lib;
  chartInstance->chartInfo.disableChart = sf_opaque_disable_c10_dynamics_lib;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c10_dynamics_lib;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c10_dynamics_lib;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c10_dynamics_lib;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c10_dynamics_lib;
  chartInstance->chartInfo.mdlSetWorkWidths = mdlSetWorkWidths_c10_dynamics_lib;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c10_dynamics_lib(chartInstance);
}

void c10_dynamics_lib_method_dispatcher(SimStruct *S, int_T method, void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c10_dynamics_lib(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c10_dynamics_lib(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c10_dynamics_lib(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c10_dynamics_lib_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
