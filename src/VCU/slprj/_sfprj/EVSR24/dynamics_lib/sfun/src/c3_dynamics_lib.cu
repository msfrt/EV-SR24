/* Include files */

#include "dynamics_lib_sfun.h"
#include "c3_dynamics_lib.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c3_dynamics_lib(SFc3_dynamics_libInstanceStruct
  *chartInstance);
static void initialize_params_c3_dynamics_lib(SFc3_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_start_c3_dynamics_lib(SFc3_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_terminate_c3_dynamics_lib(SFc3_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_setup_runtime_resources_c3_dynamics_lib
  (SFc3_dynamics_libInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c3_dynamics_lib
  (SFc3_dynamics_libInstanceStruct *chartInstance);
static void enable_c3_dynamics_lib(SFc3_dynamics_libInstanceStruct
  *chartInstance);
static void disable_c3_dynamics_lib(SFc3_dynamics_libInstanceStruct
  *chartInstance);
static void sf_gateway_c3_dynamics_lib(SFc3_dynamics_libInstanceStruct
  *chartInstance);
static void ext_mode_exec_c3_dynamics_lib(SFc3_dynamics_libInstanceStruct
  *chartInstance);
static void c3_do_animation_call_c3_dynamics_lib(SFc3_dynamics_libInstanceStruct
  *chartInstance);
static const mxArray *get_sim_state_c3_dynamics_lib
  (SFc3_dynamics_libInstanceStruct *chartInstance);
static void set_sim_state_c3_dynamics_lib(SFc3_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c3_st);
static void initSimStructsc3_dynamics_lib(SFc3_dynamics_libInstanceStruct
  *chartInstance);
static void initSubchartIOPointersc3_dynamics_lib
  (SFc3_dynamics_libInstanceStruct *chartInstance);
static real32_T c3_eML_blk_kernel(SFc3_dynamics_libInstanceStruct *chartInstance,
  real_T c3_b_frontARBStiffness, real_T c3_b_frontTrack, real32_T
  c3_b_frontARBMotionRatio);
static real32_T c3_emlrt_marshallIn(SFc3_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c3_nullptr, const char_T *c3_identifier);
static real32_T c3_b_emlrt_marshallIn(SFc3_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c3_u, const emlrtMsgIdentifier *c3_parentId);
static uint8_T c3_c_emlrt_marshallIn(SFc3_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c3_nullptr, const char_T *c3_identifier);
static uint8_T c3_d_emlrt_marshallIn(SFc3_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c3_u, const emlrtMsgIdentifier *c3_parentId);
static void init_dsm_address_info(SFc3_dynamics_libInstanceStruct *chartInstance);
static void init_simulink_io_address(SFc3_dynamics_libInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c3_dynamics_lib(SFc3_dynamics_libInstanceStruct
  *chartInstance)
{
  emlrtLicenseCheckR2022a(chartInstance->c3_fEmlrtCtx,
    "EMLRT:runTime:MexFunctionNeedsLicense", "distrib_computing_toolbox", 2);
  sf_is_first_init_cond(chartInstance->S);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  emlrtInitGPU(chartInstance->c3_fEmlrtCtx);
  hipGetLastError();
}

static void initialize_params_c3_dynamics_lib(SFc3_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void mdl_start_c3_dynamics_lib(SFc3_dynamics_libInstanceStruct
  *chartInstance)
{
  sim_mode_is_external(chartInstance->S);
}

static void mdl_terminate_c3_dynamics_lib(SFc3_dynamics_libInstanceStruct
  *chartInstance)
{
  hipError_t c3_errCode;
  c3_errCode = hipGetLastError();
  if (c3_errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(c3_errCode), (char_T *)
                       hipGetErrorName(c3_errCode), (char_T *)
                       hipGetErrorString(c3_errCode), (char_T *)
                       "SimGPUErrorChecks", chartInstance->c3_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c3_dynamics_lib
  (SFc3_dynamics_libInstanceStruct *chartInstance)
{
  sfSetAnimationVectors(chartInstance->S, chartInstance->c3_JITStateAnimation,
                        chartInstance->c3_JITTransitionAnimation);
}

static void mdl_cleanup_runtime_resources_c3_dynamics_lib
  (SFc3_dynamics_libInstanceStruct *chartInstance)
{
}

static void enable_c3_dynamics_lib(SFc3_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c3_dynamics_lib(SFc3_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void sf_gateway_c3_dynamics_lib(SFc3_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c3_JITTransitionAnimation[0] = 0U;
  *chartInstance->c3_antiRollFrontARB = c3_eML_blk_kernel(chartInstance,
    *chartInstance->c3_frontARBStiffness, *chartInstance->c3_frontTrack,
    *chartInstance->c3_frontARBMotionRatio);
  c3_do_animation_call_c3_dynamics_lib(chartInstance);
}

static void ext_mode_exec_c3_dynamics_lib(SFc3_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void c3_do_animation_call_c3_dynamics_lib(SFc3_dynamics_libInstanceStruct
  *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static const mxArray *get_sim_state_c3_dynamics_lib
  (SFc3_dynamics_libInstanceStruct *chartInstance)
{
  const mxArray *c3_b_y = NULL;
  const mxArray *c3_c_y = NULL;
  const mxArray *c3_st;
  const mxArray *c3_y = NULL;
  c3_st = NULL;
  c3_st = NULL;
  c3_y = NULL;
  sf_mex_assign(&c3_y, sf_mex_createcellmatrix(2, 1), false);
  c3_b_y = NULL;
  sf_mex_assign(&c3_b_y, sf_mex_create("y", chartInstance->c3_antiRollFrontARB,
    1, 0U, 0U, 0U, 0), false);
  sf_mex_setcell(c3_y, 0, c3_b_y);
  c3_c_y = NULL;
  sf_mex_assign(&c3_c_y, sf_mex_create("y",
    &chartInstance->c3_is_active_c3_dynamics_lib, 3, 0U, 0U, 0U, 0), false);
  sf_mex_setcell(c3_y, 1, c3_c_y);
  sf_mex_assign(&c3_st, c3_y, false);
  return c3_st;
}

static void set_sim_state_c3_dynamics_lib(SFc3_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c3_st)
{
  const mxArray *c3_u;
  c3_u = sf_mex_dup(c3_st);
  *chartInstance->c3_antiRollFrontARB = c3_emlrt_marshallIn(chartInstance,
    sf_mex_dup(sf_mex_getcell(c3_u, 0)), "antiRollFrontARB");
  chartInstance->c3_is_active_c3_dynamics_lib = c3_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c3_u, 1)),
     "is_active_c3_dynamics_lib");
  sf_mex_destroy(&c3_u);
  sf_mex_destroy(&c3_st);
}

static void initSimStructsc3_dynamics_lib(SFc3_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void initSubchartIOPointersc3_dynamics_lib
  (SFc3_dynamics_libInstanceStruct *chartInstance)
{
}

const mxArray *sf_c3_dynamics_lib_get_eml_resolved_functions_info()
{
  const mxArray *c3_nameCaptureInfo = NULL;
  c3_nameCaptureInfo = NULL;
  sf_mex_assign(&c3_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c3_nameCaptureInfo;
}

static real32_T c3_eML_blk_kernel(SFc3_dynamics_libInstanceStruct *chartInstance,
  real_T c3_b_frontARBStiffness, real_T c3_b_frontTrack, real32_T
  c3_b_frontARBMotionRatio)
{
  real_T c3_a;
  c3_a = c3_b_frontTrack / 1000.0;
  return static_cast<real32_T>(c3_b_frontARBStiffness * 1000.0 * (c3_a * c3_a) *
    1.5574077246549023) / (c3_b_frontARBMotionRatio * c3_b_frontARBMotionRatio);
}

static real32_T c3_emlrt_marshallIn(SFc3_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c3_nullptr, const char_T *c3_identifier)
{
  emlrtMsgIdentifier c3_thisId;
  real32_T c3_y;
  c3_thisId.fIdentifier = const_cast<const char_T *>(c3_identifier);
  c3_thisId.fParent = NULL;
  c3_thisId.bParentIsCell = false;
  c3_y = c3_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c3_nullptr), &c3_thisId);
  sf_mex_destroy(&c3_nullptr);
  return c3_y;
}

static real32_T c3_b_emlrt_marshallIn(SFc3_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c3_u, const emlrtMsgIdentifier *c3_parentId)
{
  real32_T c3_f;
  real32_T c3_y;
  sf_mex_import(c3_parentId, sf_mex_dup(c3_u), &c3_f, 0, 1, 0U, 0, 0U, 0);
  c3_y = c3_f;
  sf_mex_destroy(&c3_u);
  return c3_y;
}

static uint8_T c3_c_emlrt_marshallIn(SFc3_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c3_nullptr, const char_T *c3_identifier)
{
  emlrtMsgIdentifier c3_thisId;
  uint8_T c3_y;
  c3_thisId.fIdentifier = const_cast<const char_T *>(c3_identifier);
  c3_thisId.fParent = NULL;
  c3_thisId.bParentIsCell = false;
  c3_y = c3_d_emlrt_marshallIn(chartInstance, sf_mex_dup(c3_nullptr), &c3_thisId);
  sf_mex_destroy(&c3_nullptr);
  return c3_y;
}

static uint8_T c3_d_emlrt_marshallIn(SFc3_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c3_u, const emlrtMsgIdentifier *c3_parentId)
{
  uint8_T c3_b_u;
  uint8_T c3_y;
  sf_mex_import(c3_parentId, sf_mex_dup(c3_u), &c3_b_u, 1, 3, 0U, 0, 0U, 0);
  c3_y = c3_b_u;
  sf_mex_destroy(&c3_u);
  return c3_y;
}

static void init_dsm_address_info(SFc3_dynamics_libInstanceStruct *chartInstance)
{
}

static void init_simulink_io_address(SFc3_dynamics_libInstanceStruct
  *chartInstance)
{
  chartInstance->c3_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c3_frontARBStiffness = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c3_antiRollFrontARB = (real32_T *)ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c3_frontTrack = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c3_frontARBMotionRatio = (real32_T *)
    ssGetInputPortSignal_wrapper(chartInstance->S, 2);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c3_dynamics_lib_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(626859362U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(2458528930U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(2688538396U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(373436868U);
}

mxArray *sf_c3_dynamics_lib_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c3_dynamics_lib_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("chartInfo");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c3_dynamics_lib_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c3_dynamics_lib(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiDmYGJgYAPRQMzEAAGsUD4jVIwRLs4CF1cA4pLKglSQeHFRsmcKkM5LzAXzE0s"
    "rPPPS8sHmWzAgzGfDYj4jkvmcUHEI+GBPmX4RB5B+ByT9LFj0CyDpF4DixLySzKD8nBy3ovy8Es"
    "cgJz6oPGXuUXCgTD/E/gAC/pFE8w+In1kcn5hcklmWGp9sHJ9SCYymzOTi+JzMJCRzQQAAsKEen"
    "A=="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c3_dynamics_lib_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sGGEig4Er6ZO31InGfGRTDB";
}

static void sf_opaque_initialize_c3_dynamics_lib(void *chartInstanceVar)
{
  initialize_params_c3_dynamics_lib((SFc3_dynamics_libInstanceStruct*)
    chartInstanceVar);
  initialize_c3_dynamics_lib((SFc3_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c3_dynamics_lib(void *chartInstanceVar)
{
  enable_c3_dynamics_lib((SFc3_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c3_dynamics_lib(void *chartInstanceVar)
{
  disable_c3_dynamics_lib((SFc3_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c3_dynamics_lib(void *chartInstanceVar)
{
  sf_gateway_c3_dynamics_lib((SFc3_dynamics_libInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c3_dynamics_lib(SimStruct* S)
{
  return get_sim_state_c3_dynamics_lib((SFc3_dynamics_libInstanceStruct *)
    sf_get_chart_instance_ptr(S));     /* raw sim ctx */
}

static void sf_opaque_set_sim_state_c3_dynamics_lib(SimStruct* S, const mxArray *
  st)
{
  set_sim_state_c3_dynamics_lib((SFc3_dynamics_libInstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c3_dynamics_lib(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc3_dynamics_libInstanceStruct*) chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_dynamics_lib_optimization_info();
    }

    mdl_cleanup_runtime_resources_c3_dynamics_lib
      ((SFc3_dynamics_libInstanceStruct*) chartInstanceVar);
    ((SFc3_dynamics_libInstanceStruct*) chartInstanceVar)->
      ~SFc3_dynamics_libInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c3_dynamics_lib(void *chartInstanceVar)
{
  mdl_start_c3_dynamics_lib((SFc3_dynamics_libInstanceStruct*) chartInstanceVar);
  if (chartInstanceVar) {
    sf_reset_warnings_ChartRunTimeInfo(((SFc3_dynamics_libInstanceStruct*)
      chartInstanceVar)->S);
  }
}

static void sf_opaque_mdl_terminate_c3_dynamics_lib(void *chartInstanceVar)
{
  mdl_terminate_c3_dynamics_lib((SFc3_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c3_dynamics_lib(SimStruct *S)
{
  mdlProcessParamsCommon(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c3_dynamics_lib((SFc3_dynamics_libInstanceStruct*)
      sf_get_chart_instance_ptr(S));
    initSubchartIOPointersc3_dynamics_lib((SFc3_dynamics_libInstanceStruct*)
      sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c3_dynamics_lib_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [21] = {
    "eNrtWE+P20QUn2SXVYvasgckWgmJ3uCC1HYBcUB0N4kTIu2y0TpbUC+rif0Sj3Y8dudPsul34Mi",
    "ZM9+AKzc+Ah+DIyfEG8dJgxMSjyNBi7DkOGP7N7/33rx/Y1LrnhE87uH5+X1CDvB6C886mR1v5e",
    "Pa0jm7v0++yMe/vE1IkIQwAuGb4ZDdELdDmLhHJY0VcT8EjeECVMKNZonoimFSHsvEECSIACdIE",
    "6mdeBWLDWfium1EYJnVNxELIj9KDA8bOCENzwWf/h1vanQPGVtMQqDbAKGOZGJGUZvT0WYrSD1p",
    "RhBcKxM720qB9k1qVVVnhmuWcvBuIOgKpSlaQW3R19dUQ1PfuBnZ6qv8OTqJU86oKG/riCofUvQ",
    "ODZdpiL/nRqP1SvIi34AJqhPJKPdi3oyoLIvtcZTzDN2aO9tZx7oFAzMaMTGy1pUmBoH6o5+UsN",
    "WwmYxB0hGcCzfewGrn3WQLvPDLkljNYnhG5UmAfqAgdItBdGDlU1xa6OM0TlgQdMChq/qSjXGNX",
    "PNG14ZSpbxh4pknqUrYjNcb46qqirztQDQp58oN20/SUxgDz/hbVNMK2Bm/A1gpFvYT9A4buY7R",
    "bwR7YSDHNhMRsvJeOS6gsiLxNSb8EnAW2zCAEM28EH0x0bY4MkoncRNDv3V6WpJvFdsVGuSQBlA",
    "6X0vKFKDAmV858oZM2UBCNFpJZ1qWnmEWg5WgRA2NaE0SeY02di0Mr2xlI8ENDeEIWqAhS3Ieev",
    "czyk1JmWOFWdm6x6XCLOvGi1gbP5XAAQ0iCG0NZBzOMM/iBGWXWNnyeYLajpmetkAFkqVlI8lgQ",
    "sfiZ63Un6ZwKa5FMhFtmcR+3sVs8CsAzBpUCixlDSylctpG4ctJLeFFP8vurg2DtTPVnA6sb3RA",
    "YDW0utrqTQOMKk9gu4kC7YL12UtsR4RiSmMLOPWyGAizPviYvOqD99f0we8s9cGH+Tg4ugqn2Iy",
    "yQF1xNsjmebQ0z50S/fS8D9+EIys4ssDNrx8t4WtreMnStch3u/7X9/cKfHW8U6vVVux0t8CzX8",
    "Ad5Lb64cM/LtPv9sbf/nz3x/73jZfb+Gsr/LXsv8X9VHfbr9zLx+/Pe6VFZh6vJC/77ldb/OC9g",
    "h/Ysep0PDb6xJOfPT8/etwVnWHnot9qZPPd2iJvvSDv/P5D259h4GZ5QAbdMN/32DE1s348278t",
    "yXuwxR63l/yJkN+e7oZ/97hM3BwW7GVPLMvsIuEcs5HQJxeNZT+uLs/D493wM/7eFn0eFPR5kPX",
    "vV9RmabgqZoTVOK3q9664N4Xvf1z5dShTT/Yq4qrykX8Yt6t+rnXyTX//0YZ8RgrvH77GemzKoy",
    "790uum16/ErZ/5IB9/udijNyPGwzW7hPwxNvLDdU//I/79u6P95v2bZ+2Xfxh9fnQiKJ/itmC27",
    "cpv96T9Jrd4JIGq9Xuvf6OezK/b+q87hfi24wkTYTJRHz9+8umTXerTnzExCfU=",
    ""
  };

  static char newstr [1489] = "";
  newstr[0] = '\0';
  for (i = 0; i < 21; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c3_dynamics_lib(SimStruct *S)
{
  const char* newstr = sf_c3_dynamics_lib_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(1442719647U));
  ssSetChecksum1(S,(1979944304U));
  ssSetChecksum2(S,(2836250968U));
  ssSetChecksum3(S,(2051183956U));
}

static void mdlRTW_c3_dynamics_lib(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c3_dynamics_lib(SimStruct *S)
{
  SFc3_dynamics_libInstanceStruct *chartInstance;
  chartInstance = (SFc3_dynamics_libInstanceStruct *)utMalloc(sizeof
    (SFc3_dynamics_libInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc3_dynamics_libInstanceStruct));
  chartInstance = new (chartInstance) SFc3_dynamics_libInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway = sf_opaque_gateway_c3_dynamics_lib;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c3_dynamics_lib;
  chartInstance->chartInfo.mdlStart = sf_opaque_mdl_start_c3_dynamics_lib;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c3_dynamics_lib;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c3_dynamics_lib;
  chartInstance->chartInfo.enableChart = sf_opaque_enable_c3_dynamics_lib;
  chartInstance->chartInfo.disableChart = sf_opaque_disable_c3_dynamics_lib;
  chartInstance->chartInfo.getSimState = sf_opaque_get_sim_state_c3_dynamics_lib;
  chartInstance->chartInfo.setSimState = sf_opaque_set_sim_state_c3_dynamics_lib;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c3_dynamics_lib;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c3_dynamics_lib;
  chartInstance->chartInfo.mdlSetWorkWidths = mdlSetWorkWidths_c3_dynamics_lib;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c3_dynamics_lib(chartInstance);
}

void c3_dynamics_lib_method_dispatcher(SimStruct *S, int_T method, void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c3_dynamics_lib(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c3_dynamics_lib(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c3_dynamics_lib(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c3_dynamics_lib_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
