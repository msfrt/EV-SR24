/* Include files */

#include "dynamics_lib_sfun.h"
#include "c15_dynamics_lib.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c15_dynamics_lib(SFc15_dynamics_libInstanceStruct
  *chartInstance);
static void initialize_params_c15_dynamics_lib(SFc15_dynamics_libInstanceStruct *
  chartInstance);
static void mdl_start_c15_dynamics_lib(SFc15_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_terminate_c15_dynamics_lib(SFc15_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_setup_runtime_resources_c15_dynamics_lib
  (SFc15_dynamics_libInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c15_dynamics_lib
  (SFc15_dynamics_libInstanceStruct *chartInstance);
static void enable_c15_dynamics_lib(SFc15_dynamics_libInstanceStruct
  *chartInstance);
static void disable_c15_dynamics_lib(SFc15_dynamics_libInstanceStruct
  *chartInstance);
static void sf_gateway_c15_dynamics_lib(SFc15_dynamics_libInstanceStruct
  *chartInstance);
static void ext_mode_exec_c15_dynamics_lib(SFc15_dynamics_libInstanceStruct
  *chartInstance);
static void c15_do_animation_call_c15_dynamics_lib
  (SFc15_dynamics_libInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c15_dynamics_lib
  (SFc15_dynamics_libInstanceStruct *chartInstance);
static void set_sim_state_c15_dynamics_lib(SFc15_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c15_st);
static void initSimStructsc15_dynamics_lib(SFc15_dynamics_libInstanceStruct
  *chartInstance);
static void initSubchartIOPointersc15_dynamics_lib
  (SFc15_dynamics_libInstanceStruct *chartInstance);
static real32_T c15_eML_blk_kernel(SFc15_dynamics_libInstanceStruct
  *chartInstance, real_T c15_b_antiRollRearSprings, real_T c15_b_antiRollRearARB,
  real32_T c15_b_antiRollRearTires);
static real32_T c15_emlrt_marshallIn(SFc15_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c15_nullptr, const char_T *c15_identifier);
static real32_T c15_b_emlrt_marshallIn(SFc15_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c15_u, const emlrtMsgIdentifier *c15_parentId);
static uint8_T c15_c_emlrt_marshallIn(SFc15_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c15_nullptr, const char_T *c15_identifier);
static uint8_T c15_d_emlrt_marshallIn(SFc15_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c15_u, const emlrtMsgIdentifier *c15_parentId);
static void init_dsm_address_info(SFc15_dynamics_libInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc15_dynamics_libInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c15_dynamics_lib(SFc15_dynamics_libInstanceStruct
  *chartInstance)
{
  emlrtLicenseCheckR2022a(chartInstance->c15_fEmlrtCtx,
    "EMLRT:runTime:MexFunctionNeedsLicense", "distrib_computing_toolbox", 2);
  sf_is_first_init_cond(chartInstance->S);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  emlrtInitGPU(chartInstance->c15_fEmlrtCtx);
  hipGetLastError();
}

static void initialize_params_c15_dynamics_lib(SFc15_dynamics_libInstanceStruct *
  chartInstance)
{
}

static void mdl_start_c15_dynamics_lib(SFc15_dynamics_libInstanceStruct
  *chartInstance)
{
  sim_mode_is_external(chartInstance->S);
}

static void mdl_terminate_c15_dynamics_lib(SFc15_dynamics_libInstanceStruct
  *chartInstance)
{
  hipError_t c15_errCode;
  c15_errCode = hipGetLastError();
  if (c15_errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(c15_errCode), (char_T *)
                       hipGetErrorName(c15_errCode), (char_T *)
                       hipGetErrorString(c15_errCode), (char_T *)
                       "SimGPUErrorChecks", chartInstance->c15_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c15_dynamics_lib
  (SFc15_dynamics_libInstanceStruct *chartInstance)
{
  sfSetAnimationVectors(chartInstance->S, chartInstance->c15_JITStateAnimation,
                        chartInstance->c15_JITTransitionAnimation);
}

static void mdl_cleanup_runtime_resources_c15_dynamics_lib
  (SFc15_dynamics_libInstanceStruct *chartInstance)
{
}

static void enable_c15_dynamics_lib(SFc15_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c15_dynamics_lib(SFc15_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void sf_gateway_c15_dynamics_lib(SFc15_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c15_JITTransitionAnimation[0] = 0U;
  *chartInstance->c15_antiRollRearAxleAndTire = c15_eML_blk_kernel(chartInstance,
    *chartInstance->c15_antiRollRearSprings, *chartInstance->c15_antiRollRearARB,
    *chartInstance->c15_antiRollRearTires);
  c15_do_animation_call_c15_dynamics_lib(chartInstance);
}

static void ext_mode_exec_c15_dynamics_lib(SFc15_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void c15_do_animation_call_c15_dynamics_lib
  (SFc15_dynamics_libInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static const mxArray *get_sim_state_c15_dynamics_lib
  (SFc15_dynamics_libInstanceStruct *chartInstance)
{
  const mxArray *c15_b_y = NULL;
  const mxArray *c15_c_y = NULL;
  const mxArray *c15_st;
  const mxArray *c15_y = NULL;
  c15_st = NULL;
  c15_st = NULL;
  c15_y = NULL;
  sf_mex_assign(&c15_y, sf_mex_createcellmatrix(2, 1), false);
  c15_b_y = NULL;
  sf_mex_assign(&c15_b_y, sf_mex_create("y",
    chartInstance->c15_antiRollRearAxleAndTire, 1, 0U, 0U, 0U, 0), false);
  sf_mex_setcell(c15_y, 0, c15_b_y);
  c15_c_y = NULL;
  sf_mex_assign(&c15_c_y, sf_mex_create("y",
    &chartInstance->c15_is_active_c15_dynamics_lib, 3, 0U, 0U, 0U, 0), false);
  sf_mex_setcell(c15_y, 1, c15_c_y);
  sf_mex_assign(&c15_st, c15_y, false);
  return c15_st;
}

static void set_sim_state_c15_dynamics_lib(SFc15_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c15_st)
{
  const mxArray *c15_u;
  c15_u = sf_mex_dup(c15_st);
  *chartInstance->c15_antiRollRearAxleAndTire = c15_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c15_u, 0)),
     "antiRollRearAxleAndTire");
  chartInstance->c15_is_active_c15_dynamics_lib = c15_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c15_u, 1)),
     "is_active_c15_dynamics_lib");
  sf_mex_destroy(&c15_u);
  sf_mex_destroy(&c15_st);
}

static void initSimStructsc15_dynamics_lib(SFc15_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void initSubchartIOPointersc15_dynamics_lib
  (SFc15_dynamics_libInstanceStruct *chartInstance)
{
}

const mxArray *sf_c15_dynamics_lib_get_eml_resolved_functions_info()
{
  const mxArray *c15_nameCaptureInfo = NULL;
  c15_nameCaptureInfo = NULL;
  sf_mex_assign(&c15_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c15_nameCaptureInfo;
}

static real32_T c15_eML_blk_kernel(SFc15_dynamics_libInstanceStruct
  *chartInstance, real_T c15_b_antiRollRearSprings, real_T c15_b_antiRollRearARB,
  real32_T c15_b_antiRollRearTires)
{
  return static_cast<real32_T>(c15_b_antiRollRearSprings + c15_b_antiRollRearARB)
    * c15_b_antiRollRearTires / (static_cast<real32_T>(c15_b_antiRollRearSprings
    + c15_b_antiRollRearARB) + c15_b_antiRollRearTires);
}

static real32_T c15_emlrt_marshallIn(SFc15_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c15_nullptr, const char_T *c15_identifier)
{
  emlrtMsgIdentifier c15_thisId;
  real32_T c15_y;
  c15_thisId.fIdentifier = const_cast<const char_T *>(c15_identifier);
  c15_thisId.fParent = NULL;
  c15_thisId.bParentIsCell = false;
  c15_y = c15_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c15_nullptr),
    &c15_thisId);
  sf_mex_destroy(&c15_nullptr);
  return c15_y;
}

static real32_T c15_b_emlrt_marshallIn(SFc15_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c15_u, const emlrtMsgIdentifier *c15_parentId)
{
  real32_T c15_f;
  real32_T c15_y;
  sf_mex_import(c15_parentId, sf_mex_dup(c15_u), &c15_f, 0, 1, 0U, 0, 0U, 0);
  c15_y = c15_f;
  sf_mex_destroy(&c15_u);
  return c15_y;
}

static uint8_T c15_c_emlrt_marshallIn(SFc15_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c15_nullptr, const char_T *c15_identifier)
{
  emlrtMsgIdentifier c15_thisId;
  uint8_T c15_y;
  c15_thisId.fIdentifier = const_cast<const char_T *>(c15_identifier);
  c15_thisId.fParent = NULL;
  c15_thisId.bParentIsCell = false;
  c15_y = c15_d_emlrt_marshallIn(chartInstance, sf_mex_dup(c15_nullptr),
    &c15_thisId);
  sf_mex_destroy(&c15_nullptr);
  return c15_y;
}

static uint8_T c15_d_emlrt_marshallIn(SFc15_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c15_u, const emlrtMsgIdentifier *c15_parentId)
{
  uint8_T c15_b_u;
  uint8_T c15_y;
  sf_mex_import(c15_parentId, sf_mex_dup(c15_u), &c15_b_u, 1, 3, 0U, 0, 0U, 0);
  c15_y = c15_b_u;
  sf_mex_destroy(&c15_u);
  return c15_y;
}

static void init_dsm_address_info(SFc15_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc15_dynamics_libInstanceStruct
  *chartInstance)
{
  chartInstance->c15_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c15_antiRollRearAxleAndTire = (real32_T *)
    ssGetOutputPortSignal_wrapper(chartInstance->S, 1);
  chartInstance->c15_antiRollRearSprings = (real_T *)
    ssGetInputPortSignal_wrapper(chartInstance->S, 0);
  chartInstance->c15_antiRollRearARB = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c15_antiRollRearTires = (real32_T *)
    ssGetInputPortSignal_wrapper(chartInstance->S, 2);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c15_dynamics_lib_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(460049933U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(14324852U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(820815212U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(3565248949U);
}

mxArray *sf_c15_dynamics_lib_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c15_dynamics_lib_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("chartInfo");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c15_dynamics_lib_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c15_dynamics_lib(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiAWYGJgYAPSHEDMxAABrFA+I1SMES7OAhdXAOKSyoJUkHhxUbJnCpDOS8wF8xN"
    "LKzzz0vLB5lswIMxnw2I+I5L5nFBxCPhgT5l+EQeQfg8k/SxY9Isj6ReA8hPzSjKD8nNyglITix"
    "wrclId81JCMotSwf6BhRv57lJwoEw/xP4AAv6SQvMXiJ9ZHJ+YXJJZlhqfbGgan1IJjK/M5OL4n"
    "MwkhLkgAAAUqSHM"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c15_dynamics_lib_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sd1kXtovyIdHqHeTwYna1wE";
}

static void sf_opaque_initialize_c15_dynamics_lib(void *chartInstanceVar)
{
  initialize_params_c15_dynamics_lib((SFc15_dynamics_libInstanceStruct*)
    chartInstanceVar);
  initialize_c15_dynamics_lib((SFc15_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c15_dynamics_lib(void *chartInstanceVar)
{
  enable_c15_dynamics_lib((SFc15_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c15_dynamics_lib(void *chartInstanceVar)
{
  disable_c15_dynamics_lib((SFc15_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c15_dynamics_lib(void *chartInstanceVar)
{
  sf_gateway_c15_dynamics_lib((SFc15_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c15_dynamics_lib(SimStruct* S)
{
  return get_sim_state_c15_dynamics_lib((SFc15_dynamics_libInstanceStruct *)
    sf_get_chart_instance_ptr(S));     /* raw sim ctx */
}

static void sf_opaque_set_sim_state_c15_dynamics_lib(SimStruct* S, const mxArray
  *st)
{
  set_sim_state_c15_dynamics_lib((SFc15_dynamics_libInstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c15_dynamics_lib(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc15_dynamics_libInstanceStruct*) chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_dynamics_lib_optimization_info();
    }

    mdl_cleanup_runtime_resources_c15_dynamics_lib
      ((SFc15_dynamics_libInstanceStruct*) chartInstanceVar);
    ((SFc15_dynamics_libInstanceStruct*) chartInstanceVar)->
      ~SFc15_dynamics_libInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c15_dynamics_lib(void *chartInstanceVar)
{
  mdl_start_c15_dynamics_lib((SFc15_dynamics_libInstanceStruct*)
    chartInstanceVar);
  if (chartInstanceVar) {
    sf_reset_warnings_ChartRunTimeInfo(((SFc15_dynamics_libInstanceStruct*)
      chartInstanceVar)->S);
  }
}

static void sf_opaque_mdl_terminate_c15_dynamics_lib(void *chartInstanceVar)
{
  mdl_terminate_c15_dynamics_lib((SFc15_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c15_dynamics_lib(SimStruct *S)
{
  mdlProcessParamsCommon(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c15_dynamics_lib((SFc15_dynamics_libInstanceStruct*)
      sf_get_chart_instance_ptr(S));
    initSubchartIOPointersc15_dynamics_lib((SFc15_dynamics_libInstanceStruct*)
      sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c15_dynamics_lib_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [21] = {
    "eNrtWE2P20QYdrbLiqJS7QEJVFWiRySE1C2qxAGVTfPBRtqwUZMt0Mtq4nkTjzKe8c5HPnrgBjf",
    "EX+En9Mg/gP/AhSNH3nGcNHVC4nEkaBGWHGdsP/O87zvv1ziotNoBHrfxPP0gCI7w+jaeB8H8eC",
    "sbV1bO+f3D4PNs/Ms7QRBKCkMQXTsYsGngdwgbd4gisQ78D0FieAJacmuYFC0xkMWxTAxAgQhxg",
    "kQq48WrWWw5E6OmFaFj1l9HLIy6kbScPsYJCb0QfPZ3vIk1HWSsMwWhaQJQEylph1GTk+F2Kygz",
    "qUUQjrSNvW2lwXRt4lTVbcsNSzg0phC2hDYEraB36Ns1xEDNTP2M7PTV3QVaxglnRBS3dUR0FxL",
    "0DgOXCcXfC2vQegV5ka/PBDFSMcIbMa9FRBXFdjjK2Ua35t52NrGpQ98Oh0wMnXWVjUGg/ugnBW",
    "w1qMkxKDKEC+HHGzrtGtN0gZd+WRBrWAxPiaqG6AcaqF8MogPrLsGlhR5O44UFQfocWrqn2BjXy",
    "DdvtFwolcobNp57ki6FTXkbY1xVXZK3GYoa4Vz7YXsyOYcx8JS/TgwpgZ3ze4C1ZrQn0Ttc5HpG",
    "vxXs2kKGrUlBWXGvHOdQaZH4ChN+ATiLXRgARTMvRV9OtCuOrDYyrmHo18/PC/KtY1vCgBqQEAr",
    "na0WYBhQ49StPXsq0CyREo5VMqmXhGeYxWAoa6IEV9YlUI7Sxb2F4aSsXCX5ooEOog4E0yTXQu5",
    "8SbgvKHGvMys49LjVmWT9exLr4KQUOSRgBdTWQcWhjnsUJii6xduWzitqOmZnVQYeKJUUjyWJCx",
    "+LnrNSbJXApRkJORFPJuJt1MVv8CgCzBlECS9ljLKVq1kThi0mt4LqXZnffhsHZmRhO+s43vgSB",
    "1dDp6qo3CTGqGgLbTRRoH2yXPcd2RGimDbaAs0YaA3TeBwcv++DDDX3w8UoffJyd4cnDKzrDbpS",
    "F+oqzvpvn/so8twr004s+fBsuWMMFS9zi+tEKvrKBN1i55vluHrz6/o0c3wHeqVQqa3Z6N8dzmM",
    "MdZXY6+p3+9P35sxcf//pz+7sfWz/s4q+s8VfS/w734sBvv3I7G99d9ErLzDxeS17u3bMdfvB+z",
    "g/cWNOT0TdGjmctenZ9Br3Jt4KcTBrpfMc75D3Iybu4f8/1Zxi4aR5QYYtm+x43Jnbej7v5P1uR",
    "92iHPW6u+FMQ/PHFfvj3TsvaC8syeyI5xx2Tqk45VAXt4a7oFX8uL9e90/3wc/7ODr3u5PS6k/b",
    "xV8Rla7jKZ4b1eC3r/764N4Xvf1zxdShSV26UxJXlC/5h3L76+dbLN/39+1vyWZB7//g11mNbHv",
    "Xpm143vX4L/PqaD7Pxo+VevRYxTjfsFrLH2NAPNj39j/j3n572W/QlDWe/7APps0+rgvAZbg/m2",
    "6/sdke5b3PLRwqI3rwH+zfqyfI7/o5+5VYuvt14wgSVE/3JyYOHD/apT38BYyIPhA==",
    ""
  };

  static char newstr [1493] = "";
  newstr[0] = '\0';
  for (i = 0; i < 21; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c15_dynamics_lib(SimStruct *S)
{
  const char* newstr = sf_c15_dynamics_lib_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(2405755654U));
  ssSetChecksum1(S,(3227143301U));
  ssSetChecksum2(S,(1303435051U));
  ssSetChecksum3(S,(2252967294U));
}

static void mdlRTW_c15_dynamics_lib(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c15_dynamics_lib(SimStruct *S)
{
  SFc15_dynamics_libInstanceStruct *chartInstance;
  chartInstance = (SFc15_dynamics_libInstanceStruct *)utMalloc(sizeof
    (SFc15_dynamics_libInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc15_dynamics_libInstanceStruct));
  chartInstance = new (chartInstance) SFc15_dynamics_libInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway = sf_opaque_gateway_c15_dynamics_lib;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c15_dynamics_lib;
  chartInstance->chartInfo.mdlStart = sf_opaque_mdl_start_c15_dynamics_lib;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c15_dynamics_lib;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c15_dynamics_lib;
  chartInstance->chartInfo.enableChart = sf_opaque_enable_c15_dynamics_lib;
  chartInstance->chartInfo.disableChart = sf_opaque_disable_c15_dynamics_lib;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c15_dynamics_lib;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c15_dynamics_lib;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c15_dynamics_lib;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c15_dynamics_lib;
  chartInstance->chartInfo.mdlSetWorkWidths = mdlSetWorkWidths_c15_dynamics_lib;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c15_dynamics_lib(chartInstance);
}

void c15_dynamics_lib_method_dispatcher(SimStruct *S, int_T method, void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c15_dynamics_lib(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c15_dynamics_lib(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c15_dynamics_lib(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c15_dynamics_lib_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
