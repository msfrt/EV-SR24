/* Include files */

#include "dynamics_lib_sfun.h"
#include "c18_dynamics_lib.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c18_dynamics_lib(SFc18_dynamics_libInstanceStruct
  *chartInstance);
static void initialize_params_c18_dynamics_lib(SFc18_dynamics_libInstanceStruct *
  chartInstance);
static void mdl_start_c18_dynamics_lib(SFc18_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_terminate_c18_dynamics_lib(SFc18_dynamics_libInstanceStruct
  *chartInstance);
static void mdl_setup_runtime_resources_c18_dynamics_lib
  (SFc18_dynamics_libInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c18_dynamics_lib
  (SFc18_dynamics_libInstanceStruct *chartInstance);
static void enable_c18_dynamics_lib(SFc18_dynamics_libInstanceStruct
  *chartInstance);
static void disable_c18_dynamics_lib(SFc18_dynamics_libInstanceStruct
  *chartInstance);
static void sf_gateway_c18_dynamics_lib(SFc18_dynamics_libInstanceStruct
  *chartInstance);
static void ext_mode_exec_c18_dynamics_lib(SFc18_dynamics_libInstanceStruct
  *chartInstance);
static void c18_do_animation_call_c18_dynamics_lib
  (SFc18_dynamics_libInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c18_dynamics_lib
  (SFc18_dynamics_libInstanceStruct *chartInstance);
static void set_sim_state_c18_dynamics_lib(SFc18_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c18_st);
static void initSimStructsc18_dynamics_lib(SFc18_dynamics_libInstanceStruct
  *chartInstance);
static void initSubchartIOPointersc18_dynamics_lib
  (SFc18_dynamics_libInstanceStruct *chartInstance);
static real32_T c18_eML_blk_kernel(SFc18_dynamics_libInstanceStruct
  *chartInstance, real_T c18_b_suspendedMass, real_T c18_b_MM5_ay, real_T
  c18_b_deltaZ, real32_T c18_b_antiRollRearAxleAndTire, real32_T
  c18_b_antiRollTotal, real_T c18_b_rearTrack);
static real32_T c18_emlrt_marshallIn(SFc18_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c18_nullptr, const char_T *c18_identifier);
static real32_T c18_b_emlrt_marshallIn(SFc18_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c18_u, const emlrtMsgIdentifier *c18_parentId);
static uint8_T c18_c_emlrt_marshallIn(SFc18_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c18_nullptr, const char_T *c18_identifier);
static uint8_T c18_d_emlrt_marshallIn(SFc18_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c18_u, const emlrtMsgIdentifier *c18_parentId);
static void init_dsm_address_info(SFc18_dynamics_libInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc18_dynamics_libInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c18_dynamics_lib(SFc18_dynamics_libInstanceStruct
  *chartInstance)
{
  emlrtLicenseCheckR2022a(chartInstance->c18_fEmlrtCtx,
    "EMLRT:runTime:MexFunctionNeedsLicense", "distrib_computing_toolbox", 2);
  sf_is_first_init_cond(chartInstance->S);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  emlrtInitGPU(chartInstance->c18_fEmlrtCtx);
  hipGetLastError();
}

static void initialize_params_c18_dynamics_lib(SFc18_dynamics_libInstanceStruct *
  chartInstance)
{
}

static void mdl_start_c18_dynamics_lib(SFc18_dynamics_libInstanceStruct
  *chartInstance)
{
  sim_mode_is_external(chartInstance->S);
}

static void mdl_terminate_c18_dynamics_lib(SFc18_dynamics_libInstanceStruct
  *chartInstance)
{
  hipError_t c18_errCode;
  c18_errCode = hipGetLastError();
  if (c18_errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(c18_errCode), (char_T *)
                       hipGetErrorName(c18_errCode), (char_T *)
                       hipGetErrorString(c18_errCode), (char_T *)
                       "SimGPUErrorChecks", chartInstance->c18_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c18_dynamics_lib
  (SFc18_dynamics_libInstanceStruct *chartInstance)
{
  sfSetAnimationVectors(chartInstance->S, chartInstance->c18_JITStateAnimation,
                        chartInstance->c18_JITTransitionAnimation);
}

static void mdl_cleanup_runtime_resources_c18_dynamics_lib
  (SFc18_dynamics_libInstanceStruct *chartInstance)
{
}

static void enable_c18_dynamics_lib(SFc18_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c18_dynamics_lib(SFc18_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void sf_gateway_c18_dynamics_lib(SFc18_dynamics_libInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c18_JITTransitionAnimation[0] = 0U;
  *chartInstance->c18_rearElWT = c18_eML_blk_kernel(chartInstance,
    *chartInstance->c18_suspendedMass, *chartInstance->c18_MM5_ay,
    *chartInstance->c18_deltaZ, *chartInstance->c18_antiRollRearAxleAndTire,
    *chartInstance->c18_antiRollTotal, *chartInstance->c18_rearTrack);
  c18_do_animation_call_c18_dynamics_lib(chartInstance);
}

static void ext_mode_exec_c18_dynamics_lib(SFc18_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void c18_do_animation_call_c18_dynamics_lib
  (SFc18_dynamics_libInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static const mxArray *get_sim_state_c18_dynamics_lib
  (SFc18_dynamics_libInstanceStruct *chartInstance)
{
  const mxArray *c18_b_y = NULL;
  const mxArray *c18_c_y = NULL;
  const mxArray *c18_st;
  const mxArray *c18_y = NULL;
  c18_st = NULL;
  c18_st = NULL;
  c18_y = NULL;
  sf_mex_assign(&c18_y, sf_mex_createcellmatrix(2, 1), false);
  c18_b_y = NULL;
  sf_mex_assign(&c18_b_y, sf_mex_create("y", chartInstance->c18_rearElWT, 1, 0U,
    0U, 0U, 0), false);
  sf_mex_setcell(c18_y, 0, c18_b_y);
  c18_c_y = NULL;
  sf_mex_assign(&c18_c_y, sf_mex_create("y",
    &chartInstance->c18_is_active_c18_dynamics_lib, 3, 0U, 0U, 0U, 0), false);
  sf_mex_setcell(c18_y, 1, c18_c_y);
  sf_mex_assign(&c18_st, c18_y, false);
  return c18_st;
}

static void set_sim_state_c18_dynamics_lib(SFc18_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c18_st)
{
  const mxArray *c18_u;
  c18_u = sf_mex_dup(c18_st);
  *chartInstance->c18_rearElWT = c18_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getcell(c18_u, 0)), "rearElWT");
  chartInstance->c18_is_active_c18_dynamics_lib = c18_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c18_u, 1)),
     "is_active_c18_dynamics_lib");
  sf_mex_destroy(&c18_u);
  sf_mex_destroy(&c18_st);
}

static void initSimStructsc18_dynamics_lib(SFc18_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void initSubchartIOPointersc18_dynamics_lib
  (SFc18_dynamics_libInstanceStruct *chartInstance)
{
}

const mxArray *sf_c18_dynamics_lib_get_eml_resolved_functions_info()
{
  const mxArray *c18_nameCaptureInfo = NULL;
  c18_nameCaptureInfo = NULL;
  sf_mex_assign(&c18_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c18_nameCaptureInfo;
}

static real32_T c18_eML_blk_kernel(SFc18_dynamics_libInstanceStruct
  *chartInstance, real_T c18_b_suspendedMass, real_T c18_b_MM5_ay, real_T
  c18_b_deltaZ, real32_T c18_b_antiRollRearAxleAndTire, real32_T
  c18_b_antiRollTotal, real_T c18_b_rearTrack)
{
  return static_cast<real32_T>(c18_b_suspendedMass * c18_b_MM5_ay * c18_b_deltaZ)
    * (c18_b_antiRollRearAxleAndTire / c18_b_antiRollTotal) /
    static_cast<real32_T>(c18_b_rearTrack);
}

static real32_T c18_emlrt_marshallIn(SFc18_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c18_nullptr, const char_T *c18_identifier)
{
  emlrtMsgIdentifier c18_thisId;
  real32_T c18_y;
  c18_thisId.fIdentifier = const_cast<const char_T *>(c18_identifier);
  c18_thisId.fParent = NULL;
  c18_thisId.bParentIsCell = false;
  c18_y = c18_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c18_nullptr),
    &c18_thisId);
  sf_mex_destroy(&c18_nullptr);
  return c18_y;
}

static real32_T c18_b_emlrt_marshallIn(SFc18_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c18_u, const emlrtMsgIdentifier *c18_parentId)
{
  real32_T c18_f;
  real32_T c18_y;
  sf_mex_import(c18_parentId, sf_mex_dup(c18_u), &c18_f, 0, 1, 0U, 0, 0U, 0);
  c18_y = c18_f;
  sf_mex_destroy(&c18_u);
  return c18_y;
}

static uint8_T c18_c_emlrt_marshallIn(SFc18_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c18_nullptr, const char_T *c18_identifier)
{
  emlrtMsgIdentifier c18_thisId;
  uint8_T c18_y;
  c18_thisId.fIdentifier = const_cast<const char_T *>(c18_identifier);
  c18_thisId.fParent = NULL;
  c18_thisId.bParentIsCell = false;
  c18_y = c18_d_emlrt_marshallIn(chartInstance, sf_mex_dup(c18_nullptr),
    &c18_thisId);
  sf_mex_destroy(&c18_nullptr);
  return c18_y;
}

static uint8_T c18_d_emlrt_marshallIn(SFc18_dynamics_libInstanceStruct
  *chartInstance, const mxArray *c18_u, const emlrtMsgIdentifier *c18_parentId)
{
  uint8_T c18_b_u;
  uint8_T c18_y;
  sf_mex_import(c18_parentId, sf_mex_dup(c18_u), &c18_b_u, 1, 3, 0U, 0, 0U, 0);
  c18_y = c18_b_u;
  sf_mex_destroy(&c18_u);
  return c18_y;
}

static void init_dsm_address_info(SFc18_dynamics_libInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc18_dynamics_libInstanceStruct
  *chartInstance)
{
  chartInstance->c18_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c18_rearElWT = (real32_T *)ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c18_suspendedMass = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c18_MM5_ay = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c18_deltaZ = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 2);
  chartInstance->c18_antiRollRearAxleAndTire = (real32_T *)
    ssGetInputPortSignal_wrapper(chartInstance->S, 3);
  chartInstance->c18_antiRollTotal = (real32_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 4);
  chartInstance->c18_rearTrack = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 5);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c18_dynamics_lib_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(3878572157U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(325660021U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(2063972600U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(1311154427U);
}

mxArray *sf_c18_dynamics_lib_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c18_dynamics_lib_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("chartInfo");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c18_dynamics_lib_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c18_dynamics_lib(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0izig62fBop8DSb8AlF+UmljkmhMewgcVp8wdCg6U6Y"
    "fYH0DAH1Jo/gDxM4vjE5NLMstS45MNLeJTKoHxk5lcHJ+TmYQwFwQAK5sblw=="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c18_dynamics_lib_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sxh5L5hbwm5Sj8oFJtl3WXE";
}

static void sf_opaque_initialize_c18_dynamics_lib(void *chartInstanceVar)
{
  initialize_params_c18_dynamics_lib((SFc18_dynamics_libInstanceStruct*)
    chartInstanceVar);
  initialize_c18_dynamics_lib((SFc18_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c18_dynamics_lib(void *chartInstanceVar)
{
  enable_c18_dynamics_lib((SFc18_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c18_dynamics_lib(void *chartInstanceVar)
{
  disable_c18_dynamics_lib((SFc18_dynamics_libInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c18_dynamics_lib(void *chartInstanceVar)
{
  sf_gateway_c18_dynamics_lib((SFc18_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c18_dynamics_lib(SimStruct* S)
{
  return get_sim_state_c18_dynamics_lib((SFc18_dynamics_libInstanceStruct *)
    sf_get_chart_instance_ptr(S));     /* raw sim ctx */
}

static void sf_opaque_set_sim_state_c18_dynamics_lib(SimStruct* S, const mxArray
  *st)
{
  set_sim_state_c18_dynamics_lib((SFc18_dynamics_libInstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c18_dynamics_lib(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc18_dynamics_libInstanceStruct*) chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_dynamics_lib_optimization_info();
    }

    mdl_cleanup_runtime_resources_c18_dynamics_lib
      ((SFc18_dynamics_libInstanceStruct*) chartInstanceVar);
    ((SFc18_dynamics_libInstanceStruct*) chartInstanceVar)->
      ~SFc18_dynamics_libInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c18_dynamics_lib(void *chartInstanceVar)
{
  mdl_start_c18_dynamics_lib((SFc18_dynamics_libInstanceStruct*)
    chartInstanceVar);
  if (chartInstanceVar) {
    sf_reset_warnings_ChartRunTimeInfo(((SFc18_dynamics_libInstanceStruct*)
      chartInstanceVar)->S);
  }
}

static void sf_opaque_mdl_terminate_c18_dynamics_lib(void *chartInstanceVar)
{
  mdl_terminate_c18_dynamics_lib((SFc18_dynamics_libInstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c18_dynamics_lib(SimStruct *S)
{
  mdlProcessParamsCommon(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c18_dynamics_lib((SFc18_dynamics_libInstanceStruct*)
      sf_get_chart_instance_ptr(S));
    initSubchartIOPointersc18_dynamics_lib((SFc18_dynamics_libInstanceStruct*)
      sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c18_dynamics_lib_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [21] = {
    "eNrtWFtvG0UUHptgpahEEUICVQj6yAtSL4qUBwRJfaGuEmKxToP6Eo13j71DZme2c/EF9YV/0Z/",
    "AI8/8An4CP4NHHjmzXrtm7do7tgQtYqXNZnbnm3PON+c2JpX2OcHrAO/jjwmp4XMf7yqZXu/m48",
    "rCPX2/R77Mx7+9R0goIxiACGy/z8bE7xI26VBFE038L0ET+A605NYwKdqiL8tjmeiDAhHiAqlUx",
    "kuuZonlTNy0rAidZH0VszAOYml59AgXpNGF4JPXyU2t6aDEBlMQmhZAZGIl7SBucTpYz4Iyo3oM",
    "4Y22iTdXGkxgU2eqPrfcsJRDcwxhW2hDkQW9wd7AUAN1M/Yj2dmrgxlaJilnVJTnOqY6gBS9w8B",
    "lGuHfC2uQvZJyUV6PCWqkYpQ3E16PqSqL7XDU8xzdmnvzbBLTgJ4dDJgYOHaVTUCg/egnJbjq1+",
    "UQFB3AhfCTGzrrmuNsg+d+WRJrWAJPqToN0Q80RH4xiA6sA4pbC11cxgsLgvY4tHVXsSHukW/ea",
    "LtQ2ipv2GTqSXorbCa3OcRd1VvKbYWiTjnXftiuTM9gCDyT36CGboGdyvcAa82irkTvcJHrGf1W",
    "sOcWcmxdioiV98phAZUViW8x4ZeAs8SFAURI81z1+UKb4shqI5M6hn7j7KykvGVsWxhQfRpC6Xy",
    "tKNOACmd+5Sk3YtoFEqKRJZNZWXqFaQxuBSW6b0VjJNUNcuxbGF5x5SLBDw3RABpgIEtyTfTup5",
    "TbkjonGrOyc49LjVnWTy5iXfxsBQ5pGEPkaiDjcI55Fhcou8Xalc9TtHbIzKQBOlQsLRtJFhM6F",
    "j/HUneSwqW4EXIkWkomQd7FrPErAMwaVAksZY+wlKpJC5Uvp7WC590su/s2DI5najjtOd/4BgRW",
    "Q2erq940xKhqCmw3UaFdsAH7EdsRoZk22AJOmlkMRFkffEJe9cF7K/rgw4U++DC/w/vH19EEu1E",
    "W6mvOem6dewvr3C7RT8/68HU4soQjc9zs+fkCvrJCLll4Zn3/wvxb1b/PrxXkVWfvsoss8fV+Qd",
    "5eAV/L+Xrx8uef+Isn77z89Uo++/SDx0W7i3pUlvSoZP873C9Vv3PLQT7+ZNYzzTP0cCmJubmPN",
    "/jDRwV/cGM9jo/OjuLeKDkKfjiWrSeGP7z6vjndpw36Vgv6zt7fdX0aBnCWD1TYjvLzjxtTO+3L",
    "i/tZ28DHrQW/IuSPr3fDf3hSxK/ia7/A136WK6hq8qvuwWv80k+Puye74afyOxvsuFOw407Wv19",
    "Tl6XhupgRluN0W3/3xb0t8v7Hld+HMvWktiVuW3nkH8btap9vnXzb599bk89IYf7hG2zHujzq0y",
    "+9aXb9Tvz6mM/y8VfzM3o9ZjxacUrIP2Mj31/19T/i33968jfr25qOv/yH0WcPTwXlEzwWTI9d+",
    "euOcr/JzT9ht6JXn73+jXoye246t9wuxLcbj5iI5Eh/cf/B0YNd6tNfub4HXA==",
    ""
  };

  static char newstr [1489] = "";
  newstr[0] = '\0';
  for (i = 0; i < 21; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c18_dynamics_lib(SimStruct *S)
{
  const char* newstr = sf_c18_dynamics_lib_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(2175309948U));
  ssSetChecksum1(S,(55213164U));
  ssSetChecksum2(S,(1868020376U));
  ssSetChecksum3(S,(1209146970U));
}

static void mdlRTW_c18_dynamics_lib(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c18_dynamics_lib(SimStruct *S)
{
  SFc18_dynamics_libInstanceStruct *chartInstance;
  chartInstance = (SFc18_dynamics_libInstanceStruct *)utMalloc(sizeof
    (SFc18_dynamics_libInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc18_dynamics_libInstanceStruct));
  chartInstance = new (chartInstance) SFc18_dynamics_libInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway = sf_opaque_gateway_c18_dynamics_lib;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c18_dynamics_lib;
  chartInstance->chartInfo.mdlStart = sf_opaque_mdl_start_c18_dynamics_lib;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c18_dynamics_lib;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c18_dynamics_lib;
  chartInstance->chartInfo.enableChart = sf_opaque_enable_c18_dynamics_lib;
  chartInstance->chartInfo.disableChart = sf_opaque_disable_c18_dynamics_lib;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c18_dynamics_lib;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c18_dynamics_lib;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c18_dynamics_lib;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c18_dynamics_lib;
  chartInstance->chartInfo.mdlSetWorkWidths = mdlSetWorkWidths_c18_dynamics_lib;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c18_dynamics_lib(chartInstance);
}

void c18_dynamics_lib_method_dispatcher(SimStruct *S, int_T method, void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c18_dynamics_lib(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c18_dynamics_lib(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c18_dynamics_lib(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c18_dynamics_lib_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
